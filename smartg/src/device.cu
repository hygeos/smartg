#include "hip/hip_runtime.h"

/**********************************************************
*
*			device.cu
*
*	> Kernel
*	> Modélisation phénomènes physiques
*	> Initialisation de données dans le device
*	> Fonctions liées au générateur aléatoire
*
***********************************************************/

/*************************************************************/
/*************************************************************/
/*          MENTION LICENCE POUR RNGs                        */
/*************************************************************/
/*         Philox 4x32 7                                     */
/*
Copyright 2010-2011, D. E. Shaw Reseach.
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright
  notice, this list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions, and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

* Neither the name of D. E. Shaw Research nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/***************************************************************/
/*          FIN LICENCES RNGs                                  */
/***************************************************************/



/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "device.h"
#ifdef OBJ3D
#include "geometry.h"
#include "shapes.h"
#endif
#include "transform.h"

#include <math.h>

#include <hip/hip_vector_types.h>
#include <stdio.h>
/**********************************************************
*	> Kernel
***********************************************************/


extern "C" {
	__global__ void launchKernel(
							 struct Spectrum *spectrum, float *X0,
							 struct Phase *faer, struct Phase *foce,
							 unsigned long long *errorcount, int *nThreadsActive, void *tabPhotons, void *tabDist, void *tabHist, 
							 unsigned long long *Counter,
							 unsigned long long *NPhotonsIn,
							 unsigned long long *NPhotonsOut,
							 float *tabthv, float *tabphi, struct Sensor *tab_sensor,
							 struct Profile *prof_atm,
							 struct Profile *prof_oc,
							 long long *wl_proba_icdf,
							 void *rng_state
							 #ifdef OBJ3D
							 , void *tabObjInfo,
							 struct IObjets *myObjets,
							 unsigned long long *nbPhCat,
							 double *wPhCat
							 #endif
							 ) {

    // current thread index
	int idx = blockIdx.x *blockDim.x + threadIdx.x;
	// Old thred index :
	// int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
	// int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
	int loc_prev;
	int count_level;
	int this_thread_active = 1;
	unsigned long long iloop = 0;
    float dth=0.F;

    struct RNG_State rngstate;
    #ifdef PHILOX
	// philox_data:
	// index 0: seed (config)
	// index 1 to last: status

	// Paramètres de la fonction random en mémoire locale
	//la clef se defini par l'identifiant global (unique) du thread...
	//...et par la clef utilisateur ou clef par defaut
	//ce systeme garanti l'existence de 2^32 generateurs differents par run et...
	//...la possiblite de reemployer les memes sequences a partir de la meme clef utilisateur
	//(plus d'infos dans "communs.h")
	philox4x32_key_t configThr = {{idx, ((unsigned int *)rng_state)[0]}};
	//le compteur se defini par trois mots choisis au hasard (il parait)...
	//...et un compteur definissant le nombre d'appel au generateur
	//ce systeme garanti l'existence de 2^32 nombres distincts pouvant etre genere par thread,...
	//...et ce sur l'ensemble du process (et non pas 2^32 par thread par appel au kernel)
	//(plus d'infos dans "communs.h")
	philox4x32_ctr_t etatThr = {{((unsigned int *)rng_state)[idx+1], 0xf00dcafe, 0xdeadbeef, 0xbeeff00d}};

    rngstate.configThr = configThr;
    rngstate.etatThr = etatThr;

    #endif
    #ifdef CURAND_PHILOX
    // copy RNG state in local memory
    rngstate.state = ((hiprandStatePhilox4_32_10_t *)rng_state)[idx];
    #endif

	
	// Création de variable propres à chaque thread
	unsigned long long nbPhotonsThr = 0; 	// Nombre de photons traités par le thread
	
	Photon ph, ph_le; 		// On associe une structure de photon au thread

	//bool geoIntersect = false;  // S'il y a intersection avec une géométrie = true
	#ifdef OBJ3D
	IGeo geoStruc;
	bigCount = 1;   // Initialisation de la variable globale bigCount (voir geometry.h)
    #endif
	
	atomicAdd(nThreadsActive, 1);

	// double3 dp1 = make_double3(12.54, 0., 47.2589);
	// double3 dp2 = make_double3(0.0025875, 547., 5.1111111);
	// if (idx == 0) printf("double3 marche!! dp1=(%lf, %f, %f), dp2=(%f, %f, %f)", dp1.x, dp1.y, dp1.z, dp2.x, dp2.y, dp2.z);

    //
    // main loop
    //
	while (*nThreadsActive > 0) {
		iloop += 1;
		

		/* ************************************************************************************************** */
		/* si on simule des objs on utilise cette astuce pour lancer exactement le nombre souhaité de photons */
		#ifdef OBJ3D
		// Si le nombre de ph lancés NBLOOPd > 256000 et que le compteur devient > (NBLOOPd-256000) alors
		// on commence à diminuer le nombre de threads actif... ici à 999+1 = 1000 threads actif
		if ((NBLOOPd > 50000) && idx > 999 && this_thread_active && Counter[0] >= (NBLOOPd-50000) && *nThreadsActive > 1000)
		{
			this_thread_active = 0;
            atomicAdd(nThreadsActive, -1);
		}
		else if ((NBLOOPd > 5000) && idx > 99 && this_thread_active && Counter[0] >= (NBLOOPd-5000) && *nThreadsActive > 100)
		{
			this_thread_active = 0;
            atomicAdd(nThreadsActive, -1);
		}
		else if((NBLOOPd > 500) && idx > 9 && this_thread_active &&Counter[0] >= (NBLOOPd-500) && *nThreadsActive > 10)
		{
			this_thread_active = 0;
            atomicAdd(nThreadsActive, -1);
		}
		else if((NBLOOPd > 50) && idx > 0 && this_thread_active &&Counter[0] >= (NBLOOPd-50) && *nThreadsActive > 1)
		{
			this_thread_active = 0;
            atomicAdd(nThreadsActive, -1);
		}
		#endif
        /* ************************************************************************************************** */
		
		// Avant ">" et maintenant ">=" car si Counter = au nombre de ph lancés NBLOOPd, il faut s'arrêter !
        if (((Counter[0] >= NBLOOPd)
			 && this_thread_active
			 && (ph.loc == NONE))
			|| (iloop > MAX_LOOP)  // avoid infinite loop
                                       // when photons don't end
			) {
            this_thread_active = 0;
            atomicAdd(nThreadsActive, -1);
        }
		
        // Si le photon est à NONE on l'initialise et on le met à la localisation correspondant à la simulaiton en cours
        if((ph.loc == NONE) && this_thread_active){

            initPhoton(&ph, prof_atm, prof_oc, tab_sensor, spectrum, X0, NPhotonsIn, wl_proba_icdf, tabthv, tabphi, &rngstate
					   #ifdef OBJ3D
					   , myObjets
					   #endif
				);
			
            iloop = 1;
            #ifdef DEBUG_PHOTON
			if (idx==0) {printf("\n");}
            display("INIT", &ph);
            #endif

        }

        //
		// Deplacement
		//
		// -> Si OCEAN ou ATMOS
		loc_prev = ph.loc;
		if( (ph.loc == ATMOS) || (ph.loc == OCEAN)){
        #ifdef SPHERIQUE
        if (ph.loc == ATMOS)
           move_sp(&ph, prof_atm, 0, 0 , &rngstate);
        else 
        #endif

        #ifdef ALT_PP
        move_pp2(&ph, prof_atm, prof_oc, 0, 0 , &rngstate);
        #else
        move_pp(&ph, prof_atm, prof_oc, &rngstate
				#ifdef OBJ3D
				, &geoStruc, myObjets, tabObjInfo
				#endif
			);
        #endif

        #ifdef DEBUG_PHOTON
		display("MOVE", &ph);
        #endif
		}

        //
        // count after move:
        // count the photons in space and reaching surface from above or below
        //
		count_level = -1;
		if (ph.loc == SPACE) {
            count_level = UPTOA;
			
            // increment the photon counter
            // (for this thread)
            nbPhotonsThr++;
            // reset the photon location (always)
            ph.loc = NONE;
            #ifdef DEBUG_PHOTON
            display("SPACE", &ph);
            #endif

        } else if ((ph.loc == SURF0P) && (loc_prev != SURF0P)) {
            count_level = DOWN0P;
        } else if ((ph.loc == SURF0M) && (loc_prev != SURF0M)) {
            count_level = UP0M; 
        } else if (ph.loc == SEAFLOOR) {
            count_level = DOWNB;
        }

		// count the photons
        
		/* Cone Sampling */
		if (LEd ==0) countPhoton(&ph, prof_atm, prof_oc, tabthv, tabphi, count_level,
            errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

		__syncthreads();
		
		//
		// Scatter
		//
		// -> dans ATMOS ou OCEAN
		if( ((ph.loc == ATMOS) || (ph.loc == OCEAN)) ) {

			/* Choose the scatterer */
            choose_scatterer(&ph, prof_atm, prof_oc,  spectrum, 
							 &rngstate); 
            #ifdef DEBUG_PHOTON
            display("CHOOSE SCAT", &ph);
            #endif

            /* Scattering Local Estimate */
            if (LEd == 1) {
			    int NK, up_level, down_level, count_level_le;
			    int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
			    int iph0 = idx%NBPHId;
			    if (ph.loc == ATMOS) {
			        NK=2;
			        up_level = UPTOA;
			        down_level = DOWN0P;
		        }
			    if (ph.loc == OCEAN) {
			        NK=2;
                    up_level = UP0M;
			        down_level = DOWNB;
		        }

                // Loop on levels for counting (for upward and downward)
			    for(int k=0; k<NK; k++){
			        if (k==0) count_level_le = up_level;
			        else count_level_le = down_level;

                    // Double Loop on directions
                    for (int ith=0; ith<NBTHETAd; ith++){
                        for (int iph=0; iph<NBPHId; iph++){
                            // Copy of the propagation photon to to the virtual, local estimate photon
                            copyPhoton(&ph, &ph_le);
                            // Computation of the index of the direction
                            ph_le.ith = (ith + ith0)%NBTHETAd;
                            if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                            else ph_le.iph =  ph_le.ith;

                            /*#ifdef SPHERIQUE
                            // in case of atmospheric refraction determine the outgoing direction
                            if (REFRACd && ph_le.loc==ATMOS) {
                                float phi = tabphi[ph_le.iph];
                                float thv = tabthv[ph_le.ith];
                                float3 v;
                                v.x = cosf(phi) * sinf(thv);
                                v.y = sinf(phi) * sinf(thv);
                                v.z = cosf(thv);
                                ph_le.v = v;
                                move_sp(&ph_le, prof_atm, 1, UPTOA , &rngstate);
                                dth = -acosf(dot(ph_le.v,v));
                                copyPhoton(&ph, &ph_le);
                                ph_le.ith = (ith + ith0)%NBTHETAd;
                                if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                                else ph_le.iph =  ph_le.ith;
                            }
                            else dth=0.F;
                            #endif*/

                            // Scatter the virtual photon, using le=1, and count_level for the scattering angle computation
                            scatter(&ph_le, prof_atm, prof_oc, faer, foce,
                                    1, dth, tabthv, tabphi,
                                    count_level_le, &rngstate);

                            #ifdef DEBUG_PHOTON
                            if (k==0) display("SCATTER LE UP", &ph_le);
                            else display("SCATTER LE DOWN", &ph_le);
                            #endif

                            #ifdef SPHERIQUE
                            if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, count_level_le , &rngstate);
                            #ifdef DEBUG_PHOTON
                            display("MOVE LE", &ph_le);
                            #endif
                            #endif
                            #ifdef ALT_PP
                            if ((ph_le.loc==ATMOS) || (ph_le.loc==OCEAN)) move_pp2(&ph_le, prof_atm, prof_oc, 1, count_level_le , &rngstate);
                            #endif

                            // Finally count the virtual photon
                            countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, count_level_le,
                                    errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

                        } //directions
                    } // directions
                } // levels
            } // LE

            /* Scattering Propagation , using le=0 and propagation photon */
            scatter(&ph, prof_atm, prof_oc, faer, foce,
                    0, 0.F, tabthv, tabphi, 0,
                    &rngstate);
            #ifdef DEBUG_PHOTON
            display("SCATTER", &ph);
            #endif

		} // photon in ATMOS or OCEAN
		__syncthreads();


        //
		// Reflection
        //
        // -> in SURFACE
        loc_prev = ph.loc;
        if ((ph.loc == SURF0M) || (ph.loc == SURF0P)){
           // Eventually evaluate Downward 0+ and Upward 0- radiance

           // if not environment effects 
           if( ENVd==0 ) { 

           // if not a Lambertian surface
			if( DIOPTREd!=3 ) {
                /* Surface Local Estimate (not evaluated if atmosphere only simulation)*/
                if (LEd == 1 && SIMd != ATM_ONLY) {
                ///* TEST Double LE */
                  int NK, count_level_le;
                  if (NOCEd==0) NK=1;
                  else NK=2;
                  int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
                  int iph0 = idx%NBPHId;
                  for(int k=0; k<NK; k++){
                    if (k==0) count_level_le = UP0P;
                    else count_level_le = DOWN0M;

                    for (int ith=0; ith<NBTHETAd; ith++){
                      for (int iph=0; iph<NBPHId; iph++){
                        copyPhoton(&ph, &ph_le);
                        ph_le.ith = (ith + ith0)%NBTHETAd;
                        if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                        else ph_le.iph =  ph_le.ith;

                        // Reflect or Tramsit the virtual photon, using le=1, and count_level for the scattering angle computation
                        if (BRDFd != 0)
                            surfaceBRDF(&ph_le, 1, tabthv, tabphi,
                                      count_level_le, &rngstate);
                        else 
                            surfaceAgitee(&ph_le, 1, tabthv, tabphi,
                                      count_level_le, &rngstate);

                        #ifdef DEBUG_PHOTON
                        if (k==0) display("SURFACE LE UP", &ph_le);
                        else display("SURFACE LE DOWN", &ph_le);
                        #endif

                        // Count the photon up to the counting levels (at the surface UP0P or DOW0M)
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, count_level_le, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

                        // Only for upward photons count also them up to TOA
                        if (k==0) { 
                            #ifdef SPHERIQUE
                            if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
                            #endif
                            #ifdef ALT_PP
                            if (ph_le.loc==ATMOS) move_pp2(&ph_le, prof_atm, prof_oc, 1, UPTOA, &rngstate);
                            #endif
                            // Final counting at the TOA
                            countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA , errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        }
                        // Only for downward photons count also them up to Bottom 
                        if (k==1) { 
                            // Final counting at the B 
                            #ifdef ALT_PP
                            if (ph_le.loc==OCEAN) move_pp2(&ph_le, prof_atm, prof_oc, 1, DOWNB, &rngstate);
                            #endif
                            countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, DOWNB , errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        }
                      }//direction
                    }//direction
                  }// counting levels
                } //LE

                // Propagation of photon using le=0
                if (BRDFd != 0)
				    surfaceBRDF(&ph, 0, tabthv, tabphi,
                              count_level, &rngstate);
                else
				    surfaceAgitee(&ph, 0, tabthv, tabphi,
                              count_level, &rngstate);
            } // Not lambertian


            // Lambertian case
			else { 
                if (LEd == 1 && SIMd != ATM_ONLY) {
                  int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
                  int iph0 = idx%NBPHId;
                  for (int ith=0; ith<NBTHETAd; ith++){
                    for (int iph=0; iph<NBPHId; iph++){
                        copyPhoton(&ph, &ph_le);
                        ph_le.ith = (ith + ith0)%NBTHETAd;
                        if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                        else ph_le.iph =  ph_le.ith;
				        surfaceLambert(&ph_le, 1, tabthv, tabphi, spectrum, &rngstate);
                        // Only two levels for counting by definition
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UP0P,  errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        #ifdef SPHERIQUE
                        if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
                        #endif
                        #ifdef ALT_PP
                        if (ph_le.loc==ATMOS) move_pp2(&ph_le, prof_atm, prof_oc, 1, UPTOA , &rngstate);
                        #endif
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                    }//direction
                  }//direction
                } //LE
				//Propagation of Lambertian reflection with le=0
				surfaceLambert(&ph, 0, tabthv, tabphi, spectrum, &rngstate);
            } // Lambertian (DIOPTRE=!3)
           } // ENV=0

           // Environment effects, no LE computed yet
           else {
                float dis=0;
                dis = sqrtf((ph.pos.x-X0d)*(ph.pos.x-X0d) +(ph.pos.y-Y0d)*(ph.pos.y-Y0d));
                if( dis > ENV_SIZEd) {
                 if (LEd == 1 && SIMd != ATM_ONLY) {
                  int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
                  int iph0 = idx%NBPHId;
                  for (int ith=0; ith<NBTHETAd; ith++){
                    for (int iph=0; iph<NBPHId; iph++){
                        copyPhoton(&ph, &ph_le);
                        ph_le.ith = (ith + ith0)%NBTHETAd;
                        if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                        else ph_le.iph =  ph_le.ith;
				        surfaceLambert(&ph_le, 1, tabthv, tabphi, spectrum, &rngstate);
                        // Only two levels for counting by definition
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UP0P,  errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        #ifdef SPHERIQUE
                        if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
                        #endif
                        #ifdef ALT_PP
                        if (ph_le.loc==ATMOS) move_pp2(&ph_le, prof_atm, prof_oc, 1, UPTOA , &rngstate);
                        #endif
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                    }//direction
                  }//direction
                 } //LE
                 //Propagation of Lambertian reflection with le=0
                    surfaceLambert(&ph, 0, tabthv, tabphi, spectrum, &rngstate);
                }// dis
                else {
                 if (LEd == 1 && SIMd != ATM_ONLY) {
                 ///* TEST Double LE */
                  int NK, count_level_le;
                  if (NOCEd==0) NK=1;
                  else NK=2;
                  int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
                  int iph0 = idx%NBPHId;
                  for(int k=0; k<NK; k++){
                    if (k==0) count_level_le = UP0P;
                    else count_level_le = DOWN0M;

                    for (int ith=0; ith<NBTHETAd; ith++){
                      for (int iph=0; iph<NBPHId; iph++){
                        copyPhoton(&ph, &ph_le);
                        ph_le.ith = (ith + ith0)%NBTHETAd;
                        if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                        else ph_le.iph =  ph_le.ith;

                        // Reflect or Tramsit the virtual photon, using le=1, and count_level for the scattering angle computation
                        if (BRDFd != 0)
                            surfaceBRDF(&ph_le, 1, tabthv, tabphi,
                                      count_level_le, &rngstate);
                        else
                            surfaceAgitee(&ph_le, 1, tabthv, tabphi,
                                      count_level_le, &rngstate);

                        #ifdef DEBUG_PHOTON
                        if (k==0) display("SURFACE LE UP", &ph_le);
                        else display("SURFACE LE DOWN", &ph_le);
                        #endif

                        // Count the photon up to the counting levels (at the surface UP0P or DOW0M)
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, count_level_le, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

                        // Only for upward photons count also them up to TOA

                        #ifdef DEBUG_PHOTON
                        if (k==0) display("SURFACE LE UP", &ph_le);
                        else display("SURFACE LE DOWN", &ph_le);
                        #endif

                        // Count the photon up to the counting levels (at the surface UP0P or DOW0M)
                        countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, count_level_le, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

                        // Only for upward photons count also them up to TOA
                        if (k==0) { 
                            #ifdef SPHERIQUE
                            if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
                            #endif
                            #ifdef ALT_PP
                            if (ph_le.loc==ATMOS) move_pp2(&ph_le, prof_atm, prof_oc, 1, UPTOA , &rngstate);
                            #endif
                            // Final counting at the TOA
                            countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA , errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        }
                        // Only for downward photons count also them up to Bottom 
                        if (k==1) { 
                            // Final counting at the B 
                             #ifdef ALT_PP                          
                             if (ph_le.loc==OCEAN) move_pp2(&ph_le, prof_atm, prof_oc, 1, DOWNB , &rngstate); 
                             #endif 
                            countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, DOWNB , errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                        }
                      }//direction
                    }//direction
                  }// counting levels
                 } //LE
                // Propagation of photon using le=0
                    if (BRDFd != 0)
                        surfaceBRDF(&ph, 0, tabthv, tabphi, count_level, &rngstate);
                    else
                        surfaceAgitee(&ph, 0, tabthv, tabphi, count_level, &rngstate);
                } //dis
           } // ENV=1

           #ifdef DEBUG_PHOTON
           display("SURFACE", &ph);
           #endif
		}

		__syncthreads();

        //
		// Reflection
        //
        // -> in SEAFLOOR
        if(ph.loc == SEAFLOOR){
           if (LEd == 1 && SIMd != ATM_ONLY) {
              int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
              int iph0 = idx%NBPHId;
              for (int ith=0; ith<NBTHETAd; ith++){
                for (int iph=0; iph<NBPHId; iph++){
                    copyPhoton(&ph, &ph_le);
                    ph_le.ith = (ith + ith0)%NBTHETAd;
                    if (!ZIPd) ph_le.iph = (iph + iph0)%NBPHId;
                    else ph_le.iph =  ph_le.ith;
				    surfaceLambert(&ph_le, 1, tabthv, tabphi, spectrum, &rngstate);
                    //  contribution to UP0M level
                    #ifdef ALT_PP                          
                    if (ph_le.loc==OCEAN) move_pp2(&ph_le, prof_atm, prof_oc, 1, UP0M, &rngstate); 
                    #endif
                    countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UP0M,   errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                }
              }
            } //LE

			surfaceLambert(&ph, 0, tabthv, tabphi, spectrum, &rngstate);
            #ifdef DEBUG_PHOTON
            display("SEAFLOOR", &ph);
            #endif
         }
        __syncthreads();

		#ifdef OBJ3D
		//
		// Reflection
        //
        // -> in OBJSURF
        if(ph.loc == OBJSURF)
		{
			if (geoStruc.type == 2) // this is a receiver
			{ countPhotonObj3D(&ph, tabObjInfo, &geoStruc, nbPhCat, wPhCat);}

			if (geoStruc.material == 1) // Lambertian Mirror
			{
				if (LEd == 1)
				{				
					int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
					int iph0 = idx%NBPHId;
					for (int ith=0; ith<NBTHETAd; ith++){
						for (int iph=0; iph<NBPHId; iph++){
							copyPhoton(&ph, &ph_le);
							ph_le.iph = (iph + iph0)%NBPHId;
							ph_le.ith = (ith + ith0)%NBTHETAd;
							surfaceLambertienne3D(&ph_le, 1, tabthv, tabphi, spectrum,
												  &rngstate, &geoStruc);			
							// Only two levels for counting by definition
							countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UP0P,
										errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                            #ifdef SPHERIQUE
							// for spherical case attenuation if performed usin move_sp
							if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
						    #endif
							countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA,
										errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
						}//direction
					}//direction
				} //LE
				surfaceLambertienne3D(&ph, 0, tabthv, tabphi, spectrum,
									  &rngstate, &geoStruc);
			} // END Lambertian Mirror
			else if (geoStruc.material == 2) // Matte
			{
				ph.loc = ABSORBED;
			} // End Matte
			else if (geoStruc.material == 3) // Mirror
			{	
				if (LEd == 1)
				{				
					int ith0 = idx%NBTHETAd; //index shifts in LE geometry loop
					int iph0 = idx%NBPHId;
					for (int ith=0; ith<NBTHETAd; ith++){
						for (int iph=0; iph<NBPHId; iph++){
							copyPhoton(&ph, &ph_le);
							ph_le.iph = (iph + iph0)%NBPHId;
							ph_le.ith = (ith + ith0)%NBTHETAd;
							surfaceRugueuse3D(&ph_le, &geoStruc, &rngstate);
							// Only two levels for counting by definition
							countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UP0P,
										errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
                            #ifdef SPHERIQUE
							// for spherical case attenuation if performed usin move_sp
							if (ph_le.loc==ATMOS) move_sp(&ph_le, prof_atm, 1, UPTOA, &rngstate);
						    #endif
							countPhoton(&ph_le, prof_atm, prof_oc, tabthv, tabphi, UPTOA,
										errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);
						}//direction
					}//direction
				} //LE
				surfaceRugueuse3D(&ph, &geoStruc, &rngstate);
			} // End Mirror
			else {ph.loc = REMOVED;} // unknow material
			
			#ifdef DEBUG_PHOTON
			display("OBJSURF", &ph);
            #endif
		}
        __syncthreads();
		#endif
		
        //
        // count after surface:
        // count the photons leaving the surface towards the ocean or atmosphere
        //
        count_level = -1;
        if ((loc_prev == SURF0M) || (loc_prev == SURF0P)) {
            if ((ph.loc == ATMOS) || (ph.loc == SPACE)
				#ifdef OBJ3D
				|| (ph.loc == OBJSURF)
				#endif
				)
				count_level = UP0P;
            if (ph.loc == OCEAN) count_level = DOWN0M;
        }
		
        /* Cone Sampling */
        if (LEd == 0) countPhoton(&ph, prof_atm, prof_oc, tabthv, tabphi, count_level, errorcount, tabPhotons, tabDist, tabHist, NPhotonsOut);

		if(ph.loc == ABSORBED){
			ph.loc = NONE;
			nbPhotonsThr++;
		}
		if(ph.loc == REMOVED){
			ph.loc = NONE;
		}
		__syncthreads();

		

        // from time to time, transfer the per-thread photon counter to the
        // global counter
        // if (nbPhotonsThr % 100 == 0) {
        //     atomicAdd(Counter, nbPhotonsThr);
        //     nbPhotonsThr = 0;
        // }
		//nbPhotonsThr = 1;
		// if (idx == 0) printf("nombre de photons par thread : %llu et counter : %d\n", nbPhotonsThr, Counter[0]);

		atomicAdd(Counter, nbPhotonsThr);
		nbPhotonsThr = 0;
		__syncthreads();
		
	// }
	}

	// Après la boucle on rassemble les nombres de photons traités par chaque thread
	atomicAdd(Counter, nbPhotonsThr);
	
    if (ph.loc != NONE) {
        atomicAdd(errorcount+ERROR_MAX_LOOP, 1);
    }

    #ifdef PHILOX
	// Sauvegarde de l'état du random pour que les nombres
    // ne soient pas identiques à chaque appel du kernel
    ((unsigned int *)rng_state)[idx+1] = rngstate.etatThr[0];
    #endif
    #ifdef CURAND_PHILOX
    ((hiprandStatePhilox4_32_10_t *)rng_state)[idx] = rngstate.state;
    #endif

} /* launchKernel*/
} /* extern C*/




/**********************************************************
*	> Kernel 2 : Wavelength dependent absorption summation
***********************************************************/

extern "C" {
	__global__ void launchKernel2( void *tabPhotons, float *tabHist, struct Profile *prof_atm, 
            unsigned long long *NPhotonsOut
							 ) {

    // current thread index
	int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
    int DL  = (NATMd-1)/(NLOWd-1);
    // Wavelength index (high resolution)
    int il = idx%1000;
    // Wavelength index (low resolution)
    int ik = il/DL;

    int KK2 = NBTHETAd*NBPHId*NSENSORd*(NATMd+4+NLOWd);
    //int KK2 = NBTHETAd*NBPHId*(NATMd+4+NLOWd);
    int KKK2= KK2 * MAX_HIST;
    unsigned long long LL,LLL;
    int ith=0;
    int iphi=0;
    int is=0;
    unsigned long long counter2=0, NPH;
    float wabs,wsca,a,b;
    int count_level = UPTOA;
    unsigned long long II = NBTHETAd*NBPHId*NLAMd*NSENSORd;
    //unsigned long long II = NBTHETAd*NBPHId*NLAMd;
    unsigned long long JJ = is*NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi;
    //unsigned long long JJ = il*NBTHETAd*NBPHId + ith*NBPHId + iphi;
    NPH = NPhotonsOut[(((count_level*NSENSORd + is)*NLAMd + 0)*NBTHETAd + ith)*NBPHId + iphi];
    //NPH = NPhotonsOut[((count_level*NLAMd + 0)*NBTHETAd + ith)*NBPHId + iphi];
    // Start Loop on photons;
    while (counter2< NPH) {
      LL =  counter2*NSENSORd*NBTHETAd*NBPHId*(NATMd+4+NLOWd) +  count_level*KKK2;
      //LL =  counter2*NBTHETAd*NBPHId*(NATMd+4+NLOWd) +  count_level*KKK2;
      // Get scattering corrections factors
      a=tabHist[LL +  (ik+  NATMd+4)*NBTHETAd*NBPHId + ith*NBPHId + iphi];
      //if (a==0) break;
      b=tabHist[LL +  (ik+1+NATMd+4)*NBTHETAd*NBPHId + ith*NBPHId + iphi];
      if (il != NLAMd-1) wsca = __fdividef((il-ik*DL)*1.0f,DL*1.0f) * (b - a) + a;
      else wsca = tabHist[LL +(NLOWd-1+NATMd+4)*NBTHETAd*NBPHId + ith*NBPHId + iphi];

      // Computation of the absorption along photon history with cumulative distances CD in layers
      // tabHist from 0 to NATMd-1 stores CD
      wabs = 0.F;
      for (int n=0; n<(NATMd); n++){
        LLL = LL +  n*NBTHETAd*NBPHId + ith*NBPHId + iphi;
        wabs += abs(__fdividef(prof_atm[n+1   + il*(NATMd+1)].OD_abs -
                               prof_atm[n + il*(NATMd+1)].OD_abs,
                               prof_atm[n+1].z  - prof_atm[n].z) ) * tabHist[LLL];
      }

      // Get I,Q,U,V;
      float4 s = make_float4(tabHist[LL +  (NATMd+0)*NBTHETAd*NBPHId + ith*NBPHId + iphi],
                             tabHist[LL +  (NATMd+1)*NBTHETAd*NBPHId + ith*NBPHId + iphi],
                             tabHist[LL +  (NATMd+2)*NBTHETAd*NBPHId + ith*NBPHId + iphi],
                             tabHist[LL +  (NATMd+3)*NBTHETAd*NBPHId + ith*NBPHId + iphi]);

      #ifdef DOUBLE 
      double *tabCount = (double*)tabPhotons + count_level*NPSTKd*NBTHETAd*NBPHId*NLAMd*NSENSORd;
      //double *tabCount = (double*)tabPhotons + count_level*NPSTKd*NBTHETAd*NBPHId*NLAMd;
      double4 ds = make_double4(s.x, s.y, s.z, s.w);
      double dwsca=(double)wsca;
      double dwabs=(double)wabs;

	  #if __CUDA_ARCH__ >= 600
      atomicAdd(tabCount+(0*II+JJ), dwsca * dwabs * ds.x);
      atomicAdd(tabCount+(1*II+JJ), dwsca * dwabs * ds.y);
      atomicAdd(tabCount+(2*II+JJ), dwsca * dwabs * ds.z);
      atomicAdd(tabCount+(3*II+JJ), dwsca * dwabs * ds.w);
	  #else
	  DatomicAdd(tabCount+(0*II+JJ), dwsca * dwabs * ds.x);
      DatomicAdd(tabCount+(1*II+JJ), dwsca * dwabs * ds.y);
      DatomicAdd(tabCount+(2*II+JJ), dwsca * dwabs * ds.z);
      DatomicAdd(tabCount+(3*II+JJ), dwsca * dwabs * ds.w);
	  #endif

      #else
      float *tabCount = (float*)tabPhotons + count_level*NPSTKd*NBTHETAd*NBPHId*NLAMd*NSENSORd;
      //float *tabCount = (float*)tabPhotons + count_level*NPSTKd*NBTHETAd*NBPHId*NLAMd;
      atomicAdd(tabCount+(0*II+JJ), wsca * wabs * s.x);
      atomicAdd(tabCount+(1*II+JJ), wsca * wabs * s.y);
      atomicAdd(tabCount+(2*II+JJ), wsca * wabs * s.z);
      atomicAdd(tabCount+(3*II+JJ), wsca * wabs * s.w);
      #endif    
      counter2++;
    }

} /* launchKernel2*/
} /* extern C*/




/**********************************************************
*	> Phyisical Processes
***********************************************************/
/* initPhoton
*/

__device__ void initPhoton(Photon* ph, struct Profile *prof_atm, struct Profile *prof_oc, struct Sensor *tab_sensor,
                           struct Spectrum *spectrum, float *X0, unsigned long long *NPhotonsIn,
                           long long *wl_proba_icdf, float* tabthv, float* tabphi,
                           struct RNG_State *rngstate
						   #ifdef OBJ3D
						   , struct IObjets *myObjets
						   #endif
	) {
    float dz, dz_i, delta_i, epsilon;
    float cTh, sTh, phi;
    int ilayer;
	
	int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
	
    #ifdef OBJ3D
	ph->direct = 0;
	ph->H = 0;
	ph->E = 0;
	ph->S = 0;
    #endif
	
    ph->nint = 0;
	ph->weight = WEIGHTINIT;

	// Stokes parameters initialization according to natural sunlight
	ph->stokes.x = 0.5F;
	ph->stokes.y = 0.5F;
	ph->stokes.z = 0.F;
	ph->stokes.w = 0.F;

	ph->scatterer = UNDEF;
	
    // Sensor index initialization
    ph->is = __float2uint_rz(RAND * NSENSORd);

    // Wavelength initialization
    if (NWLPROBA == 0) { 
        #ifdef ALIS
        // NJACd=0 : no jacobian -> one unperturbed profile
        ph->ilam = __float2uint_rz(RAND * NLAMd/(NJACd+1));
        #else
        ph->ilam = __float2uint_rz(RAND * NLAMd);
        #endif
    } else {
        ph->ilam = wl_proba_icdf[__float2uint_rz(RAND * NWLPROBA)];
    }
	ph->wavel = spectrum[ph->ilam].lambda;

    // Position and optical thicknesses initializations

    ph->pos = make_float3(tab_sensor[ph->is].POSX,
                          tab_sensor[ph->is].POSY,
                          tab_sensor[ph->is].POSZ);
	
	#ifdef OBJ3D
	Transform TRotZ; char mPP[]="Point";
	TRotZ = TRotZ.RotateZ(tab_sensor[ph->is].PHDEG-180.);
	ph->pos = TRotZ(ph->pos, mPP);
	#endif
	
    ph->loc = tab_sensor[ph->is].LOC;

    #ifdef SPHERIQUE
	ph->radius = length(ph->pos);
    #endif

    if(ph->loc == SURF0P){
        ph->layer   = NATMd;
        ph->tau     = 0.F;
        ph->tau_abs = 0.F;
        epsilon     = 0.F;
        ph->pos.z   = 0.F;
        #ifdef SPHERIQUE
        ph->pos.z = RTER;
        #endif
        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        for (int k=0; k<NLOWd; k++) {
            ph->tau_sca[k] = 0.F;
        }
        #endif
    }

    if(ph->loc == SURF0M){
        ph->layer   = 0;
        ph->tau     = 0.F;
        ph->tau_abs = 0.F;
        epsilon     = 0.F;
        ph->pos.z   = 0.F;
        #ifdef SPHERIQUE
        ph->pos.z = RTER;
        #endif
        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        for (int k=0; k<NLOWd; k++) {
            ph->tau_sca[k] = 0.F; ;
        }
        #endif
    }

    if(ph->loc == SEAFLOOR){
        ph->layer   = NOCEd;
        ph->tau     = get_OD(BEERd, prof_oc[NOCEd +ph->ilam*(NOCEd+1)]);
        ph->tau_abs = prof_oc[NOCEd +ph->ilam*(NOCEd+1)].OD_abs;
        epsilon     = 0.F;
        ph->pos.z   = prof_oc[NOCEd].z;
        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        int DL=(NLAMd-1)/(NLOWd-1);
        for (int k=0; k<NLOWd; k++) {
            ph->tau_sca[k] = get_OD(1,prof_oc[NOCEd + k*DL*(NOCEd+1)]) ;
        }
        #endif
    }

    if(ph->loc == OCEAN){
        ilayer = 1;
        while (( prof_oc[ilayer].z > tab_sensor[ph->is].POSZ) && (ilayer < NOCEd)) {
            ilayer++;
        }
        ph->layer = ilayer;
        dz_i    = fabs(prof_oc[ilayer].z - prof_oc[ilayer-1].z);
        dz      = fabs(tab_sensor[ph->is].POSZ - prof_oc[ilayer-1].z) ;
        epsilon = fabs(__fdividef(dz,dz_i));

        delta_i = fabs(get_OD(BEERd, prof_oc[ilayer+ph->ilam*(NOCEd+1)]) - get_OD(BEERd, prof_oc[ilayer-1+ph->ilam*(NOCEd+1)]));
        ph->tau = epsilon * delta_i + (get_OD(BEERd, prof_oc[0+ph->ilam*(NOCEd+1)])-
                                       get_OD(BEERd, prof_oc[ilayer-1+ph->ilam*(NOCEd+1)])); 

        delta_i = fabs(prof_oc[ilayer+ph->ilam*(NOCEd+1)].OD_abs - prof_oc[ilayer-1+ph->ilam*(NOCEd+1)].OD_abs);
        ph->tau_abs = epsilon * delta_i + (prof_oc[0+ph->ilam*(NOCEd+1)].OD_abs -
                                           prof_oc[ilayer-1+ph->ilam*(NOCEd+1)].OD_abs); 

        if(idx==0) printf("%i %f %f %f %f\n",ilayer, prof_oc[ilayer].z, ph->tau, delta_i, epsilon);
        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        int DL=(NLAMd-1)/(NLOWd-1);
        for (int k=0; k<NLOWd; k++) {
            delta_i = fabs(get_OD(BEERd, prof_oc[ilayer+k*DL*(NOCEd+1)]) - get_OD(BEERd, prof_oc[ilayer-1+k*DL*(NOCEd+1)]));
            ph->tau_sca[k] = epsilon * delta_i + (get_OD(1,prof_oc[NOCEd + k*DL*(NOCEd+1)])-
                                                  get_OD(1,prof_oc[ilayer + k*DL*(NOCEd+1)]));
        }
        #endif
    }

    if((ph->loc == ATMOS)
	   #ifdef OBJ3D
	   || (ph->loc == OBJSURF)
	   #endif
		){
        ilayer = 1;
        float POSZd_alt; 
        #ifdef SPHERIQUE
        POSZd_alt = tab_sensor[ph->is].POSZ - RTER;
        #else
        POSZd_alt = tab_sensor[ph->is].POSZ;
        #endif
        while (( prof_atm[ilayer].z > POSZd_alt) && (ilayer < NATMd)) {
            ilayer++;
        }
        ph->layer = ilayer;
        dz_i    = fabs(prof_atm[ilayer-1].z - prof_atm[ilayer].z);
        dz      = fabs(POSZd_alt - prof_atm[ilayer].z) ;
        epsilon = fabs(__fdividef(dz,dz_i));

        delta_i = fabs(get_OD(BEERd, prof_atm[ilayer+ph->ilam*(NATMd+1)]) - get_OD(BEERd, prof_atm[ilayer-1+ph->ilam*(NATMd+1)]));
        ph->tau = epsilon * delta_i + (get_OD(BEERd, prof_atm[NATMd+ph->ilam*(NATMd+1)])-
                                       get_OD(BEERd, prof_atm[ilayer+ph->ilam*(NATMd+1)])); 
        delta_i = fabs(prof_atm[ilayer+ph->ilam*(NATMd+1)].OD_abs - prof_atm[ilayer-1+ph->ilam*(NATMd+1)].OD_abs);
        ph->tau_abs = epsilon * delta_i + (prof_atm[NATMd+ph->ilam*(NATMd+1)].OD_abs -
                                           prof_atm[ilayer+ph->ilam*(NATMd+1)].OD_abs); 
        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        int DL=(NLAMd-1)/(NLOWd-1);
        for (int k=0; k<NLOWd; k++) {
            delta_i = fabs(get_OD(BEERd, prof_atm[ilayer+k*DL*(NATMd+1)]) - get_OD(BEERd, prof_atm[ilayer-1+k*DL*(NATMd+1)]));
            ph->tau_sca[k] = epsilon * delta_i + (get_OD(1,prof_atm[NATMd + k*DL*(NATMd+1)])-
                                                  get_OD(1,prof_atm[ilayer + k*DL*(NATMd+1)]));
        }
        #endif
    }

    // Direction initialization
    if (tab_sensor[ph->is].TYPE != 0) {
        // Standard sampling of zenith angle for lambertian emittor (for planar flux)
	    cTh = sqrtf(1.F-RAND*sinf(tab_sensor[ph->is].FOV*DEUXPI/360.));
        // for spherical flux, adjust weight as a function of cTh
        float weight_irr = fabs(cTh);
        if (tab_sensor[ph->is].TYPE == 2 && weight_irr > 0.001f) ph->weight /= weight_irr;
        
	    phi = RAND*DEUXPI;
        sTh = sqrtf(1.F - cTh*cTh);
	    ph->v.x   = cosf(phi)*sTh;
	    ph->v.y   = sinf(phi)*sTh;
	    ph->v.z   = cTh;
	    // Initialization of the orthogonal vector to the propagation
        ph->u.x   = cosf(phi)*cTh;
        ph->u.y   = sinf(phi)*cTh;
	    ph->u.z   = -sTh;
    }
    else {
        // One fixed direction (for radiance)
        ph->v.x = 0.F;
        ph->v.y = 0.F;
        ph->v.z = 1.F;
	    // Initialization of the orthogonal vector to the propagation
        ph->u.x = 1.F;
        ph->u.y = 0.F;
	    ph->u.z = 0.F;
    }

    // Rotations of v and u in the detector direction THDEG,PHDEG

	float cPh, sPh;
	
    #ifdef OBJ3D
	float THRAD, PHRAD;
	#endif
	
    /*if (MId != 0) { // Multiple Init Direction
        if (MId <=0) { 
            // Random selection of Zenith init angle
            ph->ith = __float2uint_rz(RAND * NBTHETAd);
            // Random selection of Azimuth init angle
            if (MId == -1) ph->iph = __float2uint_rz(RAND * NBPHId);
            else {
                ph->iph = ph->ith; // Zip option iph follows ith;
            }
        }
        else {
            // Random selection of Zenith and (zipped) Azimuth init angles according to MId and NLAMd
            int NL = NLAMd/MId;
            int NT = NBTHETAd/MId;
            int offset = ph->ilam/NL;
            ph->ith = __float2uint_rz(RAND * NT) + offset*NT;
            ph->iph = ph->ith;
        }
		#ifdef OBJ3D
		THRAD = DEUXPI/2. - tabthv[ph->ith];
		PHRAD = DEUXPI/2. - tabphi[ph->iph];
        cTh = cosf(THRAD);
        cPh = cosf(PHRAD);
    }*/
    
    ph->ith = 0;
    ph->iph = 0;

	#ifdef OBJ3D
	THRAD = tab_sensor[ph->is].THDEG*DEUXPI/360.;
	PHRAD = tab_sensor[ph->is].PHDEG*DEUXPI/360.;
	cTh = cosf(THRAD);
	cPh = cosf(PHRAD);

	// Permet d'utiliser un angle azimuth variable
	sTh = sinf(THRAD);
	sPh = sinf(PHRAD);
    #else 
	cTh = cosf(tab_sensor[ph->is].THDEG*DEUXPI/360.);
	cPh = cosf(tab_sensor[ph->is].PHDEG*DEUXPI/360.);
	
	//Attention! Marche parceque l'angle zenith compris entre 0 et 180
	sTh = sqrtf(1.F - cTh*cTh);
	sPh = sqrtf(1.F - cPh*cPh);
	#endif

	float3x3 LTh = make_float3x3(
		cTh,  0.F,  sTh,                
		0.F,  1.F,  0.F,                 
	   -sTh,  0.F,  cTh                 
        );
	float3x3 LPh = make_float3x3(
		cPh, -sPh,  0.F,                
		sPh,  cPh,  0.F,                 
		0.F,  0.F,  1.F                 
        );
	ph->v = mul(LTh,ph->v);
	ph->v = mul(LPh,ph->v);
	ph->u = mul(LTh,ph->u);
	ph->u = mul(LPh,ph->u);


    // init specific ALIS quantities
    #ifdef ALIS
    #if !defined(ALT_PP) && !defined(SPHERIQUE)
    ph->nevt = 0;
    if (ph->loc == ATMOS) ph->layer_prev[ph->nevt]   = ph->layer;
    if (ph->loc == OCEAN || ph->loc == SURF0M) ph->layer_prev[ph->nevt]   = -ph->layer;

    ph->vz_prev[ph->nevt]      = ph->v.z;
    ph->epsilon_prev[ph->nevt] = epsilon;
    #else
    for (int k=0; k<(NATMd+1); k++) ph->cdist_atm[k]= 0.F;
    for (int k=0; k<(NOCEd+1); k++) ph->cdist_oc[k] = 0.F;
    #endif
    for (int k=0; k<NLOWd; k++) ph->weight_sca[k] = 1.0F;
    #endif

    // Init photon counters
    #ifdef ALIS
    for (int k=0; k<NLAMd; k++) atomicAdd(NPhotonsIn + NLAMd*ph->is + k, 1);
    #else
    atomicAdd(NPhotonsIn + NLAMd*ph->is + ph->ilam, 1);
    #endif

    #ifdef BACK
    ph->M = make_diag_float4x4 (1.F);
    //ph->Mf= make_diag_float4x4 (1.F);
    #endif

    #ifdef OBJ3D
	if (CFMODEd == 0) // Marche que pour le mode forward restreint
	{		
		/* ***************************************************************************************** */
		/* Créer la surface en TOA qui visera un reflecteur avec prise en compte des transformations */
		/* ***************************************************************************************** */
        #ifdef DOUBLE
		// Valeurs de l'angle zenital Theta et l'angle azimutal Phi (ici Phi pour l'instant imposé à 0)
		double sunTheta = 180-tab_sensor[ph->is].THDEG, sunPhi=0;

        // One fixed direction (for radiance) inverse of the initiale pos of the obj
		double3 vdouble = make_double3(0., 0., -1.);
		
	    // Initialization of the orthogonal vector to the propagation
		double3 udouble = make_double3(-1., 0., 0.);
		
		// Creation des tranformations (pour le calcul de la direction du photon)	
		Transformd ThetaPhid, TThetad, TPhid;
		TThetad = ThetaPhid.RotateY(sunTheta);
		TPhid = ThetaPhid.RotateZ(sunPhi);
		ThetaPhid = TThetad * TPhid; // Regroupement des transformations		

		// Application des transformation sur les vecteurs u et v en fonction de Theta et Phi
		char myV[]="Vector";
		vdouble = ThetaPhid(vdouble, myV);
		udouble = ThetaPhid(udouble, myV);

		ph->v = make_float3(float(vdouble.x), float(vdouble.y), float(vdouble.z));
		ph->u = make_float3(float(udouble.x), float(udouble.y), float(udouble.z));
		#else // IF NOT DOUBLE
		// Valeurs de l'angle zenital Theta et l'angle azimutal Phi (ici Phi pour l'instant imposé à 0)
		float sunTheta = 180-tab_sensor[ph->is].THDEG, sunPhi=0;

        // One fixed direction (for radiance)
		float3 vfloat = make_float3(0., 0., -1.);
		
	    // Initialization of the orthogonal vector to the propagation
		float3 ufloat = make_float3(-1., 0., 0.);
		
		//Creation des tranformations (pour le calcul de la direction du photon)
		Transform ThetaPhi, TTheta, TPhi;
		TTheta = ThetaPhi.RotateY(sunTheta);
		TPhi = ThetaPhi.RotateZ(sunPhi);
		ThetaPhi = TTheta * TPhi; // Regroupement des transformations		

		// Application des transformation sur les vecteurs u et v en fonction de Theta et Phi
		char myV[]="Vector";
		vfloat = ThetaPhi(vfloat, myV);
		ufloat = ThetaPhi(ufloat, myV);

		ph->v = vfloat;
		ph->u = ufloat;
        #endif // END IF DOUBLE OR FLOAT
	
		// Récupération de l'objet réflecteur
		IObjets objP;
		objP.type = 0;

		// Permet de choisir aléatoirement un miroir
		float randMirPrev = -1;
		float randMir;		
		for (int i=0; i<nObj; i++)
		{
			if (myObjets[i].type == 1) // if equal to reflector
			{
				randMir = RAND;
				if (randMir > randMirPrev)
				{
					randMirPrev = randMir;
					objP = myObjets[i];
				}
			}
		}
    
		if (objP.type == 1) // S'il y a un réflecteur 
		{
			#ifdef DOUBLE
			// // Création des transformations depuis les valeurs python du reflecteur		
			Transformd Tid;
			double posxd, posyd; 
			if (objP.mvRx != 0) { // si diff de 0 alors il y a une rot en x
				Transformd TmRXd;
				TmRXd = Tid.RotateX(objP.mvRx);
				Tid = Tid*TmRXd;}
			if (objP.mvRy != 0) { // si diff de 0 alors il y a une rot en y
				Transformd TmRYd;
				//TmRY = Ti.RotateY(objP.mvRy - (180-THDEGd)); // 180-THDEGd = Theta Sun in degree
				TmRYd = Tid.RotateY(objP.mvRy);
				Tid = Tid*TmRYd;}
			if (objP.mvRz != 0) { // si diff de 0 alors il y a une rot en z
				Transformd TmRZd;
				TmRZd = Tid.RotateZ(objP.mvRz);
				Tid = Tid*TmRZd;}
			if (objP.mvTz != 0) { // si diff de 0 alors il y a une translation en z
				double timeOned;
				timeOned = (tab_sensor[ph->is].POSZ-objP.mvTz)/vdouble.z;
				posxd = timeOned*vdouble.x;
				posyd = timeOned*vdouble.y;
			} // Les Translations en x et y sont prises en compte à la fin

			// Si l'objet plan est un rectangle avec p0 le point min et p3 le point max, nous pouvons faire ce qui suit
			double xMinPd = objP.p0x, yMinPd = objP.p0y, xMaxPd = objP.p3x, yMaxPd = objP.p3y;
			
			// Tirer aléatoirement une position sur la surface du miroir dans sa position initiale
			double3 posTransd = make_double3(   (  ( (xMaxPd-xMinPd)*double(RAND) ) + xMinPd  ), (  ( (yMaxPd-yMinPd)*double(RAND) ) + yMinPd  ), 0.  );
			
			// Application des transfos de rot du miroir à cette entité
			char myP[]="Point";		
			posTransd = Tid(posTransd, myP);
			
			// Projection des positions x et y suivant la direction solaire sur la surface de base de l'entité
			double timeTwod;
			timeTwod = posTransd.z/vdouble.z;
			posTransd.x -= timeTwod*vdouble.x;
			posTransd.y -= timeTwod*vdouble.y;

			// On veut lancer les photons depuis TOA + prise en compte des transfos de translation en x et y			
			posTransd.x +=  posxd + double(objP.mvTx);
			posTransd.y +=  posyd + double(objP.mvTy);			
			posTransd.z = tab_sensor[ph->is].POSZ;
			
			// mise à jour de la position finale du photon
			ph->pos=make_float3(float(posTransd.x), float(posTransd.y), float(posTransd.z));		
			#else // IF NOT DOUBLE
			// // Création des transformations depuis les valeurs python du reflecteur
			Transform Ti;			
			if (objP.mvRx != 0) { // si diff de 0 alors il y a une rot en x
				Transform TmRX;
				TmRX = Ti.RotateX(objP.mvRx);
				Ti = Ti*TmRX;}
			if (objP.mvRy != 0) { // si diff de 0 alors il y a une rot en y
				Transform TmRY;
				//TmRY = Ti.RotateY(objP.mvRy - (180-THDEGd)); // 180-THDEGd = Theta Sun in degree
				TmRY = Ti.RotateY(objP.mvRy);
				Ti = Ti*TmRY;}
			if (objP.mvRz != 0) { // si diff de 0 alors il y a une rot en z
				Transform TmRZ;
				TmRZ = Ti.RotateZ(objP.mvRz);
				Ti = Ti*TmRZ;}
			if (objP.mvTz != 0) { // si diff de 0 alors il y a une translation en z
				float timeOne;
				timeOne = (tab_sensor[ph->is].POSZ-objP.mvTz)/ph->v.z;
				ph->pos.x = timeOne*ph->v.x;
				ph->pos.y = timeOne*ph->v.y;
			} // Les Translations en x et y sont prises en compte à la fin			

			// Si l'objet plan est un rectangle avec p0 le point min et p3 le point max, nous pouvons faire ce qui suit
			float xMinP = objP.p0x, yMinP = objP.p0y, xMaxP = objP.p3x, yMaxP = objP.p3y;
			
			// Tirer aléatoirement une position sur la surface du miroir dans sa position initiale
			float3 posTrans = make_float3(   (  ( (xMaxP-xMinP)*RAND ) + xMinP  ), (  ( (yMaxP-yMinP)*RAND ) + yMinP  ), 0.  );
			
			// Application des transfos de rot du miroir à cette entité
			char myP[]="Point";
			posTrans = Ti(posTrans, myP);
			
			// Projection des positions x et y suivant la direction solaire sur la surface de base de l'entité
			float timeTwo;
			timeTwo = posTrans.z/ph->v.z;
			posTrans.x -= timeTwo*ph->v.x;
			posTrans.y -= timeTwo*ph->v.y;

			// On veut lancer les photons depuis TOA + prise en compte des transfos de translation en x et y
			posTrans.x +=  ph->pos.x + objP.mvTx;
			posTrans.y +=  ph->pos.y + objP.mvTy;			
			posTrans.z = tab_sensor[ph->is].POSZ;
			
			// mise à jour de la position finale du photon
			ph->pos=posTrans;
			#endif // END IF DOUBLE OR FLOAT
		}
		/* ***************************************************************************************** */		
	} // CFMODE == 0

	else if (CFMODEd == 1)
	{
		float3 cusForwPos = make_float3( ((CFXd * RAND) - 0.5*CFXd), ((CFYd * RAND) - 0.5*CFYd), 0.);
		ph->pos.x += cusForwPos.x + 1.;
		ph->pos.y += cusForwPos.y;
		ph->pos.z = tab_sensor[ph->is].POSZ;
	} //END CFMODEd == 1
    #endif //END OBJ3D
    }


#ifdef SPHERIQUE
__device__ void move_sp(Photon* ph, struct Profile *prof_atm, int le, int count_level,
                        struct RNG_State *rngstate) {
	
    float tauRdm;
    float hph = 0.;  // cumulative optical thickness
    float vzn, delta1, h_cur, tau_cur, epsilon, AMF;
    #ifndef ALIS
    float h_cur_abs;
    #endif
    float d;
    float rat;
    int sign_direction;
    int i_layer_fw, i_layer_bh; // index or layers forward and behind the photon
    float costh, sinth2;
    float3 no;
    int ilam = ph->ilam*(NATMd+1);  // wavelength offset in optical thickness table

    if (ph->layer == 0) ph->layer = 1;

    // Random Optical Thickness to go through
    if (!le) tauRdm = -logf(1.F-RAND);
    // if called with le mode, it serves to compute the transmission
    // from photon last intercation position to TOA, thus 
    // photon is forced to exit upward or downward and tauRdm is chosen to be an upper limit
    else tauRdm = 1e6;

    vzn = __fdividef( dot(ph->v, ph->pos), ph->radius);

    // a priori value for sign_direction:
    // sign_direction may change sign from -1 to +1 if the photon does not
    // cross lower layer
    if (vzn <= 0) sign_direction = -1;
    else sign_direction = 1;

    while (1) {

        //
        // stopping criteria
        //
        if (ph->layer == NATMd+1) {
            ph->loc = SURF0P;
            ph->tau = 0.;
            ph->layer -= 1;  // next time photon enters move_sp, it's at layers NATM
            break;
        }
        if (ph->layer <= 0) {
            ph->loc = SPACE;
            break;
        }

        //
        // determine the index of the next potential layer
        //
        if (sign_direction < 0) {
            // photon goes down
            // (towards higher indices)
            i_layer_fw = ph->layer;
            i_layer_bh = ph->layer - 1;
        } else {
            // photon goes up
            // (towards lower indices)
            i_layer_fw = ph->layer - 1;
            i_layer_bh = ph->layer;
        }

        // initializations
        costh = vzn;
        sinth2 = 1.f-costh*costh;
        //
        // calculate the distance d to the fw layer
        // from the current position
        //
        // ri : radius of next layer boundary ri=zi+RTER
        // r  : radius of current point along the path 
        // costh: angle between the position vector and the direction vector
        // In the triangle we have ri² = d² + r² + 2*d*r*costh
        // or: d**2 + 2*r*costh*d + r**2-ri**2 = 0 , to be solved for d
        // delta = 4.r².costh² - 4(r²-ri²) = 4*r²*((ri/r)²-sinth²) = 4*r²*delta1
        // with delta1 = (ri/r)²-sinth²
        rat = (prof_atm[i_layer_fw].z+RTER)/ph->radius;
        delta1 = rat*rat - sinth2;   // same sign as delta

        if (delta1 < 0) {
            if (sign_direction > 0) {
                // because of numerical uncertainties, a downward photon may
                // not strictly be between zi and zi+1
                // in rare case of grazing angle there is sometimes no intersection
                // with current layer because photon is actually slightly above it.
                // therefore we consider that delta=0 such that the photon is
                // tangent to the layer
                delta1 = 0.;
            } else {
                // no intersection, with lower layer, we should go towards higher layer
                sign_direction = 1;
                continue;
            }
        }

        /* Now, there are two real solutions for d
        *  The solution is the smallest positive one
        *
        * if photon goes towards higher layers (sign_direction == 1) and costh>0
        * => we keep the smallest solution in abs. val   (both terms are of opposite signs)
        *
        * if photon goes towards lower layers (sign_direction == -1) and costh<0
        * => we keep the smallest solution in abs. val   (both terms are of opposite signs)
        *
        * if photon goes towards higher layers (sign_direction == 1) and costh<0
        * => we keep the greatest solution in abs. val   (both terms are of same signs)
        *
        */
        /* d = 0.5f*(-2.*ph->radius*costh + sign_direction*2*ph->radius*sqrtf(delta1)); simplified to: */
        d = ph->radius*(-costh + sign_direction*sqrtf(delta1));
        AMF = __fdividef(d, abs(prof_atm[i_layer_bh].z - prof_atm[i_layer_fw].z)); // Air Mass Factor

        //
        // calculate the optical thicknesses h_cur and h_cur_abs to the next layer
        // We compute the layer extinction coefficient of the layer DTau/Dz and multiply by the distance within the layer
        //
        tau_cur = abs(get_OD(BEERd,prof_atm[i_layer_bh+ilam]) - get_OD(BEERd,prof_atm[i_layer_fw+ilam]));
        h_cur   = tau_cur * AMF;
        #ifndef ALIS
        h_cur_abs = abs(prof_atm[i_layer_bh+ilam].OD_abs - prof_atm[i_layer_fw+ilam].OD_abs) *AMF;
        #endif

        //
        // update photon position
        //
        if (hph + h_cur > tauRdm) {
            // photon stops within the layer
            epsilon = (tauRdm - hph)/h_cur;
            d *= epsilon;
            AMF*= epsilon;
            ph->pos = operator+(ph->pos, ph->v*d);
            ph->radius = length(ph->pos);
            #ifndef ALIS
            if (BEERd == 1) ph->weight *= __expf(-( epsilon * h_cur_abs));
            #else
            float tau;
            ph->cdist_atm[ph->layer] += d;
            int DL=(NLAMd-1)/(NLOWd-1);
            for (int k=0; k<NLOWd; k++) {
                tau = abs(get_OD(1,prof_atm[i_layer_bh + k*DL*(NATMd+1)]) - get_OD(1,prof_atm[i_layer_fw + k*DL*(NATMd+1)]));
			    ph->weight_sca[k] *= exp(-(tau-tau_cur)*AMF);
            }
            #endif
            break;

        } else {
            // photon advances to the next layer
            hph += h_cur;
            ph->pos = operator+(ph->pos, ph->v*d);
            ph->radius = length(ph->pos);
            no  = operator/(ph->pos, ph->radius);
            vzn = dot(ph->v, no);
            //float psi=0.F;
            if (REFRACd) {
                // We update photon direction at the interface due to refraction
                // 1. sin_i just to verify if refraction occurs
                float sTh  = sqrtf(1.F - vzn*vzn);
                // 2. determine old and new refraction indices from old and new layer indices
                float nind = __fdividef(prof_atm[i_layer_fw+ilam].n, prof_atm[i_layer_bh+ilam].n);
                float sign;
                if (nind > 1) sign=1;
                else sign=-1.F;
                if (sTh!=0.F && nind!=1.F) { // in case of refraction
                  float3 v1;
                  float3 n = sign*no; // See convention as for Rough Surface 
                  float cTh  = -dot(n, ph->v);
	              if( sTh<=nind){ // no total reflection
                    sTh  = sqrtf(1.F - cTh*cTh);
                    // 3. Snell Descartes law :
		            float temp = __fdividef(sTh,nind);
		            float cot  = sqrtf(1.F - temp*temp);
                    float alpha= __fdividef(cTh, nind) - cot;
                    // 4. Update photons direction cosines and u
                    v1=operator+(operator/(ph->v, nind), alpha*n);
                    ph->v = v1;
                    vzn = dot(ph->v, no);
                  }
                  else { //in case of total reflection we continue with refraction but tangent direction
                    v1=operator+(ph->v, (2*cTh)*n);
                    ph->v = v1;
                    vzn = dot(ph->v, no);
                  } // no total reflection 
                } // no refraction computation necessary

            } // No Refraction

            #ifndef ALIS
            if (BEERd == 1) ph->weight *= __expf(-( h_cur_abs));
            #else
            float tau;
            ph->cdist_atm[ph->layer] += d;
            int DL=(NLAMd-1)/(NLOWd-1);
            for (int k=0; k<NLOWd; k++) {
                tau = abs(get_OD(1,prof_atm[i_layer_bh + k*DL*(NATMd+1)]) - get_OD(1,prof_atm[i_layer_fw + k*DL*(NATMd+1)]));
			    ph->weight_sca[k] *= __expf(-(tau-tau_cur)*AMF);
            }
            #endif

            ph->layer -= sign_direction;
        } // photon advances to next layer

    } // while loop

    if (le) {
        if (( (count_level==UPTOA)  && (ph->loc==SPACE ) ) || ( (count_level==DOWN0P) && (ph->loc==SURF0P) )) ph->weight *= __expf(-(hph + h_cur));
        else ph->weight = 0.;
    }

    if ((BEERd == 0) && (ph->loc == ATMOS)) ph->weight *= prof_atm[ph->layer+ilam].ssa;
    //if (BEERd == 0) ph->weight *= prof_atm[ph->layer+ilam].ssa;
}
#endif // SPHERIQUE



#ifdef ALT_PP
__device__ void move_pp2(Photon* ph, struct Profile *prof_atm, struct Profile *prof_oc, int le, int count_level,
                        struct RNG_State *rngstate) {

    float tauRdm;
    float hph = 0.;  // cumulative optical thickness
    float vzn, h_cur, tau_cur, epsilon, AMF;
    #ifndef ALIS
    float h_cur_abs;
    #endif
    float d;
    int sign_direction;
    int i_layer_fw, i_layer_bh; // index or layers forward and behind the photon
    int ilam; 
    struct Profile *prof;
    int  NL;
	//int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
    if (ph->loc==OCEAN) {
        NL   = NOCEd+1;
        prof = prof_oc;
    }
    if (ph->loc==ATMOS) {
        NL   = NATMd+1;
        prof = prof_atm;
    }
    ilam = ph->ilam*NL;  // wavelength offset in optical thickness table

    if (ph->layer == 0) ph->layer = 1;

    // Random Optical Thickness to go through
    if (!le) tauRdm = -logf(1.F-RAND);
    // if called with le mode, it serves to compute the transmission
    // from photon last intercation position to TOA, thus 
    // photon is forced to exit upward or downward and tauRdm is chosen to be an upper limit
    else tauRdm = 1e6;

    vzn = ph->v.z;

    // a priori value for sign_direction:
    // sign_direction may change sign from -1 to +1 if the photon does not
    // cross lower layer
    if (vzn <= 0) sign_direction = -1;
    else sign_direction = 1;
    int count=0;

    while (1) {

        //
        // stopping criteria
        //
        if (ph->loc == ATMOS) {
         if (ph->layer == NATMd+1) {
            ph->loc = SURF0P;
            ph->layer -= 1;  // next time photon enters move_pp2, it's at layers NATM
            break;
         }
         if (ph->layer <= 0) {
            ph->loc = SPACE;
            break;
         }
        } 
        if (ph->loc == OCEAN) {
         if (ph->layer == NOCEd+1) {
            ph->loc = SEAFLOOR;
            ph->layer -= 1;  // next time photon enters move_pp2, it's at layers NOCE
            break;
         }
         if (ph->layer <= 0) {
            ph->loc = SURF0M;
            ph->layer= 0;
            break;
         }
        }

        //
        // determine the index of the next potential layer
        //
        if (sign_direction < 0) {
            // photon goes down
            // (towards higher indices)
            i_layer_fw = ph->layer;
            i_layer_bh = ph->layer - 1;
        } else {
            // photon goes up
            // (towards lower indices)
            i_layer_fw = ph->layer - 1;
            i_layer_bh = ph->layer;
        }

        //
        // calculate the distance d to the fw layer
        // from the current position
        d   = __fdividef(abs(ph->pos.z - prof[i_layer_fw].z), fabs(ph->v.z));
        AMF = __fdividef(d, abs(prof[i_layer_bh].z - prof[i_layer_fw].z)); // Air Mass Factor

        //
        // calculate the optical thicknesses h_cur and h_cur_abs to the next layer
        // We compute the layer extinction coefficient of the layer DTau/Dz and multiply by the distance within the layer
        //
        tau_cur = abs(get_OD(BEERd,prof[i_layer_bh+ilam]) - get_OD(BEERd,prof[i_layer_fw+ilam]));
        h_cur   = tau_cur * AMF;
        #ifndef ALIS
        h_cur_abs = abs(prof[i_layer_bh+ilam].OD_abs - prof[i_layer_fw+ilam].OD_abs) *AMF;
        #endif

        //
        // update photon position
        //
        if (hph + h_cur > tauRdm) {
            // photon stops within the layer
            epsilon = (tauRdm - hph)/h_cur;
            d *= epsilon;
            AMF*= epsilon;
            ph->pos = operator+(ph->pos, ph->v*d);
            #ifndef ALIS
            if (BEERd == 1) ph->weight *= __expf(-( epsilon * h_cur_abs));
            #else
            float tau;
            if (ph->loc==ATMOS) ph->cdist_atm[ph->layer] += d;
            if (ph->loc==OCEAN) ph->cdist_oc[ ph->layer] += d;
            int DL=(NLAMd-1)/(NLOWd-1);
            for (int k=0; k<NLOWd; k++) {
                tau = abs(get_OD(1,prof[i_layer_bh + k*DL*NL]) - get_OD(1,prof[i_layer_fw + k*DL*NL]));
			    ph->weight_sca[k] *= exp(-(tau-tau_cur)*AMF);
            }
            #endif
            break;

        } else {
            // photon advances to the next layer
            hph += h_cur;
            ph->pos = operator+(ph->pos, ph->v*d);

            #ifndef ALIS
            if (BEERd == 1) ph->weight *= __expf(-( h_cur_abs));
            #else
            float tau;
            if (ph->loc==ATMOS) ph->cdist_atm[ph->layer] += d;
            if (ph->loc==OCEAN) ph->cdist_oc[ ph->layer] += d;
            int DL=(NLAMd-1)/(NLOWd-1);
            for (int k=0; k<NLOWd; k++) {
                tau = abs(get_OD(1,prof[i_layer_bh + k*DL*NL]) - get_OD(1,prof[i_layer_fw + k*DL*NL]));
			    ph->weight_sca[k] *= __expf(-(tau-tau_cur)*AMF);
            }
            #endif

            ph->layer -= sign_direction;
            count++;
        } // photon advances to next layer

    } // while loop

    if (le) {
        if (( (count_level==UPTOA)  && (ph->loc==SPACE ) ) || 
            ( (count_level==DOWN0P) && (ph->loc==SURF0P) ) ||
            ( (count_level==UP0M)   && (ph->loc==SURF0M) ) ||
            ( (count_level==DOWNB)  && (ph->loc==SEAFLOOR) ) ) 
            ph->weight *= __expf(-(hph + h_cur));
        else ph->weight = 0.;
    }

    if ((BEERd == 0) && ((ph->loc == ATMOS) || (ph->loc == OCEAN))) {
        ph->weight *= prof[ph->layer+ilam].ssa;
        //if (idx==0) printf("%d %d %d %f\n",ph->loc, ph->layer, ph->ilam, prof[ph->layer+ilam].ssa);
    }
}
#endif // ALT_PP



__device__ void move_pp(Photon* ph, struct Profile *prof_atm, struct Profile *prof_oc,
                        struct RNG_State *rngstate
						#ifdef OBJ3D
						, IGeo *geoS, struct IObjets *myObjets, void *tabObjInfo
						#endif
	) {

	float delta_i=0.f, delta=0.f, epsilon;
	float tauR, prev_tau, tauBis, phz; //rdist
    int ilayer;
	
    #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
    float dsca_dl, dsca_dl0=-ph->tau ;
    int DL=(NLAMd-1)/(NLOWd-1);
    #endif
    #ifndef ALIS
    float ab;
    #endif

	prev_tau = ph->tau;        // previous value of tau photon
	tauR = -logf(1.f - RAND);  // optical distance reached calculated randomly
	ph->tau += (tauR*ph->v.z); // the value of tau photon at the reached point

	if (ph->loc == OCEAN){  
        // If tau>0 photon is reaching the surface 
        if (ph->tau > 0) {

            #ifndef ALIS
            if (BEERd == 1) {// absorption between start and stop
                ab =  0.F;
                ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            }
            #else

            #if !defined(ALT_PP) && !defined(SPHERIQUE)
            dsca_dl0 += 0.F;
            for (int k=0; k<NLOWd; k++) {
                dsca_dl = 0.F;
                dsca_dl -= ph->tau_sca[k]; 
                ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl)-fabs(dsca_dl0),  fabs(ph->v.z)));
                ph->tau_sca[k] = 0.F;
            }
            #endif

            #endif

            ph->tau = 0.F;
            ph->tau_abs = 0.F;
            ph->loc = SURF0M;
            if (SIMd == OCEAN_ONLY){
              ph->loc = SPACE;
            }
            ph->layer = NOCEd;

            // move the photon forward up to the surface
            // the linear distance is ph->z/ph->vz
            operator+=(ph->pos, ph->v * fabs(ph->pos.z/ph->v.z));

            #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
            ph->nevt++;
            ph->layer_prev[ph->nevt] = -ph->layer;
            ph->vz_prev[ph->nevt] = ph->v.z;
            ph->epsilon_prev[ph->nevt] = 1.f;
            #endif
           return;
        }
        // If tau<TAUOCEAN photon is reaching the sea bottom
        else if( ph->tau < get_OD(BEERd, prof_oc[NOCEd + ph->ilam *(NOCEd+1)]) ){

            #ifndef ALIS
            if (BEERd == 1) {// absorption between start and stop
                ab = prof_oc[NOCEd + ph->ilam *(NOCEd+1)].OD_abs;
                ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            }
            #else

            #if !defined(ALT_PP) && !defined(SPHERIQUE)
            dsca_dl0 += get_OD(1,prof_oc[NOCEd + ph->ilam*(NOCEd+1)]) ; 
            for (int k=0; k<NLOWd; k++) {
                dsca_dl = get_OD(1,prof_oc[NOCEd + k*DL*(NOCEd+1)]);
                dsca_dl -= ph->tau_sca[k]; 
                ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl) - fabs(dsca_dl0), fabs(ph->v.z)));
                ph->tau_sca[k] = get_OD(1,prof_oc[NOCEd + k*DL*(NOCEd+1)]);
            }
            #endif

            #endif

            ph->loc = SEAFLOOR;
            ph->tau = get_OD(BEERd, prof_oc[NOCEd + ph->ilam *(NOCEd+1)]);
            ph->tau_abs = prof_oc[NOCEd + ph->ilam *(NOCEd+1)].OD_abs;
            ph->layer = 0;

			// move the photon forward down to the seafloor
            operator+=(ph->pos, ph->v * fabs( (ph->pos.z - prof_oc[NOCEd].z) /ph->v.z));

            #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
            ph->nevt++;
            ph->layer_prev[ph->nevt] = -ph->layer;
            ph->vz_prev[ph->nevt] = ph->v.z;
            ph->epsilon_prev[ph->nevt] = 0.f;
            #endif
            return;
        }

        //computing photons layer number
        ilayer = 1;
        while (( get_OD(BEERd, prof_oc[ilayer+ ph->ilam *(NOCEd+1)]) > (ph->tau)) && (ilayer < NOCEd)) {
            ilayer++;
        }
        ph->layer = ilayer;

        delta_i= fabs(get_OD(BEERd, prof_oc[ilayer+ph->ilam*(NOCEd+1)]) - get_OD(BEERd, prof_oc[ilayer-1+ph->ilam*(NOCEd+1)]));
        delta= fabs(ph->tau - get_OD(BEERd, prof_oc[ilayer-1+ph->ilam*(NOCEd+1)])) ;
        epsilon = __fdividef(delta,delta_i);


        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        ph->nevt++;
        ph->layer_prev[ph->nevt] = -ph->layer;
        ph->vz_prev[ph->nevt] = ph->v.z;
        ph->epsilon_prev[ph->nevt] = epsilon;
        #endif
            
        #ifndef ALIS
        if (BEERd == 0) ph->weight *= prof_oc[ph->layer+ph->ilam*(NOCEd+1)].ssa;
        else { // We compute the cumulated absorption OT at the new postion of the photon
            // photon new position in the layer
            ab = prof_oc[ilayer-1+ph->ilam*(NOCEd+1)].OD_abs + epsilon * (prof_oc[ilayer+ph->ilam*(NOCEd+1)].OD_abs - prof_oc[ilayer-1+ph->ilam*(NOCEd+1)].OD_abs);
            // absorption between start and stop
            ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            ph->tau_abs = ab;
        }
        #else

        #if !defined(ALT_PP) && !defined(SPHERIQUE)
        // cumulated scattering OD at reference wavelength
        dsca_dl0 += get_OD(1,prof_oc[NOCEd + ph->ilam*(NOCEd+1)]) - 
            (epsilon * (get_OD(1,prof_oc[ilayer+ph->ilam*(NOCEd+1)]) - get_OD(1,prof_oc[ilayer-1+ph->ilam*(NOCEd+1)])) +
            get_OD(1,prof_oc[ilayer-1+ph->ilam*(NOCEd+1)]));
        for (int k=0; k<NLOWd; k++) {
           // cumulated scattering relative OD wrt reference wavelength
            float tautmp = get_OD(1,prof_oc[NOCEd + k*DL*(NOCEd+1)]) - 
                (epsilon * (get_OD(1,prof_oc[ilayer+k*DL*(NOCEd+1)]) - get_OD(1,prof_oc[ilayer-1+k*DL*(NOCEd+1)])) +
                get_OD(1,prof_oc[ilayer-1+k*DL*(NOCEd+1)])) ;
            dsca_dl  = tautmp - ph->tau_sca[k]; 
            ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl) -fabs(dsca_dl0), fabs(ph->v.z)));
            ph->tau_sca[k] = tautmp;
        }
        #endif

        #endif
		
        // calculate new photon position
        phz =  prof_oc[ilayer-1].z + epsilon * ( prof_oc[ilayer].z - prof_oc[ilayer-1].z); 
		// move the photon to new position
		operator+=(ph->pos, ph->v * fabs( (ph->pos.z - phz) / ph->v.z));

    } // Ocean



	


    if (ph->loc == ATMOS) {
        float rdist;
	    #ifdef OBJ3D
		// ========================================================================================================
		// Here geometry modification in the function move_pp
		// ========================================================================================================
		float timeT;                                 // the time from the parametric form of a ray
		bool mytest = false;                         // initiate the boolean of the intersection test
		float3 phit=make_float3(0.f, 0.f, 0.f);      // initiate the intersection point 


		// Launch the function geoTest to see if there are an intersection with the 
		// geometry, return true/false and give the position phit of the intersection
		// nObj = le nombre d'objets, si = 0 alors le test n'est pas nécessaire.
	    if (nObj > 0){
			mytest = geoTest(ph->pos, ph->v, ph->locPrev, &phit, geoS, myObjets);
			if (ph->direct == 0 && ph->pos.z == 120. && mytest == false && CFMODEd == 0) {ph->loc=NONE; return;}
		}
		//if (ph->pos.z == 120. && mytest == false) {ph->loc=NONE; return;}
		//if (ph->pos.z == 120. && mytest == true && geoS->type != 1) {ph->loc=NONE; return;}
		//if (ph->pos.z < 1.2 && mytest == true && geoS->type == 3) {ph->loc=ABSORBED; return;}
		// if mytest = true (intersection with the geometry) and the position of the intersection is in
		// the atmosphere (0 < Z < 120), then: Begin to analyse is there is really an intersection
		if(mytest && phit.z >= 0.F && phit.z <= 120.F)
		{
			float tauHit = 0.f; // Optical depth distance (from the initial position of the photon to phit)
			int ilayer2 = ph->layer;
			if (ilayer2==0) {ilayer2=1;} // Be sure that we're not out of the atmosphere

			if((phit.z >= prof_atm[ilayer2].z) && (phit.z < prof_atm[ilayer2-1].z)) // 1 layer case: n = 1
			{
				// delta_i is: Delta(tau)1 = |tau(i-1) - tau(i)|
				delta_i = fabs(get_OD(BEERd, prof_atm[ilayer2+ph->ilam*(NATMd+1)]) - 
							   get_OD(BEERd, prof_atm[ilayer2-1+ph->ilam*(NATMd+1)]));
				// tauHit = (Delat(D1)/Delat(Z1))*delta_i
				tauHit += (length(ph->pos, phit)/fabs(prof_atm[ilayer2-1].z - prof_atm[ilayer2].z))*delta_i;
			}
			else // several layers case: n >= 2
			{
				// Find the layer where there is intersection
				ilayer2 = 1;
				while(prof_atm[ilayer2].z > phit.z && prof_atm[ilayer2].z > 0.F)
				{
					ilayer2 ++;
				} 

				float3 newP, oldP;
				bool higher = false;

				ilayer = ph->layer;          // initialise with the actual layer
				if (ilayer==0) {ilayer=1;}   // be sure that we're not out of the atmosphere
				oldP = ph->pos;                // initialise with the actual position

				// check if the photon come from higher or lower layers
				if(ilayer < ilayer2) // true if the photon come from higher layers
					higher =  true;

				while(ilayer != ilayer2)
				{
					if(higher){timeT = fabs(prof_atm[ilayer].z - oldP.z)/fabs(ph->v.z);}
					else{timeT = fabs(prof_atm[ilayer-1].z - oldP.z)/fabs(ph->v.z);}
					newP = oldP + timeT*ph->v;
					delta_i = fabs(get_OD(BEERd, prof_atm[ilayer+ph->ilam*(NATMd+1)]) - 
								   get_OD(BEERd, prof_atm[ilayer-1+ph->ilam*(NATMd+1)]));
					tauHit += (length(newP, oldP)/fabs(prof_atm[ilayer - 1].z - prof_atm[ilayer].z))*delta_i;
					
					// the photon come from higher layers
					if(higher){ilayer++;}
					// the photon come from lower layers
					else{ilayer--;}
					oldP = newP; //Update the position of the photon
				}

				// Calculate and add the last tau distance when ilayer is equal to ilayer2
				delta_i = fabs(get_OD(BEERd, prof_atm[ilayer2+ph->ilam*(NATMd+1)]) - 
							   get_OD(BEERd, prof_atm[ilayer2-1+ph->ilam*(NATMd+1)]));
				tauHit += (length(phit, oldP)/fabs(prof_atm[ilayer2 - 1].z - prof_atm[ilayer2].z))*delta_i;
			}


			// if tauHit (optical distance to hit the geometry) < tauR, then: there is interaction.
			if (tauHit < tauR || IsAtm == 0) // IsAtm == 0 (without atm for objects) -> we don't care about tau
			{
				// to see
				ph->layer = ilayer2;
				if (IsAtm == 0) ph->weight *= 1;
				else if (BEERd == 0) ph->weight *= prof_atm[ph->layer+ph->ilam*(NATMd+1)].ssa;
				else
				{ // We compute the cumulated absorption OT at the new postion of the photon
					// see move photon paper eq 11
					ab = prof_atm[NATMd+ph->ilam*(NATMd+1)].OD_abs - 
						(epsilon * (prof_atm[ilayer2+ph->ilam*(NATMd+1)].OD_abs - prof_atm[ilayer2-1+ph->ilam*(NATMd+1)].OD_abs) +
						 prof_atm[ilayer2-1+ph->ilam*(NATMd+1)].OD_abs);
					// absorption between start and stop
					ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
					ph->tau_abs = ab;
				}
				ph->loc = OBJSURF;                      // update of the loc of the photon 
				ph->tau = prev_tau + tauHit * ph->v.z;  // update the value of tau photon
				ph->pos = phit;                         // update the position of the photon
				return;
				
				// Photon phBis;
				
				// phBis.weight = 1.;
				// phBis.pos = phit;
				// if (IsAtm == 0) phBis.weight = ph->weight * 1;
				// else if (BEERd == 0) phBis.weight =  ph->weight * prof_atm[ph->layer+ph->ilam*(NATMd+1)].ssa;
				// else
				// { // We compute the cumulated absorption OT at the new postion of the photon
				// 	// see move photon paper eq 11
				// 	ab = prof_atm[NATMd+ph->ilam*(NATMd+1)].OD_abs - 
				// 		(epsilon * (prof_atm[ilayer2+ph->ilam*(NATMd+1)].OD_abs - prof_atm[ilayer2-1+ph->ilam*(NATMd+1)].OD_abs) +
				// 		 prof_atm[ilayer2-1+ph->ilam*(NATMd+1)].OD_abs);
				// 	// absorption between start and stop
				// 	phBis.weight = ph->weight * exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
				// }
				// //atomicAdd(CounterIntObj, 1);
				// phBis.v = ph->v;
				// countPhotonObj3D(&phBis, tabObjInfo, geoS);
				
			}
		} // End of mytest = true

		// Case where atm is false in special case with objetcs
		// if there is not an intersect with an objet we have a special treatment
		if (nObj > 0 && IsAtm == 0 && !mytest)
		{
			BBox boite(make_float3(Pmin_x, Pmin_y, Pmin_z), make_float3(Pmax_x, Pmax_y, Pmax_z));
			Ray Rayon(ph->pos, ph->v, 0);
			float intTime0=0., intTime1=0.;
			bool intersectBox;
			float3 intersectPoint = make_float3(-1., -1., -1.);
			
			intersectBox = boite.IntersectP(Rayon, &intTime0, &intTime1);
			
			if (intersectBox && intTime0 < 0.) {intersectPoint = Rayon.o * intTime1;}
			else if (intersectBox && intTime0 >= 0.) {intersectPoint = Rayon.o * intTime0;}
			else {printf("error1 in move_pp geo!! \n"); return;}

			if (intersectPoint.z > 119.999)
			{
				ph->loc = SPACE;
				ph->layer = 0;
				ph->weight *= 1;
				return;
			}
			else if (intersectPoint.z < 0.0001)
			{
				ph->loc = SURF0P;
				ph->tau = 0.F;
				ph->tau_abs = 0.F;
				ph->pos.z = 0.;
				ph->layer = NATMd;
				ph->weight *= 1;
				return;
			}
			else {ph->loc = NONE;return;}
		}
		// ========================================================================================================
        #endif //END OBJ3D
		
        // If tau<0 photon is reaching the surface 
        if(ph->tau < 0.F){

            #ifndef ALIS
            if (BEERd == 1) {// absorption between start and stop
                ab =  0.F;
                ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            }
            #else

            #if !defined(ALT_PP) && !defined(SPHERIQUE)
            dsca_dl0 += 0.F;
            for (int k=0; k<NLOWd; k++) {
                dsca_dl = 0.F;
                dsca_dl -= ph->tau_sca[k]; 
                ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl)-fabs(dsca_dl0),  fabs(ph->v.z)));
                ph->tau_sca[k] = 0.F;
            }
            #endif

            #endif

            ph->loc = SURF0P;
            ph->tau = 0.F;
            ph->tau_abs = 0.F;
            // move the photon forward down to the surface
            // the linear distance is ph->z/ph->vz
            operator+=(ph->pos, ph->v * fabs(ph->pos.z/ph->v.z));
            ph->pos.z = 0.;
            ph->layer = NATMd;

            #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
            ph->nevt++;
            ph->layer_prev[ph->nevt] = ph->layer;
            ph->vz_prev[ph->nevt] = ph->v.z;
            ph->epsilon_prev[ph->nevt] = 1.f;
            #endif
        return;
        }
        // If tau>TAUATM photon is reaching space
        else if( ph->tau > get_OD(BEERd, prof_atm[NATMd + ph->ilam *(NATMd+1)]) ){

            #ifndef ALIS
		    if (BEERd == 1) {// absorption between start and stop
                ab = prof_atm[NATMd + ph->ilam *(NATMd+1)].OD_abs;
                ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            }
            #else

            #if !defined(ALT_PP) && !defined(SPHERIQUE)
            dsca_dl0 += get_OD(1,prof_atm[NATMd + ph->ilam*(NATMd+1)]) ; 
            for (int k=0; k<NLOWd; k++) {
                dsca_dl = get_OD(1,prof_atm[NATMd + k*DL*(NATMd+1)]);
                dsca_dl -= ph->tau_sca[k]; 
                ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl) - fabs(dsca_dl0), fabs(ph->v.z)));
                ph->tau_sca[k] = get_OD(1,prof_atm[NATMd + k*DL*(NATMd+1)]);
            }
            #endif

            #endif

            ph->loc = SPACE;
            ph->layer = 0;

            #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
            ph->nevt++;
            ph->layer_prev[ph->nevt] = ph->layer;
            ph->vz_prev[ph->nevt] = ph->v.z;
            ph->epsilon_prev[ph->nevt] = 0.f;
            #endif

            return;
        }
        
        // Sinon il reste dans l'atmosphère, et va subit une nouvelle diffusion
        // Calcul de la layer dans laquelle se trouve le photon
        tauBis =  get_OD(BEERd, prof_atm[NATMd + ph->ilam *(NATMd+1)]) - ph->tau;
        ilayer = 1;
        
        while (( get_OD(BEERd, prof_atm[ilayer+ ph->ilam *(NATMd+1)]) < (tauBis)) && (ilayer < NATMd)) {
            ilayer++;
        }
        
        ph->layer = ilayer;

        delta_i= fabs(get_OD(BEERd, prof_atm[ilayer+ph->ilam*(NATMd+1)]) - get_OD(BEERd, prof_atm[ilayer-1+ph->ilam*(NATMd+1)]));
        delta= fabs(tauBis - get_OD(BEERd, prof_atm[ilayer-1+ph->ilam*(NATMd+1)])) ;
        epsilon = __fdividef(delta,delta_i);

        #if defined(ALIS) && !defined(ALT_PP) && !defined(SPHERIQUE)
        ph->nevt++;
        ph->layer_prev[ph->nevt] = ph->layer;
        ph->vz_prev[ph->nevt] = ph->v.z;
        ph->epsilon_prev[ph->nevt] = epsilon;
        #endif

        #ifndef ALIS
        if (BEERd == 0) ph->weight *= prof_atm[ph->layer+ph->ilam*(NATMd+1)].ssa;
        else { // We compute the cumulated absorption OT at the new postion of the photon
            // photon new position in the layer
            ab = prof_atm[NATMd+ph->ilam*(NATMd+1)].OD_abs - 
                (epsilon * (prof_atm[ilayer+ph->ilam*(NATMd+1)].OD_abs - prof_atm[ilayer-1+ph->ilam*(NATMd+1)].OD_abs) +
                prof_atm[ilayer-1+ph->ilam*(NATMd+1)].OD_abs);
            // absorption between start and stop
            ph->weight *= exp(-fabs(__fdividef(ab-ph->tau_abs, ph->v.z)));
            ph->tau_abs = ab;
        }
        #else

        #if !defined(ALT_PP) && !defined(SPHERIQUE)
        // cumulated scattering OD at reference wavelength
        dsca_dl0 += get_OD(1,prof_atm[NATMd + ph->ilam*(NATMd+1)]) - 
            (epsilon * (get_OD(1,prof_atm[ilayer+ph->ilam*(NATMd+1)]) - get_OD(1,prof_atm[ilayer-1+ph->ilam*(NATMd+1)])) +
            get_OD(1,prof_atm[ilayer-1+ph->ilam*(NATMd+1)]));
        for (int k=0; k<NLOWd; k++) {
           // cumulated scattering relative OD wrt reference wavelength
            float tautmp = get_OD(1,prof_atm[NATMd + k*DL*(NATMd+1)]) - 
                (epsilon * (get_OD(1,prof_atm[ilayer+k*DL*(NATMd+1)]) - get_OD(1,prof_atm[ilayer-1+k*DL*(NATMd+1)])) +
                get_OD(1,prof_atm[ilayer-1+k*DL*(NATMd+1)])) ;
            dsca_dl  = tautmp - ph->tau_sca[k]; 
            ph->weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl) -fabs(dsca_dl0), fabs(ph->v.z)));
            ph->tau_sca[k] = tautmp;
        }
        #endif

        #endif

        // calculate new photon position
        phz = epsilon * (prof_atm[ilayer].z - prof_atm[ilayer-1].z) + prof_atm[ilayer-1].z; 
		rdist=  fabs(__fdividef(phz-ph->pos.z, ph->v.z));
        operator+= (ph->pos, ph->v*rdist);
        ph->pos.z = phz;

    } //ATMOS

}


__device__ void scatter(Photon* ph,
       struct Profile *prof_atm, struct Profile *prof_oc,
        struct Phase *faer, struct Phase *foce,
        int le, float dth,
        float* tabthv, float* tabphi, int count_level,
        struct RNG_State *rngstate) {

	float cTh=0.f;
	float zang=0.f, theta=0.f;
	int iang, ilay, ipha;
	float psi, sign;
	struct Phase *func;
	float P11, P12, P22, P33, P43, P44;

	#ifdef OBJ3D
	ph->direct += 1;
	ph->S += 1;
	#endif
	
    if (le){
        /* in case of LE the photon units vectors, scattering angle and Psi rotation angle are determined by output zenith and azimuth angles*/
        float thv, phi;
        float3 v;
        float EPS = 1e-12;

        if (count_level==DOWN0P || count_level==DOWNB) sign = -1.0F;
        else sign = 1.0F;
        phi = tabphi[ph->iph];
        thv = tabthv[ph->ith] + dth;
        if (thv < EPS) thv = EPS;
        v.x = cosf(phi) * sinf(thv);
        v.y = sinf(phi) * sinf(thv);
        v.z = sign * cosf(thv);
        theta = ComputeTheta(ph->v, v);
        cTh = __cosf(theta);
		if (cTh < -1.0) cTh = -1.0;
		if (cTh >  1.0) cTh =  1.0;
        ComputePsiLE(ph->u, ph->v, v, &psi, &ph->u); 
        ph->v = v;
    }

    /* Scattering in atmosphere */
	if(ph->loc!=OCEAN){

			ilay = ph->layer + ph->ilam*(NATMd+1); // atm layer index
			
			func = faer; // atm phases
			
			/************************************/
			/* Rayleigh or ptcle scattering */
			/************************************/
			if( ph->scatterer == RAY ){ipha  = 0;}   // Rayleigh index
			else if(ph->scatterer == PTCLE ){ipha  = prof_atm[ilay].iphase + 1;} // particle index
		
		}
	/* Scattering in ocean */
	else {

			ilay = ph->layer + ph->ilam*(NOCEd+1); // oce layer index

			func = foce; // oce phases
			
			if (ph->scatterer == RAY){ipha  = 0;}	// Rayleigh index
			else if(ph->scatterer == PTCLE ){ ipha  = prof_oc[ilay].iphase + 1;} // particle index

    }


	if ( (ph->scatterer == RAY) || (ph->scatterer == PTCLE) ){

		if(!le) {

			/* in the case of propagation (not LE) the photons scattering angle and Psi
			   rotation angle are determined randomly */
			/////////////
			// Get Theta from Cumulative Distribution Function
			zang = RAND*(NF-1);
			iang= __float2int_rd(zang);
			zang = zang - iang;

			theta = (1.-zang)*func[ipha*NF+iang].p_ang + zang*func[ipha*NF+iang+1].p_ang;
			cTh = __cosf(theta);

			/////////////
			// Get Scattering matrix from CDF
			P11 = (1-zang)*func[ipha*NF+iang].p_P11 + zang*func[ipha*NF+iang+1].p_P11;
			P12 = (1-zang)*func[ipha*NF+iang].p_P12 + zang*func[ipha*NF+iang+1].p_P12;
			P22 = (1-zang)*func[ipha*NF+iang].p_P22 + zang*func[ipha*NF+iang+1].p_P22;
			P33 = (1-zang)*func[ipha*NF+iang].p_P33 + zang*func[ipha*NF+iang+1].p_P33;
			P43 = (1-zang)*func[ipha*NF+iang].p_P43 + zang*func[ipha*NF+iang+1].p_P43;
			P44 = (1-zang)*func[ipha*NF+iang].p_P44 + zang*func[ipha*NF+iang+1].p_P44;

            #ifndef BIAS
			/////////////
			//  Get Psi
			//  Rejection method for sampling psi 
			float fpsi_cond=0.F; 
			float fpsi=0.F; 
			float gamma=0.F; 
			float Q = ph->stokes.x - ph->stokes.y;
			float U = ph->stokes.z;
			float DoLP = __fdividef(sqrtf(Q*Q+U*U), ph->stokes.x + ph->stokes.y);
			float K = __fdividef(P11-P22,P11+P22+2*P12);
			if (abs(Q) > 0.F) gamma   = 0.5F * atan2(-(double)U,(double)Q);
			float fpsi_cond_max = (1.F + DoLP * fabs(K) )/DEUXPI;
			int niter=0;
			while (fpsi >= fpsi_cond)
				{
					niter++;
					psi = RAND * DEUXPI;	
					fpsi= RAND * fpsi_cond_max;
					fpsi_cond = (1.F + DoLP * K * cosf(2*(psi-gamma)) )/DEUXPI;
					if (niter >= 100) {
						// safety check
                        #ifdef DEBUG
						printf("Warning, photon rejected in scatter while loop\n");
						printf("%i  S=(%f,%f), DoLP, gamma=(%f,%f) psi,theta=(%f,%f) \n",
							   niter,
							   Q,
							   U,
							   DoLP,
							   gamma,
							   psi/PI*180,
							   theta/PI*180
							   );
                        #endif
						ph->loc = NONE;
						break;
					}
				}

            #else
			/////////////
			//  Get Phi
			//  Biased sampling scheme for psi 1)
			psi = RAND * DEUXPI;	
            #endif


		}else {
	
			/////////////
			// Get Index of scattering angle and Scattering matrix directly 
			zang = theta * (NF-1)/PI ;
			iang = __float2int_rd(zang);
			zang = zang - iang;

			P11 = (1-zang)*func[ipha*NF+iang].a_P11 + zang*func[ipha*NF+iang+1].a_P11;
			P12 = (1-zang)*func[ipha*NF+iang].a_P12 + zang*func[ipha*NF+iang+1].a_P12;
			P22 = (1-zang)*func[ipha*NF+iang].a_P22 + zang*func[ipha*NF+iang+1].a_P22;
			P33 = (1-zang)*func[ipha*NF+iang].a_P33 + zang*func[ipha*NF+iang+1].a_P33;
			P43 = (1-zang)*func[ipha*NF+iang].a_P43 + zang*func[ipha*NF+iang+1].a_P43;
			P44 = (1-zang)*func[ipha*NF+iang].a_P44 + zang*func[ipha*NF+iang+1].a_P44;

		}

		// Stokes vector rotation
		rotateStokes(ph->stokes, psi, &ph->stokes);

		// Scattering matrix multiplication
		float4x4 P_scatter = make_float4x4(
										   P11, P12, 0. , 0.  ,
										   P12, P22, 0. , 0.  ,
										   0. , 0. , P33, -P43,
										   0. , 0. , P43, P44
										   );

		ph->stokes = mul(P_scatter, ph->stokes);

        #ifdef BACK
		float4x4 L;
		rotationM(-psi,&L);
		ph->M   = mul(ph->M,mul(L,P_scatter));
		//float4x4 Lf;
		//rotationM(psi,&Lf);
		//ph->Mf  = mul(mul(P_scatter,Lf),ph->Mf);
        #endif

		if (!le){
			float debias = 1.F;
            #ifdef BIAS
			// Bias sampling scheme 2): Debiasing and normalizing
			debias = __fdividef(2.F, P11 + P22 + 2*P12 ); // Debias is equal to the inverse of the phase function
            #else
			debias = __fdividef(1.F, ph->stokes.x + ph->stokes.y);
            #endif

			operator*=(ph->stokes, debias); 
            #ifdef BACK
			ph->M  = mul(ph->M ,   make_diag_float4x4(debias)); // Bias sampling scheme only for backward mode
			//ph->Mf = mul(ph->Mf ,  make_diag_float4x4(debias));
            #endif
		}

		else {
			ph->weight /= 4.F; // Phase function normalization
		}

	}

	else if (ph->scatterer == CHLFLUO){ 

		/////////////////
		// Fluorescence

		if (!le){

			// isotropic point source
			// see Leathers, R. A.; Downes, T. V.; Davis, C. O. & Davis, C. D. Monte Carlo Radiative Transfer Simulations for Ocean Optics: A Practical Guide Naval Research Laboratory, 2004, section 5.1.3
			float phi;
			float sTh;
			cTh = 1.0-2.0*RAND;
			phi = RAND*DEUXPI;
			sTh = sqrtf(1.F - cTh*cTh);
			ph->v.x   = cosf(phi)*sTh;
			ph->v.y   = sinf(phi)*sTh;
			ph->v.z   = cTh;
			// Initialization of the orthogonal vector to the propagation
			ph->u.x   = cosf(phi)*cTh;
			ph->u.y   = sinf(phi)*cTh;
			ph->u.z   = -sTh;
			
		}else{
			ph->weight /= 4.0  ;    // Phase function normalization	
		}


		// Depolarisation du Photon
		float4x4 L = make_float4x4(
								   0.5F, 0.5F, 0.F, 0.F,
								   0.5F, 0.5F, 0.F, 0.F,
								   0.0F, 0.0F, 0.F, 0.F,
								   0.0F, 0.0F, 0.F, 0.F 
								   );
		ph->stokes = mul(L,ph->stokes);
		}

    #ifdef ALIS
	if ( (ph->scatterer == RAY) or (ph->scatterer == PTCLE) ){
        Profile *prof;                                             
        int layer_end;                                             

        if(ph->loc == ATMOS){                                      
             layer_end = NATMd;                                     
             prof = prof_atm;                                       
        }                                                          
        if(ph->loc == OCEAN){                                      
             layer_end = NOCEd;                                     
             prof = prof_oc;                                        
        } 

		int DL=(NLAMd-1)/(NLOWd-1);
		float P11_aer_ref, P11_ray, P22_aer_ref, P22_ray, P12_aer_ref, P12_ray, P_ref;
		float pmol= prof[ph->layer+ ph->ilam*(layer_end+1)].pmol;
       
		if (pmol < 1.) {
			zang = theta * (NF-1)/PI ;
			iang = __float2int_rd(zang);
			zang = zang - iang;
			int ipharef = prof[ph->layer+ph->ilam*(layer_end+1)].iphase + 1; 
			// Phase functions of particles and molecules, and mixture of both at reference wavelength
			P11_aer_ref = (1-zang)*func[ipharef*NF+iang].a_P11 + zang*func[ipharef*NF+iang+1].a_P11;
			P11_ray     = (1-zang)*func[0      *NF+iang].a_P11 + zang*func[0      *NF+iang+1].a_P11;
			P22_aer_ref = (1-zang)*func[ipharef*NF+iang].a_P22 + zang*func[ipharef*NF+iang+1].a_P22;
			P22_ray     = (1-zang)*func[0      *NF+iang].a_P22 + zang*func[0      *NF+iang+1].a_P22;
			P12_aer_ref = (1-zang)*func[ipharef*NF+iang].a_P12 + zang*func[ipharef*NF+iang+1].a_P12;
			P12_ray     = (1-zang)*func[0      *NF+iang].a_P12 + zang*func[0      *NF+iang+1].a_P12;
			P_ref       = (P11_ray+P22_ray+2.F*P12_ray) * pmol + (P11_aer_ref+P22_aer_ref+2.F*P12_aer_ref) * (1.-pmol);
		}

		for (int k=0; k<NLOWd; k++) {
			ph->weight_sca[k] *= __fdividef(get_OD(1,prof[ph->layer   + k*DL*(layer_end+1)]) - 
                                            get_OD(1,prof[ph->layer-1 + k*DL*(layer_end+1)]) , 
											get_OD(1,prof[ph->layer   + ph->ilam*(layer_end+1)]) - 
                                            get_OD(1,prof[ph->layer-1 + ph->ilam*(layer_end+1)]));
			if (pmol < 1.) {
				int iphak    = prof[ph->layer + k*DL*(layer_end+1)].iphase + 1; 
				float pmol_k = prof[ph->layer + k*DL*(layer_end+1)].pmol;
				// Phase functions of particles  at other wavelengths, molecular is supposed to be constant with wavelength
				float P11_aer = (1-zang)*func[iphak*NF+iang].a_P11 + zang*func[iphak*NF+iang+1].a_P11;
				float P22_aer = (1-zang)*func[iphak*NF+iang].a_P22 + zang*func[iphak*NF+iang+1].a_P22;
				float P12_aer = (1-zang)*func[iphak*NF+iang].a_P12 + zang*func[iphak*NF+iang+1].a_P12;
				// Phase functions of the mixture of particles and molecules at other wavelengths
				float P_k   = (P11_ray+P22_ray+2.F*P12_ray) * pmol_k + (P11_aer+P22_aer+2.F*P12_aer) * (1.-pmol_k);
				ph->weight_sca[k] *= __fdividef(P_k, P_ref);
			}
		}
	}

    #endif

	if (!le){
		if (RRd==1){
			/** Russian roulette for propagating photons **/
			if( ph->weight < WEIGHTRRd ){
				if( RAND < __fdividef(ph->weight,WEIGHTRRd) ){ph->weight = WEIGHTRRd;}
				else{ph->loc = ABSORBED;}
			}
		}
		if (ph->scatterer != CHLFLUO) { modifyUV( ph->v, ph->u, cTh, psi, &ph->v, &ph->u) ;}
	}
    else {
		
        ph->weight /= fabs(ph->v.z); 

    }

	ph->scatterer = UNDEF;
	
}


__device__ void choose_scatterer(Photon* ph,
        struct Profile *prof_atm, struct Profile *prof_oc,
		struct Spectrum *spectrum,
        struct RNG_State *rngstate) {

	//int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
	
    ph->nint += 1;
  
	float pmol;
	float pine;
	
	if(ph->loc!=OCEAN){
		/* Scattering in atmosphere */
		pmol = 1.f - prof_atm[ph->layer+ph->ilam*(NATMd+1)].pmol;
		/* Elastic scattering    */
		if ( pmol < RAND ){
			ph->scatterer = RAY; // Rayleigh index
		} else {
			ph->scatterer = PTCLE;	; // particle index
		}	
	}else{
		/* Scattering in ocean */
		pmol = 1.f - prof_oc[ph->layer+ph->ilam*(NOCEd+1)].pmol;
		pine = 1.f - prof_oc[ph->layer+ph->ilam*(NOCEd+1)].pine;

		if (pine  < RAND){
			/* inelastic scattering    */
			ph->scatterer =CHLFLUO ;
		}else{
			/* Elastic scattering    */
			if ( pmol < RAND ){
				ph->scatterer = RAY; // Rayleigh index
			} else {
				ph->scatterer = PTCLE;	; // particle index
			}
		}

	}
	

	if (ph->scatterer == CHLFLUO){

		////////////////////
		// Chl Fluorescence
		
		/* Wavelength change */
		float sigmac   = 10.6;
		float lambdac0 = 685.0; 
		float new_wavel;
		int new_ilam;
		float rand1 = RAND;
		float rand2 = RAND;
		new_wavel = lambdac0 + sigmac * sqrtf(-2.0*logf(RAND)) * cosf(DEUXPI * rand2);

		ph->weight /= ph->wavel / new_wavel;
		ph->wavel = new_wavel;

		if (ph->wavel > spectrum[NLAMd-1].lambda ){

			ph->weight = 0.0;
			ph->loc = ABSORBED;

		}else if (ph->wavel < spectrum[0].lambda ) {

			ph->weight = 0.0;
			ph->loc = ABSORBED;

		} else {
			
			// get new lamb index
			//new_ilam = __float2int_rd(__fdividef( (ph->wavel -  spectrum[0].lambda)* NLAMd, spectrum[NLAMd-1].lambda - spectrum[0].lambda ));
            new_ilam=0;
            while ((ph->wavel>spectrum[new_ilam].lambda) && (new_ilam<NLAMd)) new_ilam++;
			// update tau photon coordinates according to its new wavelength
			ph->tau_abs = ph->tau_abs * prof_oc[ph->layer + new_ilam *(NOCEd+1)].OD_abs / prof_oc[ph->layer + ph->ilam *(NOCEd+1)].OD_abs;				
			ph->tau     = ph->tau     * get_OD(BEERd, prof_oc[ph->layer + new_ilam *(NOCEd+1)]) /  get_OD(BEERd, prof_oc[ph->layer + ph->ilam *(NOCEd+1)]);
			ph->ilam = new_ilam;
		}

	}
	
}


__device__ void surfaceAgitee(Photon* ph, int le,
                              float* tabthv, float* tabphi, int count_level,
                              struct RNG_State *rngstate) {

	if( SIMd == ATM_ONLY){ // Atmosphère , la surface absorbe tous les photons
		ph->loc = ABSORBED;
		return;
	}
    ph->nint += 1;

    #ifdef OBJ3D
	ph->E += 1;
	#endif

	// Réflexion sur le dioptre agité
	float theta;	// Angle de deflection polaire de diffusion [rad]
	float psi;		// Angle azimutal de diffusion [rad]
	float cTh, sTh;	//cos et sin de l'angle d'incidence du photon sur le dioptre
	
	float sig, sig2  ;
	float beta = 0.F;	// Angle par rapport à la verticale du vecteur normal à une facette de vagues 
	float sBeta;
	float cBeta;
	
	float alpha ;	//Angle azimutal du vecteur normal a une facette de vagues
	
	float nind; // relative index of refrection 
	float temp;
	
    // coordinates of the normal to the wave facet in the original axis
	float3 no;
    // coordinates of the half direction vector of the wave facet in the original axis (see Walter 2007)
    float3 half;
    // coordinates of the normal to the wave facet in the local axis (Nx, Ny, Nz)
	float3 n_l;

	float rpar, rper, rparper, rparper_cross;	// Coefficient de reflexion parallèle et perpendiculaire
	float rpar2;		// Coefficient de reflexion parallèle au carré
	float rper2;		// Coefficient de reflexion perpendiculaire au carré
	float rat  ;	    // Reflection coefficient for unpolarized light
	int ReflTot;		// Flag pour la réflexion totale sur le dioptre
	float cot;			// Cosinus de l'angle de réfraction du photon
	float ncot, ncTh;	// ncot = nind*cot, ncoi = nind*cTh
	float tpar, tper, tparper, tpar2, tper2;	//
    int iter=0;
    float vzn;  // projection of V on the local vertical
    float thv, phi;
	float3 v, v_l;

    // Reflection  and Transmission Matrices
    float4x4 R, T;

    // Determination of the relative refractive index
    // a: air, b: water , Mobley 2015 nind = nba = nb/na
    // in general nind = n_t/n_i or no/ni (transmitted over incident or output versus input)
    // and sign for further computation, sign positive for upward facet normal for reflection
    float sign;
    if (ph->loc == SURF0M)  {
        nind = __fdividef(1.f,NH2Od);
        sign = -1;
    }
    else  {
        nind = NH2Od;
        sign = 1;
    }
     
	
    #ifdef SPHERIQUE
    // define 3 vectors Nx, Ny and Nz in cartesian coordinates which define a
    // local orthonormal basis at the impact point.
    // Nz is the local vertical direction, the direction of the 2 others does not matter
    // because the azimuth is chosen randomly
	float3 Nx, Ny, Nz;
	Nz = ph->pos; // Nz is the vertical at the impact point
    // Ny is chosen arbitrarily by cross product of Nz with axis X = (1,0,0)
	Ny = cross(Nz, make_float3(1.0,0.0,0.0));
    // Nx is the cross product of Ny and Nz
	Nx = cross(Ny, Nz);
	// Normalization
	Nx = normalize(Nx);
	Ny = normalize(Ny);
	Nz = normalize(Nz);

    #ifdef DEBUG
    // we check that there is no upward photon reaching surface0+
    if ((ph->loc == SURF0P) && (dot(ph->v, ph->pos) > 0)) {
        // upward photon when reaching the surface at (0+)
        printf("Warning, vzn>0 (vzn=%f) with SURF0+ in surfaceAgitee, %f %f %f %f %f %f\n", dot(ph->v, ph->pos),Nz.x,Nz.y,Nz.z, ph->pos.x,ph->pos.y,ph->pos.z);
    }
    #endif

    /* Compute the photon v vector in the local frame */
    v_l.x = dot(ph->v,Nx);
    v_l.y = dot(ph->v,Ny);
    v_l.z = dot(ph->v,Nz);

    #else
    v_l = ph->v;
    #endif

    if (ph->loc==SURF0M) v_l = ph->v;

	/** **/
    //  Estimation of the probability P of interaction of the photon with zentih angle theta with a facet of slope beta and azimut alpha	
    //  P_alpha_beta : Probability of occurence of a given azimuth and slope
    //  P_alpha_beta = P_Cox_Munk(beta) * P(alpha | beta), conditional probability, for normal incidence, independent variables and P(alpha|beta)=P(alpha)=1/2pi
    //  following Plass75:
    //  Pfacet : Probability of occurence of a facet
    //  Pfacet = projected area of the facet divided by unit area of the possible interaction surface * P_alpha_beta
    //  Pfacet = P_alpha_beta / cos(beta)
    //  for non normal incident angle, the probability of interaction between the photon and the facet is proportional to the surface of the facet seen by the photon so
    //  that is cosine of incident angle of photon on the facet theta_inc=f(alpha,beta,theta)
    //  P # Pfacet * cos(theta_inc) for cos(theta_inc) >0
    //  P = 0 for cos(theta_inc)<=0
    //  for having a true probability, one has to normalize this to 1. The A normalization factor depends on theta and is the sum on all alpha and beta with the condition
    //  cos(theta_inc)>0 (visible facet)
    //  A = Sum_0_2pi Sumr_0_pi/2 P_alpha_beta /cos(beta) cos(theta_inc) dalpha dbeta
    //  Finally P = 1/A * P_alpha_beta  /cos(beta) cos(theta_inc)


    sig2 = 0.003F + 0.00512f *WINDSPEEDd;
    sig  = sqrtf(sig2);

    /* SAMPLING */

    if (!le) {
	 if( DIOPTREd !=0 ){
        // Rough surface

        theta = DEMIPI;
        //  Computation of P_alpha_beta = P_Cox_Munk(beta) * P(alpha | beta)
        //  we draw beta first according to Cox_Munk isotropic and then draw alpha, conditional probability
        //  rejection method: to exclude unphysical azimuth (leading to incident angle theta >=PI/2)
        //  we continue until acceptable value for alpha


        while (theta >= DEMIPI) {
           iter++;
           if (iter >= 100) {
                // safety check
                #ifdef DEBUG
                printf("Warning, photon rejected in RoughSurface while loop\n");
                printf("%i  V=(%f,%f,%f) beta,alpha=(%f,%f) \n",
                        iter,
                        ph->v.x,
                        ph->v.y,
                        ph->v.z,
                        beta/PI*180,
                        alpha/PI*180
                      );
                #endif
                ph->loc = NONE;
                break;
           }
           beta = atanf( sig*sqrtf(-__logf(RAND)) );
           alpha = DEUXPI * RAND;
           sBeta = __sinf( beta );
           cBeta = __cosf( beta );

           // Normal of the facet in the local frame
           n_l.x = sign * sBeta * __cosf( alpha );
           n_l.y = sign * sBeta * __sinf( alpha );
           n_l.z = sign * cBeta;

           // Compute incidence angle //
           cTh = -(dot(n_l,v_l));
           theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), cTh ) ));
        } // while

     } else {
        // Flat surface
        beta  = 0.F;
        cBeta = 1.F;
        n_l.x   = 0.F;
        n_l.y   = 0.F;
        n_l.z   = sign;

        cTh = -(dot(n_l, v_l));
        theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), cTh ) ));
     }
    } /* not le*/

    if (le) {
     float sign_le = 1.F;
     if (count_level==DOWN0M) sign_le = -1.F;
     phi = tabphi[ph->iph];
     thv = tabthv[ph->ith];
     v.x  = cosf(phi) * sinf(thv);
     v.y  = sinf(phi) * sinf(thv);
     v.z  = sign_le * cosf(thv);  
     
     // Normal to the facet in the global frame
     // We refer to Walter 2007
     // i : input unit vector, directed outward facet, so i=-ph->v
     // o : output unit vector, so o=v

     // 1) Determination of the half direction vector
     if ((ph->loc==SURF0P) && (count_level==DOWN0M) ||
         (ph->loc==SURF0M) && (count_level==UP0P))   { // Refraction geometry
        // vector equation for determining the half direction half = - (no*o + ni*i)
        // or half = - (nind*o + i)
        // The convention in Walter is h pointing towards the medieum with lowest index of refraction
        /*****/
        // So
		 half = operator-(v*nind, ph->v) *(-1.F*sign);
         // test : exclude facets whose normal are not on the same side as incoming photons
         if ((half.z * sign) < 0) {
             ph->loc=REMOVED;
             return;
         }
     }
     if ((ph->loc==SURF0P) && (count_level==UP0P) ||
         (ph->loc==SURF0M) && (count_level==DOWN0M)) { // Reflection geometry
        // vector equation for determining the half direction h = (o + i)
		 half = operator-(v, ph->v);
     }


     // 2) Normalization of the half direction vector: facet normal unit vector
     no=normalize(half);
     //no=normalize(no);

     // Incidence angle
     cTh = fabs(-dot(no, ph->v));
     theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), cTh ) ));

     #ifdef SPHERIQUE
     // facet slope
     cBeta = fabs(dot(no, Nz));
     beta  = fabs(acosf(cBeta));
     #else
     cBeta = fabs(no.z);
     beta  = acosf(no.z);
     #endif
	 if( (DIOPTREd == 0) && (fabs(beta) >= 1e-6)) {  //  for a flat ocean beta shall be stricly zero 
        ph->weight = 0.;
        return;
     }
    
    } /*le*/

	sTh = __sinf(theta);

    // express the coordinates of the normal to the wave facet in the original
    // axis instead of local axis (Nx, Ny, Nz)
    if (!le) {
    #ifdef SPHERIQUE
	no = operator+(operator+(n_l.x*Nx, n_l.y*Ny), n_l.z*Nz);
    #else
    no = n_l;
    #endif
    }


    #ifdef SPHERIQUE
    // avz is the projection of V on the local vertical
	float avz = fabs(dot(Nz, ph->v));
    #else
    float avz = fabs(ph->v.z);
    #endif

	// Rotation of Stokes parameters

	// temp = dot(cross(ph->v,ph->u),normalize(cross(ph->v,no)));
    // Simplification :
	temp = __fdividef(dot(no, ph->u), sTh);
	psi = acosf( fmin(1.00F, fmax( -1.F, temp ) ));	

	if( dot(no, cross(ph->u, ph->v)) <0 ){
		psi = -psi;
	}

    rotateStokes(ph->stokes, psi, &ph->stokes);
    #ifdef BACK
    float4x4 L = make_diag_float4x4 (1.F);
    rotationM(-psi,&L);
    float tpar_b, tper_b, tpar2_b, tper2_b, tparper_b;
    float rpar2_b, rper2_b, rat_b;
    float cTh_b;
    #endif

	if( sTh<=nind){
		temp = __fdividef(sTh,nind);
		cot = sqrtf( 1.0F - temp*temp );
		ncTh = nind*cTh;
		ncot = nind*cot;
		rpar = __fdividef(ncTh - cot,ncTh  + cot); // DR Mobley 2015 sign convention
		rper = __fdividef(cTh - ncot,cTh + ncot);
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
        rparper = rpar * rper;
        rparper_cross = 0.F;
		tpar = __fdividef( 2.F*cTh,ncTh+ cot);
		tper = __fdividef( 2.F*cTh,cTh+ ncot);
        tpar2= tpar * tpar;
        tper2= tper * tper;
        tparper = tpar * tper;
        // DR rat is the energetic reflection factor used to normalize the R and T matrix (see Xun 2014)
        #ifdef BIAS
		rat =  __fdividef(rpar2 + rper2, 2.F);
        #else
        rat =  __fdividef(ph->stokes.x*rpar2 + ph->stokes.y*rper2, ph->stokes.x+ph->stokes.y);
        #endif
		ReflTot = 0;
        #ifdef BACK
        // in backward mode, nind -> 1/nind and incidence angle <-> emergence angle
        cTh_b = cot;
        float cot_b = cTh;
        float nind_b= 1.F/nind;
        float ncTh_b  = nind_b * cTh_b;
        float ncot_b  = nind_b * cot_b;
		rpar2_b = __fdividef(ncTh_b - cot_b,ncTh_b  + cot_b);
        rpar2_b *= rpar2_b;
		rper2_b = __fdividef(cTh_b - ncot_b,cTh_b + ncot_b);
        rper2_b *= rper2_b;
        rat_b = __fdividef(rpar2_b + rper2_b, 2.F);
		tpar_b = __fdividef( 2.F*cTh_b,ncTh_b+ cot_b);
		tper_b = __fdividef( 2.F*cTh_b,cTh_b+ ncot_b);
        tpar2_b= tpar_b * tpar_b;
        tper2_b= tper_b * tper_b;
        tparper_b = tpar_b * tper_b;
        #endif
	}
	else{
		cot = 0.f;
		rpar = 1.f;
		rper = 1.f;
        rat = 1.f;
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
        rparper = __fdividef(2.*sTh*sTh*sTh*sTh, 1.-(1.+nind * nind)*cTh*cTh) - 1.; //  Mobley 2015
        rparper_cross = -__fdividef(2.*cTh*sTh*sTh*sqrtf(sTh*sTh-nind*nind), 1.-(1.+nind * nind)*cTh*cTh); //  Mobley 2015
        tpar = 0.;
        tper = 0.;
        tpar2 =0.;
        tper2 =0.;
        tparper =0.;
		ReflTot = 1;
	}

    // Weighting
    float p,qv,LambdaS,LambdaR,jac;

    // Lambda shadowing Source direction
    LambdaS  =  LambdaM(avz,sig2*0.5);

    //
    // Local Estimate part
    //
    if (le && (DIOPTREd!=0)) {
     // The weight depends on the normalized VISIBLE interaction PDF qv (Ross 2005) 
     // Compute p 
     float cBeta2 = cBeta*cBeta;
     p =  __fdividef( __expf(-(1.F-cBeta2)/(cBeta2*sig2)) , cBeta2*cBeta * sig2); 

     if ((ph->loc==SURF0P) && (count_level==UP0P) ||
         (ph->loc==SURF0M) && (count_level==DOWN0M)) { // Reflection geometry
            qv  = __fdividef(p * fabs(cTh), cBeta * fabs(v.z));
            // Multiplication by the reflection Jacobian
            jac = __fdividef(1.F, 4.F * fabs(cTh) );
     }
     if ((ph->loc==SURF0P) && (count_level==DOWN0M) ||
         (ph->loc==SURF0M) && (count_level==UP0P))   { // Refraction geometry
            if (sTh <= nind) {
                qv  =  __fdividef(p * fabs(cTh), cBeta * fabs(v.z));
                // Multiplication by the refraction Jacobian
                #ifndef BACK
                jac = __fdividef(nind*nind * cot, (ncot - cTh)*(ncot - cTh)); // See Zhai et al., 2010
                #else
                jac = __fdividef(cTh, nind*nind * (cTh/nind - cot)*(cTh/nind - cot));
                #endif
            }
            else qv = 0.F;
     }

     // Reflected/Refracted direction, Normalization of qv
     LambdaR  =  LambdaM(fabs(v.z),sig2*0.5);

     float norma;
     if (WAVE_SHADOWd) norma = 1. + LambdaS + LambdaR;
     else norma = 1. + LambdaS;
     qv /= norma;

     // apply the BRDF to the weight
     ph->weight *= __fdividef(qv * jac , avz);

    } /*le */

    //
    // Propagation part
    //

    // 1. Reflection
    int condR=1;
    if (!le) condR = (SURd==3)&&(RAND<rat);
	if (  (!le && (condR || (SURd==1) || ReflTot) )
       || ( le && (ph->loc==SURF0M) && (count_level == DOWN0M) )
       || ( le && (ph->loc==SURF0P) && (count_level == UP0P)   )
       ){	// Reflection

	    R= make_float4x4(
		    rpar2, 0., 0., 0.,
		    0., rper2, 0., 0.,
		    0., 0.,  rparper, rparper_cross,
		    0., 0., -rparper_cross, rparper
		    );

        ph->stokes = mul(R,ph->stokes);
        #ifndef BIAS
        ph->weight *= ph->stokes.x + ph->stokes.y;
		operator/=(ph->stokes, ph->stokes.x + ph->stokes.y); 
        #endif

        #ifdef BACK
        ph->M   = mul(ph->M,mul(L,R));
        #endif
		
        if (le) { ph->v = v; }
        else { operator+=(ph->v, (2.F*cTh)*no); }

		ph->u = operator/(operator-(no, cTh*ph->v), sTh);	

        //  Normalization of the reflexion matrix
        //  the reflection coefficient is taken into account:
        //  once in the random selection (Rand < rat)
        //  once in the reflection matrix multiplication
        //  so twice and thus we normalize by rat (Xun 2014).
        //  not to be applied for forced reflection (SUR=1 or total reflection) where there is no random selection
		if (SURd==3 && !ReflTot && !le) {
            ph->weight /=rat;
			}

        #ifdef SPHERIQUE
        vzn = dot(ph->v, Nz);
        #else
        vzn = ph->v.z;
        #endif

        //
        // photon next location
        //
        if (ph->loc == SURF0P) {
            if (vzn > 0) {  // avoid multiple reflexion above the surface
                // SURF0P becomes ATM or SPACE
                if( SIMd==SURF_ONLY || SIMd==OCEAN_SURF ){
                    ph->loc = SPACE;
                } else{
                    ph->loc = ATMOS;
                    ph->layer = NATMd;
                }
            } // else, no change of location
            else if (SINGLEd) ph->loc = REMOVED;
        } else {
            if (vzn < 0) {  // avoid multiple reflexion under the surface
               // SURF0M becomes OCEAN or ABSORBED
               if( SIMd==SURF_ATM ){
                  ph->loc = ABSORBED;
               } else{
                  ph->loc = OCEAN;
                  ph->layer = 0;
               }
            } // else, no change of location
            else if (SINGLEd) ph->loc = REMOVED;
        }

		#ifdef OBJ3D
		if (ph->loc == OBJSURF)
		{
			if (vzn > 0) {ph->loc = ATMOS;}
			else {ph->loc = OCEAN;}
		}
		#endif

     } // Reflection


    // 2. Transmission
    else if (  (!le && !condR) 
        || ( le && (ph->loc==SURF0M) && (count_level == UP0P  ) && !ReflTot )
        || ( le && (ph->loc==SURF0P) && (count_level == DOWN0M) && !ReflTot )
        ){	// Transmission

        T= make_float4x4(
        tpar2, 0., 0., 0.,
        0., tper2, 0., 0.,
        0., 0., tparper, 0.,
        0., 0., 0., tparper
        );
        ph->stokes = mul(T,ph->stokes);

        #ifndef BACK
        float geo_trans_factor = nind* cot/cTh; // DR Mobley 2015 OK , see Xun 2014, Zhai et al 2010
        #else
        float geo_trans_factor = 1./nind* cTh/cot;
        #endif
        ph->weight *= geo_trans_factor;

        #ifndef BIAS
        ph->weight *= ph->stokes.x + ph->stokes.y;
	    operator/=(ph->stokes, ph->stokes.x + ph->stokes.y); 
        #endif

        #ifdef BACK
        /* for reciprocity of transmission function see Walter 2007 */
        float4x4 T_b= make_float4x4(
        tpar2_b, 0., 0., 0.,
        0., tper2_b, 0., 0.,
        0., 0., tparper_b, 0.,
        0., 0., 0., tparper_b
        );
        ph->M   = mul(ph->M,mul(L,T_b));
        #endif
    
        alpha  = __fdividef(cTh, nind) - cot;

        if (le) { ph->v = v; }
        else { ph->v = operator+(operator/(ph->v, nind), alpha*no); }
        ph->u = operator/(operator+(no, cot*ph->v), sTh )*nind;

        #ifdef SPHERIQUE
        vzn = dot(ph->v, Nz);
        #else
        vzn = ph->v.z;
        #endif


        // Normalization of the transmission matrix
        // the transmission coefficient is taken into account:
        // once in the random selection (Rand > rat)
        // once in the transmission matrix multiplication
        // so we normalize by (1-rat) (Xun 2014).
        // Not to be applied for forced transmission (SUR=2)
        if ( (SURd == 3 ) && !le) 
            #ifndef BACK
            ph->weight /= (1-rat);
            #else
            ph->weight /= (1-rat_b);
            #endif

        //
        // photon next location
        //
        if (ph->loc == SURF0M) {
         if (vzn > 0) {
            // SURF0P becomes ATM or SPACE
            if( SIMd==SURF_ONLY || SIMd==OCEAN_SURF ){
                ph->loc = SPACE;
            } else{
                ph->loc = ATMOS;
                ph->layer = NATMd;
            }
         } else {
            // multiple transmissions (vz<0 after water->air transmission)
            ph->loc = SURF0P;
            if (SINGLEd) ph->loc = REMOVED;
         }
        } else {
           if (vzn < 0) {  // avoid multiple reflexion under the surface
              // SURF0M becomes OCEAN or ABSORBED
              if( SIMd==SURF_ONLY || SIMd==SURF_ATM ){
                ph->loc = ABSORBED;
              } else{
                ph->loc = OCEAN;
                ph->layer = 0;
              }
           } else {
              // multiple transmissions (vz<0 after water->air transmission)
              // (for symmetry, but should not happen)
              ph->loc = SURF0M;
              if (SINGLEd) ph->loc = REMOVED;
           }
        }
	} // Transmission

    LambdaR  =  LambdaM(fabs(ph->v.z),sig2*0.5);

    if (!le) {
        if (WAVE_SHADOWd) ph->weight *= __fdividef(fabs(cTh), cBeta * (1.F + LambdaS + LambdaR) * avz );
        else              ph->weight *= __fdividef(fabs(cTh), cBeta * (1.F + LambdaS) * avz );
        // Ross et al 2005, Ross and Dion, 2007, Zeisse 1995
        // Slope sampling bias correction using the normalized interaction PDF q
        // weight has to be multiplied by q/p, where p is the slope PDF
        // Coefficient Lambda for normalization of q taking into acount slope shadowing and hiding
        // Including wave shadows is performed at the end after the outgoing direction is calculated

		if (RRd==1){
			/* Russian roulette for propagating photons **/
			if( ph->weight < WEIGHTRRd ){
				if( RAND < __fdividef(ph->weight,WEIGHTRRd) ){ph->weight = WEIGHTRRd;}
				else{ph->loc = ABSORBED;}
			}
		}
    }

}

/* Fresnel Reflection Matrix*/
__device__ float4x4 FresnelR(float3 vi, float3 vr) {

    float cTh, sTh, cot, ncTh, ncot, theta, temp;
    float rpa, rpe, rpape, rpa2, rpe2, rpape_c;
    float4x4 R;
    // Determination of the relative refractive index
    // a: air, b: water , Mobley 2015 nind = nba = nb/na
    // and sign for further computation
    float nind = NH2Od;
    // vector equation for determining the half direction h = sign(i dot o) (i + o)
	float3 no = operator-(vr, vi);
    // Normalization of the half direction vector
    no=normalize(no);
    // Incidence angle in the local frame
    cTh   = fabs(dot(no, vi));
    theta = acosf(fmin(1.F-VALMIN, fmax(-(1.F-VALMIN), cTh)));
    sTh   = sinf(theta);
    // Fresnel coefficients
	temp    = __fdividef(sTh, nind);
	cot     = sqrtf(1.0F - temp*temp);
	ncTh    = nind*cTh;
	ncot    = nind*cot;
	rpa     = __fdividef(ncTh - cot, ncTh  + cot); // DR Mobley 2015 sign convention
	rpe     = __fdividef(cTh - ncot, cTh + ncot);
	rpa2    = rpa*rpa;
	rpe2    = rpe*rpe;
    rpape   = rpa*rpe;
    rpape_c = 0.F;

	R   = make_float4x4(
	          rpa2, 0.  , 0.     , 0.,
	          0.  , rpe2, 0.     , 0.,
	          0.  , 0.  , rpape  , rpape_c,
	          0.  , 0.  ,-rpape_c, rpape
	          );

    return R;
}

/* Surface BRDF */
__device__ void surfaceBRDF_old(Photon* ph, int le,
                              float* tabthv, float* tabphi, int count_level,
                              struct RNG_State *rngstate) {
	
	if( SIMd == ATM_ONLY){ // Atmosphere only, surface absorbs all
		ph->loc = ABSORBED;
		return;
	}
    ph->nint += 1;

    #ifdef OBJ3D
	ph->E += 1;
    #endif

	// Réflexion sur le dioptre agité
	float theta;	// Angle de deflection polaire de diffusion [rad]
	float psi;		// Angle azimutal de diffusion [rad]
	float cTh, sTh;	//cos et sin de l'angle d'incidence du photon sur le dioptre
	
	float sig2;
	float cBeta;
	
	float nind; // relative index of refrection 
	float temp;
	
    // coordinates of the normal to the wave facet in the original axis
	float3 no;

	float rpar, rper, rparper, rparper_cross;	// Coefficient de reflexion parallèle et perpendiculaire
	float rpar2;		// Coefficient de reflexion parallèle au carré
	float rper2;		// Coefficient de reflexion perpendiculaire au carré
	float cot;			// Cosinus de l'angle de réfraction du photon
	float ncot, ncTh;	// ncot = nind*cot, ncoi = nind*cTh
    float thv, phi;
	float3 v;

    // Reflection  and Transmission Matrices
    float4x4 R;

    // Determination of the relative refractive index
    // a: air, b: water , Mobley 2015 nind = nba = nb/na
    // and sign for further computation
    nind = NH2Od;
     
    #ifdef SPHERIQUE
    // define 3 vectors Nx, Ny and Nz in cartesian coordinates which define a
    // local orthonormal basis at the impact point.
    // Nz is the local vertical direction, the direction of the 2 others does not matter
    // because the azimuth is chosen randomly
	float3 Nx, Ny, Nz;
	Nz = ph->pos; // Nz is the vertical at the impact point
    // Ny is chosen arbitrarily by cross product of Nz with axis X = (1,0,0)
	Ny = cross(Nz, make_float3(1.0,0.0,0.0));
    // Nx is the cross product of Ny and Nz
	Nx = cross(Ny, Nz);
	// Normalizatioin
	Nx = normalize(Nx);
	Ny = normalize(Ny);
	Nz = normalize(Nz);
    #endif

    sig2 = 0.003F + 0.00512f *WINDSPEEDd;

    // Rough surface
    if (le) {
     phi = tabphi[ph->iph];
     thv = tabthv[ph->ith];
    }
    else {
	 phi = RAND*DEUXPI;
	 thv = acosf(sqrtf( RAND ));
    }

    v.x  = cosf(phi) * sinf(thv);
    v.y  = sinf(phi) * sinf(thv);
    v.z  = cosf(thv);  
     
    // vector equation for determining the half direction h = sign(i dot o) (i + o)
	no = operator-(v, ph->v);

    // 2) Normalization of the half direction vector
    no=normalize(no);

    // Incidence angle in the local frame
    cTh = fabs(dot(no, ph->v));
    theta = acosf(fmin(1.F-VALMIN, fmax(-(1.F-VALMIN), cTh)));

    #ifdef SPHERIQUE
    // facet slope
    cBeta = fabs(dot(no, Nz));
    #else
    cBeta = fabs(no.z);
    #endif


    #ifdef SPHERIQUE
    // avz is the projection of V on the local vertical
	float avz = fabs(dot(Nz, ph->v));
    #else
    float avz = fabs(ph->v.z);
    #endif

	// Rotation of Stokes parameters
	sTh  = __sinf(theta);
    temp = __fdividef(dot(no, ph->u), sTh);
	psi  = acosf(fmin(1.F, fmax(-1.F, temp)));	

	if( dot(no, cross(ph->u, ph->v)) <0 ){
		psi = -psi;
	}

    rotateStokes(ph->stokes, psi, &ph->stokes);
    #ifdef BACK
    float4x4 L = make_diag_float4x4(1.F);
    rotationM(-psi,&L);
    #endif

	temp = __fdividef(sTh,nind);
	cot = sqrtf( 1.0F - temp*temp );
	ncTh = nind*cTh;
	ncot = nind*cot;
	rpar = __fdividef(ncTh - cot,ncTh  + cot); // DR Mobley 2015 sign convention
	rper = __fdividef(cTh - ncot,cTh + ncot);
	rpar2 = rpar*rpar;
	rper2 = rper*rper;
    rparper = rpar * rper;
    rparper_cross = 0.F;

    // BRDF Weighting
    float cBeta2 = cBeta*cBeta;
    ph->weight *=  __fdividef( __expf(-(1.F-cBeta2)/(cBeta2*sig2)), 4.F * cBeta2*cBeta2 * avz * fabs(v.z) * sig2);

	R= make_float4x4(
	   rpar2, 0., 0., 0.,
	   0., rper2, 0., 0.,
	   0., 0., rparper, rparper_cross,
	   0., 0., -rparper_cross, rparper
	   );

    ph->stokes = mul(R,ph->stokes);
    #ifndef BIAS
    ph->weight *= ph->stokes.x + ph->stokes.y;
	operator/=(ph->stokes, ph->stokes.x + ph->stokes.y); 
    #endif

    #ifdef BACK
    ph->M   = mul(ph->M,mul(L,R));
    #endif
		
    ph->v = v;
	ph->u = operator/(operator-(no, cTh*ph->v), sTh);	

        // photon next location
    if( SIMd==SURF_ONLY || SIMd==OCEAN_SURF ){
        ph->loc = SPACE;
    } else {
          ph->loc = ATMOS;
          ph->layer = NATMd;
    }

    if (WAVE_SHADOWd) {
        // Add Wave shadowing
        // compute wave shadow outgoing photon
        float LambdaR, LambdaS;
        LambdaS  =  LambdaM(avz,sig2*0.5);
        LambdaR  =  LambdaM(fabs(v.z),sig2*0.5);
        ph->weight *= __fdividef(1.F, 1.F + LambdaR + LambdaS);
    }

    if (!le) {
		if (RRd==1){
			// Russian roulette for propagating photons 
			if( ph->weight < WEIGHTRRd ){
				if( RAND < __fdividef(ph->weight,WEIGHTRRd) ){ph->weight = WEIGHTRRd;}
				else{ph->loc = ABSORBED;}
			}
		}
    }

}

/* Surface Lambert */
__device__ void surfaceLambert(Photon* ph, int le,
                              float* tabthv, float* tabphi, struct Spectrum *spectrum,
                              struct RNG_State *rngstate) {
	
	if( SIMd == ATM_ONLY){ // Atmosphere only, surface absorbs all
		ph->loc = ABSORBED;
		return;
	}

    ph->nint += 1;
	
	#ifdef OBJ3D
	ph->direct += 1;
	ph->E += 1;
	#endif
  	
    float thv, phi;
	float3 v_n, u_n; // photon outgoing direction in the LOCAL frame

    #ifdef SPHERIQUE
    // define 3 vectors Nx, Ny and Nz in cartesian coordinates which define a
    // local orthonormal basis at the impact point.
    // Nz is the local vertical direction, the direction of the 2 others does not matter
    // because the azimuth is chosen randomly
	float3 Nx, Ny, Nz;
    MakeLocalFrame(ph->pos, &Nx, &Ny, &Nz);
    #endif

    /***************************************************/
    /* Computation of outgoing direction */
    /***************************************************/
    if (le) {
     // Outgoing direction in GLOBAL frame
     phi = tabphi[ph->iph];
     thv = tabthv[ph->ith];
     DirectionToUV(thv, phi, &ph->v, &ph->u);

     #ifdef SPHERIQUE
     float weight = dot(ph->v, Nz);
     if ((weight <= 0.) && (ph->loc != SEAFLOOR)) { /*[Eq. 40]*/
         ph->loc = ABSORBED;
         return;
     }
     else ph->weight *= weight/fabs(ph->v.z); /*[Eq. 39]*/
     #endif
    }
    else {
     // Cosine of the LOCAL zenith angle sampling for Lambertian reflector
	 phi = RAND*DEUXPI;
	 thv = acosf(sqrtf(RAND));
     DirectionToUV(thv, phi, &v_n, &u_n);

     // Computation of the outgoing direction in GLOBAL frame
     #ifdef SPHERIQUE
     if (ph->loc == SEAFLOOR) {
         ph->v = v_n;
         ph->u = u_n;
     }
     else {
         /* LOCAL to GLOBAL frame */
         ph->v = LocalToGlobal(Nx, Ny, Nz, v_n);
         ph->u = LocalToGlobal(Nx, Ny, Nz, u_n);
     }
     #else
     ph->v = v_n;
     ph->u = u_n;
     #endif
    }

    /***************************************************/
    /* Update of Stokes vector  */
    /***************************************************/
    // Reflection Matrix
    float4x4 RL = make_float4x4(
                    0.5F, 0.5F, 0.F, 0.F,
                    0.5F, 0.5F, 0.F, 0.F,
                    0.0F, 0.0F, 0.F, 0.F,
                    0.0F, 0.0F, 0.F, 0.F 
            );
    ph->stokes = mul(RL, ph->stokes); /*[Eq. 15,39]*/

    #ifdef BACK
    ph->M = mul(ph->M, RL);
    #endif

    #ifdef OBJ3D
	ph->loc = ATMOS;
	#else
	/***************************************************/
	/* Update of photon location and weight */
	/***************************************************/
	if (ph->loc == SURF0P){
		bool test_s = ( SIMd == SURF_ONLY);
		ph->loc = SPACE*test_s + ATMOS*(!test_s);
		ph->layer = NATMd;
		ph->weight *= spectrum[ph->ilam].alb_surface;  /*[Eq. 16,39]*/
	}
	else
	{
		ph->loc = OCEAN;
		ph->layer = NOCEd; 
		ph->weight *= spectrum[ph->ilam].alb_seafloor; /*[Eq. 16,39]*/
	}
    #endif

} //surfaceLambert

/* Surface BRDF */
__device__ void surfaceBRDF(Photon* ph, int le,
							float* tabthv, float* tabphi, int count_level,
							struct RNG_State *rngstate) {
	if( SIMd == ATM_ONLY){ // Atmosphere only, surface absorbs all
		ph->loc = ABSORBED;
		return;
	}
    ph->nint += 1;
	
    float thv, phi, psi;
    float sig2, temp;
	float3 vr; // photon outgoing direction in the LOCAL frame
	float3 vi; // photon ingoing  direction in the LOCAL frame
	float3 v , u;  // photon outgoing direction in the GLOBAL frame
    float3 no_n, no; // normal to the facet LOCAL and GLOBAL frame
    float3 w_ne, w_ol;
    float cBeta2; //facet slope squared
    float4x4 R; // Fresnel Reflection Matrix

    #ifdef SPHERIQUE
    // define 3 vectors Nx, Ny and Nz in cartesian coordinates which define a
    // local orthonormal basis at the impact point.
    // Nz is the local vertical direction, the direction of the 2 others does not matter
    // because the azimuth is chosen randomly
	float3 Nx, Ny, Nz;
    float weight;
    MakeLocalFrame(ph->pos, &Nx, &Ny, &Nz);
    /* Transformation of ingoing direction in the local frame*/
    vi = GlobalToLocal(Nx, Ny, Nz, ph->v);
    #else
    vi = ph->v;
    #endif

    /***************************************************/
    /* Computation of outgoing direction */
    /***************************************************/
    if (le) {
     // Outgoing direction in GLOBAL frame
     phi = tabphi[ph->iph];
     thv = tabthv[ph->ith];
     DirectionToUV(thv, phi, &v, &u);
     #ifdef SPHERIQUE
     // Test if outgoing direction is in Earth s shadow
     if ((dot(v, Nz) <= 0.) && (ph->loc != SEAFLOOR)) { /*[Eq. 40]*/
         ph->loc = ABSORBED;
         return;
     }
     /*Transformation in the local frame*/
     vr = GlobalToLocal(Nx, Ny, Nz, v);
     #else
     vr = v;
     #endif
    }

    else {
     // Cosine of the LOCAL zenith angle sampling for Lambertian reflector
	 phi = RAND*DEUXPI;
	 thv = acosf(sqrtf(RAND));
     DirectionToUV(thv, phi, &vr, &u);
    }

    // Computation of the outgoing direction in GLOBAL frame
    #ifdef SPHERIQUE
    if (ph->loc == SEAFLOOR) {
         v = vr;
    }
    else {
         /* LOCAL to GLOBAL frame */
         v = LocalToGlobal(Nx, Ny, Nz, vr);
    }
    #else
    v = vr;
    #endif

    /***************************************************/
    /* Computation of slope and weight */
    /***************************************************/
	no_n   = operator-(vr, vi);
    no_n   = normalize(no_n);
    cBeta2 = no_n.z * no_n.z;
    sig2   = 0.003F + 0.00512F * WINDSPEEDd;
    if (le) ph->weight *= __fdividef( __expf(-(1.F-cBeta2)/(cBeta2*sig2)), 4.F * cBeta2*cBeta2 * fabs(vi.z) * fabs(v.z)  * sig2);
    else    ph->weight *= __fdividef( __expf(-(1.F-cBeta2)/(cBeta2*sig2)), 4.F * cBeta2*cBeta2 * fabs(vi.z) * fabs(vr.z) * sig2);

    if (WAVE_SHADOWd) {
        // Add wave shadowing computed in the local frame
        float LambdaR, LambdaS;
        LambdaS = LambdaM(fabs(vi.z), sig2*0.5);
        LambdaR = LambdaM(fabs(vr.z), sig2*0.5);
        ph->weight *= __fdividef(1.F, 1.F + LambdaR + LambdaS);
    }


    /***************************************************/
    /* Update of Stokes vector  */
    /***************************************************/
	// Psi determination
    #ifdef SPHERIQUE
    no   = LocalToGlobal(Nx, Ny, Nz, no_n);
    #else
    no   = no_n;
    #endif
    float cTh = fabs(dot(no, ph->v));
    float theta = acosf(fmin(1.F-VALMIN, fmax(-(1.F-VALMIN), cTh)));
    float sTh = sinf(theta);
    temp = __fdividef(dot(no, ph->u), sTh);
	psi  = acosf(fmin(1.F, fmax(-1.F, temp)));	
	if(dot(no, cross(ph->u, ph->v)) < 0 ) psi = -psi;

    /*w_ne = normalize(cross(ph->v, no));
    w_ol = cross(ph->v, ph->u);
    temp = dot(w_ol, w_ne);
	psi  = acosf(fmin(1.F, fmax(-1.F, temp)));	
	if(dot(no, cross(ph->u, ph->v)) < 0 ) psi = -psi;*/

    // Stokes rotation
    rotateStokes(ph->stokes, psi, &ph->stokes);
    #ifdef BACK
    float4x4 L;
    rotationM(-psi, &L);
    #endif

    /*float3x3 M = rotation3D(psi, ph->v);
    u = mul(M, ph->u);
    M = rotation3D(acosf(dot(vr, -vi)), w_ne);
    u = mul(M, u);*/

    // Fresnel Matrix Multiplication
    R = FresnelR(vi, vr);
    ph->stokes = mul(R, ph->stokes);

    #ifdef BACK
    ph->M = mul(ph->M, mul(L, R));
    #endif

    /***************************************************/
    /* Update of photon direction, location  */
    /***************************************************/
	u     = operator/(operator-(no, cTh*ph->v), sTh);	
    ph->v = v;
    ph->u = u;

    if (SIMd==SURF_ONLY || SIMd==OCEAN_SURF){
        ph->loc   = SPACE;
    } else {
        ph->loc   = ATMOS;
        ph->layer = NATMd;
    }

    // Russian roulette for propagating photons 
    if (!le && RRd==1) {
		if (ph->weight < WEIGHTRRd){
			if (RAND < __fdividef(ph->weight,WEIGHTRRd)) ph->weight = WEIGHTRRd;
			else ph->loc = ABSORBED;
		}
    }

} //surfaceBRDF

#ifdef OBJ3D
__device__ void surfaceLambertienne3D(Photon* ph, int le, float* tabthv, float* tabphi,
									  struct Spectrum *spectrum, struct RNG_State *rngstate, IGeo* geoS)
{
	ph->nint += 1;

	if (geoS->type == 1)
	{
		if (  isBackward( make_double3(geoS->normalBase.x, geoS->normalBase.y, geoS->normalBase.z),
						  make_double3(ph->v.x, ph->v.y, ph->v.z) )  ) // AV
		{ ph->H += 1; }
		else { ph->E += 1; } // AR traité comme environnement
	}
	else if ( geoS->type == 2)
	{ ph->E += 1; }
	
	float3 u_n, v_n;	// Vecteur du photon après reflexion
    float phi;
    float cTh, sTh, cPhi, sPhi;

    if (le)
	{
		cTh  = cosf(tabthv[ph->ith]);  
        phi  = tabphi[ph->iph];
    }
    else
	{
        float ddis=0.0F;
        if ((LEd==0) || (LEd==1 && RAND>ddis))
		{
            // Standard sampling
	        cTh = sqrtf( RAND );
	        phi = RAND*DEUXPI;
        }
        else
		{
            // DDIS sampling , Buras and Mayer
            float Om = 0.001;
	        cTh = sqrtf(1.F-RAND*Om);
            phi = RAND*DEUXPI;
            ph->weight *= DEUXPI*(1. -sqrtf(1.F-Om));
        }
    }

	sTh = sqrtf( 1.0F - cTh*cTh );
	cPhi = __cosf(phi);
	sPhi = __sinf(phi);
	
	/** calcul u,v new **/
	v_n.x = cPhi*sTh;
	v_n.y = sPhi*sTh;
	v_n.z = cTh;
	
	u_n.x = cPhi*cTh;
	u_n.y = sPhi*cTh;
	u_n.z = -sTh;

	// Depolarisation du Photon
    float4x4 L = make_float4x4(
                    0.5F, 0.5F, 0.F, 0.F,
                    0.5F, 0.5F, 0.F, 0.F,
                    0.0F, 0.0F, 0.F, 0.F,
                    0.0F, 0.0F, 0.F, 0.F 
		);
    ph->stokes = mul(L,ph->stokes);

    #ifdef BACK
    ph->M = mul(ph->M,L);
    #endif
	
	ph->locPrev = OBJSURF;
	ph->loc = ATMOS;
	
	// // Unit vectors which form a second base and where e3 is the geo normal
	float3 e1, e2, e3;
	e3 = normalize(geoS->normal);         // be sure that e3 is normalized
	coordinateSystem(e3, &e1, &e2);  // create e1, e2 orthogonal unit vectors 
	// e1 = normalize(e1);
	// e2 = normalize(e2);

	// The passage matrix and his transpose tM, ie. for a vector X: $X = M X' and X' = tM X$
	// - Works only if the two basis have the same origin
	float4x4 M = make_float4x4(
		e1.x, e2.x, e3.x, 0.f,
		e1.y, e2.y, e3.y, 0.f,
		e1.z, e2.z, e3.z, 0.f,
		0.f , 0.f , 0.f , 1.f
		);
	float4x4 tM = transpose(M);
		
	// Create the transforms obect to world
	Transform oTw(M, tM);
	char myV[]="Vector";

	// apply the transformation
	v_n = oTw(v_n, myV);
	u_n = oTw(u_n, myV);

	// 
	if ( (isnan(v_n.x)) || (isnan(v_n.y)) || (isnan(v_n.z)) || (isnan(u_n.x)) || (isnan(u_n.y)) || (isnan(u_n.z)) )
	{
		ph->loc = REMOVED;
		return;
	}
	
	// Update the value of u and v of the photon	
	ph->v = v_n;
	ph->u = u_n;

	ph->weight *= geoS->reflectivity;
		
    if (!le)
	{
		if (RRd==1){
			/* Russian roulette for propagating photons **/
			if( ph->weight < WEIGHTRRd ){
				if( RAND < __fdividef(ph->weight,WEIGHTRRd) ){ph->weight = WEIGHTRRd;}
				else{ph->loc = ABSORBED;}
			}
		}
	} // not le
} // Function lamb3D

__device__ void surfaceRugueuse3D(Photon* ph, IGeo* geoS, struct RNG_State *rngstate)
{
    ph->nint += 1;

	if (geoS->type == 1)
	{
		if (  isBackward( make_double3(geoS->normalBase.x, geoS->normalBase.y, geoS->normalBase.z),
						  make_double3(ph->v.x, ph->v.y, ph->v.z) )  ) // AV
		{ ph->H += 1; }
		else { ph->E += 1; } // AR traité comme environnement
	}
	else if ( geoS->type == 2)
	{ ph->E += 1; }
	
	float3 u_n, v_n;	// Vecteur du photon après reflexion

	v_n = ph->v;
	u_n = ph->u;

	// Rotation of Stokes parameters
	float temp, psi;
	
	temp = dot(cross(ph->v, ph->u),normalize(cross(ph->v, geoS->normal)));
	psi = acosf( fmin(1.00F, fmax( -1.F, temp ) ));	

	if( dot(geoS->normal, cross(ph->u, ph->v)) <0 ){
		psi = -psi;
	}
	
	rotateStokes(ph->stokes, psi, &ph->stokes);
    #ifdef BACK
    float4x4 L = make_diag_float4x4 (1.F);
    rotationM(-psi,&L);
    #endif
	
	float4x4 R = make_float4x4(
		1., 0.,  0.,  0.,
		0., 1.,  0.,  0.,
		0., 0., -1.,  0.,
		0., 0.,  0., -1. 
		);
	
	ph->stokes = mul(R, ph->stokes);
	
    #ifdef BACK
	ph->M   = mul(ph->M,mul(L,R));
    #endif
	Transform transfo, invTransfo, aRot;
	char myV[]="Vector";
	transfo = geoS->mvTF;
	aRot = aRot.RotateZ(180);
	invTransfo = transfo.Inverse(transfo);
	
	v_n = invTransfo(v_n, myV);
	u_n = invTransfo(u_n, myV);

	v_n = aRot(v_n, myV);
	u_n = aRot(u_n, myV);

	v_n = make_float3(-v_n.x, -v_n.y, -v_n.z);
	u_n = make_float3(-u_n.x, -u_n.y, -u_n.z);

	v_n = transfo(v_n, myV);
	u_n = transfo(u_n, myV);
	
	if ( (isnan(v_n.x)) || (isnan(v_n.y)) || (isnan(v_n.z)))
	{
		ph->loc = REMOVED;
		return;
	}
	
	// Update the value of u and v of the photon	
	ph->locPrev = OBJSURF;
	ph->loc = ATMOS;
	ph->v = normalize(v_n);
	ph->u = normalize(u_n);
	
	ph->weight *= geoS->reflectivity;
	
	if (RRd==1){
		/* Russian roulette for propagating photons **/
		if( ph->weight < WEIGHTRRd ){
			if( RAND < __fdividef(ph->weight,WEIGHTRRd) ){ph->weight = WEIGHTRRd;}
			else{ph->loc = ABSORBED;}
		}
	}
	
} // FUNCTION SURFACEAGITE3D


__device__ void countPhotonObj3D(Photon* ph, void *tabObjInfo, IGeo* geoS, unsigned long long *nbPhCat, double *wPhCat)
{
	Transform transfo, invTransfo;
	char myP[]="Point";
	
    double *tabCountObj;
	double weight;
	float3 p_t;

	int indI = 0;
	int indJ = 0;

	float sizeX = nbCx*TCd;
	float sizeY = nbCy*TCd;

	// if (!isBackward(geoS->normalBase, ph->v)) return;
	if (   isForward(  make_double3(geoS->normalBase.x, geoS->normalBase.y, geoS->normalBase.z),
				  make_double3(ph->v.x, ph->v.y, ph->v.z)  )   ) return;

	p_t = ph->pos;
	transfo = geoS->mvTF;
	invTransfo = transfo.Inverse(transfo);
	p_t = invTransfo(p_t, myP);

    // ancienne implementation = mauvaise
    // Transform rotz;
	// rotz = rotz.RotateZ(0);
	// p_t = rotz(p_t, myP);
	// indJ = floorf((p_t.x/TCd) + (sizeX/(2*TCd)));
	// indI = ceilf((p_t.y/TCd) + (sizeY/(2*TCd)));
	// if (indJ == nbCx) indJ -= 1;
	// indI = (nbCy-1) - (indI-1);

    // new implementation = bonne
    // x (axe vers le haut ^); y (axe vers la gauche <--)
	indJ = floorf( (-(p_t.y/TCd)) + (sizeY/(2*TCd)) );
	indI = floorf( (-(p_t.x/TCd)) + (sizeX/(2*TCd)) );
	if (indJ == nbCy) indJ -= 1;
	if (indI == nbCx) indI -= 1;
	
    #ifdef DOUBLE
	tabCountObj = (double*)tabObjInfo;
	weight = (double)ph->weight;

	if(isnan(weight))
	{
		printf("Care weight is nan !! \n");
		return;
	}

	#if __CUDA_ARCH__ >= 600
	// All the beams reaching a receiver
	atomicAdd(tabCountObj+(nbCy*indI)+indJ, weight);

	// Les huit catégories
	if (ph->H == 0 && ph->E == 0 && ph->S == 0) 
	{ // CAT 1 : aucun changement de trajectoire avant de toucher le R.
	    atomicAdd(wPhCat, weight); // comptage poids
		atomicAdd(nbPhCat, 1);     // comptage nombre de photons
		atomicAdd(tabCountObj+(nbCy*nbCx)+(nbCy*indI)+indJ, weight); // distri
	}
	else if ( ph->H > 0 && ph->E == 0 && ph->S == 0)
	{ // CAT 2 : only H avant de toucher le R.
		atomicAdd(wPhCat+1, weight);
		atomicAdd(nbPhCat+1, 1);
		atomicAdd(tabCountObj+(2*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H == 0 && ph->E > 0 && ph->S == 0)
	{ // CAT 3 : only E avant de toucher le R.
		atomicAdd(wPhCat+2, weight);
		atomicAdd(nbPhCat+2, 1);
		atomicAdd(tabCountObj+(3*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H == 0 && ph->E == 0 && ph->S > 0)
	{ // CAT 4 : only S avant de toucher le R.
		atomicAdd(wPhCat+3, weight);
		atomicAdd(nbPhCat+3, 1);
		atomicAdd(tabCountObj+(4*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H > 0 && ph->E == 0 && ph->S > 0)
	{ // CAT 5 : 2 proc. H et S avant de toucher le R.
		atomicAdd(wPhCat+4, weight);
		atomicAdd(nbPhCat+4, 1);
		atomicAdd(tabCountObj+(5*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H > 0 && ph->E > 0 && ph->S == 0)
	{ // CAT 6 : 2 proc. H et E avant de toucher le R.
		atomicAdd(wPhCat+5, weight);
		atomicAdd(nbPhCat+5, 1);
		atomicAdd(tabCountObj+(6*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
        //printf("H = %d, E = %d, S = %d", ph->H, ph->E, ph->S);
	}
	else if ( ph->H == 0 && ph->E > 0 && ph->S > 0)
	{ // CAT 7 : 2 proc. E et S avant de toucher le R.
		atomicAdd(wPhCat+6, weight);
		atomicAdd(nbPhCat+6, 1);
		atomicAdd(tabCountObj+(7*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}	
	else if ( ph->H > 0 && ph->E > 0 && ph->S > 0)
	{ // CAT 8 : 3 proc. H, E et S avant de toucher le R.
		atomicAdd(wPhCat+7, weight);
		atomicAdd(nbPhCat+7, 1);
		atomicAdd(tabCountObj+(8*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}	
	
	#else
	DatomicAdd(tabCountObj+(nbCy*indI)+indJ, weight);

	// Les huit catégories
	if (ph->H == 0 && ph->E == 0 && ph->S == 0) 
	{ // CAT 1 : aucun changement de trajectoire avant de toucher le R.
	    DatomicAdd(wPhCat, weight); // comptage poids
		atomicAdd(nbPhCat, 1);     // comptage nombre de photons
		DatomicAdd(tabCountObj+(nbCy*nbCx)+(nbCy*indI)+indJ, weight); // distri
	}
	else if ( ph->H > 0 && ph->E == 0 && ph->S == 0)
	{ // CAT 2 : only H avant de toucher le R.
		DatomicAdd(wPhCat+1, weight);
		atomicAdd(nbPhCat+1, 1);
		DatomicAdd(tabCountObj+(2*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
		printf("H = %d, E = %d, S = %d", ph->H, ph->E, ph->S);
	}
	else if ( ph->H == 0 && ph->E > 0 && ph->S == 0)
	{ // CAT 3 : only E avant de toucher le R.
		DatomicAdd(wPhCat+2, weight);
		atomicAdd(nbPhCat+2, 1);
		DatomicAdd(tabCountObj+(3*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H == 0 && ph->E == 0 && ph->S > 0)
	{ // CAT 4 : only S avant de toucher le R.
		DatomicAdd(wPhCat+3, weight);
		atomicAdd(nbPhCat+3, 1);
		DatomicAdd(tabCountObj+(4*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H > 0 && ph->E == 0 && ph->S > 0)
	{ // CAT 5 : 2 proc. H et S avant de toucher le R.
		DatomicAdd(wPhCat+4, weight);
		atomicAdd(nbPhCat+4, 1);
		DatomicAdd(tabCountObj+(5*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	else if ( ph->H > 0 && ph->E > 0 && ph->S == 0)
	{ // CAT 6 : 2 proc. H et E avant de toucher le R.
		DatomicAdd(wPhCat+5, weight);
		atomicAdd(nbPhCat+5, 1);
		DatomicAdd(tabCountObj+(6*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
        //printf("H = %d, E = %d, S = %d", ph->H, ph->E, ph->S);
	}
	else if ( ph->H == 0 && ph->E > 0 && ph->S > 0)
	{ // CAT 7 : 2 proc. E et S avant de toucher le R.
		DatomicAdd(wPhCat+6, weight);
		atomicAdd(nbPhCat+6, 1);
		DatomicAdd(tabCountObj+(7*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}	
	else if ( ph->H > 0 && ph->E > 0 && ph->S > 0)
	{ // CAT 8 : 3 proc. H, E et S avant de toucher le R.
		DatomicAdd(wPhCat+7, weight);
		atomicAdd(nbPhCat+7, 1);
		DatomicAdd(tabCountObj+(8*nbCy*nbCx)+(nbCy*indI)+indJ, weight);
	}
	#endif
    #endif
}
#endif


__device__ void countPhoton(Photon* ph,
        struct Profile *prof_atm, struct Profile *prof_oc,
        float *tabthv, float *tabphi,
        int count_level,
		unsigned long long *errorcount,
        void *tabPhotons, void *tabDist, void *tabHist, unsigned long long *NPhotonsOut
        ) {

    if (count_level < 0 || ph->loc==REMOVED || ph->loc==ABSORBED) {
        // don't count anything
        return;
    }

    // don't count the photons directly transmitted
    if (ph->nint == 0) {
        return;
    }

    // Declaration for double
    #ifdef DOUBLE 
     double *tabCount;                   // pointer to the "counting" array:
     double dweight;
	 double4 ds;                         // Stokes vector casted to double 
     #ifdef ALIS
      double dwsca, dwabs;                // General ALIS variables 
      //!!!!!!!!!
      #if ( defined(SPHERIQUE) || defined(ALT_PP) )
       double *tabCount2;                  // Specific ALIS counting array pointer for path implementation (cumulative distances)
      #endif
     #endif
     //!!!!!!!!!

    // Declaration for single
    #else                              
     float *tabCount; 
     //!!!!!!!!!
     #if ( defined(SPHERIQUE) || defined(ALT_PP) ) && defined(ALIS)
      float *tabCount2;
     //!!!!!!!!!
     #endif
    #endif

    #if ( defined(SPHERIQUE) || defined(ALT_PP) ) && defined(ALIS)
     float *tabCount3; // Specific ALIS counting array pointer for path implementation (distances histograms)
    #endif

    // We dont count UPTOA photons leaving in boxes outside SZA range
    if ((LEd==0) && (count_level==UPTOA) && (acosf(ph->v.z) > (SZA_MAXd*90./DEMIPI))) return;

    float theta = acosf(fmin(1.F, fmax(-1.F, ph->v.z)));

	float psi=0.;
	int ith=0, iphi=0, il=0, is=ph->is;
    float4 st; // replace s1, s2, s3, s4
    unsigned long long II, JJ, JJJ;


    if ((theta != 0.F) && (theta!= acosf(-1.F))) {
       ComputePsi(ph, &psi, theta);
    }
    else {
       if (LEd == 0) {
          atomicAdd(errorcount+ERROR_THETA, 1);
		  // Permet de visualiser les photons aux zenith dans le cas où il y a au moins un obj
		  #ifdef OBJ3D
		  return;
		  #endif
       }
       else {
          // Compute Psi in the special case of zenith
          ComputePsiZenith(ph,&psi,tabphi[ph->iph]);
       }
    }

    rotateStokes(ph->stokes, psi, &st);
    st.w = ph->stokes.w;

    #ifdef BACK
    float4x4 L;
    float4 stback = make_float4(0.5F, 0.5F, 0., 0.);
    rotationM(-psi,&L);
	ph->M = mul(ph->M,L);
    stback = mul(ph->M, stback);
    st = stback;
    /*float4 stforw = make_float4(0.5F, 0.5F, 0., 0.);
    rotationM(psi,&L);
	ph->Mf = mul(L,ph->Mf);
    stforw = mul(ph->Mf,stforw);
    st = stforw;*/
    #endif

	float weight = ph->weight;
    #ifdef ALIS
        float weight_sca[MAX_NLOW];
        for (int k=0; k<NLOWd; k++) {
            weight_sca[k] = ph->weight_sca[k];
        }
    #endif

	// Compute Box for outgoing photons in case of cone sampling
	if (LEd == 0) { 
        // if compute box returns 0, it excluded the photon (outside sun disc for example), so we dont count it
        if (!ComputeBox(&ith, &iphi, &il, ph, errorcount, count_level)) return;
    }


    // For virtual (LE) photons the direction is stored within photon structure
    // Moreover we compute also final attenuation for LE 
    else {
        ith = ph->ith;
        if (!ZIPd) iphi= ph->iph;
        il  = ph->ilam;

        if (!(   (SIMd==SURF_ONLY) 
              || (NATMd==0 && (count_level==UPTOA || count_level==UP0P)) 
              || (NOCEd==0 && count_level==UP0P)
             )
           ){

        // Computation of final attenutation only in fast PP
        #if !defined(SPHERIQUE) && !defined(ALT_PP)
        int layer_le;
        float tau_le;
        Profile *prof;
        int layer_end;

        // define correct start and end layers and profiles for LE
        if (count_level==UPTOA) {
            layer_le = 0; 
            layer_end= NATMd;
            prof = prof_atm;
        }
        if ((count_level==DOWN0P) || (count_level==DOWN0M) || (count_level==UP0P) || (count_level==UP0M) ) {
            if ((ph->loc == ATMOS) || (ph->loc == SURF0M) || (ph->loc == SURF0P)
				#ifdef OBJ3D
				|| (ph->loc == OBJSURF)
				#endif
				) {
                layer_le = NATMd;
                layer_end= NATMd;
                prof = prof_atm;
            }
            if ((ph->loc == OCEAN) || (ph->loc == SEAFLOOR)) {
                layer_le = NOCEd;
                layer_end= NOCEd;
                prof = prof_oc;
            }
        }
        if (count_level==DOWNB) {
            layer_le = 0;
            layer_end= NOCEd;
            prof = prof_oc;
        }

        #ifndef ALIS
        // Attenuation of the current photon
        // First get the extinction optical depth at the counting level
        tau_le = prof[(layer_end-layer_le) + ph->ilam *(layer_end+1)].OD;
        // if BEER=0, photon variable tau corresponds to extinction
        if (BEERd == 0) weight *= expf(-fabs(__fdividef(tau_le - ph->tau, ph->v.z))); // LE attenuation to count_level
        // if BEER=1, photon variable tau corresponds to scattering only, need to add photon absorption variable
        else weight *= expf(-fabs(__fdividef(tau_le - (ph->tau+ph->tau_abs), ph->v.z))); // LE attenuation to count_level

        // Specific computation for ALIS
        #else
        float dsca_dl, dsca_dl0;
        int DL=(NLAMd-1)/(NLOWd-1);

        // Complete photon history toward space for further final absorption computation
        ph->layer = 0;
        ph->nevt++;
        // ph->layer_prev[ph->nevt] = ph->layer;
        if (ph->loc == ATMOS) ph->layer_prev[ph->nevt]   = ph->layer;
        if (ph->loc == OCEAN || ph->loc == SURF0M) ph->layer_prev[ph->nevt]   = -ph->layer;
        ph->vz_prev[ph->nevt] = ph->v.z;
        ph->epsilon_prev[ph->nevt] = 0.f;
        
        // Attenuation by scattering only of the main 'central' or 'reference' photon
        // First get the scattering optical depth at the counting level
        tau_le = prof[(layer_end-layer_le) + ph->ilam *(layer_end+1)].OD_sca;
        // LE attenuation to count_level without absorption, central wavelength
        dsca_dl0 = tau_le - ph->tau; 
        weight *= expf(-fabs(__fdividef(dsca_dl0, ph->v.z)));

        // Differential LE scattering attenuation to count_level for others 'scattering' wavelengths
        for (int k=0; k<NLOWd; k++) {
           dsca_dl = prof[(layer_end-layer_le) + k*DL*(layer_end+1)].OD_sca - ph->tau_sca[k]; 
           weight_sca[k] *= exp(-__fdividef(fabs(dsca_dl) -fabs(dsca_dl0), fabs(ph->v.z)));
        }
        #endif // NOT ALIS
        #endif // NOT SPHERIQUE && NOT ALT_PP
     } // SIMd  

    }   //LE
	
    float weight_irr = fabs(ph->v.z);
    // In Forward mode, and in case of spherical flux, update the weight
	if (FLUXd==2 && LEd==0 & weight_irr > 0.001f) weight /= weight_irr;
    if (count_level == UPTOA && HORIZd == 0) weight *= weight_irr;

    #ifdef DEBUG
	int idx = blockIdx.x *blockDim.x + threadIdx.x;
    if (isnan(weight)) printf("(idx=%d) Error, weight is NaN, %d\n", idx,ph->loc);
    if (isnan(st.x)) printf("(idx=%d) Error, s1 is NaN\n", idx);
    if (isnan(st.y)) printf("(idx=%d) Error, s2 is NaN\n", idx);
    if (isnan(st.z)) printf("(idx=%d) Error, s3 is NaN\n", idx);
    #endif

    II = NBTHETAd*NBPHId*NLAMd*NSENSORd;
    JJJ= NPSTKd*II;

    // Regular counting procedure
    #ifndef ALIS //=========================================================================================================
	if(((ith >= 0) && (ith < NBTHETAd)) && ((iphi >= 0) && (iphi < NBPHId)) && (il >= 0) && (il < NLAMd) && (!isnan(weight)))
	{
      JJ = is*NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi;

      #ifdef DOUBLE 
      // select the appropriate level (count_level)
      tabCount = (double*)tabPhotons + count_level*JJJ;
      dweight = (double)weight;
      ds = make_double4(st.x, st.y, st.z, st.w);

	  #if __CUDA_ARCH__ >= 600
	  // If GTX 1000 or more recent use native double atomic add
      atomicAdd(tabCount+(0*II+JJ), dweight*(ds.x+ds.y));
      atomicAdd(tabCount+(1*II+JJ), dweight*(ds.x-ds.y));
      atomicAdd(tabCount+(2*II+JJ), dweight*ds.z);
      atomicAdd(tabCount+(3*II+JJ), dweight*ds.w);
      #else
      DatomicAdd(tabCount+(0*II+JJ), dweight*(ds.x+ds.y));
	  DatomicAdd(tabCount+(1*II+JJ), dweight*(ds.x-ds.y));
	  DatomicAdd(tabCount+(2*II+JJ), dweight*ds.z);
	  DatomicAdd(tabCount+(3*II+JJ), dweight*ds.w);
	  #endif

      #else
      tabCount = (float*)tabPhotons + count_level*JJJ;
      atomicAdd(tabCount+(0*II+JJ), weight * (st.x+st.y));
      atomicAdd(tabCount+(1*II+JJ), weight * (st.x-st.y));
      atomicAdd(tabCount+(2*II+JJ), weight * st.z);
      atomicAdd(tabCount+(3*II+JJ), weight * st.w);
      #endif

      atomicAdd(NPhotonsOut + (((count_level*NSENSORd + is)*NLAMd + il)*NBTHETAd + ith)*NBPHId + iphi, 1);
	}
	else
	{
	  atomicAdd(errorcount+ERROR_CASE, 1);
	}
    //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

    #else //ALIS ===========================================================================================================
    int DL=(NLAMd-1)/(NLOWd-1);
	if(((ith >= 0) && (ith < NBTHETAd)) && ((iphi >= 0) && (iphi < NBPHId)) && (!isnan(weight)))
    {
     if(HISTd==0) {
      // For all wavelengths
      for (il=0; il<NLAMd; il++) {
          float wabs = 1.0f;
          JJ = is*NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi;

          // Linear interpolation upon wavelength of the scattering correction
          int ik=il/DL;
          float wsca;
          if (il != NLAMd-1) wsca = __fdividef((il-ik*DL)*1.0f,DL*1.0f) * (weight_sca[ik+1] - weight_sca[ik]) +
                          weight_sca[ik]; 
          else wsca = weight_sca[NLOWd-1];
          
          //  OR Polynomial fit for scattering correction, !!DEV
          /* 
          float wsca = 0.;
          for (int k=0; k<NLOWd; k++){
            float acc = 1.f;
            for (int j=0; j< NLOWd; j++) {
                if (j!=k) acc *= __fdividef((float)il-(float)j*DL,(float)k*DL-(float)j*DL); 
            }
            wsca += ph->weight_sca[k] * acc;
           }
          */
        
          #if !defined(SPHERIQUE) && !defined(ALT_PP)
          Profile *prof;
          // Computation of the absorption along photon history with heights and direction cosines 
          for (int n=0; n<ph->nevt; n++){
              //Computing absorption optical depths form start to stop for all segments
              float tau_abs1, tau_abs2;
              int ilayer, NL;
              if (ph->layer_prev[n+1] == 0) tau_abs2 = 0.;
              else {
               if (ph->layer_prev[n+1] < 0)  { 
                  prof = prof_oc;
                  ilayer=-ph->layer_prev[n+1];
                  NL = NOCEd+1;
               }
               if (ph->layer_prev[n+1] > 0)  { 
                  prof = prof_atm;
                  ilayer=ph->layer_prev[n+1];
                  NL = NATMd+1;
               }
                tau_abs2 = (prof[ilayer   + il *NL].OD_abs -
                               prof[ilayer-1 + il *NL].OD_abs) *
                               ph->epsilon_prev[n+1] + prof[ilayer-1 + il *NL].OD_abs;
              }


              if (ph->layer_prev[n]   == 0) tau_abs1 = 0.;
              else {
               if (ph->layer_prev[n] < 0)  { 
                  prof = prof_oc;
                  ilayer=-ph->layer_prev[n];
                  NL = NOCEd+1;
               }
               if (ph->layer_prev[n] > 0)  { 
                  prof = prof_atm;
                  ilayer=ph->layer_prev[n];
                  NL = NATMd+1;
               }
               tau_abs1 = (prof[ilayer   + il *NL].OD_abs -
                               prof[ilayer-1 + il *NL].OD_abs) *
                               ph->epsilon_prev[n+1] + prof[ilayer-1 + il *NL].OD_abs;
              }

              wabs *= exp(-fabs(__fdividef(tau_abs2 - tau_abs1 , ph->vz_prev[n+1])));
          }

          #else
          // Computation of the absorption along photon history with cumulative distances in layers
          wabs = 0.F;
          for (int n=1; n<(NATMd+1); n++){
              wabs += abs(__fdividef(prof_atm[n   + il*(NATMd+1)].OD_abs -
                                     prof_atm[n-1 + il*(NATMd+1)].OD_abs,
                                     prof_atm[n].z  - prof_atm[n-1].z) ) * ph->cdist_atm[n];
          }
          for (int n=1; n<(NOCEd+1); n++){
              wabs += abs(__fdividef(prof_oc[n   + il*(NOCEd+1)].OD_abs -
                                     prof_oc[n-1 + il*(NOCEd+1)].OD_abs,
                                     prof_oc[n].z  - prof_oc[n-1].z) ) * ph->cdist_oc[n];
          }
          wabs = exp(-wabs);
          #endif

          #ifdef DOUBLE 
          tabCount = (double*)tabPhotons + count_level*JJJ;
          dweight = (double)weight;
          ds = make_double4(st.x, st.y, st.z, st.w);
          dwsca=(double)wsca;
          dwabs=(double)wabs;

		  #if __CUDA_ARCH__ >= 600
          atomicAdd(tabCount+(0*II+JJ), dweight * dwsca * dwabs * (ds.x+ds.y));
          atomicAdd(tabCount+(1*II+JJ), dweight * dwsca * dwabs * (ds.x-ds.y));
		  atomicAdd(tabCount+(2*II+JJ), dweight * dwsca * dwabs * ds.z);
          atomicAdd(tabCount+(3*II+JJ), dweight * dwsca * dwabs * ds.w);
		  #else
		  // If GTX 1000 or more recent use native double atomic add
          DatomicAdd(tabCount+(0*II+JJ), dweight * dwsca * dwabs * (ds.x+ds.y));
          DatomicAdd(tabCount+(1*II+JJ), dweight * dwsca * dwabs * (ds.x-ds.y));
          DatomicAdd(tabCount+(2*II+JJ), dweight * dwsca * dwabs * ds.z);
          DatomicAdd(tabCount+(3*II+JJ), dweight * dwsca * dwabs * ds.w);
		  #endif		  

          #else
          tabCount = (float*)tabPhotons + count_level*JJJ;
          atomicAdd(tabCount+(0*II+JJ), weight * wsca * wabs * (st.x+st.y));
          atomicAdd(tabCount+(1*II+JJ), weight * wsca * wabs * (st.x-st.y));
          atomicAdd(tabCount+(2*II+JJ), weight * wsca * wabs * st.z);
          atomicAdd(tabCount+(3*II+JJ), weight * wsca * wabs * st.w);
          #endif    

          atomicAdd(NPhotonsOut + (((count_level*NSENSORd +is)*NLAMd + il)*NBTHETAd + ith)*NBPHId + iphi, 1);
      } // wavelength loop 
     } //  if HISTd==0

     #if ( defined(SPHERIQUE) || defined(ALT_PP) )
     unsigned long long K   = NBTHETAd*NBPHId*NSENSORd;
     unsigned long long KK  = K*NATMd;
     unsigned long long LL;
     if (HISTd==1) { // Histories stored for absorption computation afterward (only spherical or alt_pp)
          unsigned long long counter2=atomicAdd(NPhotonsOut + (((count_level*NSENSORd + is)*NLAMd + 0)*NBTHETAd + ith)*NBPHId + iphi, 1);
          if (counter2 >= MAX_HIST) return;
          //int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
          unsigned long long KK2 = K*(NATMd+4+NLOWd);
          unsigned long long KKK2= KK2 * MAX_HIST;
          unsigned long long LL2;
          tabCount3   = (float*)tabHist     + count_level*KKK2;
          for (int n=0; n<NOCEd; n++){
                LL2 = counter2*KK2 +  n*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
                atomicAdd(tabCount3+LL2, ph->cdist_oc[n]);
          }
          for (int n=0; n<NATMd; n++){
                LL2 = counter2*KK2 +  (n+NOCEd)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
                atomicAdd(tabCount3+LL2, ph->cdist_atm[n]);
          }
          LL2 = counter2*KK2 +  (NATMd+0)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
          atomicAdd(tabCount3+LL2, weight * (st.x+st.y));
          LL2 = counter2*KK2 +  (NATMd+1)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
          atomicAdd(tabCount3+LL2, weight * (st.x-st.y));
          LL2 = counter2*KK2 +  (NATMd+2)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
          atomicAdd(tabCount3+LL2, weight * (st.z));
          LL2 = counter2*KK2 +  (NATMd+3)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
          atomicAdd(tabCount3+LL2, weight * (st.w));

          for (int n=0; n<NLOWd; n++){
                LL2 = counter2*KK2 +  (n+NATMd+4)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
                atomicAdd(tabCount3+LL2, weight_sca[n]);
          }
       } // HISTd==1

       #ifdef DOUBLE
          tabCount2   = (double*)tabDist     + count_level*KK;
          for (int n=0; n<NOCEd; n++){
            LL = n*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
            atomicAdd(tabCount2+LL, (double)ph->cdist_oc[n]);
            //DatomicAdd(tabCount2+LL, (double)ph->cdist_oc[n]);
          }
          for (int n=0; n<NATMd; n++){
            LL = (n+NOCEd)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
            atomicAdd(tabCount2+LL, (double)ph->cdist_atm[n]);
            //DatomicAdd(tabCount2+LL, (double)ph->cdist_atm[n]);
          }
       #else
          tabCount2   = (float*)tabDist     + count_level*KK;
          for (int n=0; n<NOCEd; n++){
            LL = n*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
            atomicAdd(tabCount2+LL, (double)ph->cdist_oc[n]);
          }
          for (int n=0; n<NATMd; n++){
            LL = (n+NOCEd)*K + is*NBPHId*NBTHETAd + ith*NBPHId + iphi;
            atomicAdd(tabCount2+LL, ph->cdist_atm[n]);
          }
       #endif
      #endif // SPHERIQUE or ALT_PP

    } // correct output box
	else
	{
		atomicAdd(errorcount+ERROR_CASE, 1);
	}
    #endif //ALIS
    //+++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

}



//
// Rotation of the stokes parameters by an angle psi between the incidence and
// the emergence planes
// input: float4 stokes parameters
//        rotation angle psi in radians
// output: float 4 rotated stokes parameters
//
__device__ void rotateStokes(float4 s, float psi, float4 *sr)
{
    float cPsi = __cosf(psi); float sPsi = __sinf(psi); float cPsi2 = cPsi * cPsi; float sPsi2 = sPsi * sPsi;
	float twopsi = 2.F*psi;  float s2Psi = __sinf(twopsi); float a = 0.5f*s2Psi;

	float3x3 L = make_float3x3(
		cPsi2, sPsi2, -a,                
		sPsi2, cPsi2, a,                 
		s2Psi, -s2Psi, __cosf(twopsi)   
		);

    // Since s(4) do not change by the rotation, multiply the 3x3 matrix L(psi) by the 3 first terms of s
	(*sr) = mul(L,s); // see the function "mul" in hip/hip_vector_types.h for more infos
}

//
// Rotation Matrix L from an angle psi between the incidence and
// the emergence planes
__device__ void rotationM(float psi, float4x4 *L)
{
    float cPsi = __cosf(psi); float sPsi = __sinf(psi); float cPsi2 = cPsi * cPsi; float sPsi2 = sPsi * sPsi;
	float twopsi = 2.F*psi;  float s2Psi = __sinf(twopsi); float a = 0.5f*s2Psi;

	*L = make_float4x4(
		cPsi2, sPsi2, -a, 0.f,               
		sPsi2, cPsi2, a, 0.f,                 
		s2Psi, -s2Psi, __cosf(twopsi), 0.f,
        0.f, 0.f, 0.f, 1.f
		);
}

// Rotation Matrix of angle theta around unit vector u
__device__ float3x3 rotation3D(float theta, float3 u)
{
    // Rodrigues rotation formula
    float ct=cosf(theta);
    float st=sinf(theta);
    float3x3 A, B, C, R;
    A = make_diag_float3x3(1.F);
    B = make_float3x3( 0.F,-u.z, u.y,
                       u.z, 0.F,-u.x,
                      -u.y, u.x, 0.F
                     );
    /*C = make_float3x3(u.x*u.x, u.x*u.y, u.x*u.z,
                      u.x*u.y, u.y*u.y, u.y*u.z,
                      u.x*u.z, u.y*u.z, u.z*u.z
                     );*/
    //R = add(add(mul(A, ct), mul(B, st)), mul(C, 1.F-ct)); 

    C = mul(B, B);
    R = add(add(A, mul(B, st)), mul(C, 1.F-ct)); 

    return R;
}


/* ComputePsi
*/
__device__ void ComputePsi(Photon* ph, float* psi, float theta)
{
    // see Rammella et al. Three Monte Carlo programs of polarized light transport into scattering media: part I Optics Express, 2005, 13, 4420
    double wz;
    wz = (double)ph->v.x * (double)ph->u.y - (double)ph->v.y * (double)ph->u.x;
    *psi = atan2(wz, -1.e+00*(double)ph->u.z); 
}

/* ComputePsiZenith
*/
__device__ void ComputePsiZenith(Photon* ph, float* psi, float phi)
{
        // Compute Psi in the special case of zenith
        float ux_phi;
        float uy_phi;
        float cos_psi;
        float sin_psi;
        float eps=1e-2;
        //float eps=1e-4;

        ux_phi  = cosf(phi);
        uy_phi  = sinf(phi);
        cos_psi = (ux_phi*ph->u.x + uy_phi*ph->u.y);
        if( cos_psi >  1.0) cos_psi =  1.0;
        if( cos_psi < -1.0) cos_psi = -1.0;
        sin_psi = sqrtf(1.0 - (cos_psi*cos_psi));
        if( (abs((ph->u.x*cos_psi-ph->u.y*sin_psi)-ux_phi) < eps) && (abs((ph->u.x*sin_psi+ph->u.y*cos_psi)-uy_phi) < eps) ) {
                *psi = -acosf(cos_psi);
        }
        else{
                *psi = acosf(cos_psi);
        } 
}


/* ComputeBox
*/
__device__ int ComputeBox(int* ith, int* iphi, int* il,
                           Photon* photon, unsigned long long *errorcount, int count_level)
{
	// vxy est la projection du vecteur vitesse du photon sur (x,y)
	float vxy = sqrtf(photon->v.x * photon->v.x + photon->v.y * photon->v.y);

	// Calcul de la valeur de ithv
	// _rn correspond à round to the nearest integer
    #ifndef SPHERIQUE
	*ith = __float2int_rd(__fdividef(acosf(fabsf(photon->v.z)) * NBTHETAd, DEMIPI));
    #else
    if (count_level==UPTOA) *ith = __float2int_rd(__fdividef(acosf(photon->v.z) * NBTHETAd, SZA_MAXd/90.*DEMIPI));
    else                    *ith = __float2int_rd(__fdividef(acosf(fabsf(photon->v.z)) * NBTHETAd, DEMIPI));
    #endif


	// Calcul de la valeur de il
    *il = photon->ilam;

	/* Si le photon ressort très près du zénith on ne peut plus calculer iphi,
	 on est à l'intersection de toutes les cases du haut */
	
	if(vxy >= VALMIN)
	{	//on calcule iphi
	
		// On place d'abord le photon dans un demi-cercle
		float cPhiP = __fdividef(photon->v.x, vxy); //cosPhiPhoton
		// Cas limite où phi est très proche de 0, la formule générale ne marche pas
		//if(cPhiP >= 1.F) *iphi = 0;
		// Cas limite où phi est très proche de PI, la formule générale ne marche pas
		//else if(cPhiP <= -1.F) *iphi = (NBPHId) - 1;
		// Size of the angular boxes
        float dphi = __fdividef(2.F*PI,NBPHId);

        // Boxes centred on 0., dphi, 2dphi, ..., 180-dphi, 180., 180.+dphi,...., 360-dphi .
        // Boxes indices 0, 1, 2, ..., NBPHI/2-1, NBPHI/2, NBPHI/2 +1,..., NBPHI-2, NBPHI -1
        // So 2 boxes on 0 and 180 + NBPHI/2-1 boxes with vy>0 and NBPHI/2 -1 boxes with vy<0
        // Total NBPHI boxes from 0 to NBPHI -1; NBPHI has to be even
        // if the azimuth is within the zeroth boxe centered on 0. of width dphi/2 (half width dphi/4)
        if(cPhiP >= cosf(dphi/2.)) *iphi = 0;
        // if the azimuth is in the middle box centered on 180.
        else if(cPhiP <= -cosf(dphi/2.)) *iphi = NBPHId/2;
		else {
            /* otherwise it lies in a dphi box whose index (starting from 1) is given by the ratio of
             Phi -dphi/4. to the possible phi range that is PI-dphi/2. multiplied by the number of boxes NBPHId/2-1*/
            *iphi = __float2int_rd(__fdividef((acosf(cPhiP)-dphi/2.) * (NBPHId/2-1.0F), PI-dphi)) + 1;
		
		    // Puis on place le photon dans l'autre demi-cercle selon vy, utile uniquement lorsque l'on travail sur tous l'espace
   		    if(photon->v.y < 0.F) *iphi = NBPHId - *iphi;
            }
		// Lorsque vy=0 on décide par défaut que le photon reste du côté vy>0
		if(photon->v.y == 0.F) atomicAdd(errorcount+ERROR_VXY, 1);
	}
	
	else{
		// Photon très près du zenith
		atomicAdd(errorcount+ERROR_VXY, 1);
// 		/*if(photon->vy < 0.F) *iphi = NBPHId - 1;
// 		else*/ *iphi = 0;
		if(photon->v.y >= 0.F)  *iphi = 0;
		else *iphi = NBPHId - 1;
	}
    if (SUN_DISCd <= 0) return 1;	

    float phi = *iphi * __fdividef(2.F*PI, NBPHId);
    float dth =  __fdividef(SZA_MAXd * PI, 180.F * NBTHETAd);
    float cth = cosf((*ith + 0.5F) * dth);
    float sth = sqrtf(1.F - cth*cth);
    float3 center_dir = make_float3(cosf(phi)*sth, sinf(phi)*sth, cth);
    if ((abs(acosf(dot(photon->v, center_dir)))*180.F/PI) >  SUN_DISCd ) {

        //int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
        //if (idx==0 && count_level==UPTOA) printf("center %f %f %f, phot %f %f %f \n", center_dir.x, center_dir.y, center_dir.z, photon->v.x, photon->v.y, photon->v.z);
        return 0;
    }
    return 1;
}

#ifdef DEBUG_PHOTON
__device__ void display(const char* desc, Photon* ph) {
    //
    // display the status of the photon (only for thread 0)
    //
	int idx = blockIdx.x *blockDim.x + threadIdx.x;

    if (idx == 0) {
		
		printf("%16s %4i X=(%9.4f,%9.4f,%9.4f) V=(%6.3f,%6.3f,%6.3f) U=(%6.3f,%6.3f,%6.3f) S=(%6.3f,%6.3f,%6.3f,%6.3f) tau=%8.3f tau_abs=%8.3f wvl=%6.3f weight=%11.3e ",
               desc,
			   ph->nint,
               ph->pos.x, ph->pos.y, ph->pos.z,
               ph->v.x,ph->v.y,ph->v.z,
			   ph->u.x,ph->u.y,ph->u.z,
               ph->stokes.x, ph->stokes.y,
               ph->stokes.z, ph->stokes.w,
               ph->tau,ph->tau_abs, ph->wavel, ph->weight, ph->scatterer
               );

        switch(ph->scatterer) {
            case -1: printf("scatterer =   UNDEF"); break;
            case 0: printf("scatterer =     RAY"); break;
            case 1: printf("scatterer =   PTCLE"); break;
            case 2: printf("scatterer = CHLFLUO"); break;
            default:
                    printf("scatterer =   UNDEF");
        }
        switch(ph->loc) {
            case 0: printf(" loc=   SPACE"); break;
            case 1: printf(" loc=   ATMOS"); break;
            case 2: printf(" loc=  SURF0P"); break;
            case 3: printf(" loc=  SURF0M"); break;
            case 4: printf(" loc=ABSORBED"); break;
            case 5: printf(" loc=    NONE"); break;
            case 6: printf(" loc=   OCEAN"); break;
            case 7: printf(" loc=SEAFLOOR"); break;
            case 8: printf(" loc= OBJSURF"); break;
		    case 9: printf(" loc= REMOVED"); break;
            default:
                    printf(" loc=UNDEFINED");
        }
        #ifdef ALIS
        printf(" wsca=");
        for (int k=0; k<NLOWd; k++) printf("%7.5f ",ph->weight_sca[k]);
        #if !defined(ALT_PP) && !defined(SPHERIQUE)
        printf(" nevt=%2d",ph->nevt);
        printf(" dtausca=");
        for (int k=0; k<NLOWd; k++) printf("%7.5f ",ph->tau_sca[k]);
        printf(" layers=");
        for (int k=0; k<ph->nevt+1; k++) printf("%3d ",ph->layer_prev[k]);
        printf(" vz=");
        for (int k=0; k<ph->nevt+1; k++) printf("%7.5f ",ph->vz_prev[k]);
        printf(" delta=");
        for (int k=0; k<ph->nevt+1; k++) printf("%7.5f ",ph->epsilon_prev[k]);
        #endif
        #endif
        printf("\n");
    }
}
#endif

__device__ void modifyUV( float3 v0, float3 u0, float cTh, float psi, float3 *v1, float3 *u1){ 
    float sTh, cPsi, sPsi;
	float3 v, u, w;

    sPsi = __sinf(psi);
    cPsi = __cosf(psi);
    sTh = sqrtf(1.F - cTh*cTh);
	w = cross(u0, v0); // w : cross product entre l'ancien vec u et l'ancien vec v du photon
	v = operator+(cTh * v0, sTh * (operator+(cPsi * u0, sPsi * w))); // v est le nouveau vecteur v du photon
	// Changement du vecteur u (orthogonal au vecteur vitesse du photon)
    if (cTh <= -1.F) {
		u = -u0;}
    else if (cTh >= 1.F){
        u  = u0;}
    else {u = operator-(cTh * v, v0);}

	*v1 = normalize(v); // v1 = v normalized
	*u1 = normalize(u); // u1 = u normalized
}

__device__ void ComputePsiLE( float3 u0, float3 v0, float3 v1, float* psi, float3* u1){
	float prod_scal, den, y1, cpsi, spsi;	
	float EPS6 = 1e-4;	
	float3 w0, w1;

	// compute former w
	w0 = cross(u0, v0); // w : cross product entre l'ancien vec u et l'ancien vec v du photon
	w1 = cross(v1, v0);	// compute the normal to the new scattering plan i.e. new w vector

	den = length(w1); // Euclidean length also called L2-norm
	if (den < EPS6) {
		prod_scal =  dot(v0, v1);
		if (prod_scal < 0.0)
			w1 = w0;       // diffusion vers l'avant
		else{ w1 = -w0; }   // diffusion vers l'arriere
	}

	else{ operator/=(w1, den); }
	
	cpsi = dot(w0,w1); 	//  Compute the scalar product between w0 and w1

	if (cpsi >  1.0) 
		cpsi =  1.0;
	if (cpsi < -1.0) 
		cpsi = -1.0;
	spsi = sqrtf(1.0 - cpsi * cpsi);
	if (spsi >  1.0) 
		spsi =  1.0;

	// Change of reference frame, look for the expression of  {vx1, vy1, vz1}
	// in the base linked to the photon before the scattering event = old
	// scattering plan. 
	// Let say that x1, y1, z1 are the new coordinate of cos_dir_sensor
	y1 = dot(w0,v1);
	// --- Sign of spsi
	if (y1 < 0.0) 
		spsi = -spsi;

	*psi = acosf(cpsi);
	if (spsi<0)
		*psi = 2*PI - *psi;

	// get the new u vector
	*u1 = cross(v1, w1);	
}

__device__ float ComputeTheta(float3 v0, float3 v1){
	// compute the diffusion angle theta between
	// to direction cosine {vx0, vy0, vz0} and {vx1, vy1, vz1} 

	float cs;
	float theta;
	
	//--- Find cos(theta) and sin(theta)
	cs =  dot(v1,v0)  ;//  produit scalaire
	
	// test cs to avois acos(cs)=NaN
	if(cs>+1) cs = 1.00;
	if(cs<-1) cs = -1.00;
		
	//--- compute theta
	
	theta = acosf(cs);

	return(theta);		
}

__device__ void copyPhoton(Photon* ph, Photon* ph_le) {
    ph_le->v = ph->v; //float3
    ph_le->u = ph->u; // float3
    ph_le->stokes = ph->stokes; //float4
    ph_le->loc = ph->loc;
    ph_le->tau = ph->tau;
    ph_le->tau_abs = ph->tau_abs;
    ph_le->layer = ph->layer;
    ph_le->weight = ph->weight;
    ph_le->wavel = ph->wavel;
    ph_le->ilam = ph->ilam;
	ph_le->scatterer=ph->scatterer;
    ph_le->pos = ph->pos; // float3
    ph_le->nint = ph->nint;
    ph_le->is = ph->is;
    #ifdef SPHERIQUE
    ph_le->radius = ph->radius;
    #endif

    #ifdef ALIS
    int k; 
    #if !defined(ALT_PP) && !defined(SPHERIQUE)
    int kmax=ph->nevt+1;
    ph_le->nevt = ph->nevt;
    for (k=0; k<kmax; k++) ph_le->layer_prev[k] = ph->layer_prev[k];
    for (k=0; k<kmax; k++) ph_le->vz_prev[k] = ph->vz_prev[k];
    for (k=0; k<kmax; k++) ph_le->epsilon_prev[k] = ph->epsilon_prev[k];
    for (k=0; k<NLOWd; k++) ph_le->tau_sca[k] = ph->tau_sca[k];
    #else
    for (k=0; k<(NATMd+1); k++) ph_le->cdist_atm[k] = ph->cdist_atm[k];
    for (k=0; k<(NOCEd+1); k++) ph_le->cdist_oc[k]  = ph->cdist_oc[k];
    #endif
    for (k=0; k<NLOWd; k++) ph_le->weight_sca[k] = ph->weight_sca[k];
    #endif

    #ifdef BACK
    int kk;
    for (kk=0; kk<16; kk++) ph_le->M[kk] = ph->M[kk];
    //for (kk=0; kk<16; kk++) ph_le->Mf[kk] = ph->Mf[kk];
    #endif

}

__device__ float get_OD(int BEERd, struct Profile prof) {  
    if (BEERd == 1) return prof.OD_sca;
    else            return prof.OD;
}


__device__ float Lambda(float avz, float sig) {
    float l;
    if (avz == 1.F) l = 0.F;
    else {
        float nu = __fdividef(1.F, tanf(acosf(avz))*(sqrtf(2.) * sig));
        l = __fdividef(__expf(-nu*nu) - nu * sqrtf(PI) * erfcf(nu),2.F * nu * sqrtf(PI));
    }
    return l;
}

__device__ double LambdaM(double avz, double sig2) {
    // Mischenko implementation
    double l;
    if (avz == 1.F) l = 0.;
    else {
        double s1,s2,s3,xi,xxi,dcot,t1,t2;
        s1 = __dsqrt_rn(2.*(double)sig2/(double)PI);
        s3 = __drcp_rn(__dsqrt_rn(2.*(double)sig2));
        s2 = s3*s3;
        xi = (double)avz;
        xxi=xi*xi;
        dcot =  xi *__drcp_rn(__dsqrt_rn(1.-xxi));
        t1 = exp(-dcot*dcot*s2);
        t2 = erfc(dcot*s3);
        l  = 0.5*(s1*t1/dcot-t2);
    }
    return l;
}

__device__ void DirectionToUV(float th, float phi, float3* v, float3* u) {
     *v = make_float3(cosf(phi) * sinf(th),
                      sinf(phi) * sinf(th),
                      cosf(th));  
	 *u = make_float3(cosf(phi) * cosf(th),
	                  sinf(phi) * cosf(th),
	                  -sinf(th));
}

__device__ float3 LocalToGlobal(float3 Nx, float3 Ny, float3 Nz, float3 v) {
     float3x3 B = make_float3x3(
                Nx.x, Ny.x, Nz.x,
                Nx.y, Ny.y, Nz.y,
                Nx.z, Ny.z, Nz.z
                );
     return mul(B, v);
}

__device__ float3 GlobalToLocal(float3 Nx, float3 Ny, float3 Nz, float3 v) {
     float3x3 B = make_float3x3(
                Nx.x, Nx.y, Nx.z,
                Ny.x, Ny.y, Ny.z,
                Nz.x, Nz.y, Nz.z
                );
     return mul(B, v);
}

__device__ void MakeLocalFrame(float3 pos, float3* Nx, float3* Ny, float3* Nz) {
	*Nz = normalize(pos); // Nz is the vertical at the impact point
	*Ny = normalize(cross(*Nz, make_float3(1.0, 0.0, 0.0)));
	*Nx = normalize(cross(*Ny, *Nz));
}

#ifdef PHILOX

/**********************************************************
*	> Fonctions liées au générateur aléatoire
***********************************************************/


/* randomPhilox4x32_7float
* Fonction random Philox-4x32-7 qui renvoit un float dans ]0;1]
*/
__device__ float randomPhilox4x32_7float(philox4x32_ctr_t* ctr, philox4x32_key_t* key)
{
    //Recuperation d'un unsigned int pour retourner un float dans ]0;1]
    return __fdividef(__uint2float_rz(randomPhilox4x32_7uint(ctr, key)) + 1.0f, 4294967296.0f);
}

/* randomPhilox4x32_7uint
* Fonction random Philox-4x32-7 qui renvoit un uint à partir d'un generateur (etat+config)
* TODO A noter que 4 valeurs sont en fait generees, un seul uint peut etre renvoye, donc 3 sont perdus
* En pratique les valeurs generees sont des int32. Il y a donc une conversion vers uint32 de realisee
*/
__device__ unsigned int randomPhilox4x32_7uint(philox4x32_ctr_t* ctr, philox4x32_key_t* key)
{
    //variable de retour
    philox4x32_ctr_t res;
    //generation de 4 int32
    res = philox4x32_R(7, *ctr, *key);
    //increment du premier mot de 32bits du compteurs
    (*ctr).v[0]++;
    //conversion d'un des mots generes sous forme d'unsigned int
    return (unsigned int) res[0];
}
#endif

#if defined(DOUBLE) && (__CUDA_ARCH__ < 600)
__device__ double DatomicAdd(double* address, double val)
{
        unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
        do {
           assumed = old;
           old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                __longlong_as_double(assumed)));

                // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
}
#endif


/**********************************************************
*	> Fonctions liées à la création de géométries
***********************************************************/
#ifdef OBJ3D
/* geoTest
* Vérifie s'il y a une intersection avec au moins un objet, si il y a intersection avec plusieurs objets alors
* retourne les infos d'intersection de l'objet dans la distance de parcours est la plus proche
*/
__device__ bool geoTest(float3 o, float3 dir, int phLocPrev, float3* phit, IGeo *GeoV, struct IObjets *ObjT)
{
	Ray R1(o, dir, 0); // initialisation du rayon pour l'étude d'intersection
	// ******************interval d'étude******************
	BBox interval(make_float3(Pmin_x, Pmin_y, Pmin_z),
				  make_float3(Pmax_x, Pmax_y, Pmax_z));
	
	if (!interval.IntersectP(R1))
	{
		*(phit) = make_float3(-1, -1, -1);
	    GeoV->normal = make_float3(0, 0, 0);
		return false;
	}
	// *****************************************************
	
	// *************commun avec tous les objets*************
	float myT = HIP_INF_F; // myT = time
	bool myB = false;
	DifferentialGeometry myDg;
	float3 tempPhit; // Phit temporaire
    // *****************************************************
	
	// *******Propre aux objets de type surface plane*******
	int vi[6] = {0, 1, 2,  // vertices index for triangle 1
				 2, 3, 1}; // vertices index for triangle 2
	Transform nothing; // transformation "nulle"
	// *****************************************************

	for (int i = 0; i < nObj; ++i)
	{
		float myTi = HIP_INF_F;
		bool myBi = false;
		DifferentialGeometry myDgi;
		// *****************************First Step********************************
		// prise en compte de tte les tranformations existantes de l'objet(i)
		Transform Ti, invTi; // déclare la tranfo de l'objet i et son inverse

		/* !!! Nous notons qu'il est important de commencer avec la translation
		   car s'il y a une rotation, alors le repère change (axe x ou y ou z) !!! */

		// si une valeur en x, y ou z diff de 0 alors il y a une translation
		if (ObjT[i].mvTx != 0 or ObjT[i].mvTy != 0 or ObjT[i].mvTz != 0) {
			Transform TmT;
			TmT = Ti.Translate(make_float3(ObjT[i].mvTx, ObjT[i].mvTy,
										   ObjT[i].mvTz));
			Ti = TmT; }
		
		if (ObjT[i].mvRx != 0) { // si diff de 0 alors il y a une rot en x
			Transform TmRX;
			TmRX = Ti.RotateX(ObjT[i].mvRx);
			Ti = Ti*TmRX;}
		if (ObjT[i].mvRy != 0) { // si diff de 0 alors il y a une rot en y
			Transform TmRY;
			TmRY = Ti.RotateY(ObjT[i].mvRy);
			Ti = Ti*TmRY;}
		if (ObjT[i].mvRz != 0) { // si diff de 0 alors il y a une rot en z
			Transform TmRZ;
			TmRZ = Ti.RotateZ(ObjT[i].mvRz);
			Ti = Ti*TmRZ;}


		invTi = Ti.Inverse(Ti); // inverse de la tranformation
		// ***********************************************************************
		
		// ******************************Second Step******************************
		// on voit s'il y a une intersection avec l'objet(i)
		if (ObjT[i].geo == 1) // cas d'un objet de type sphere
		{
			Sphere myObject(&Ti, &invTi, ObjT[i].myRad, ObjT[i].z0,
							ObjT[i].z1, ObjT[i].phi);
		
			BBox myBBox = myObject.WorldBoundSphere();

			if (myBBox.IntersectP(R1))
				myBi = myObject.Intersect(R1, &myTi, &myDgi);
		}
		else if (ObjT[i].geo == 2) // cas d'un objet de type surface plane
		{
			// declaration of a table of float3 which contains P0, P1, P2, P3
			float3 Pvec[4] = {make_float3(ObjT[i].p0x, ObjT[i].p0y, ObjT[i].p0z),
							  make_float3(ObjT[i].p1x, ObjT[i].p1y, ObjT[i].p1z),
							  make_float3(ObjT[i].p2x, ObjT[i].p2y, ObjT[i].p2z),
							  make_float3(ObjT[i].p3x, ObjT[i].p3y, ObjT[i].p3z)};
			
			// Create the triangleMesh (2 = number of triangle ; 4 = number of vertices)
			TriangleMesh myObject(&Ti, &invTi, 2, 4, vi, Pvec);
			
			BBox myBBox = myObject.WorldBoundTriangleMesh();
			if (myBBox.IntersectP(R1))
				myBi = myObject.Intersect(R1, &myTi, &myDgi);				
		}
		// ***********************************************************************
		
		// ******************************third Step*******************************
		// s'il y a intersection avec plusieurs objets, assure qu'on garde l'objet
		// le plus proche du point de départ du photon
		if (myBi & (myT > myTi)) // si intercect objet(i) + time(i-1) > time(i)
		{ // si objet(i) plus proche que objet(i-1) alors remplacement des données
		    tempPhit = R1(myTi); // valeur temporaire de phit
			
			// this condition enable to correct an important bug in case of reflection
			// Without this condition, the photon where the initial position is assimilated to phit
			// will be reflected...
			if ((fabs(tempPhit.x-o.x) > 1e-3) || (fabs(tempPhit.y-o.y) > 1e-3) ||
				(fabs(tempPhit.z-o.z) > 1e-3) || (phLocPrev != OBJSURF))
			{
				myB = true;
				myT = myTi;
				myDg = myDgi;
				//GeoV->material = ObjT[i].material;
				//GeoV->reflectivity = ObjT[i].reflect;
				GeoV->normal = faceForward(myDg.nn, -1.*R1.d);
				GeoV->normalBase = make_float3(ObjT[i].nBx, ObjT[i].nBy, ObjT[i].nBz);
				// if(isBackward(GeoV->normal, dir))
				if(  isBackward( make_double3(GeoV->normalBase.x, GeoV->normalBase.y, GeoV->normalBase.z),
								 make_double3(dir.x, dir.y, dir.z) )  )
				{
					GeoV->material = ObjT[i].materialAV;
					GeoV->reflectivity = ObjT[i].reflectAV;
				}
				else
				{
					GeoV->material = ObjT[i].materialAR; //AR
					GeoV->reflectivity = ObjT[i].reflectAR;
				}
				*(phit) = tempPhit;
				GeoV->mvTF = Ti;
				GeoV->type = ObjT[i].type;
				GeoV->mvR = make_float3(ObjT[i].mvRx, ObjT[i].mvRy, ObjT[i].mvRz);
			}
		}
		// ***********************************************************************
	} // FIN BOUCLE FOR (PARCOURANT LES OBJETS)
	
	if (myB) { // Il y a intersection avec au moins un objet
		return true; }
	else { // Il y a pas d'intersection avec un objet
		*(phit) = make_float3(-1, -1, -1);
		return false; }	
} // FIN DE LA FONCTION GEOTEST()
#endif
