#include "hip/hip_runtime.h"

/**********************************************************
*
*			device.cu
*
*	> Kernel
*	> Modélisation phénomènes physiques
*	> Initialisation de données dans le device
*	> Fonctions liées au générateur aléatoire
*
***********************************************************/

/*************************************************************/
/*************************************************************/
/*          MENTION LICENCE POUR RNGs                        */
/*************************************************************/
/*         Philox 4x32 7                                     */
/*
Copyright 2010-2011, D. E. Shaw Research.
All rights reserved.

Redistribution and use in source and binary forms, with or without
modification, are permitted provided that the following conditions are
met:

* Redistributions of source code must retain the above copyright
  notice, this list of conditions, and the following disclaimer.

* Redistributions in binary form must reproduce the above copyright
  notice, this list of conditions, and the following disclaimer in the
  documentation and/or other materials provided with the distribution.

* Neither the name of D. E. Shaw Research nor the names of its
  contributors may be used to endorse or promote products derived from
  this software without specific prior written permission.

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS AND CONTRIBUTORS
"AS IS" AND ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT
LIMITED TO, THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR
A PARTICULAR PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT
OWNER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL,
SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT
LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY
THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
(INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/
/***************************************************************/
/*          FIN LICENCES RNGs                                  */
/***************************************************************/



/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "device.h"
#include <math.h>


/**********************************************************
*	> Kernel
***********************************************************/

/* lancementKernel
* Kernel de lancement et gestion de la simulation
* Les fonctions de plus bas niveau sont appelées en fonction de la localisation du photon
* Il peut être important de rappeler que le kernel lance tous les threads mais effectue des calculs similaires. La boucle de la
* fonction va donc être effectuée pour chaque thread du block de la grille
* A TESTER: Regarder pour effectuer une réduction de l'atomicAdd
*/



__device__ void launchKernel(Variables* var, Tableaux tab
		, Init* init
			       )
{
	// idx est l'indice du thread considéré
	int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
    int loc_prev;
    int count_level;
    int this_thread_active = 1;


	// Paramètres de la fonction random en mémoire locale
	#ifdef RANDMWC
	unsigned long long etatThr;
	unsigned int configThr;
	configThr = tab.config[idx];
	etatThr = tab.etat[idx];
	#endif
	#if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
        curandSTATE etatThr;
	etatThr = tab.etat[idx];
	#endif
	#ifdef RANDMT
	ConfigMT configThr;
	EtatMT etatThr;
	configThr = tab.config[idx];
	etatThr = tab.etat[idx];
	#endif
        #ifdef RANDPHILOX4x32_7
        //la clef se defini par l'identifiant global (unique) du thread...
        //...et par la clef utilisateur ou clef par defaut
        //ce systeme garanti l'existence de 2^32 generateurs differents par run et...
        //...la possiblite de reemployer les memes sequences a partir de la meme clef utilisateur
        //(plus d'infos dans "communs.h")
        philox4x32_key_t configThr = {{idx, tab.config}};
        //le compteur se defini par trois mots choisis au hasard (il parait)...
        //...et un compteur definissant le nombre d'appel au generateur
        //ce systeme garanti l'existence de 2^32 nombres distincts pouvant etre genere par thread,...
        //...et ce sur l'ensemble du process (et non pas 2^32 par thread par appel au kernel)
        //(plus d'infos dans "communs.h")
        philox4x32_ctr_t etatThr = {{tab.etat[idx], 0xf00dcafe, 0xdeadbeef, 0xbeeff00d}};
        #endif

	
	// Création de variable propres à chaque thread
	unsigned long long nbPhotonsThr = 0; 	// Nombre de photons traités par le thread
	
	#ifdef PROGRESSION
	unsigned int nbPhotonsSorThr = 0; 		// Nombre de photons traités par le thread et ressortis dans l'espace
	#endif
	
	Photon ph, ph_le; 		// On associe une structure de photon au thread
	ph.loc = NONE;	// Initialement le photon n'est nulle part, il doit être initialisé
	
    atomicAdd(&(var->nThreadsActive), 1);

    //
    // main loop
    //
    while (var->nThreadsActive > 0) {

        if ((var->nbPhotons > NBLOOPd) && this_thread_active) {
            this_thread_active = 0;
            atomicAdd(&(var->nThreadsActive), -1);
        }

		// Si le photon est à NONE on l'initialise et on le met à la localisation correspondant à la simulaiton en cours
		if((ph.loc == NONE) && this_thread_active){
			
			initPhoton(&ph, tab
				, init
			    , &etatThr
			    #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
			    , &configThr
			    #endif
					);
            #ifdef DEBUG_PHOTON
            display("INIT", &ph);
            #endif
			
		}
		

        //
		// Deplacement
        //
        // -> Si OCEAN ou ATMOS
        loc_prev = ph.loc;
		if( (ph.loc == ATMOS) || (ph.loc == OCEAN)){

            #ifdef SPHERIQUE
            if (ph.loc == ATMOS)
                move_sp(&ph, tab, init
                        , &etatThr
                        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
                        , &configThr
                        #endif
                                );
            else 
            #endif
                move_pp(&ph,tab.z, tab.h, tab.pMol , tab.abs , tab.ho, &etatThr
                        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
                        , &configThr
                        #endif
                                );
            #ifdef DEBUG_PHOTON
            display("MOVE", &ph);
            #endif
                /*move_spp(&ph, tab, init
                        , &etatThr
                        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
                        , &configThr
                        #endif
                                );*/
		}

        //
        // count after move:
        // count the photons in space and reaching surface from above or below
        //
        count_level = -1;
        if (ph.loc == SPACE) {
            count_level = UPTOA;

            // increment the photon counter
            // (for this thread)
            nbPhotonsThr++;

            #ifdef PROGRESSION
            nbPhotonsSorThr++;
            #endif

            // reset the photon location (always)
            ph.loc = NONE;
            #ifdef DEBUG_PHOTON
            display("SPACE", &ph);
            #endif
        } else if ((ph.loc == SURF0M) || (ph.loc == SURF0P)) {
            if ((loc_prev == ATMOS) || (loc_prev == SPACE)) count_level = DOWN0P;
            if (loc_prev == OCEAN) count_level = UP0M;
        }

        // count the photons
        
        /* Cone Sampling */
        if (LEd ==0) countPhoton(&ph, tab, count_level
                #ifdef PROGRESSION
                , var
                #endif
                );


		syncthreads();

		
        //
		// Scatter
        //
        // -> dans ATMOS ou OCEAN
		if( (ph.loc == ATMOS) || (ph.loc == OCEAN)){

            /* Local Estimate */
            if (LEd == 1) {
			 for (int iph=0; iph<NBPHId; iph++){
			  for (int ith=0; ith<NBTHETAd; ith++){
                copyPhoton(&ph, &ph_le);
                ph_le.iph = iph;
                ph_le.ith = ith;
                scatter(&ph_le, tab.faer, tab.ssa , tab.foce , tab.sso, tab.ip, tab.ipo, 1, tab.thv, tab.phi, &etatThr
			    #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
			    , &configThr
			    #endif
				);
                #ifdef DEBUG_PHOTON
                display("SCATTER LE", &ph_le);
                #endif
                countPhoton(&ph_le, tab, UPTOA
                #ifdef PROGRESSION
                , var
                #endif
                );
              }
             }
            }

			scatter(&ph, tab.faer, tab.ssa , tab.foce , tab.sso, tab.ip, tab.ipo, 0, tab.thv, tab.phi, &etatThr 
			#if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
			, &configThr
			#endif
				);
            #ifdef DEBUG_PHOTON
            display("SCATTER", &ph);
            #endif

		}
		syncthreads();


        //
		// Reflection
        //
        // -> in SURFACE
        loc_prev = ph.loc;
        if ((ph.loc == SURF0M) || (ph.loc == SURF0P)){
           // Eventually evaluate Downward 0+ and Upward 0- radiance


           if( ENVd==0 ) { // si pas d effet d environnement
			if( DIOPTREd!=3 )
				surfaceAgitee(&ph, tab.alb, &etatThr
					#if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
					, &configThr
					#endif
						);

			else
				surfaceLambertienne(&ph, tab.alb, &etatThr
                                        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
					, &configThr
					#endif
						);
           }

           else {
                float dis=0;
                dis = sqrtf((ph.x-X0d)*(ph.x-X0d) +(ph.y-Y0d)*(ph.y-Y0d));
                if( dis > ENV_SIZEd) {
				     surfaceLambertienne(&ph, tab.alb, &etatThr
                                        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
					 , &configThr
					      #endif
						);
                }
                else {
				     surfaceAgitee(&ph, tab.alb, &etatThr
					        #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
					 , &configThr
					        #endif
						);
                }
           }
            #ifdef DEBUG_PHOTON
             display("SURFACE", &ph);
            #endif
		}
		syncthreads();

        //
		// Reflection
        //
        // -> in SEAFLOOR
		if(ph.loc == SEAFLOOR){
		     surfaceLambertienne(&ph, tab.alb, &etatThr
                                    #if defined(RANDMWC) || defined(RANDMT) || defined(RANDPHILOX4x32_7)
			 , &configThr
			      #endif
			);
            #ifdef DEBUG_PHOTON
            display("SEAFLOOR", &ph);
            #endif
        }
		syncthreads();


        //
        // count after surface:
        // count the photons leaving the surface towards the ocean or atmosphere
        //
        count_level = -1;
        if ((loc_prev == SURF0M) || (loc_prev == SURF0P)) {
            if ((ph.loc == ATMOS) || (ph.loc == SPACE)) count_level = UP0P;
            if (ph.loc == OCEAN) count_level = DOWN0M;
        }
        
        /* Cone Sampling */
        if (LEd == 0) countPhoton(&ph, tab, count_level
                #ifdef PROGRESSION
                , var
                #endif
                );



		if(ph.loc == ABSORBED){
			ph.loc = NONE;
			nbPhotonsThr++;
		}
		syncthreads();

		

        // from time to time, transfer the per-thread photon counter to the
        // global counter
        if (nbPhotonsThr % 100 == 0) {
            atomicAdd(&(var->nbPhotons), nbPhotonsThr);
            nbPhotonsThr = 0;
        }

	}


	// Après la boucle on rassemble les nombres de photons traités par chaque thread

	atomicAdd(&(var->nbPhotons), nbPhotonsThr);

	#ifdef PROGRESSION
	// On rassemble les nombres de photons traités et sortis de chaque thread
	atomicAdd(&(var->nbPhotonsSor), nbPhotonsSorThr);

	// On incrémente avncement qui compte le nombre d'appels du Kernel
	atomicAdd(&(var->nbThreads), 1);
	#endif

        #ifdef RANDPHILOX4x32_7
	// Sauvegarde de l'état du random pour que les nombres ne soient pas identiques à chaque appel du kernel
	tab.etat[idx] = etatThr[0];
        #else
	// Sauvegarde de l'état du random pour que les nombres ne soient pas identiques à chaque appel du kernel
	tab.etat[idx] = etatThr;
        #endif


}


/**********************************************************
*	> Modélisation phénomènes physiques
***********************************************************/

/* initPhoton
* Initialise le photon dans son état initial avant l'entrée dans l'atmosphère
*/
__device__ void initPhoton(Photon* ph, Tableaux tab
		,  Init* init
		#ifdef RANDMWC
		, unsigned long long* etatThr, unsigned int* configThr
		#endif
		#if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                , curandSTATE* etatThr
        #endif
		#ifdef RANDMT
		, EtatMT* etatThr, ConfigMT* configThr
		#endif
		#ifdef RANDPHILOX4x32_7
                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
		#endif
		    )
{
	// Initialisation du vecteur vitesse
	ph->vx = - STHVd;
	ph->vy = 0.F;
	ph->vz = - CTHVd;

	
	// Initialisation du vecteur orthogonal au vecteur vitesse
	ph->ux = -ph->vz;
	ph->uy = 0.F;
	ph->uz = ph->vx;
	
    // Initialisation de la longueur d onde
     //mono chromatique
	ph->ilam = __float2uint_rz(RAND * NLAMd);
	ph->wavel = tab.lambda[ph->ilam];
    atomicAdd(tab.nbPhotonsInter+ph->ilam, 1);

    if ((SIMd == -2) || (SIMd == 1) || (SIMd == 2)) {

        //
        // Initialisation du photon au sommet de l'atmosphère
        //

        ph->x = init->x0;
        ph->y = init->y0;
        ph->z = init->z0;
        ph->couche = 0;   // top of atmosphere

        #ifdef SPHERIQUE
        ph->rayon = sqrtf(ph->x*ph->x + ph->y*ph->y + ph->z*ph->z );
        #endif

        // !! DEV on ne calucle pas d ep optique ici
        ph->loc = ATMOS;
        ph->tau = tab.h[NATMd + ph->ilam*(NATMd+1)]; 

    } else if ((SIMd == -1) || (SIMd == 0) || (SIMd == 3)) {

        //
        // Initialisation du photon à la surface ou dans l'océan
        //
        ph->x = 0.;
        ph->y = 0.;
        #ifdef SPHERIQUE
        ph->z = RTER;
        #else
        ph->z = 0;
        #endif

        ph->tau = 0.f;

        if (SIMd == 3) {
            ph->loc = OCEAN;
        } else {
            ph->loc = SURF0P;
        }

    } else ph->loc = NONE;
	

	ph->weight = WEIGHTINIT;
	
	// Initialisation des paramètres de stokes du photon
	ph->stokes1 = 0.5F;
	ph->stokes2 = 0.5F;
	ph->stokes3 = 0.F;
	ph->stokes4 = 0.F;

}



#ifdef SPHERIQUE
__device__ void move_sp(Photon* ph, Tableaux tab, Init* init
        #ifdef RANDMWC
        , unsigned long long* etatThr, unsigned int* configThr
        #endif
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                , curandSTATE* etatThr
        #endif
        #ifdef RANDMT
        , EtatMT* etatThr, ConfigMT* configThr
        #endif
        #ifdef RANDPHILOX4x32_7
                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
        #endif
            ) {

    float tauRdm;
    float hph = 0.;  // cumulative optical thickness
    float vzn, delta1, h_cur;
    float d_tot = 0.;
    float d;
    float rat;
    int sign_direction;
    int i_layer_fw, i_layer_bh; // index or layers forward and behind the photon
    float costh, sinth2;
    int ilam = ph->ilam*(NATMd+1);  // wavelength offset in optical thickness table

    if (ph->couche == 0) ph->couche = 1;

    #ifdef DEBUG
    int niter = 0;
    // ph->couche is indexed
    // from 1 (TOA layer between interfaces 0 and 1)
    // to NATM (bottom layer between interfaces NATM-1 to NATM)
    if ((ph->couche > NATMd) || (ph->couche <= 0)) {
        printf("Fatal error, wrong index (%d)\n", ph->couche);
    }
    #endif

    // Random Optical Thickness to go through
    tauRdm = -logf(1.F-RAND);

    vzn = __fdividef( ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z , ph->rayon);
    #ifndef ALT_MOVE
    costh = vzn;
    sinth2 = 1.f-costh*costh;
    #endif

    // a priori value for sign_direction:
    // sign_direction may change sign from -1 to +1 if the photon does not
    // cross lower layer
    if (vzn <= 0) sign_direction = -1;
    else sign_direction = 1;

    while (1) {

        #ifdef DEBUG
        niter++;

        if (niter > 2*NATMd+1) {
            printf("niter=%d break\n", niter);
            break;
        }
        #endif

        //
        // stopping criteria
        //
        if (ph->couche == NATMd+1) {
            ph->loc = SURF0P;
            ph->tau = 0.;
            ph->couche -= 1;  // next time photon enters move_sp, it's at layers NATM
            #ifdef DEBUG
            if (ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z > 0) {
                printf("Warning, vzn > 0 at SURF0P in move_sp (vzn=%f)\n", vzn);
            }
            #endif
            break;
        }
        if (ph->couche <= 0) {
            ph->loc = SPACE;
            break;
        }

        //
        // determine the index of the next potential layer
        //
        if (sign_direction < 0) {
            // photon goes down
            // (towards higher indices)
            i_layer_fw = ph->couche;
            i_layer_bh = ph->couche - 1;
        } else {
            // photon goes up
            // (towards lower indices)
            i_layer_fw = ph->couche - 1;
            i_layer_bh = ph->couche;
        }

        #ifdef ALT_MOVE
        // initializations
        costh = vzn;
        sinth2 = 1.f-costh*costh;
        #endif

        //
        // calculate the distance d to the fw layer
        #ifndef ALT_MOVE
        // from the initial position
        #else
        // from the current position
        #endif
        //
        // ri : radius of next layer boundary ri=zi+RTER
        // r  : radius of current point along the path 
        // costh: angle between the position vector and the direction vector
        // In the triangle we have ri² = d² + r² + 2*d*r*costh
        // or: d**2 + 2*r*costh*d + r**2-ri**2 = 0 , to be solved for d
        // delta = 4.r².costh² - 4(r²-ri²) = 4*r²*((ri/r)²-sinth²) = 4*r²*delta1
        // with delta1 = (ri/r)²-sinth²
        rat = (tab.z[i_layer_fw]+RTER)/ph->rayon;
        delta1 = rat*rat - sinth2;   // same sign as delta

        if (delta1 < 0) {
            if (sign_direction > 0) {
                #ifdef DEBUG
                printf("Warning sign_direction (niter=%d, lay=%d, delta1=%f, alt=%f zlay1=%f zlay2=%f vzn=%f)\n",
                        niter, ph->couche, delta1, ph->rayon-RTER,
                        tab.z[i_layer_fw],
                        tab.z[i_layer_bh],
                        vzn);
                #endif

                // because of numerical uncertainties, a downward photon may
                // not strictly be between zi and zi+1
                // in rare case of grazing angle there is sometimes no intersection
                // with current layer because photon is actually slightly above it.
                // therefore we consider that delta=0 such that the photon is
                // tangent to the layer
                delta1 = 0.;
            } else {
                // no intersection, with lower layer, we should go towards higher layer
                sign_direction = 1;
                continue;
            }
        }

        /* Now, there are two real solutions for d
        *  The solution is the smallest positive one
        *
        * if photon goes towards higher layers (sign_direction == 1) and costh>0
        * => we keep the smallest solution in abs. val   (both terms are of opposite signs)
        *
        * if photon goes towards lower layers (sign_direction == -1) and costh<0
        * => we keep the smallest solution in abs. val   (both terms are of opposite signs)
        *
        * if photon goes towards higher layers (sign_direction == 1) and costh<0
        * => we keep the greatest solution in abs. val   (both terms are of same signs)
        *
        */
        /* d = 0.5f*(-2.*ph->rayon*costh + sign_direction*2*ph->rayon*sqrtf(delta1)); simplified to: */
        d = ph->rayon*(-costh + sign_direction*sqrtf(delta1));
        #ifdef DEBUG
        if (d < 0) {
            #ifndef ALT_MOVE
            printf("Warning in move_sp (d=%f < 0 ; vzn=%f, sqrt(delta1)=%f)\n",
                d, vzn, sqrtf(delta1));
            #else
            printf("(alt_move) Warning in move_sp (d=%f < 0 ; vzn=%f, sqrt(delta1)=%f)\n",
                d, vzn, sqrtf(delta1));
            #endif
        } else if (d_tot > d) {
            printf("Error in move_sp (d_tot=%f > d=%f)\n", d_tot, d);
        }
        #endif


        //
        // calculate the optical thickness h_cur to the next layer
        // We compute the layer extinction coefficient of the layer DTau/Dz and multiply by the distance within the layer
        //
        #ifndef ALT_MOVE
        h_cur = __fdividef(abs(tab.h[i_layer_bh+ilam] - tab.h[i_layer_fw+ilam])*(d - d_tot),
                          abs(tab.z[i_layer_bh] - tab.z[i_layer_fw]));
        #else
        h_cur = __fdividef(abs(tab.h[i_layer_bh+ilam] - tab.h[i_layer_fw+ilam])*d,
                          abs(tab.z[i_layer_bh] - tab.z[i_layer_fw]));
        #endif


        //
        // update photon position
        //
        if (hph + h_cur > tauRdm) {
            // photon stops within the layer
            #ifndef ALT_MOVE
            d_tot += (d - d_tot)*(tauRdm - hph)/h_cur;
            #else
            d *= (tauRdm-hph)/h_cur;
            ph->x = ph->x + ph->vx*d;
            ph->y = ph->y + ph->vy*d;
            ph->z = ph->z + ph->vz*d;
            ph->rayon = sqrtf(ph->x*ph->x + ph->y*ph->y + ph->z*ph->z);
            ph->weight *= 1.f - tab.abs[ph->couche+ilam];
            ph->prop_aer = 1.f - tab.pMol[ph->couche+ilam];

            #ifdef DEBUG
            vzn = __fdividef( ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z , ph->rayon);
            #endif
            #endif

            break;
        } else {
            // photon advances to the next layer
            hph += h_cur;
            ph->couche -= sign_direction;
            #ifndef ALT_MOVE
            d_tot = d;
            #else
            ph->x = ph->x + ph->vx*d;
            ph->y = ph->y + ph->vy*d;
            ph->z = ph->z + ph->vz*d;
            ph->rayon = sqrtf(ph->x*ph->x + ph->y*ph->y + ph->z*ph->z);
            vzn = __fdividef( ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z , ph->rayon);
            #endif
        }

    }
    #ifndef ALT_MOVE
    //
    // update the position of the photon
    //
    ph->x = ph->x + ph->vx*d_tot;
    ph->y = ph->y + ph->vy*d_tot;
    ph->z = ph->z + ph->vz*d_tot;
    ph->rayon = sqrtf(ph->x*ph->x + ph->y*ph->y + ph->z*ph->z);
    ph->weight *= 1.f - tab.abs[ph->couche+ilam];
    ph->prop_aer = 1.f - tab.pMol[ph->couche+ilam];
    #endif
}
#endif // SPHERIQUE

__device__ void move_spp(Photon* ph, Tableaux tab, Init* init
        #ifdef RANDMWC
        , unsigned long long* etatThr, unsigned int* configThr
        #endif
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                , curandSTATE* etatThr
        #endif
        #ifdef RANDMT
        , EtatMT* etatThr, ConfigMT* configThr
        #endif
        #ifdef RANDPHILOX4x32_7
                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
        #endif
            ) {

    float tauRdm;
    float hph = 0., aph=0.;  // cumulative optical thicknesses scattering and absorption
    float vzn, tau_tot;
    float d_cur, h_cur, a_cur;
    int sign_direction;
    int i_layer_fw, i_layer_bh ; // index or layers forward and behind the photon
    float costh;
    int ilam = ph->ilam*(NATMd+1);  // wavelength offset in optical thickness table

    // Random Optical Thickness to go through
    tauRdm = -logf(1.F-RAND);

    vzn = ph->vz;
    costh = vzn;

    if (vzn <= 0) {
        sign_direction = -1;
    }
    else {
        sign_direction = 1;
    }

    while (1) {

        //
        // stopping criteria
        //
        if (ph->couche == NATMd) {
            ph->loc = SURF0P;
            ph->couche -= 1;  // next time photon enters move_sp, it's at layer NATM-1
            break;
        }
        if (ph->couche < 0) {
            ph->loc = SPACE;
            break;
        }

        i_layer_fw = ph->couche + (1-sign_direction)/2;
        i_layer_bh = ph->couche + (1+sign_direction)/2;

        d_cur = __fdividef(abs(ph->z - tab.z[i_layer_fw]),abs(costh));

        // calculate the extinction optical thickness h_cur to the next layer
        // We compute the layer extinction coefficient of the layer DTau/Dz and multiply by the distance within the layer

        tau_tot = __fdividef(abs(tab.h[i_layer_fw+ilam] - tab.h[i_layer_bh+ilam]),
                          abs(tab.z[i_layer_fw] - tab.z[i_layer_bh])) * d_cur;

        h_cur = tau_tot * (1.- tab.abs[ph->couche+ilam]); // extinction OT (without gaseous absorption)
        a_cur = tau_tot * tab.abs[ph->couche+ilam]; // gaseous absorption OT

        //
        // update photon position
        //
        if (hph + h_cur > tauRdm) {
            // photon stops within the layer
            d_cur *= (tauRdm-hph)/h_cur;
            a_cur *= (tauRdm-hph)/h_cur;
            ph->x = ph->x + ph->vx*d_cur;
            ph->y = ph->y + ph->vy*d_cur;
            ph->z = ph->z + ph->vz*d_cur;
            ph->weight *= expf(-(aph + a_cur)); // Total gaseous absorption
            ph->prop_aer = 1.f - tab.pMol[ph->couche+ilam];
            break;
        } else {
            // photon advances to the next layer
            hph += h_cur;
            aph += a_cur;
            ph->x = ph->x + ph->vx*d_cur;
            ph->y = ph->y + ph->vy*d_cur;
            ph->z = ph->z + ph->vz*d_cur;
            ph->couche -= sign_direction;
        }
    }
}


__device__ void move_pp(Photon* ph,float*z, float* h, float* pMol , float *abs , float* ho
		#ifdef RANDMWC
		, unsigned long long* etatThr, unsigned int* configThr
		#endif
		#if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                , curandSTATE* etatThr
                #endif
		#ifdef RANDMT
		, EtatMT* etatThr, ConfigMT* configThr
		#endif
		#ifdef RANDPHILOX4x32_7
                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
		#endif
		    ) {


	float Dsca=0.f, dsca=0.f;

	ph->tau += -logf(1.f - RAND)*ph->vz;

	float tauBis;
    int icouche;

	if (ph->loc == OCEAN){  
        if (ph->tau > 0) {
           ph->tau = 0.F;
           ph->loc = SURF0M;
           if (SIMd == 3){
              ph->loc = SPACE;
           }
           return;
        }
        // Si tau<TAUOCEAN le photon atteint le fond 
        else if( ph->tau < ho[NOCEd + ph->ilam *(NOCEd+1)] ){
            ph->loc = SEAFLOOR;
            ph->tau = ho[NOCEd + ph->ilam *(NOCEd+1)];
            return;
        }

        // Calcul de la couche dans laquelle se trouve le photon
        tauBis =  ho[NOCEd + ph->ilam *(NOCEd+1)] - ph->tau;
        icouche = 1;

        while ((ho[icouche+ ph->ilam *(NOCEd+1)] > (tauBis)) && (icouche < NOCEd)) {
            icouche++;
        }
        ph->couche = icouche;

    }

    if (ph->loc == ATMOS) {

        // Si tau<0 le photon atteint la surface
        if(ph->tau < 0.F){
            ph->loc = SURF0P;
            ph->tau = 0.F;

            // move the photon forward down to the surface
            // the linear distance is ph->z/ph->vz
            ph->x += ph->vx * fabs(ph->z/ph->vz);
            ph->y += ph->vy * fabs(ph->z/ph->vz);
            ph->z = 0.;
        return;
        }
        // Si tau>TAUATM le photon atteint l'espace
        else if( ph->tau > h[NATMd + ph->ilam *(NATMd+1)] ){
            ph->loc = SPACE;
            return;
        }
        
        // Sinon il reste dans l'atmosphère, et va subit une nouvelle diffusion
        
        // Calcul de la couche dans laquelle se trouve le photon
        tauBis =  h[NATMd + ph->ilam *(NATMd+1)] - ph->tau;
        icouche = 1;
        
        while ((h[icouche+ ph->ilam *(NATMd+1)] < (tauBis)) && (icouche < NATMd)) {
            icouche++;
        }
        
        ph->couche = icouche;
        ph->prop_aer = 1.f - pMol[ph->couche+ph->ilam*(NATMd+1)];
        ph->weight = ph->weight * (1.f - abs[ph->couche+ph->ilam*(NATMd+1)]);


        float phz,rdist;
        Dsca= fabs(h[icouche] - h[icouche-1]) ;
        dsca= fabs(tauBis - h[icouche-1]) ;

        //calcul de la nouvelle altitude du photon
        phz=z[icouche-1]+(dsca/Dsca)*(z[icouche]-z[icouche-1]);
        rdist=(phz-ph->z)/ph->vz;
        ph->z = phz;
        ph->x = ph->x + ph->vx*rdist;
        ph->y = ph->y + ph->vy*rdist;

    }

}


__device__ void scatter( Photon* ph, float* faer, float* ssa , float* foce , float* sso, int* ip, int* ipo, int le, float* tabthv, float* tabphi
			#ifdef RANDMWC
			, unsigned long long* etatThr, unsigned int* configThr
			#endif
			#if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                        , curandSTATE* etatThr
                        #endif
			#ifdef RANDMT
			, EtatMT* etatThr, ConfigMT* configThr
			#endif
                        #ifdef RANDPHILOX4x32_7
                        , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
                        #endif
			){

	float cTh=0.f ;
	float zang=0.f, theta=0.f;
	int iang, ilay, ipha;
	float stokes1, stokes2, stokes3, stokes4;
	float cTh2, psi;
	float prop_aer = ph->prop_aer;
	
    /* Scattering in atmosphere */
	if(ph->loc!=OCEAN){
        ilay = ph->couche + ph->ilam*(NATMd+1); // atm layer index
        ipha  = ip[ilay]; // atm phase function index
        if (le){
            /* in case of LE the photon units vectors, scattering angle and Psi rotation angle are determined by output zenith and azimuth angles*/
            float thv, phi;
            float vx ,vy ,vz;
            /*int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);
            int ipack = idx/(NBTHETAd*NBPHId);
            int iang  = idx - ipack * (NBTHETAd*NBPHId);
            int ith = iang/NBTHETAd;
            int iph = iang - ith*NBTHETAd ;*/
            phi =  __fdividef(((float)ph->iph) * 2 * PI, NBPHId);
            thv =  __fdividef(((float)ph->ith) * DEMIPI, NBTHETAd);
            vx = __cosf(phi) * __sinf(thv);
            vy = __sinf(phi) * __sinf(thv);
            vz = __cosf(thv);
            theta = calculTheta(ph->vx, ph->vy, ph->vz, vx, vy, vz);
            cTh = __cosf(theta);
            cTh2 = cTh * cTh;
            calculPsiLE(ph->ux , ph->uy, ph->uz, ph->vx , ph->vy, ph->vz, vx, vy, vz, &psi, &ph->ux, &ph->uy, &ph->uz); 
            ph->vx = vx;
            ph->vy = vy;
            ph->vz = vz;
        }

		if( prop_aer<RAND ){
            /***********************/
            /* Rayleigh scattering */
            /***********************/
            if(!le) {
                /* in the case of propagation (not LE) the photons scattering angle and Psi rotation angle are determined randomly*/
			    /////////////
			    // Get Theta (see Wang et al., 2012)
			    float b = (RAND - 4.0 * ALPHAd - BETAd) / (2.0 * ALPHAd);
			    float expo = 1./2.;
			    float base = ACUBEd + b*b;
			    float tmp  = pow(base, expo);
			    expo = 1./3.;
			    base = -b + tmp;
			    float u = pow(base,expo);
			    cTh     = u - Ad / u;  						       
			    if (cTh < -1.0) cTh = -1.0;
			    if (cTh >  1.0) cTh =  1.0;
			    cTh2 = cTh * cTh;
			
			    /////////////
			    //  Get Phi
			    //  Biased sampling scheme for psi 1)
			    psi = RAND * DEUXPI;
            }

			// Stokes vector rotation
			rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3,  psi,
				     &ph->stokes1, &ph->stokes2, &ph->stokes3 );

			// Scattering matrix multiplication
			float cross_term;
			stokes1 = ph->stokes1;
			stokes2 = ph->stokes2;
			cross_term  = DELTA_PRIMd * (ph->stokes1 + ph->stokes2);
			ph->stokes1 = 3./2. * (  DELTAd  * stokes1 + cross_term );
			ph->stokes2 = 3./2. * (  DELTAd  * cTh2 * stokes2 + cross_term );			
			ph->stokes3 = 3./2. * (  DELTAd  * cTh  * ph->stokes3 );
			ph->stokes4 = 3./2. * (  DELTAd  * DELTA_SECOd * cTh * ph->stokes4 );

            if (!le){
			    // Bias sampling scheme 2): Debiasing
			    float phase_func;
			    phase_func = 3./4. * DELTAd * (cTh2+1.0) + 3.0 * DELTA_PRIMd;
			    ph->stokes1 /= phase_func;  
			    ph->stokes2 /= phase_func;  
			    ph->stokes3 /= phase_func;     		
			    ph->stokes4 /= phase_func;     		
            }

		}
		else{
            /***********************/
            /* Aerosols scattering */
            /***********************/
            float P11,P22,P33,P43;
            if(!le) {
                /* in the case of propagation (not LE) the photons scattering angle and Psi rotation angle are determined randomly*/
			    /////////////
                // Get Theta from Cumulative Distribution Function
                // (column number 4 of faer)
			    zang = RAND*(NFAERd-2);
			    iang= __float2int_rd(zang);
			    zang = zang - iang;
			    theta = faer[ipha*NFAERd*10+iang*10+4]+ zang*( faer[ipha*NFAERd*10+(iang+1)*10+4]-faer[ipha*NFAERd*10+iang*10+4] );
			    //theta = faer[ipha*NFAERd*5+iang*5+4]+ zang*( faer[ipha*NFAERd*5+(iang+1)*5+4]-faer[ipha*NFAERd*5+iang*5+4] );
			    cTh = __cosf(theta);

			    /////////////
			    //  Get Phi
			    //  Biased sampling scheme for psi 1)
			    psi = RAND * DEUXPI;	

                /////////////
                // Get Scattering matrix from CDF
                // (column 0 -> 3 of faer)
                P11 = faer[ipha*NFAERd*10+iang*10+0];
                P22 = faer[ipha*NFAERd*10+iang*10+1];
                P33 = faer[ipha*NFAERd*10+iang*10+2];
                P43 = faer[ipha*NFAERd*10+iang*10+3];
            }

            else {
                /////////////
                // Get Index of scattering angle and Scattering matrix directly 
                // (column 6 -> 9 of faer)
                zang = theta * NFAERd/PI ;
                iang = __float2int_rd(zang);
			    zang = zang - iang;
                if (abs(cTh) < 1) {
                    P11 = faer[ipha*NFAERd*10+iang*10+6] + zang * (faer[ipha*NFAERd*10+(iang+1)*10+6] - faer[ipha*NFAERd*10+iang*10+6]);
                    P22 = faer[ipha*NFAERd*10+iang*10+7] + zang * (faer[ipha*NFAERd*10+(iang+1)*10+7] - faer[ipha*NFAERd*10+iang*10+7]);
                    P33 = faer[ipha*NFAERd*10+iang*10+8] + zang * (faer[ipha*NFAERd*10+(iang+1)*10+8] - faer[ipha*NFAERd*10+iang*10+8]);
                    P43 = faer[ipha*NFAERd*10+iang*10+9] + zang * (faer[ipha*NFAERd*10+(iang+1)*10+9] - faer[ipha*NFAERd*10+iang*10+9]);
                }
                else if (cTh >=1) {
                    P11 = faer[ipha*NFAERd*10+0*10+6];
                    P22 = faer[ipha*NFAERd*10+0*10+7];
                    P33 = faer[ipha*NFAERd*10+0*10+8];
                    P43 = faer[ipha*NFAERd*10+0*10+9];
                }
                else {
                    P11 = faer[ipha*NFAERd*10+(NFAERd-1)*10+6];
                    P22 = faer[ipha*NFAERd*10+(NFAERd-1)*10+7];
                    P33 = faer[ipha*NFAERd*10+(NFAERd-1)*10+8];
                    P43 = faer[ipha*NFAERd*10+(NFAERd-1)*10+9];
                }
            }

			// Stokes vector rotation
			rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3,   psi,
			        &ph->stokes1, &ph->stokes2, &ph->stokes3);

			// Scattering matrix multiplication
            stokes3=ph->stokes3;
            stokes4=ph->stokes4;
			ph->stokes1 *= P11;
			ph->stokes2 *= P22;
			ph->stokes3 = stokes3 * P33 - stokes4 * P43;
			ph->stokes4 = stokes4 * P33 + stokes3 * P43;

            if (!le){
			    // Bias sampling scheme 2): Debiasing
			    float debias;
			    debias = __fdividef( 2., P11 + P22 );
			    ph->stokes1 *= debias;  
			    ph->stokes2 *= debias;  
			    ph->stokes3 *= debias;  
			    ph->stokes4 *= debias;  

            }

            // Photon weight reduction due to the aerosol single scattering albedo of the current layer
			ph->weight *= ssa[ilay];
			
		}

	}
	else{	/* Photon dans l'océan */
	    float prop_raman=1., new_wavel;
        float cPsi, sPsi;
        ilay = ph->couche + ph->ilam*(NOCEd+1); // oce layer index
        ipha  = ipo[ilay]; // oce phase function index

        // we fix the proportion of Raman to 2% at 488 nm, !! DEV
        //prop_raman = 0.02 * pow ((1.e7/ph->wavel-3400.)/(1.e7/488.-3400.),5); // Raman scattering to pure water scattering ratio

	    if(prop_raman <RAND ){
            // diffusion Raman
            // Phase function similar to Rayleigh
		    // Get Teta (see Wang et al., 2012)
			float b = (RAND - 4.0 * ALPHAd - BETAd) / (2.0 * ALPHAd);
			float expo = 1./2.;
			float base = ACUBEd + b*b;
			float tmp  = pow(base, expo);
			expo = 1./3.;
			base = -b + tmp;
			float u = pow(base,expo);
			cTh     = u - Ad / u;  						       

			if (cTh < -1.0) cTh = -1.0;
			if (cTh >  1.0) cTh =  1.0;
			cTh2 = cTh * cTh;
			
			/////////////
			//  Get Phi
			// Biased sampling scheme for phi
			psi = RAND * DEUXPI;	//psiPhoton
			cPsi = __cosf(psi);	//cosPsiPhoton
			sPsi = __sinf(psi);     //sinPsiPhoton		

			// Calcul des parametres de Stokes du photon apres diffusion
			
			// Rotation des paramètres de stokes
			rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3, psi,
				     &ph->stokes1, &ph->stokes2, &ph->stokes3);

			// Calcul des parametres de Stokes du photon apres diffusion
			float cross_term;
			stokes1 = ph->stokes1;
			stokes2 = ph->stokes2;
			cross_term  = DELTA_PRIMd * (stokes1 + stokes2);
			ph->stokes1 = 3./2. * (  DELTAd  * stokes1 + cross_term );
			ph->stokes2 = 3./2. * (  DELTAd  * cTh2 * stokes2 + cross_term );			
			ph->stokes3 = 3./2. * (  DELTAd * cTh  * ph->stokes3 );
			ph->stokes4 = 3./2. * (  DELTAd * DELTA_SECOd * cTh * ph->stokes4 );
			// bias sampling scheme
			float phase_func;
			phase_func = 3./4. * DELTAd * (cTh2+1.0) + 3.0 * DELTA_PRIMd;
			ph->stokes1 /= phase_func;  
			ph->stokes2 /= phase_func;  
			ph->stokes3 /= phase_func;     		
			ph->stokes4 /= phase_func;     		

            // Changement de longueur d onde
            new_wavel  = 22.94 + 0.83 * (ph->wavel) + 0.0007 * (ph->wavel)*(ph->wavel);
            ph->weight /= new_wavel/ph->wavel;
            ph->wavel = new_wavel;
		  }

	  else{
          // diffusion elastique
		
		zang = RAND*(NFOCEd-2);
		iang = __float2int_rd(zang);
		zang = zang - iang;

		theta = foce[ipha*NFOCEd*5+iang*5+4]+ zang*( foce[ipha*NFOCEd*5+(iang+1)*5+4]-foce[ipha*NFOCEd*5+iang*5+4] );
		
		cTh = __cosf(theta);

        //////////////
        //  Get Phi

        // biased sampling scheme for phi
        psi = RAND * DEUXPI;	//psiPhoton
        cPsi = __cosf(psi);	//cosPsiPhoton
        sPsi = __sinf(psi);     //sinPsiPhoton
        // Rotation des paramètres de stokes
        rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3, psi,
                &ph->stokes1, &ph->stokes2, &ph->stokes3);


		stokes3 = ph->stokes3;
		stokes4 = ph->stokes4;
        // Calcul des parametres de Stokes du photon apres diffusion
        ph->stokes1 *= foce[ipha*NFOCEd*5+iang*5+0];
        ph->stokes2 *= foce[ipha*NFOCEd*5+iang*5+1];
        ph->stokes3 = stokes3*foce[ipha*NFOCEd*5+iang*5+2] - stokes4*foce[ipha*NFOCEd*5+iang*5+3];
        ph->stokes4 = stokes4*foce[ipha*NFOCEd*5+iang*5+2] + stokes3*foce[ipha*NFOCEd*5+iang*5+3];

        float debias;
        debias = __fdividef( 2., foce[ipha*NFOCEd*5+iang*5+0] + foce[ipha*NFOCEd*5+iang*5+1] );
        ph->stokes1 *= debias;
        ph->stokes2 *= debias;
        ph->stokes3 *= debias;
        ph->stokes4 *= debias;

		ph->weight *= sso[ilay];

	 } // elastic scattering

	/** Roulette russe **/
	if( ph->weight < WEIGHTRR ){
		if( RAND < __fdividef(ph->weight,WEIGHTRR) ){
			ph->weight = WEIGHTRR;
		}
		else{
				ph->loc = ABSORBED;
			}
		}
		
    } //photon in ocean

   ////////// Fin séparation ////////////
   
    if (!le){
        modifyUV( ph->vx, ph->vy, ph->vz, ph->ux, ph->uy, ph->uz, cTh, psi, 
                &ph->vx, &ph->vy, &ph->vz, &ph->ux, &ph->uy, &ph->uz) ;
    }

}


/* surfaceAgitee
* Reflexion sur une surface agitée ou plane en fonction de la valeur de DIOPTRE
*/
__device__ void surfaceAgitee(Photon* ph, float* alb
		#ifdef RANDMWC
		, unsigned long long* etatThr, unsigned int* configThr
		#endif
		#if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                , curandSTATE* etatThr
                #endif
		#ifdef RANDMT
		, EtatMT* etatThr, ConfigMT* configThr
		#endif
		#ifdef RANDPHILOX4x32_7
                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
		#endif
			){
	
	if( SIMd == -2){ // Atmosphère , la surface absorbe tous les photons
		ph->loc = ABSORBED;
		return;
	}
	
	// Réflexion sur le dioptre agité
	float theta;	// Angle de deflection polaire de diffusion [rad]
	float psi;		// Angle azimutal de diffusion [rad]
	float cTh, sTh;	//cos et sin de l'angle d'incidence du photon sur le dioptre
	
	float sig = 0.F;
	float beta = 0.F;	// Angle par rapport à la verticale du vecteur normal à une facette de vagues 
	float sBeta;
	float cBeta;
	
	float alpha ;	//Angle azimutal du vecteur normal a une facette de vagues
	
	float nind;
	float temp;
	
    // coordinates of the normal to the wave facet in the original axis
	float nx, ny, nz;

    // coordinates of the normal to the wave facet in the local axis (Nx, Ny, Nz)
	float n_x, n_y, n_z;

	float s1, s2, s3 ;
    float stokes3, stokes4;
	
	float rpar, rper, rparper, rparper_cross;	// Coefficient de reflexion parallèle et perpendiculaire
	float rpar2;		// Coefficient de reflexion parallèle au carré
	float rper2;		// Coefficient de reflexion perpendiculaire au carré
	float rat;			// Rapport des coefficients de reflexion perpendiculaire et parallèle
	float ReflTot;		// Flag pour la réflexion totale sur le dioptre
	float cot;			// Cosinus de l'angle de réfraction du photon
	float ncot, ncTh;	// ncot = nind*cot, ncoi = nind*cTh
	float tpar, tper;	//
    float geo_trans_factor;
    int iter=0;
    float vzn;  // projection of V on the local vertical
	
    #ifdef SPHERIQUE
    // define 3 vectors Nx, Ny and Nz in cartesian coordinates which define a
    // local orthonormal basis at the impact point.
    // Nz is the local vertical direction, the direction of the 2 others does not matter
    // because the azimuth is chosen randomly
    float Nxx, Nxy, Nxz;
    float Nyx, Nyy, Nyz;
    float Nzx, Nzy, Nzz;
    float norm;

    // Nz is the vertical at the impact point
    Nzx = ph->x/RTER;
    Nzy = ph->y/RTER;
    Nzz = ph->z/RTER;

    // Ny is chosen arbitrarily by cross product of Nz with axis X = (1,0,0)
    // and normalized
    Nyx = 0.;
    Nyy = Nzz;
    Nyz = -Nzy;
    norm = sqrt(Nyy*Nyy + Nyz*Nyz);
    Nyy /= norm;
    Nyz /= norm;

    // Nx is the cross product of Ny and Nz
    Nxx = Nzy*Nzy + Nzz*Nzz;
    Nxy = -Nzx*Nzy;
    Nxz = -Nzx*Nzz;
    norm = sqrt(Nxx*Nxx + Nxy*Nxy + Nxz*Nxz);
    Nxx /= norm;
    Nxy /= norm;
    Nxz /= norm;


    #ifdef DEBUG
    // we check that there is no upward photon reaching surface0+
    if ((ph->loc == SURF0P) && (ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z > 0)) {
        // upward photon when reaching the surface at (0+)
        printf("Warning, vzn>0 (vzn=%f) with SURF0+ in surfaceAgitee\n",
                ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z);
    }
    #endif
    #endif

	
	/** **/
    // DR Estimation of the probability P of interaction of the photon with zentih angle theta with a facet of slope beta and azimut alpha	
    // DR P_alpha_beta : Probability of occurence of a given azimuth and slope
    // DR P_alpha_beta = P_Cox_Munk(beta) * P(alpha | beta), conditional probability, for normal incidence, independent variables and P(alpha|beta)=P(alpha)=1/2pi
    // DR following Plass75:
    // DR Pfacet : Probability of occurence of a facet
    // DR Pfacet = projected area of the facet divided by unit area of the possible interaction surface * P_alpha_beta
    // DR Pfacet = P_alpha_beta / cos(beta)
    // DR for non normal incident angle, the probability of interaction between the photon and the facet is proportional to the surface of the facet seen by the photon so
    // DR that is cosine of incident angle of photon on the facet theta_inc=f(alpha,beta,theta)
    // DR P # Pfacet * cos(theta_inc) for cos(theta_inc) >0
    // DR P = 0 for cos(theta_inc)<=0
    // DR for having a true probability, one has to normalize this to 1. The A normalization factor depends on theta and is the sum on all alpha and beta with the condition
    // DR cos(theta_inc)>0 (visible facet)
    // DR A = Sum_0_2pi Sumr_0_pi/2 P_alpha_beta /cos(beta) cos(theta_inc) dalpha dbeta
    // DR Finally P = 1/A * P_alpha_beta  /cos(beta) cos(theta_inc)
	if( DIOPTREd !=0 ){
        // Rough surface

        theta = DEMIPI;
        // DR Computation of P_alpha_beta = P_Cox_Munk(beta) * P(alpha | beta)
        // DR we draw beta first according to Cox_Munk isotropic and then draw alpha, conditional probability
        // DR rejection method: to exclude unphysical azimuth (leading to incident angle theta >=PI/2)
        // DR we continue until acceptable value for alpha
        sig = sqrtf(0.003F + 0.00512f *WINDSPEEDd);
        beta = atanf( sig*sqrtf(-__logf(RAND)) );
        while (theta >= DEMIPI) {
            iter++;
            if (iter >= 100) {
                // safety check
                #ifdef DEBUG
                printf("Warning, photon rejected in RoughSurface while loop\n");
                printf("  V=(%f,%f,%f)\n",
                        ph->vx,
                        ph->vy,
                        ph->vz
                      );
                #endif
                ph->loc = NONE;
                break;
            }
           alpha = DEUXPI * RAND;
           sBeta = __sinf( beta );
           cBeta = __cosf( beta );

           // the facet has coordinates
           // (sin(beta)*cos(alpha), sin(beta)*sin(alpha), cos(beta)) in axis (Nx, Ny, Nz)
           n_x = sBeta*__cosf( alpha );
           n_y = sBeta*__sinf( alpha );

           // compute relative index of refraction
           // DR a: air, b: water , Mobley 2015 nind = nba = nb/na
           if (ph->loc == SURF0M) {
               nind = __fdividef(1.f,NH2Od);
               n_z = -cBeta;
           }
           else{
               nind = NH2Od;
               n_z = cBeta;
           }

           temp = -(n_x*ph->vx + n_y*ph->vy + n_z*ph->vz);
           theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), temp ) ));
        }
    } else {
        // Flat surface

        beta = 0;
        alpha = DEUXPI * RAND;
        sBeta = __sinf( beta );
        cBeta = __cosf( beta );
        n_x = sBeta*__cosf( alpha );
        n_y = sBeta*__sinf( alpha );

        if (ph->loc == SURF0M) {
            nind = __fdividef(1.f,NH2Od);
            n_z = -cBeta;
        }
        else{
            nind = NH2Od;
            n_z = cBeta;
        }
        temp = -(n_x*ph->vx + n_y*ph->vy + n_z*ph->vz);
        theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), temp ) ));
    }


    // express the coordinates of the normal to the wave facet in the original
    // axis instead of local axis (Nx, Ny, Nz)
    #ifdef SPHERIQUE
    nx = n_x*Nxx + n_y*Nyx + n_z*Nzx;
    ny = n_x*Nxy + n_y*Nyy + n_z*Nzy;
    nz = n_x*Nxz + n_y*Nyz + n_z*Nzz;
    #else
    nx = n_x;
    ny = n_y;
    nz = n_z;
    #endif


	cTh = __cosf(theta);
	sTh = __sinf(theta);

    // Anorm factor modelled with a simple linear fit that represents the departure from vz,
    // (Anorm-vz)
    // ^                                               +
    // |                                              + 
    // |                                             + 
    // |                                            + 
    // |                                           + 
    // ++++++++++++++++++++++++++++++++++++++++++++--------> (theta)
    // 0                                          |        90
    //                                        Theta_thres=f(Windspeed)
    // The slope of the model is constant=0.004 and threshold depends on windspeed. Below threshold on theta, all slopes
    // are possible and thus A=1/vz
    float Anorm;
    float slopeA=0.00377;
    float theta_thres;
    theta_thres = 83.46 - WINDSPEEDd; // between 1 and 15 m/s
    #ifdef SPHERIQUE
    // avz is the projection of V on the local vertical
    float avz = abs(ph->x*ph->vx + ph->y*ph->vy + ph->z*ph->vz)/RTER;
    #else
    float avz = abs(ph->vz);
    #endif
    float aavz = acosf(avz)*360./DEUXPI;
    if(aavz > theta_thres){
       Anorm = avz + slopeA * (aavz - theta_thres);
    }
    else{
       Anorm = avz;
    }

    // DR probability of slope interaction with photon corection factor, biased sampling correction of pure Cox_Munk probability function
    ph->weight *= __fdividef(abs(cTh), cBeta * Anorm);

	// Rotation of Stokes parameters
	s1 = ph->stokes1;
	s2 = ph->stokes2;
	s3 = ph->stokes3;

	if( (s1!=s2) || (s3!=0.F) ){

		temp = __fdividef(nx*ph->ux + ny*ph->uy + nz*ph->uz,sTh);
		psi = acosf( fmin(1.00F, fmax( -1.F, temp ) ));	

		if( (nx*(ph->uy*ph->vz-ph->uz*ph->vy) + ny*(ph->uz*ph->vx-ph->ux*ph->vz) + nz*(ph->ux*ph->vy-ph->uy*ph->vx) ) <0 ){
			psi = -psi;
		}

        rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3, psi,
                &ph->stokes1, &ph->stokes2, &ph->stokes3);
	}

	if( sTh<=nind){
		temp = __fdividef(sTh,nind);
		cot = sqrtf( 1.0F - temp*temp );
		ncTh = nind*cTh;
		ncot = nind*cot;
		rpar = __fdividef(ncTh - cot,ncTh  + cot); // DR Mobley 2015 sign convention
		rper = __fdividef(cTh - ncot,cTh + ncot);
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
        rparper = rpar * rper;
        rparper_cross = 0.;
        // DR rat is the energetic reflection factor used to normalize the R and T matrix (see Xun 2014)
		rat =  __fdividef(ph->stokes1*rper2 + ph->stokes2*rpar2,ph->stokes1+ph->stokes2);
		//rat = 0.5 * (rper2 + rpar2); // DR see Xun 2014, eq 15 strange ....
		ReflTot = 0;
	}
	else{
		cot = 0.f;
		rpar = 1.f;
		rper = 1.f;
        rat = 1.f;
        // DR rat is normalizing the relection matrix
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
        rparper = __fdividef(2.*sTh*sTh*sTh*sTh, 1.-(1.+nind * nind)*cTh*cTh) - 1.; // DR !! Mobley 2015
        rparper_cross = -__fdividef(2.*cTh*sTh*sTh*sqrtf(sTh*sTh-nind*nind), 1.-(1.+nind * nind)*cTh*cTh); // DR !! Mobley 2015
		ReflTot = 1;
	}

    stokes3 = ph->stokes3;	
    stokes4 = ph->stokes4;	
	if( (ReflTot==1) || (SURd==1) || ( (SURd==3)&&(RAND<rat) ) ){

		
		ph->stokes1 *= rper2;
		ph->stokes2 *= rpar2;
		ph->stokes3 = rparper*stokes3 + rparper_cross*stokes4; // DR Mobley 2015 sign convention
		ph->stokes4 = rparper*stokes4 - rparper_cross*stokes3; // DR Mobley 2015 sign convention
		
		ph->vx += 2.F*cTh*nx;
		ph->vy += 2.F*cTh*ny;
		ph->vz += 2.F*cTh*nz;
		ph->ux = __fdividef( nx-cTh*ph->vx,sTh );
		ph->uy = __fdividef( ny-cTh*ph->vy,sTh );
		ph->uz = __fdividef( nz-cTh*ph->vz,sTh );
		

        // DR Normalization of the reflexion matrix
        // DR the reflection coefficient is taken into account:
        // DR once in the random selection (Rand < rat)
        // DR once in the reflection matrix multiplication
        // DR so twice and thus we normalize by rat (Xun 2014).
        // DR not to be applied for forced reflection (SUR=1 or total reflection) where there is no random selection
		if (SURd==3 && ReflTot==0) {
			ph->weight /= rat;
			}

        #ifdef SPHERIQUE
        vzn = ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z;
        #else
        vzn = ph->vz;
        #endif

        //
        // photon next location
        //
        if (ph->loc == SURF0P) {
            if (vzn > 0) {  // avoid multiple reflexion above the surface
                // SURF0P becomes ATM or SPACE
                if( SIMd==-1 || SIMd==0 ){
                    ph->loc = SPACE;
                } else{
                    ph->loc = ATMOS;
                }
            } // else, no change of location
        } else {
            if (vzn < 0) {  // avoid multiple reflexion under the surface
                // SURF0M becomes OCEAN or ABSORBED
                if( SIMd==1 ){
                    ph->loc = ABSORBED;
                } else{
                    ph->loc = OCEAN;
                }
            } // else, no change of location
        }


	} // Reflection

	else{	// Transmission

		
        geo_trans_factor = nind* cot/cTh; // DR Mobley 2015 OK , see Xun 2014
		tpar = __fdividef( 2*cTh,ncTh+ cot);
		tper = __fdividef( 2*cTh,cTh+ ncot);
		
		ph->stokes2 *= tpar*tpar*geo_trans_factor;
		ph->stokes1 *= tper*tper*geo_trans_factor;
		ph->stokes3 *= tpar*tper*geo_trans_factor; //DR positive factor Mobley 2015
		ph->stokes4 *= tpar*tper*geo_trans_factor; //DR positive factor Mobley 2015
		
		alpha  = __fdividef(cTh,nind) - cot;
		ph->vx = __fdividef(ph->vx,nind) + alpha*nx;
		ph->vy = __fdividef(ph->vy,nind) + alpha*ny;
		ph->vz = __fdividef(ph->vz,nind) + alpha*nz;
		ph->ux = __fdividef( nx+cot*ph->vx,sTh )*nind;
		ph->uy = __fdividef( ny+cot*ph->vy,sTh )*nind;
		ph->uz = __fdividef( nz+cot*ph->vz,sTh )*nind;

        #ifdef SPHERIQUE
        vzn = ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z;
        #else
        vzn = ph->vz;
        #endif


        // DR Normalization of the transmission matrix
        // the transmission coefficient is taken into account:
        // once in the random selection (Rand > rat)
        // once in the transmission matrix multiplication
        // so we normalize by (1-rat) (Xun 2014).
        // Not to be applied for forced transmission (SUR=2)
        if ( SURd == 3) 
            ph->weight /= (1-rat);

        //
        // photon next location
        //
        if (ph->loc == SURF0M) {
            if (vzn > 0) {
                // SURF0P becomes ATM or SPACE
                if( SIMd==-1 || SIMd==0 ){
                    ph->loc = SPACE;
                } else{
                    ph->loc = ATMOS;
                }
            } else {
                // multiple transmissions (vz<0 after water->air transmission)
                ph->loc = SURF0P;
            }
        } else {
            if (vzn < 0) {  // avoid multiple reflexion under the surface
                // SURF0M becomes OCEAN or ABSORBED
                if( SIMd==-1 || SIMd==1 ){
                    ph->loc = ABSORBED;
                } else{
                    ph->loc = OCEAN;
                }
            } else {
                // multiple transmissions (vz<0 after water->air transmission)
                // (for symmetry, but should not happen)
                ph->loc = SURF0P;
            }
        }

	} // Transmission
}


/* surfaceLambertienne
* Reflexion sur une surface lambertienne
*/
__device__ void surfaceLambertienne(Photon* ph, float* alb
						#ifdef RANDMWC
						, unsigned long long* etatThr, unsigned int* configThr
						#endif
                                                #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
                                                , curandSTATE* etatThr
						#endif
						#ifdef RANDMT
						, EtatMT* etatThr, ConfigMT* configThr
						#endif
                                                #ifdef RANDPHILOX4x32_7
                                                , philox4x32_ctr_t* etatThr, philox4x32_key_t* configThr
                                                #endif
						){
	
	if( SIMd == -2){ 	// Atmosphère ou océan seuls, la surface absorbe tous les photons
		ph->loc = ABSORBED;
		return;
	}
	
	float uxn,vxn,uyn,vyn,uzn,vzn;	// Vecteur du photon après reflexion
	float cTh2 = RAND;
	float cTh = sqrtf( cTh2 );
	float sTh = sqrtf( 1.0F - cTh2 );
	
	float phi = RAND*DEUXPI;	//angle azimutal
	float cPhi = __cosf(phi);
	float sPhi = __sinf(phi);
	
    #ifdef SPHERIQUE
	float icp, isp, ict, ist;	// Sinus et cosinus de l'angle d'impact
    #endif
	

	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	/** Calcul du theta impact et phi impact **/
	
    if (ph->loc != SEAFLOOR){

	
	/** Calcul de l'angle entre l'axe z et la normale au point d'impact **/
	/*NOTE: le float pour les calculs suivant fait une erreur de 2.3% 
	* par exemple (theta_float=0.001196 / theta_double=0.0011691
	* Mais ils sont bien plus performant et cette erreur ne pose pas de problème jusqu'à présent.
	* De plus, l'angle d'impact n'est pas calculé mais directement les cosinus et sinus de cet angle.
	*/
	if( ph->z > 0. ){
		ict = __fdividef(ph->z,RTER);
		
		if(ict>1.f){
			ict = 1.f;
		}
		
		ist = sqrtf( 1.f - ict*ict );
		
		if(ph->x >= 0.f) ist = -ist;
		
		if( sqrtf(ph->x*ph->x + ph->y*ph->y)<1.e-6 ){
			/*NOTE En fortran ce test est à 1.e-8, relativement au double utilisés, peut peut être être supprimer ici*/
			icp = 1.f;
		}
		else{
			icp = __fdividef(ph->x,sqrtf(ph->x*ph->x + ph->y*ph->y));
			isp = sqrtf( 1.f - icp*icp );
			
			if( ph->y < 0.f ) isp = -isp;
		}
	}
	else{
		// Photon considéré comme perdu
		ph->loc = ABSORBED;	// Correspondant au weight=0 en Fortran
		return;
	}
	
	
	/** Il faut exprimer Vx,y,z et Ux,y,z dans le repère de la normale au point d'impact **/
	vxn= ict*icp*ph->vx - ict*isp*ph->vy + ist*ph->vz;
	vyn= isp*ph->vx + icp*ph->vy;
	vzn= -icp*ist*ph->vx + ist*isp*ph->vy + ict*ph->vz;
	
	uxn= ict*icp*ph->ux - ict*isp*ph->uy + ist*ph->uz;
	uyn= isp*ph->ux + icp*ph->uy;
	uzn= -icp*ist*ph->ux + ist*isp*ph->uy + ict*ph->uz;
	
	ph->vx = vxn;
	ph->vy = vyn;
	ph->vz = vzn;
	ph->ux = uxn;
	ph->uy = uyn;
	ph->uz = uzn;

    } // photon not seafloor
	
	#endif
	
	
	/** calcul u,v new **/
	vxn = cPhi*sTh;
	vyn = sPhi*sTh;
	vzn = cTh;
	
	uxn = cPhi*cTh;
	uyn = sPhi*cTh;
	uzn = -sTh;
	

	// Depolarisation du Photon
	float norm;
	norm = ph->stokes1 + ph->stokes2;
	ph->stokes1 = 0.5 * norm;
	ph->stokes2 = 0.5 * norm;
    ph->stokes3 = 0.0;
    ph->stokes4 = 0.0;

	
	ph->vx = vxn;
	ph->vy = vyn;
	ph->vz = vzn;
	ph->ux = uxn;
	ph->uy = uyn;
	ph->uz = uzn;
	

    if (DIOPTREd!=4 && ((ph->loc == SURF0M) || (ph->loc == SURF0P))){
	  // Si le dioptre est seul, le photon est mis dans l'espace
	  bool test_s = ( SIMd == -1);
	  ph->loc = SPACE*test_s + ATMOS*(!test_s);
    }
	
    if (ph->loc != SEAFLOOR){

	  ph->weight *= alb[0+ph->ilam*2];

	  #ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	  /** Retour dans le repère d'origine **/
	  // Re-projection vers le repères de direction de photon. L'angle à prendre pour la projection est -angleImpact
	  isp = -isp;
	  ist = -ist;
	
	  vxn= ict*icp*ph->vx - ict*isp*ph->vy + ist*ph->vz;
	  vyn= isp*ph->vx + icp*ph->vy;
	  vzn= -icp*ist*ph->vx + ist*isp*ph->vy + ict*ph->vz;
	
	  uxn= ict*icp*ph->ux - ict*isp*ph->uy + ist*ph->uz;
	  uyn= isp*ph->ux + icp*ph->uy;
	  uzn= -icp*ist*ph->ux + ist*isp*ph->uy + ict*ph->uz;
	
	  ph->vx = vxn;
	  ph->vy = vyn;
	  ph->vz = vzn;
	  ph->ux = uxn;
	  ph->uy = uyn;
	  ph->uz = uzn;
	#endif
    } // not seafloor 

    else {
	  ph->weight *= alb[1+ph->ilam*2];
      ph->loc = OCEAN;
    }
    
}



__device__ void countPhoton(Photon* ph,
        Tableaux tab,
        int count_level
		#ifdef PROGRESSION
		, Variables* var   // TODO: remove nbPhotonsSorThr
		#endif
		    ) {

    if (count_level < 0) {
        // don't count anything
        return;
    }

    // don't count the photons directly transmitted
    if ((ph->weight == WEIGHTINIT) && (ph->stokes1 == ph->stokes2) && (ph->stokes3 == 0.f) && (ph->stokes4 == 0.f)) {
        return;
    }

    #ifdef DOUBLE 
    double *tabCount;                   // pointer to the "counting" array:
    #else                               // may be TOA, or BOA down, and so on
    float *tabCount; 
    #endif

    float theta = acosf(fmin(1.F, fmax(-1.F, 0.f * ph->vx + 1.f * ph->vz)));
    #ifdef SPHERIQUE
    if(ph->vz<=0.f) {
         // do not count the downward photons leaving atmosphere
         return;
    }
    #endif

	if(theta == 0.F)
	{
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurtheta), 1);
		#endif
		//return;
	}


	float psi;
	int ith=0, iphi=0, il=0;
	// Initialisation de psi
	calculPsi(ph, &psi, theta);
	
	// Rotation of stokes parameters
    float s1, s2, s3, s4;
    rotateStokes(ph->stokes1, ph->stokes2, ph->stokes3,  psi,
            &s1, &s2, &s3);
    s4 = ph->stokes4;
	// Calcul de la case dans laquelle le photon sort
	if (LEd == 0) calculCase(&ith, &iphi, &il, ph 
			   #ifdef PROGRESSION
			   , var
			   #endif
			   );
    else {
        ith = ph->ith;
        iphi= ph->iph;
        il = ph->ilam;
        ph->weight *= __expf(__fdividef(-(tab.h[NATMd + ph->ilam *(NATMd+1)]-ph->tau),abs(ph->vz))); // LE attenuation to TOA
    }
	
  	/*if( ph->vy<0.f )
    		s3 = -s3;*/  // DR 
	
    // Change sign convention for compatibility with OS
    s3 = -s3;

	float tmp = s1;
	s1 = s2;
	s2 = tmp;
	

	float weight = ph->weight;

    #ifdef DEBUG
    int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
    if (isnan(weight)) printf("(idx=%d) Error, weight is NaN\n", idx);
    if (isnan(s1)) printf("(idx=%d) Error, s1 is NaN\n", idx);
    if (isnan(s2)) printf("(idx=%d) Error, s2 is NaN\n", idx);
    if (isnan(s3)) printf("(idx=%d) Error, s3 is NaN\n", idx);
    #endif

	// Rangement du photon dans sa case, et incrémentation de variables
	if(((ith >= 0) && (ith < NBTHETAd)) && ((iphi >= 0) && (iphi < NBPHId)) && (il >= 0) && (il < NLAMd) && (!isnan(weight)))
	{
        // select the appropriate level (count_level)
        tabCount = tab.tabPhotons + count_level*5*NBTHETAd*NBPHId*NLAMd;

        // count in that level
        #ifdef DOUBLE 
            DatomicAdd(tabCount+(0 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), (double)weight * (double)s1);
            DatomicAdd(tabCount+(1 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), (double)weight * (double)s2);
            DatomicAdd(tabCount+(2 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), (double)weight * (double)s3);
            DatomicAdd(tabCount+(3 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), (double)weight * (double)s4);
            DatomicAdd(tabCount+(4 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), (double)1.);
        #else
            atomicAdd(tabCount+(0 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), weight * s1);
            atomicAdd(tabCount+(1 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), weight * s2);
            atomicAdd(tabCount+(2 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), weight * s3);
            atomicAdd(tabCount+(3 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), weight * s4);
            atomicAdd(tabCount+(4 * NBTHETAd*NBPHId*NLAMd + il*NBTHETAd*NBPHId + ith*NBPHId + iphi), 1.);
        #endif
	}
	else
	{
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurcase), 1);
		#endif
	}

}



//
// Rotation of the stokes parameters by an angle psi between the incidence and
// the emergence planes
// input: 3 stokes parameters s1, s2, s3, (s4 does not need to be rotated)
//        rotation angle psi in radians
// output: 3 rotated stokes parameters s1r, s2r, s3r,
//
__device__ void rotateStokes(float s1, float s2, float s3, float psi,
        float *s1r, float *s2r, float *s3r)
{
    float cPsi = __cosf(psi);
    float sPsi = __sinf(psi);
    float cPsi2 = cPsi * cPsi;
    float sPsi2 = sPsi * sPsi;
    float twopsi = 2.F*psi;
    float a, s2Psi;
    s2Psi = __sinf(twopsi);
    a = 0.5f*s2Psi*s3;
    *s1r = cPsi2 * s1 + sPsi2 * s2 - a;
    *s2r = sPsi2 * s1 + cPsi2 * s2 + a;
    *s3r = s2Psi * (s1 - s2) + __cosf(twopsi) * s3;
}



/* calculPsi
* Calcul du psi pour la direction de sortie du photon
*/
__device__ void calculPsi(Photon* photon, float* psi, float theta)
{
	float sign;
// 	if (theta >= 0.05F)
// 	{
		*psi = acosf(fmin(1.F, fmax(-1.F, __fdividef(0.f * photon->ux + 1.f * photon->uz, __sinf(theta)))));
// 	}
// 	else
// 	{
// 		*psi = acosf(fmin(1.F - VALMIN, fmax(-(1.F - VALMIN), - 1.f * photon->ux + 0.f * photon->uz)));
// 	}
	
	sign = 0.f * (photon->uy * photon->vz - photon->uz * photon->vy) + 1.f * (photon->ux * photon->vy - photon->uy * photon->vx);
	if (sign < 0.F) *psi = -(*psi);
}


/* calculCase
* Fonction qui calcule la position (ith, iphi) et l'indice spectral (il) du photon dans le tableau de sortie
* La position correspond à une boite contenu dans l'espace de sortie
*/
__device__ void calculCase(int* ith, int* iphi, int* il, Photon* photon
			#ifdef PROGRESSION
			, Variables* var
			#endif 
			)
{
	// vxy est la projection du vecteur vitesse du photon sur (x,y)
	float vxy = sqrtf(photon->vx * photon->vx + photon->vy * photon->vy);

	// Calcul de la valeur de ithv
	// _rn correspond à round to the nearest integer
	*ith = __float2int_rd(__fdividef(acosf(fabsf(photon->vz)) * NBTHETAd, DEMIPI));
	//*ith = __float2int_rn(__fdividef(acosf(fabsf(photon->vz)) * NBTHETAd, DEMIPI));

	// Calcul de la valeur de il
    // DEV!!
    *il = photon->ilam;

	/* Si le photon ressort très près du zénith on ne peut plus calculer iphi,
	 on est à l'intersection de toutes les cases du haut */
	
	if(vxy >= VALMIN)
	{	//on calcule iphi
	
		// On place d'abord le photon dans un demi-cercle
		float cPhiP = __fdividef(photon->vx, vxy); //cosPhiPhoton
		// Cas limite où phi est très proche de 0, la formule générale ne marche pas
		//if(cPhiP >= 1.F) *iphi = 0;
		// Cas limite où phi est très proche de PI, la formule générale ne marche pas
		//else if(cPhiP <= -1.F) *iphi = (NBPHId) - 1;
		// Size of the angular boxes
        float dphi = __fdividef(2.F*PI,NBPHId);

        // Boxes centred on 0., dphi, 2dphi, ..., 180-dphi, 180., 180.+dphi,...., 360-dphi .
        // Boxes indices 0, 1, 2, ..., NBPHI/2-1, NBPHI/2, NBPHI/2 +1,..., NBPHI-2, NBPHI -1
        // So 2 boxes on 0 and 180 + NBPHI/2-1 boxes with vy>0 and NBPHI/2 -1 boxes with vy<0
        // Total NBPHI boxes from 0 to NBPHI -1; NBPHI has to be even
        // if the azimuth is within the zeroth boxe centered on 0. of width dphi/2 (half width dphi/4)
        if(cPhiP >= cosf(dphi/2.)) *iphi = 0;
        // if the azimuth is in the middle box centered on 180.
        else if(cPhiP <= -cosf(dphi/2.)) *iphi = NBPHId/2;
		else {
            /* otherwise it lies in a dphi box whose index (starting from 1) is given by the ratio of
             Phi -dphi/4. to the possible phi range that is PI-dphi/2. multiplied by the number of boxes NBPHId/2-1*/
            *iphi = __float2int_rd(__fdividef((acosf(cPhiP)-dphi/2.) * (NBPHId/2-1.0F), PI-dphi)) + 1;
		
		    // Puis on place le photon dans l'autre demi-cercle selon vy, utile uniquement lorsque l'on travail sur tous l'espace
   		    if(photon->vy < 0.F) *iphi = NBPHId - *iphi;
            }
		#ifdef PROGRESSION
		// Lorsque vy=0 on décide par défaut que le photon reste du côté vy>0
		if(photon->vy == 0.F) atomicAdd(&(var->erreurvy), 1);
		#endif
	}
	
	else{
		// Photon très près du zenith
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurvxy), 1);
		#endif
// 		/*if(photon->vy < 0.F) *iphi = NBPHId - 1;
// 		else*/ *iphi = 0;
		if(photon->vy >= 0.F)  *iphi = 0;
		else *iphi = NBPHId - 1;
	}
	
}

#ifdef DEBUG_PHOTON
__device__ void display(const char* desc, Photon* ph) {
    //
    // display the status of the photon (only for thread 0)
    //
    int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);

    if (idx==0) {
        printf("%16s X=(%.3g,%.3g,%.3g) \tV=(%.3g,%.3g,%.3g) \tU=(%.3g,%.3g,%.3g) \tS=(%.3g,%.3g,%.3g,%.3g) \ttau=%.3g \tweight=%.3g loc=",
               desc,
               ph->x, ph->y, ph->z,
               ph->vx,ph->vy,ph->vz,
               ph->ux,ph->uy,ph->uz,
               ph->stokes1, ph->stokes2,
               ph->stokes3, ph->stokes4,
               ph->tau, ph->weight
               );
        switch(ph->loc) {
            case 0: printf("SPACE"); break;
            case 1: printf("ATMOS"); break;
            case 2: printf("SURF0P"); break;
            case 3: printf("SURF0M"); break;
            case 4: printf("ABSORBED"); break;
            case 5: printf("NONE"); break;
            case 6: printf("OCEAN"); break;
            case 7: printf("SEAFLOOR"); break;
            default:
                    printf("UNDEFINED");
        }
        printf("\n");
    }
}
#endif

__device__ void modifyUV( float vx0, float vy0, float vz0, float ux0, float uy0, float uz0,
        float cTh, float psi, 
        float *vx1, float *vy1, float *vz1, float *ux1, float *uy1, float *uz1) { 

    float sTh, cPsi, sPsi, wx, wy, wz, vx, vy, vz, ux, uy, uz, norm;
    sPsi = __sinf(psi);
    cPsi = __cosf(psi);
    sTh = sqrtf(1.F - cTh*cTh);
	// w est le rotationnel entre l'ancien vecteur u et l'ancien vecteur v du photon
	wx = uy0 * vz0 - uz0 * vy0;
	wy = uz0 * vx0 - ux0 * vz0;
	wz = ux0 * vy0 - uy0 * vx0;
	// v est le nouveau vecteur v du photon
	vx = cTh * vx0 + sTh * ( cPsi * ux0 + sPsi * wx );
	vy = cTh * vy0 + sTh * ( cPsi * uy0 + sPsi * wy );
	vz = cTh * vz0 + sTh * ( cPsi * uz0 + sPsi * wz );
	// Changement du vecteur u (orthogonal au vecteur vitesse du photon)
    if (cTh <= -1.F) {
        ux  = -ux0;
        uy  = -uy0;
        uz  = -uz0;
        }
    else if (cTh >= 1.F){
        ux  = ux0;
        uy  = uy0;
        uz  = uz0;
    }
    else {
        ux = cTh * vx - vx0;
        uy = cTh * vy - vy0;
        uz = cTh * vz - vz0;
    }

    norm=sqrtf(vx*vx+vy*vy+vz*vz);
    *vx1 = vx/norm;
    *vy1 = vy/norm;
    *vz1 = vz/norm;
    norm=sqrtf(ux*ux+uy*uy+uz*uz);
    *ux1 = ux/norm;
    *uy1 = uy/norm;
    *uz1 = uz/norm;
}

__device__ void calculPsiLE( float ux0, float uy0, float uz0,
			     float vx0, float vy0, float vz0, 
			     float vx1, float vy1, float vz1, 
			     float* psi,
			     float* ux1, float* uy1, float* uz1)
{
	float prod_scal;
	
	float den;
	float y1;
	float cpsi;
	float spsi;

	float EPS6 = 1e-6;
	
	float wx0, wy0, wz0;
	float wx1, wy1, wz1;

	// compute former w
	// w est le rotationnel entre l'ancien vecteur u et l'ancien vecteur v du photon
	wx0 = uy0 * vz0 - uz0 * vy0;
	wy0 = uz0 * vx0 - ux0 * vz0;
	wz0 = ux0 * vy0 - uy0 * vx0;

	// compute the normal to the new scattering plan i.e. new w vector
	wx1 = vy1 * vz0 - vz1 * vy0;
	wy1 = vz1 * vx0 - vx1 * vz0;
	wz1 = vx1 * vy0 - vy1 * vx0;

	den = sqrtf( wx1* wx1 +  wy1* wy1 +  wz1* wz1);

	if (den < EPS6) {
		prod_scal =  vx0*vx1 + vy0*vy1 + vz0*vz1;
		if (prod_scal < 0.0)
			{   
				// diffusion vers l'avant
				wx1 = wx0;
				wy1 = wy0;
				wz1 = wz0;
			}
		else
			{ 
				// diffusion vers l'arriere
				wx1 = -wx0;
				wy1 = -wy0;
				wz1 = -wz0;
			}
	}
	else
		{
			wx1 = __fdividef(wx1,den);
			wy1 = __fdividef(wy1,den);
			wz1 = __fdividef(wz1,den);
		}
	
	//  Compute the scalar product between w0 and w1
	cpsi = wx0 * wx1 + wy0 * wy1 + wz0 * wz1;

	if (cpsi >  1.0) 
		cpsi =  1.0;
	if (cpsi < -1.0) 
		cpsi = -1.0;
	spsi = sqrtf(1.0 - cpsi * cpsi);
	if (spsi >  1.0) 
		spsi =  1.0;

	// Change of reference frame, look for the expression of  {vx1, vy1, vz1}
	// in the base linked to the photon before the scattering event = old
	// scattering plan. 
	// Let say that x1, y1, z1 are the new coordinate of cos_dir_sensor
	y1 = wx0*vx1 + wy0*vy1 + wz0*vz1;
	// --- Sign of spsi
	if (y1 < 0.0) 
		spsi = -spsi;

	*psi = acosf(cpsi);
	if (spsi<0)
		*psi = 2*PI - *psi;

	// get the new u vector
	*ux1 = vy1 * wz1 - vz1 * wy1 ;
	*uy1 = vz1 * wx1 - vx1 * wz1 ; 
	*uz1 = vx1 * wy1 - vy1 * wx1 ;
	
}

__device__ float calculTheta(float vx0, float vy0, float vz0, float vx1, float vy1, float vz1){

	// compute the diffusion angle theta between
	// to direction cosine {vx0, vy0, vz0} and {vx1, vy1, vz1} 

	float cs;
	float theta;
	
	//--- Find cos(theta) and sin(theta)
	cs =  vx1*vx0 + vy1*vy0 + vz1*vz0  ;//  produit scalaire
	
	// test cs to avois acos(cs)=NaN
	if(cs>+1) cs = 1.00;
	if(cs<-1) cs = -1.00;
		
	//--- compute theta
	
	theta = acosf(cs);

	return(theta);		
}

__device__ void copyPhoton(Photon* ph, Photon* ph_le) {
    //
    ph_le->vx = ph->vx;
    ph_le->vy = ph->vy;
    ph_le->vz = ph->vz;
    ph_le->ux = ph->ux;
    ph_le->uy = ph->uy;
    ph_le->uz = ph->uz;
    ph_le->stokes1 = ph->stokes1;
    ph_le->stokes2 = ph->stokes2;
    ph_le->stokes3 = ph->stokes3;
    ph_le->stokes4 = ph->stokes4;
    ph_le->loc = ph->loc;
    ph_le->tau = ph->tau;
    ph_le->couche = ph->couche;
    ph_le->weight = ph->weight;
    ph_le->wavel = ph->wavel;
    ph_le->ilam = ph->ilam;
    ph_le->prop_aer = ph->prop_aer;
    ph_le->x = ph->x;
    ph_le->y = ph->y;
    ph_le->z = ph->z;
    #ifdef SPHERIQUE
    ph_le->rayon = ph->rayon;
    ph_le->taumax = ph->taumax;
    #endif
}

/**********************************************************
*	> Fonctions liées au générateur aléatoire
***********************************************************/

#ifdef RANDCUDA
/* initRandCUDA
* Fonction qui initialise les generateurs du random cuda
*/
__global__ void initRandCUDA(hiprandState_t* etat, unsigned long long seed)
{
	// Pour chaque thread on initialise son generateur avec le meme seed mais un idx different
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
	hiprand_init(seed, idx, 0, etat+idx);
}
#endif
#if defined(RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
/* initRandCUDANDQRNGs
* Fonction qui initialise le generateur (scrambled) sobol 32 de hiprand
*/
__global__ void initRandCUDANDQRNGs
(
    curandSTATE* etat,
    hiprandDirectionVectors32_t *rngDirections
)
{
    // Pour chaque thread on initialise son generateur avec le meme seed mais un idx different
    unsigned int gID = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + blockIdx.y * gridDim.x));
    hiprand_init(
        //seule 20000 dimensions sont disponibles... le % permet de ne pas planter ici en segfault, mais...
        //...attention a la pertinence des resultats ici, si on depasse les 20000 threads !
        rngDirections[gID % 20000],
        #ifdef RANDCURANDSCRAMBLEDSOBOL32
        3, //aucune indication sur la pertinence de cette valeur...
        #endif
        /*0*/gID,
        etat+gID
               );
}
#endif


#ifdef RANDMT
/* initRandMTEtat
* Fonction qui initialise l'etat des generateurs du random Mersenne Twister (generateur = etat + config)
*/
__global__ void initRandMTEtat(EtatMT* etat, ConfigMT* config)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
	// Initialisation de l'etat du MT de chaque thread avec un seed different et aleatoire
	etat[idx].mt[0] = config[idx].seed;
	for (int i = 1; i < MT_NN; i++)
		etat[idx].mt[i] = (1812433253U * (etat[idx].mt[i - 1] ^ (etat[idx].mt[i - 1] >> 30)) + i) & MT_WMASK;
	etat[idx].iState = 0;
	etat[idx].mti1 = etat[idx].mt[0];
}


/* randomMTfloat
* Fonction random Mersenne Twister qui renvoit un float de ]0.1] à partir d'un generateur (etat+config)
*/
__device__ float randomMTfloat(EtatMT* etat, ConfigMT* config)
{
	//Convert to (0, 1] float
	return __fdividef(__uint2float_rz(randomMTuint(etat, config)) + 1.0f, 4294967296.0f);
}


/* randomMTuint
* Fonction random Mersenne Twister qui renvoit un uint à partir d'un generateur (etat+config)
*/
__device__ unsigned int randomMTuint(EtatMT* etat, ConfigMT* config)
{
	unsigned int mti;
	unsigned int mtiM;
	unsigned int x;
	int iState1;
	int iStateM;
	iState1 = etat->iState + 1;
	iStateM = etat->iState + MT_MM;
	if(iState1 >= MT_NN) iState1 -= MT_NN;
	if(iStateM >= MT_NN) iStateM -= MT_NN;
	mti  = etat->mti1;
	etat->mti1 = etat->mt[iState1];
	mtiM = etat->mt[iStateM];
	
	// MT recurrence
	x = (mti & MT_UMASK) | (etat->mti1 & MT_LMASK);
	x = mtiM ^ (x >> 1) ^ ((x & 1) ? config->matrix_a : 0);
	
	etat->mt[etat->iState] = x;
	etat->iState = iState1;
	
	//Tempering transformation
	x ^= (x >> MT_SHIFT0);
	x ^= (x << MT_SHIFTB) & config->mask_b;
	x ^= (x << MT_SHIFTC) & config->mask_c;
	x ^= (x >> MT_SHIFT1);
	return x;
}
#endif


#ifdef RANDMWC
/* randomMWCfloat
* Fonction random MWC qui renvoit un float de ]0.1] à partir d'un generateur (x+a)
*/
__device__ float randomMWCfloat(unsigned long long* x,unsigned int* a)
{
	//Generate a random number (0,1]
	*x=(*x&0xffffffffull)*(*a)+(*x>>32);
	return __fdividef(__uint2float_rz((unsigned int)(*x)) + 1.0f,(float)0x100000000);
}

#endif

#ifdef RANDPHILOX4x32_7
/* initPhilox4x32_7Compteur
* Fonction qui initialise la partie variable du compteur des philox
*/
__global__ void initPhilox4x32_7Compteur(unsigned int* tab, unsigned int compteurInit)
{
    unsigned int gID = threadIdx.x + blockDim.x * (threadIdx.y + blockDim.y * (blockIdx.x + blockIdx.y * gridDim.x));

    tab[gID] = compteurInit;
}

/* randomPhilox4x32_7float
* Fonction random Philox-4x32-7 qui renvoit un float dans ]0;1]
*/
__device__ float randomPhilox4x32_7float(philox4x32_ctr_t* ctr, philox4x32_key_t* key)
{
    //Recuperation d'un unsigned int pour retourner un float dans ]0;1]
    return __fdividef(__uint2float_rz(randomPhilox4x32_7uint(ctr, key)) + 1.0f, 4294967296.0f);
}

/* randomPhilox4x32_7uint
* Fonction random Philox-4x32-7 qui renvoit un uint à partir d'un generateur (etat+config)
* TODO A noter que 4 valeurs sont en fait generees, un seul uint peut etre renvoye, donc 3 sont perdus
* En pratique les valeurs generees sont des int32. Il y a donc une conversion vers uint32 de realisee
*/
__device__ unsigned int randomPhilox4x32_7uint(philox4x32_ctr_t* ctr, philox4x32_key_t* key)
{
    //variable de retour
    philox4x32_ctr_t res;
    //generation de 4 int32
    res = philox4x32_R(7, *ctr, *key);
    //increment du premier mot de 32bits du compteurs
    (*ctr).v[0]++;
    //conversion d'un des mots generes sous forme d'unsigned int
    return (unsigned int) res[0];
}
#endif

extern "C" {
    __global__ void lancementKernelPy(Variables* var, Tableaux *tab
	, Init* init
    ) {
        launchKernel(var, *tab,init);
    }
}

__device__ double DatomicAdd(double* address, double val)
{
        unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
        unsigned long long int old = *address_as_ull, assumed;
        do {
           assumed = old;
           old = atomicCAS(address_as_ull, assumed,
                __double_as_longlong(val +
                __longlong_as_double(assumed)));

                // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
        } while (assumed != old);

        return __longlong_as_double(old);
}
