#include "hip/hip_runtime.h"

/**********************************************************
*
*			device.cu
*
*	> Kernel
*	> Modélisation phénomènes physiques
*	> Initialisation de données dans le device
*	> Fonctions liées au générateur aléatoire
*
***********************************************************/


/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "device.h"


/**********************************************************
*	> Kernel
***********************************************************/

/* lancementKernel
* Kernel de lancement et gestion de la simulation
* Les fonctions de plus bas niveau sont appelées en fonction de la localisation du photon
* Il peut être important de rappeler que le kernel lance tous les threads mais effectue des calculs similaires. La boucle de la
* fonction va donc être effectuée pour chaque thread du block de la grille
* A TESTER: Regarder pour effectuer une réduction de l'atomicAdd
*/
__global__ void lancementKernel(Variables* var, Tableaux tab
		#ifdef SPHERIQUE
		, Init* init
		#endif
		#ifdef TABRAND
		, float* tableauRand
		#endif
		#ifdef TRAJET
		, Evnt* evnt
		#endif
			       )
{
	// idx est l'indice du thread considéré
	int idx = (blockIdx.x * YGRIDd + blockIdx.y) * XBLOCKd * YBLOCKd + (threadIdx.x * YBLOCKd + threadIdx.y);

	// Paramètres de la fonction random en mémoire locale
	#ifdef RANDMWC
	unsigned long long etatThr;
	unsigned int configThr;
	configThr = tab.config[idx];
	etatThr = tab.etat[idx];
	#endif
	#ifdef RANDCUDA
	hiprandState_t etatThr;
	etatThr = tab.etat[idx];
	#endif
	#ifdef RANDMT
	ConfigMT configThr;
	EtatMT etatThr;
	configThr = tab.config[idx];
	etatThr = tab.etat[idx];
	#endif

	#ifdef TABRAND
	// DEBUG Recuperation des nombres aleatoires generes par la fonction random utilisée
	if(idx < 5)
		if(tableauRand[50] == 0.f)
		{
			int k = 0;
			if(tableauRand[0] != 0.f) k = 50;
			for(int j = 0; j < 10; j++) tableauRand[k+idx*10+j] = RANDDEBUG;
		}
	#endif
	
	// Création de variable propres à chaque thread
	unsigned long long nbPhotonsThr = 0; 	// Nombre de photons traités par le thread
	
	#ifndef SPHERIQUE
	// Il n'y a pas de diffusion forcée pour une géométrie sphérique
	int flagDiff = DIFFFd;
	#endif
	
	#ifdef PROGRESSION
	unsigned int nbPhotonsSorThr = 0; 		// Nombre de photons traités par le thread et ressortis dans l'espace
	#endif
	
	Photon ph; 		// On associe une structure de photon au thread
	ph.loc = NONE;	// Initialement le photon n'est nulle part, il doit être initialisé
// 	float z;
	
	/** Mesure du temps d'execution **/
	#ifdef TEMPS
	clock_t start, stop;
	float time;
	#endif
	
	
	/** Boucle de calcul **/   
	// Dans cette boucle on simule le parcours du photon, puis on le réinitialise,... Le thread lance plusieurs photons
	for(unsigned int iloop= 0; iloop < NBLOOPd; iloop++)
	{
		// Si le photon est à NONE on l'initialise et on le met à la localisation correspondant à la simulaiton en cours
		if(ph.loc == NONE){
			
			#ifdef TEMPS
			if(idx==0){
				start = clock();
			}
			
			#endif
			
			initPhoton(&ph/*, &z*/
				#ifdef SPHERIQUE
				, tab, init
				#endif
				#ifdef TRAJET
				, idx, evnt
				#endif
					);

			#ifdef TEMPS
			if(idx==0){
				stop = clock();
				time = __fdividef((float) (stop-start),__int2float_rn(CLOCKS_PER_SEC));
				printf("(1) Temps de initPhoton: %f\n", time);
			}
			#endif
			
			#ifndef SPHERIQUE
			flagDiff = DIFFFd;
			#endif
			
		}
		// Chaque block attend tous ses threads avant de continuer
		syncthreads();
		
		
		// Si le photon est à ATMOS on le fait avancer jusqu'à SURFACE, ou SPACE, ou ATMOS s'il subit une diffusion
		if( (ph.loc == ATMOS) || (ph.loc == OCEAN) ){
			
			#ifdef TEMPS
			if(idx==0){
				start = clock();
			}
			#endif
			
			move(&ph/*, &z*/
				#ifndef SPHERIQUE
				,flagDiff, tab.h, tab.pMol
				#endif
				#ifdef SPHERIQUE
				, tab, init
				#endif
				, &etatThr
				#if defined(RANDMWC) || defined(RANDMT)
				, &configThr
				#endif
				#ifdef TRAJET
				, idx, evnt
				#endif
						);
						
			#ifdef TEMPS
			if(idx==0){
				stop = clock();
				time = __fdividef((float) (stop-start),__int2float_rn(CLOCKS_PER_SEC));
				printf("(2) Temps de move: %f\n", time);
			}
			#endif
		}
		// Chaque block attend tous ses threads avant de continuer
		syncthreads();
		
		// Si le photon est encore à ATMOS il subit une diffusion et reste dans ATMOS
		if( (ph.loc == ATMOS) || (ph.loc == OCEAN) ){
	
			#ifdef TEMPS
			if(idx==0){
				start = clock();
			}
			#endif
			
			// Diffusion
			scatter( &ph, tab.faer, tab.foce, &etatThr
			#if defined(RANDMWC) || defined(RANDMT)
			, &configThr
			#endif
			#ifdef TRAJET
			, idx, evnt
			#endif
				);
				
			#ifdef TEMPS
			if(idx==0){
				stop = clock();
				time = __fdividef((float) (stop-start),__int2float_rn(CLOCKS_PER_SEC));
				printf("(3) Temps de scatter: %f\n", time);
			}
			#endif

		}
		// Chaque block attend tous ses threads avant de continuer
		syncthreads();
		
		// Si le photon est à SURFACE
		if(ph.loc == SURFACE){
			
// 			if( DIOPTREd!=3 )
				surfaceAgitee(&ph, &etatThr
					#if defined(RANDMWC) || defined(RANDMT)
					, &configThr
					#endif
					#ifdef TRAJET
					, idx, evnt
					#endif
						);
						
// 			else
// 				surfaceLambertienne(&ph, &etatThr
// 					#if defined(RANDMWC) || defined(RANDMT)
// 					, &configThr
// 					#endif
// 					#ifdef TRAJET
// 					, idx, evnt
// 					#endif
// 						);
		}
		// Chaque block attend tous ses threads avant de continuer
		syncthreads();
		
		// Si le photon est dans SPACE ou ABSORBED on récupère ses infos et on le remet à NONE
		
		if(ph.loc == ABSORBED){
			ph.loc = NONE;
			nbPhotonsThr++;

		}
		syncthreads();
		
		if(ph.loc == SPACE){
			exit(&ph, tab, &nbPhotonsThr
						#ifdef PROGRESSION
						, &nbPhotonsSorThr, var
						#endif
						#ifdef TRAJET
						, idx, evnt
						#endif
						);
		}
		syncthreads();
		

		#ifndef SPHERIQUE	/* Code spécifique à une atmosphère parallèle */
		//Mise à jour du poids suite à la 1ère diffusion forcée
		if(flagDiff==1 ){
			ph.weight *= (1.F - __expf(-TAUMAXd));
			flagDiff=0;
		}
		syncthreads();
		#endif

	}// Fin boucle for
	

	// Après la boucle on rassemble les nombres de photons traités par chaque thread
	atomicAdd( &(var->nbPhotons), nbPhotonsThr );
	
	#ifdef PROGRESSION
	// On rassemble les nombres de photons traités et sortis de chaque thread
	atomicAdd(&(var->nbPhotonsSor), nbPhotonsSorThr);

	// On incrémente avncement qui compte le nombre d'appels du Kernel
	atomicAdd(&(var->nbThreads), 1);
	#endif
	
	// Sauvegarde de l'état du random pour que les nombres ne soient pas identiques à chaque appel du kernel
	tab.etat[idx] = etatThr;
}


/**********************************************************
*	> Modélisation phénomènes physiques
***********************************************************/

/* initPhoton
* Initialise le photon dans son état initial avant l'entrée dans l'atmosphère
*/
__device__ void initPhoton(Photon* ph/*, float* z*/
		#ifdef SPHERIQUE
		, Tableaux tab, Init* init
		#endif
		#ifdef TRAJET
		, int idx, Evnt* evnt
		#endif
		    )
{
	// Initialisation du vecteur vitesse
	ph->vx = - STHSd;
	ph->vy = 0.F;
	ph->vz = - CTHSd;
	
	// Initialisation du vecteur orthogonal au vecteur vitesse
	ph->ux = -ph->vz;
	ph->uy = 0.F;
	ph->uz = ph->vx;
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	ph->locPrec=NONE;
	
	// 	Paramètres initiaux calculés dans impactInit - host.cu
	ph->x = init->x0;
	ph->y = init->y0;
	ph->z = init->z0;
	ph->couche=0;	// Sommet de l'atmosphère
	ph->rayon = sqrtf(ph->x*ph->x + ph->y*ph->y + ph->z*ph->z );
	#endif
	
	#ifndef SPHERIQUE	/* Code spécifique à une atmosphère en plan parallèle */
	ph->z = TAUATMd;
	#endif
	
	// Le photon est initialement dans l'atmosphère, et tau peut être vu comme sa hauteur par rapport au sol
	if( SIMd==3 ){
		ph->loc=OCEAN;
	}
	else if( SIMd==-1 || SIMd==0 ){
		ph->loc = SURFACE;
		#ifndef SPHERIQUE
		ph->z = 0.f;
		#endif
	}
	else
		ph->loc = ATMOS;
	
	ph->weight = WEIGHTINIT;
	
	// Initialisation des paramètres de stokes du photon
	ph->stokes1 = 0.5F;
	ph->stokes2 = 0.5F;
	ph->stokes3 = 0.F;

	
	
	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
		int i = 0;
		// On cherche la première action vide du tableau
		while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
		// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
		// "1"représente l'événement "initialisation" du photon
		evnt[i].action = 1;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
}


/* move
* Effectue le déplacement du photon dans l'atmosphère
* Pour l'atmosphère sphèrique, l'algorithme est basé sur la formule de pythagore généralisée.
* Modification des coordonnées position du photon
*/
__device__ void move(Photon* ph/*, float* z*/
		#ifndef SPHERIQUE
		,int flagDiff, float* h, float* pMol
		#endif
		#ifdef SPHERIQUE
		, Tableaux tab, Init* init
		#endif
		#ifdef RANDMWC
		, unsigned long long* etatThr, unsigned int* configThr
		#endif
		#ifdef RANDCUDA
		, hiprandState_t* etatThr
		#endif
		#ifdef RANDMT
		, EtatMT* etatThr, ConfigMT* configThr
		#endif
		#ifdef TRAJET
		, int idx, Evnt* evnt
		#endif
		    )
{
	float rra;
	int icouche;
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	float rsolfi = 0.f;
	float delta;
	
	float tauRdm;	// Epaisseur optique aléatoire tirée
	
	// Permet la sauvegarde du profil parcouru par le photon
	float zph = 0.f;
	float zph_p = 0.f;
	float hph = 0.f;
	float hph_p = 0.f;
	
	float vzn;				// projeté de vz sur l'axe defini par le photon et le centre de la terre
	float sinth;			// Sinus de l'angle entre z' et Vz'
	float costh;			// Cosinus de l'angle entre z' et Vz'
	float ztangentielle;	//Altitude tangentielle du photon (distance minimale entre sa trajectoire et le centre 
							// de la terre
	int coucheTangentielle = 666;	// Initialisation arbitraire

	float rayon2;			// Rayon au carré
	float rayon;
	int icouchefi = 0;
	int icompteur = 0;
	int sens;				// Sens=1 si le photon monte, sens=-1 si il descend.
	int icoucheTemp; 		// Couche suivante que le photon va toucher
	int flagSortie = 0;		// Indique si le photon va sortir sans interaction dans l'atmosphère
	
	float rdist;
	
	#ifdef DEBUG
	double rsol1,rsol2;
	#endif
	
	/** Mesure du temps d'execution **/
	#ifdef TEMPS
	clock_t start, stop;
	float time;
	#endif
	
	/** Tirage au sort de la profondeur optique à parcourir **/
	/*  Tirage effectué lors de chaque appel de la fonction */
	// Pas de diffusion forcée en sphérique
	
	tauRdm = -logf(1.F-RAND);
	
	if( tauRdm == 0. ){
		/* Le photon ne bouge pas mais il faut tout de même considéré le fait qu'il a subit un déplacement "nul"
		 * Il va quand même intéragir.
		*/
		ph->locPrec = ATMOS;
		return;
	}


	/** Calcul puis déduction de la couche du photon **/
	#ifdef TEMPS
	if(idx==0){
		start = clock();
	}
	#endif
	
	if( ph->locPrec==NONE ){
		/* Le photon vient de l'espace et rentre pour la première fois dans l'atmosphère
		*/
		
		// Le photon descend forcement car il vient du sommet de l'atmosphère
		sens = -1;
		
		// Si tauRdm est plus élevé que Taumax, le photon va directement heurter la surface
		if( tauRdm >= (init->taumax0) ){
			flagSortie = 1;
			zph=tab.zph0[NATMd]; /* Pour retrouver le zintermax ensuite. Cette valeur signifie que le photon a traversé toute
									l'atmosphère */
		}


		while( (hph < tauRdm) && (flagSortie!=1) ){
		/* Le photon vient du sommet de l'atmosphère - On parcourt le profil initial calculé dans impact.
		*/

			#ifdef DEBUG
			if( icompteur==(NATMd+1) ){
				printf("icompteur = NATMd+1 pour premier calcul de position - tauRdm=%f - taumax=%f - hph_p=%f - hph=%f\n",\
					tauRdm, ph->taumax, hph_p, hph);
					flagSortie = 1;
					break;
			}
			#endif
			
			// Sauvegarde du calcul de la couche précédente
			hph_p = hph;
			zph_p = zph;
			
			hph = tab.hph0[icompteur];
			zph = tab.zph0[icompteur];
			
			icompteur++;
		}
		
		#ifdef TEMPS
		if(idx==0){
			stop = clock();
			time = __fdividef((float) (stop-start),__int2float_rn(CLOCKS_PER_SEC));
			printf("(2.1) Temps de move pour une 1ère intéraction: %f\n", time);
		}
		#endif
	}


	else if( ((ph->locPrec==ATMOS)||(ph->locPrec==SURFACE)) ){
		/* Le photon vient de l'atmosphère ou de la surface
		* Son profil est calculé jusqu'à arriver à la position voulue, c'est à dire que le photon parcourt l'épaisseur optique TauRdm
		* zph est la distance parcourue par le photon, hph est l'épaisseur optique parcourue.
		*/

		
		/** Changement de repère **/
		/* calcul du nouveau z', axe passant par le centre de la terre et le photon.
		* Cette axe permet de définir une projection de vz donnant la direction locale du photon
		*/
		
		rayon = ph->rayon;
		rayon2 = rayon*rayon;

		/* Calcul du Vz' par rapport à z'. Son signe donne la direction du photon
		* Vz'=vzn= V.Z/|Z|	
		*/	
		vzn = __fdividef( ph->vx*ph->x + ph->vy*ph->y + ph->vz*ph->z ,rayon);
		
		
		/** Test initial pour le photon venant de la surface **/
		/* Il faut abandonner le photon si ce test est positif
		* Il signifie que sur une surface agitée, le photon peut être réfléchi mais toujours se diriger vers la surface
		*/
		if((vzn<0.f)&&(ph->locPrec==SURFACE)){
			// Problème du à des imprécisions de calculs je pense ,également à la surface agitée
			ph->loc=ABSORBED;
			return;
		}

		/* Calcul costh= Vz'/|Vz| */
		costh = vzn;
		
		if( abs(costh)>1.f ){
			costh = rintf(costh);
		}
		
		sinth = sqrtf(1.f-costh*costh);	// Le signe n'importe pas car on prend le carré ou la valeur absolu pour ztangentielle


		/** Initialisation des paramètres du photon : couche et sens de propagation dans l'atmosphère **/
		
		/* Couche dans laquel se trouve le photon
		* La valeur de cette couche va évoluer au fur et à mesure des calculs, représentant un photon virtuel passant de couches
		* en couches
		*/
		icoucheTemp = ph->couche;
		
		// On choisit que par défaut le photon monte, ceci pour éviter un test
		sens = 1;
		
		if(vzn<0.f){
		// Le photon descend
			sens = -1;
			
			/* Calcul de la distance tangentielle, distance minimale au centre de la terre sur le parcours du photon
			* Cette couche sera la couche dans laquelle le photon va changer de direction de propagation (montant ou descandant 
			* dans l'atmosphère)
			*/
			ztangentielle = rayon*sinth;
			
			// Puis recherche de la couche correspondante
			if( ztangentielle>RTER ){
				coucheTangentielle = 0;
				while( (RTER+tab.z[coucheTangentielle])>ztangentielle){ 
					coucheTangentielle++;
					#ifdef DEBUG
					if( coucheTangentielle==(NATMd+1) ){
						printf("Arret de calcul couche ztangentielle (%lf)\n", ztangentielle);
						ph->loc = NONE;
						return;
					}
					#endif
				}
			}
			// Sinon le photon va forcement passer par la surface, on ne change pas coucheTangentielle
		}


		/** Recherche des couches à parcourir **/
		/* Le premier cas est un peu différent car le photon se trouve entre 2 couches.
		* On le positionne virtuellement sur une couche (cela ne change rien pour les calculs).
		* Le calcul est basé sur la formule de pythagore généralisée. Elle calcule la distance parcourue par le photon jusqu'à
		* une couche donnée.
		*/
		
		// Si le photon arrive dans la couche tangentielle, il change de sens de propagation
		if( (icoucheTemp==coucheTangentielle)&&(sens==-1) ){
			// Le photon va remonter dans les couches
			sens=1;
		}


		// icouchefi est la couche suivante que va toucher le photon "virtuel", elle dépend du sens
		// L'équivalent est icouchefi = icoucheTemp; if( sens== +1) icouchefi--;
		bool test_sens = (sens==1);
		icouchefi = icoucheTemp - test_sens;
		
		if( icouchefi<0 ){
			/* Cela signifie que le photon est à la limite de l'atmosphère mais qu'il va sortir car il remonte l'atmosphère
			* On le force donc à sortir en le mettant dans l'espace
			*/
			ph->loc=SPACE;
			return;
		}

		#ifdef DEBUG
		if( (icouchefi<0)||(icouchefi>NATMd) ){
			printf("OUPS#1: icouchefi=%d  sens=%d  icoucheTemp=%d  vzn=%lf  locPrec=%d  rayon=%25.20lf\n\t\
(%20.16lf , %20.16lf , %20.16lf )\n",\
			icouchefi,sens,icoucheTemp, vzn, ph->locPrec, rayon, ph->x, ph->y, ph->z);
			ph->loc=NONE;
			return;
		}
		#endif
		
		
		/** Premier calcul **/
		// Déterminant de l'équation déduite du pythagore généralisé
		delta = 4.f*( (tab.z[icouchefi]+RTER)*(tab.z[icouchefi]+RTER) - rayon2*sinth*sinth);
		
		
		if(delta<0){
			#ifdef DEBUG
				printf("OUPS rdelta #1=%lf - icoucheTemp=%d - tab.z[icoucheTemp]= %16.13lf - rayon= %17.13lf - rayon2=%20.16lf\n\t\
sinth= %20.19lf - sens=%d\n",\
				delta, icoucheTemp, tab.z[icoucheTemp], rayon, rayon2, sinth, sens);
			#endif
			ph->loc=NONE;
			return;
		}
		

		/* Calcul de la solution avec astuce
		* Sens indique si le photon monte ou descends, et la solution se déduit de ce signe. Un dessin permet de s'en convaincre
		* Si le photon monte, costh>0, il faut prendre la plus petite solution
		* Si il descend dans les couches, costh<0, il faut prendre la plus petite solution
		* Si il monte dans les couches avec costh<0, il faut prendre la plus grande solution
		*/
		rsolfi = 0.5f*( -2.f*rayon*costh + sens*sqrtf(delta) );
		
		if( abs(rsolfi) < 5e-3f ){
			rsolfi=0.f;
		}
		
		if( rsolfi<0.f ){
			#ifdef DEBUG
			printf("OUPS: rsolfi #1=%lf, (%lf,%lf) - vzn=%lf - sens=%d - locPrec=%d\n\t\
costh= %16.15lf - rayon= %16.12lf - delta= %16.10lf - icoucheTemp=%d - icouchefi=%d\n",\
rsolfi, 0.5*( -2*rayon*costh + sqrt(delta)),0.5*( -2*rayon*costh - sqrt(delta)) , vzn, sens, ph->locPrec, costh, rayon, delta,
icoucheTemp, icouchefi);
			#endif
			
			ph->loc=NONE;
			return;
		}


		// Calcul des paramètres du profil du photon au cours de son parcours
		if( icouchefi!=icoucheTemp ){
			hph = __fdividef( abs(tab.h[icoucheTemp] - tab.h[icouchefi])*rsolfi, abs(tab.z[icouchefi] - tab.z[icoucheTemp]) );
		}
		else{
			if( icouchefi==0 ){
				hph = __fdividef( abs(tab.h[1] - tab.h[0])*rsolfi, abs(tab.z[1] - tab.z[0]) );
			}
			else{
				hph = __fdividef( abs(tab.h[icouchefi-1] - tab.h[icouchefi])*rsolfi, abs(tab.z[icouchefi-1] - tab.z[icouchefi]) );
			}
		}

		zph=rsolfi;
		icoucheTemp = icouchefi;


		/** Calcul du profil total **/
		// Calcul jusqu'à sortir ou intéragir
		
		while( (hph < tauRdm) ){
			
			icompteur++;
			
			/* Vérification si le photon est sorti de l'atmosphère
			* La variable sens permettra ensuite de savoir si le photon sort vers la surface ou l'espace
			*/
			if( (icoucheTemp==0)||(icoucheTemp==NATMd) ) {
				flagSortie=1;
				break;
			}

			// Mise à jour de la couche que va toucher le photon (c'est icouchefi)
			icouchefi = icoucheTemp - sens;
			
			if( (icouchefi==coucheTangentielle)&&(sens==-1) ){
				// Le photon va remonter dans les couches
				sens=1;
				icouchefi = icoucheTemp;
			}


			// Solution de l'équation issue de pythagore généralisé
			delta = 4.f*( (tab.z[icouchefi]+RTER)*(tab.z[icouchefi]+RTER) - rayon2*sinth*sinth);
			
			
			if(delta<0){
				#ifdef DEBUG
				printf("OUPS delta #2=%lf - icouchefi=%d - tab.z[icouchefi]= %16.13lf - rayon= %17.13lf - rayon2= %20.16lf\n\t\
sinth= %20.19lf - sens=%d\n",\
				delta, icouchefi, tab.z[icouchefi], rayon, rayon2, sinth, sens);
				#endif
				ph->loc=NONE;
				return;
			}
			
			
			// Calcul de la solution avec astuce
			rsolfi= 0.5f*( -2.f*rayon*costh + sens*sqrtf(delta));

			
			// Calcul des grandeurs du profil
			hph_p = hph;
			zph_p = zph;
			
			#ifdef DEBUG
			if( icouchefi<0 ){
				printf("OUPS: icouchefi #1 = %d - rayon=%lf - icouchePhoton=%d - icoucheTemp=%d\n",\
				icouchefi, rayon, ph->couche, icoucheTemp);
				ph->loc=NONE;
				return;
			}
			if(icouchefi>NATMd){
				printf("OUPS: icouchefi #1 = %d\n",icouchefi);
				ph->loc=NONE;
				return;
			}
			#endif
			
			// Valeur de la couche actuelle
			if( icouchefi!=icoucheTemp ){
				hph += __fdividef( 	abs(tab.h[icoucheTemp] - tab.h[icouchefi])*(rsolfi-zph_p), 
									abs(tab.z[icouchefi] - tab.z[icoucheTemp]) );
			}
			else{
				if( icouchefi==0 ){
					hph += __fdividef( 	abs(tab.h[1] - tab.h[0])*(rsolfi-zph_p) , abs(tab.z[1]- tab.z[0]) );
				}
				else{
					hph += __fdividef( 	abs(tab.h[icouchefi-1] - tab.h[icouchefi])*(rsolfi-zph_p),
										abs(tab.z[icouchefi-1] - tab.z[icouchefi]) );
				}
			}

			zph=rsolfi;


			icoucheTemp = icouchefi;
			
			#ifdef DEBUG
			// Compteur de débordement
			if(icompteur==(2*NATMd+2)){
				printf("icouche = 2(NATMd+1) - (%lf,%lf,%lf) - icouchefi=%d - flagSortie=%d\n\t\
	ph->vz=%f - rsolfi=%f - tauRdm=%f - hph=%f\n",\
				ph->x,ph->y,ph->z, icouchefi,flagSortie, ph->vz,rsolfi,tauRdm,hph);
				ph->loc=NONE;
				return;
			}
			#endif
		
		}// Fin while
		
		#ifdef TEMPS
		if(idx==0){
			stop = clock();
			time = __fdividef((float) (stop-start),__int2float_rn(CLOCKS_PER_SEC));
			printf("(2.2) Temps de move pour une intéraction quelconque: %f\n", time);
		}
		#endif

	}// Fin de si photon provenant de l'atmosphere ou la surface

	
	/** Actualisation des coordonnées du photon **/
	/* Calcul des nouvelles coordonnées (x,y,z) du photon
	* Si le photon va intéragir dans l'atmosphère:
	* 		Interpolation linéaire entre les bornes de la couche car le trajet au sein de la couche est rectiligne
	* 		la 2ème ligne peut être remplacée pour la compréhension par rrb = zph - hph*rra; rdist = rra*tauRdm + rrb; mais les 
	* 		performances sont légérement réduites.
	* Sinon
	*		La distance parcourue correspond à la traversée de l'atmosphère, représentée par zph
	*/
	
	if( flagSortie==0 ){
		rra = __fdividef( zph_p - zph , hph_p - hph );
		rdist = rra*( tauRdm-hph ) + zph;
	}
	else{
		rdist = zph;
	}
	
	ph->x = ph->x + ph->vx*rdist;
	ph->y = ph->y + ph->vy*rdist;
	ph->z = ph->z + ph->vz*rdist;
	

	/** Sortie sans intéraction **/
	if( flagSortie==1 ){
		// Il n'y a pas eu d'intéraction avec l'atmosphère
		
		if(sens==-1){
			ph->loc = SURFACE;
			ph->couche = NATMd;
			ph->rayon = RTER;
		}
		else{
			ph->loc = SPACE;
		}

		return;
	}


	/** Sorti avec intéraction **/
	// Calcul du rayon
	rayon2 = ph->x*ph->x + ph->y*ph->y + ph->z*ph->z;
	rayon = sqrtf(rayon2);

	
	if(rayon < RTER){
		if( (rayon-RTER)<1.e-4f ){
			/* Ce test est parfois vrai lorsqu'il y a la surface. 
			 * Le rayon n'est pas égal à RTER, surement à cause d'erreur de calcul du GPU
			*/
			rayon=RTER;
			ph->loc=SURFACE;
			#ifdef DEBUG
			printf("MetaProblème #2: Correction du rayon\n");
			#endif
		}
		else{
			#ifdef DEBUG
			printf("MetaProblème #2: rayon=%20.16lf - (%lf,%lf,%lf) - icouchefi=%d - icompteur=%d -locPrec=%d\n\t\
rsolfi=%15.12lf - tauRdm= %lf - hph_p= %15.12lf - hph= %15.12lf - zph_p= %15.12lf - zph= %15.12lf\n",\
			rayon,ph->x, ph->y,ph->z, icouchefi, icompteur,ph->locPrec,\
			rsolfi,tauRdm, hph_p, hph, zph_p, zph);
			#endif
			ph->loc = NONE;
			return;
		}
	}

	// Boucle pour définir entre quels rayons est le photon
	icouche = 0;
	while((RTER+tab.z[icouche])>rayon){
		icouche++;
		#ifdef DEBUG
		if (icouche==NATMd+1){
			printf("Arret de calcul couche #2 (rayon=%f)\n", rayon);
			ph->loc=NONE;
			return;
		}
		#endif
	}


	ph->couche = icouche;
	ph->rayon = rayon;
	ph->locPrec=ATMOS;
	
	// On sort maintenant de la fonction et comme le photon reste dans ATMOS, le kernel appelle scatter()

	#endif	/* Fin de la partie atmosphère sphérique */


	#ifndef SPHERIQUE	/* Code spécifique à une atmosphère parallèle */
	float tauBis;
	
// 	if( ph->loc==ATMOS ){
		ph->z += -__logf( flagDiff + RAND*(1.F +(__expf(-TAUMAXd)-2.f)*flagDiff))*ph->vz;
// 	}
// 	else{
// 		ph->z += -logf(1.f - RAND)*ph->vz;
// 		if( ph->z < 0.f ){
// 			ph->loc = OCEAN;
// 		}
// 		else{
// 			ph->z = 0.F;
// 			ph->loc = SURFACE;
// 			if( SIMd==3 ){
// 				ph->loc=SPACE;
// 			}
// 		}
// 			
// 		#ifdef TRAJET
// 		// Récupération d'informations sur le premier photon traité
// 		if(idx == 0)
// 		{
// 			int i = 0;
// 			// On cherche la première action vide du tableau
// 			while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
// 			// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
// 			// "2"représente l'événement "move" du photon
// 			evnt[i].action = 2;
// 			// On récupère le tau et le poids du photon
// 			evnt[i].poids = ph->weight;
// 			evnt[i].tau = ph->z;
// 		}
// 		#endif
// 		
// 		return;
// 	}
	
	// Si tau<0 le photon atteint la surface
	if(ph->z < 0.F){
		ph->loc = SURFACE;
		ph->z = 0.F;
		return;
	}
	// Si tau>TAURAY le photon atteint l'espace
	else if(ph->z > TAUATMd){
		ph->loc = SPACE;
		return;
	}
	/////
// 	if(/*ph->*/*z < 0.F){
// 		ph->loc = SURFACE;
// 		/*ph->*/*z = 0.F;
// 		return;
// 	}
// 	// Si tau>TAURAY le photon atteint l'espace
// 	else if(/*ph->*/*z > TAUATMd){
// 		ph->loc = SPACE;
// 		return;
// 	}
	
	// Sinon il reste dans l'atmosphère, et va être traité par scatter
	
	// Calcul de la couche dans laquelle se trouve le photon
	tauBis = TAUATMd-ph->z;
	icouche = 1;
	
	while( (h[icouche] < (tauBis))&&(icouche<NATMd) ){
		icouche++;
	}
	
	ph->couche = icouche;
	#endif
	
	/** Interpolation linéaire pour connaitre la proportion d'aérosols à l'endroit où se situe le photon **/
	icouche = ph->couche;
	
	// Calcul sans interpolation
	#ifdef SPHERIQUE
	ph->prop_aer = 1.f - tab.pMol[icouche];
	#endif
	#ifndef SPHERIQUE
	ph->prop_aer = 1.f - pMol[icouche];
	#endif
	
	// Calcul avec interpolation linéaire
// 	if(icouche==0){
//    		printf("ph->couche=0 pour le calcul de proportion d'aérosols\n");
//    		ph->prop_aer = 1.f - pMol[icouche];
//    	}
// 	else{
// 		#ifdef SPHERIQUE
// 		rra = __fdividef( tab.pMol[icouche] - tab.pMol[icouche-1] , tab.h[icouche] - tab.h[icouche-1] );
// 		ph->prop_aer = 1.f - ( rra*(ph->rayon - RTER - tab.h[icouche]) + tab.pMol[icouche] );
// 		#endif
// 		#ifndef SPHERIQUE
// 		rra = __fdividef( pMol[icouche] - pMol[icouche-1] , h[icouche] - h[icouche-1] );
// 		ph->prop_aer = 1.f - ( rra*(tauBis - h[icouche]) + pMol[icouche] );
// 		#endif
// 	}
	

	
	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
		int i = 0;
		// On cherche la première action vide du tableau
		while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
		// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
		// "2"représente l'événement "move" du photon
		evnt[i].action = 2;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
}


/* scatter
* Diffusion du photon par une molécule ou un aérosol
* Modification des paramètres de stokes et des vecteurs U et V du photon (polarisation, vitesse)
*/
__device__ void scatter( Photon* ph, float* faer, float* foce
			#ifdef RANDMWC
			, unsigned long long* etatThr, unsigned int* configThr
			#endif
			#ifdef RANDCUDA
			, hiprandState_t* etatThr
			#endif
			#ifdef RANDMT
			, EtatMT* etatThr, ConfigMT* configThr
			#endif
			#ifdef TRAJET
			, int idx, Evnt* evnt
			#endif
			){

	float cTh=0.f, sTh, psi, cPsi, sPsi;
	float wx, wy, wz, vx, vy, vz;
	
	psi = RAND * DEUXPI;	//psiPhoton
	cPsi = __cosf(psi);		//cosPsiPhoton
	sPsi = __sinf(psi);		//sinPsiPhoton
	
	
	// Modification des nombres de Stokes
	modifStokes(ph, psi, cPsi, sPsi, 1);
	
	/* Les calculs qui différent pour les aérosols et les molécules sont regroupés dans cette partie.
	 * L'idée à termes est de réduire au maximum cette fonction, en calculant également la fonction de phase pour les
	 * molécules, à la manière des aérosols.
	*/

	float zang=0.f, theta=0.f;
	int iang;
	float stokes1, stokes2;
	float cTh2;
	float prop_aer = ph->prop_aer;
	
	stokes1 = ph->stokes1;
	stokes2 = ph->stokes2;
	
	
	///////// Possible de mettre dans une fonction séparée, mais attention aux performances /////////
	///////// Faire également attention à bien passer le pointeur de cTh et le modifier dans la fonction /////////
	if(ph->loc!=OCEAN){
		
		if( prop_aer<RAND ){
			// Theta calculé pour la diffusion moléculaire
			cTh =  2.F * RAND - 1.F; // cosThetaPhoton
			cTh2 = (cTh)*(cTh);
			
			// Calcul du poids après diffusion
			ph->weight *= __fdividef(1.5F * ((1.F+GAMAd)*stokes1+((1.F-GAMAd)*cTh2+2.F*GAMAd)*stokes2), (1.F+2.F*GAMAd) *
			(stokes1+stokes2));
			// Calcul des parametres de Stokes du photon apres diffusion
			ph->stokes1 += GAMAd * stokes2;
			ph->stokes2 = ( (1.F - GAMAd) * cTh2 + GAMAd) * stokes2 + GAMAd * ph->stokes1;
			ph->stokes3 *= (1.F - GAMAd) * (cTh);
		}
		else{
			// Aérosols
			zang = RAND*(NFAERd-1);
			iang= __float2int_rd(zang);
			
			zang = zang - iang;
			/* L'accès à faer[x][y] se fait par faer[y*5+x] */
			theta = faer[iang*5+4]+ zang*( faer[(iang+1)*5+4]-faer[iang*5+4] );
			
			cTh = __cosf(theta);
			
			/** Changement du poids et des nombres de stokes du photon **/
			float faer1 = faer[iang*5+0];
			float faer2 = faer[iang*5+1];
			
			// Calcul du poids après diffusion
			ph->weight *= __fdividef( 2.0F*(stokes1*faer1+stokes2*faer2) , stokes1+stokes2)*W0AERd;
			
			// Calcul des parametres de Stokes du photon apres diffusion
			ph->stokes1 *= faer1;
			ph->stokes2 *= faer2;
			ph->stokes3 *= faer[iang*5+2];
			// 		photon->stokes4 = 0.F;
		}
	}
	else{	/* Photon dans l'océan */
		float p1, p2, p3;
		float u;
		
		zang = RAND*(NFOCEd-2);
		iang = __float2int_rd(zang);
		zang = zang - iang;
		/* L'accès à foce[x][y] se fait par foce[y*5+x] */
		theta = foce[iang*5+4]+ zang*( foce[(iang+1)*5+4]-foce[iang*5+4] );
		
		cTh = __cosf(theta);
		
		// p1 et p2 sont inversés par cohérence étant donné que le code Fortran d'origine interverti stoke1 et stoke2
		p2 = foce[iang*5+0];
		p1 = foce[iang*5+1];
		p3 = foce[iang*5+2];
		
	
		ph->weight  *= 2.0f*__fdividef( (stokes1*p1+stokes2*p2) , stokes1+stokes2)*W0OCEd;
		ph->stokes1 *= 2.0f*p1;
		ph->stokes2 *= 2.0f*p2;
		u = ph->stokes3;
		ph->stokes3 = p3*u;
		
		
		/**  **/
		// if( ph->weight < WEIGHTMIN ){
		// ph->loc=ABSORBED;
		// return;
	// }
	
	/** Roulette russe **/
	if( ph->weight < WEIGHTRR ){
		if( RAND < __fdividef(ph->weight,WEIGHTRR) ){
			ph->weight = WEIGHTRR;
		}
		else{
				ph->loc = ABSORBED;
			}
		}
		
	}
   
   ////////// Fin séparation ////////////
	
	sTh = sqrtf(1.F - cTh*cTh);	// sinThetaPhoton
	
	/** Création de 2 vecteurs provisoires w et v **/
	float vx_s, vy_s, vz_s, ux_s, uy_s, uz_s;	// Parametres du photon sauves pour optimisation
	vx_s = ph->vx;
	vy_s = ph->vy;
	vz_s = ph->vz;
	ux_s = ph->ux;
	uy_s = ph->uy;
	uz_s = ph->uz;
	// w est le rotationnel entre l'ancien vecteur u et l'ancien vecteur v du photon
	wx = uy_s * vz_s - uz_s * vy_s;
	wy = uz_s * vx_s -ux_s * vz_s;
	wz = ux_s * vy_s - uy_s * vx_s;
	// v est le nouveau vecteur v du photon
	vx = cTh * vx_s + sTh * ( cPsi * ux_s + sPsi * wx );
	vy = cTh * vy_s + sTh * ( cPsi * uy_s + sPsi * wy );
	vz = cTh * vz_s + sTh * ( cPsi * uz_s + sPsi * wz );
	// Changement du vecteur u (orthogonal au vecteur vitesse du photon)
	ph->ux = __fdividef(cTh * vx - vx_s, sTh);
	ph->uy = __fdividef(cTh * vy - vy_s, sTh);
	ph->uz = __fdividef(cTh * vz - vz_s, sTh);
	
	/////
	// w est le rotationnel entre l'ancien vecteur u et l'ancien vecteur v du photon
// 	wx = ph->uy * ph->vz - ph->uz * ph->vy;
// 	wy = ph->uz * ph->vx - ph->ux * ph->vz;
// 	wz = ph->ux * ph->vy - ph->uy * ph->vx;
// 	
// 	// v est le nouveau vecteur v du photon
// 	vx = cTh * ph->vx + sTh * ( cPsi * ph->ux + sPsi * wx );
// 	vy = cTh * ph->vy + sTh * ( cPsi * ph->uy + sPsi * wy );
// 	vz = cTh * ph->vz + sTh * ( cPsi * ph->uz + sPsi * wz );
// 
// 	// Changement du vecteur u (orthogonal au vecteur vitesse du photon)
// 	ph->ux = __fdividef(cTh * vx - ph->vx, sTh);
// 	ph->uy = __fdividef(cTh * vy - ph->vy, sTh);
// 	ph->uz = __fdividef(cTh * vz - ph->vz, sTh);
	
	// Changement du vecteur v (vitesse du photon)
	ph->vx = vx;
	ph->vy = vy;
	ph->vz = vz;

	
	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
	   int i = 0;
	   // On cherche la première action vide du tableau
	   while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
	   // Et on remplit la première case vide des tableaux (tableaux de 20 cases)
		// "3"représente l'événement "scatter" du photon
		evnt[i].action = 3;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
}


/* surfaceAgitee
* Reflexion sur une surface agitée ou plane en fonction de la valeur de DIOPTRE
*/
__device__ void surfaceAgitee(Photon* ph
		#ifdef RANDMWC
		, unsigned long long* etatThr, unsigned int* configThr
		#endif
		#ifdef RANDCUDA
		, hiprandState_t* etatThr
		#endif
		#ifdef RANDMT
		, EtatMT* etatThr, ConfigMT* configThr
		#endif
		#ifdef TRAJET
		, int idx, Evnt* evnt
		#endif
			){
	
	if( SIMd == -2){ // Atmosphère ou océan seuls, la surface absorbe tous les photons
		ph->loc = ABSORBED;
		
		#ifdef TRAJET
		// Récupération d'informations sur le premier photon traité
		if(idx == 0)
		{
			int i = 0;
			// On cherche la première action vide du tableau
			while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
			// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
			// "4"représente l'événement "surface" du photon
			evnt[i].action = 4;
			// On récupère le tau et le poids du photon
			evnt[i].poids = ph->weight;
			evnt[i].tau = ph->z;
		}
		#endif
		
		return;
	}
	
	// Réflexion sur le dioptre agité
	float theta;	// Angle de deflection polaire de diffusion [rad]
	float psi;		// Angle azimutal de diffusion [rad]
	float cTh, sTh;	//cos et sin de l'angle d'incidence du photon sur le dioptre
	
	float sig = 0.F;
	float beta = 0.F;	// Angle par rapport à la verticale du vecteur normal à une facette de vagues 
	float sBeta;
	float cBeta;
	
	float alpha = DEUXPI*RAND;	//Angle azimutal du vecteur normal a une facette de vagues
	
	float nind;
	float temp;
	
	float nx, ny, nz;	// Coordonnées du vecteur normal à une facette de vague
	float s1, s2, s3;
	
	float rpar, rper;	// Coefficient de reflexion parallèle et perpendiculaire
	float rpar2;		// Coefficient de reflexion parallèle au carré
	float rper2;		// Coefficient de reflexion perpendiculaire au carré
	float rat;			// Rapport des coefficients de reflexion perpendiculaire et parallèle
	float ReflTot;		// Flag pour la réflexion totale sur le dioptre
	float cot;			// Cosinus de l'angle de réfraction du photon
	float ncot, ncTh;	// ncot = nind*cot, ncoi = nind*cTh
	float tpar, tper;	//
	
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	
	/** Calcul du theta impact et phi impact **/
	//NOTE: Dans le code Fortran, ce calcul est effectué dans atmos
	float icp, isp, ict, ist;	// Sinus et cosinus de l'angle d'impact
	float vxn, vyn, vzn, uxn, uyn, uzn;
	
	ph->locPrec = ph->loc;
	
	
	/** Calcul de l'angle entre l'axe z et la normale au point d'impact **/
	/*NOTE: le float pour les calculs suivant fait une erreur de 2.3% 
	 * par exemple (theta_float=0.001196 / theta_double=0.0011691
	 * Mais ils sont bien plus performant et cette erreur ne pose pas de problème jusqu'à présent.
	 * De plus, l'angle d'impact n'est pas calculé mais directement les cosinus et sinus de cet angle.
	*/
	if( ph->z > 0. ){
		ict = __fdividef(ph->z,RTER);
		
		if(ict>1.f){
				ict = 1.f;
		}

		ist = sqrtf( 1.f - ict*ict );
   
		if(ph->x >= 0.f) ist = -ist;
		
		if( sqrtf(ph->x*ph->x + ph->y*ph->y)<1.e-6 ){
			/*NOTE En fortran ce test est à 1.e-8, relativement au double utilisés, peut peut être être supprimer ici*/
			icp = 1.f;
		}
		else{
			icp = __fdividef(ph->x,sqrtf(ph->x*ph->x + ph->y*ph->y));
			isp = sqrtf( 1.f - icp*icp );
			
			if( ph->y < 0.f ) isp = -isp;
	   }
	}
	else{
		// Photon considéré comme perdu
		ph->loc = ABSORBED;	// Correspondant au weight=0 en Fortran
		return;
	}
	
	
	/** Il faut exprimer Vx,y,z et Ux,y,z dans le repère de la normale au point d'impact **/
	vxn= ict*icp*ph->vx - ict*isp*ph->vy + ist*ph->vz;
	vyn= isp*ph->vx + icp*ph->vy;
	vzn= -icp*ist*ph->vx + ist*isp*ph->vy + ict*ph->vz;
	
	uxn= ict*icp*ph->ux - ict*isp*ph->uy + ist*ph->uz;
	uyn= isp*ph->ux + icp*ph->uy;
	uzn= -icp*ist*ph->ux + ist*isp*ph->uy + ict*ph->uz;
	
	ph->vx = vxn;
	ph->vy = vyn;
	ph->vz = vzn;
	ph->ux = uxn;
	ph->uy = uyn;
	ph->uz = uzn;
	#endif	/* Fin de la séparation atmosphère sphérique */
	
	/** **/
	
	if( DIOPTREd !=0 ){
		sig = sqrtf(0.003F + 0.00512f *WINDSPEEDd);
		beta = atanf( sig*sqrtf(-__logf(RAND)) );
	}
	sBeta = __sinf( beta );
	cBeta = __cosf( beta );
	
	nx = sBeta*__cosf( alpha );
	ny = sBeta*__sinf( alpha );
	
	// Projection de la surface apparente de la facette sur le plan horizontal		
	if( ph->vz > 0 ){
		nind = __fdividef(1.f,NH2Od);
		nz = -cBeta;
		ph->weight *= __fdividef( abs(nx*ph->vx + ny*ph->vy + nz*ph->vz), ph->vz*cBeta);
	}
	else{
		nind = NH2Od;
		nz = cBeta;
		ph->weight *= -__fdividef( abs(nx*ph->vx + ny*ph->vy + nz*ph->vz), ph->vz*cBeta );
	}
	
	
	temp = -(nx*ph->vx + ny*ph->vy + nz*ph->vz);

	theta = acosf( fmin(1.00F-VALMIN, fmax( -(1.F-VALMIN), temp ) ));

	if(theta >= DEMIPI){
		nx = -nx;
		ny = -ny;
		theta = acosf( -(nx*ph->vx + ny*ph->vy + nz*ph->vz) );
	}

	cTh = __cosf(theta);
	sTh = __sinf(theta);

	// Rotation des paramètres de Stokes
	s1 = ph->stokes1;
	s2 = ph->stokes2;
	s3 = ph->stokes3;

	if( (s1!=s2) || (s3!=0.F) ){

		temp = __fdividef(nx*ph->ux + ny*ph->uy + nz*ph->uz,sTh);
		psi = acosf( fmin(1.00F, fmax( -1.F, temp ) ));	

		if( (nx*(ph->uy*ph->vz-ph->uz*ph->vy) + ny*(ph->uz*ph->vx-ph->ux*ph->vz) + nz*(ph->ux*ph->vy-ph->uy*ph->vx) ) <0 )
		{
			psi = -psi;
		}

	/*psi est l'angle entre le plan de diffusion et le plan de diffusion precedent. Rotation des
	parametres de Stoke du photon d'apres cet angle.*/
	modifStokes(ph, psi, __cosf(psi), __sinf(psi), 0 );
	
	}

	if( sTh<=nind){
		temp = __fdividef(sTh,nind);
		cot = sqrtf( 1.0F - temp*temp );
		ncTh = nind*cTh;
		ncot = nind*cot;
		rpar = __fdividef(cot - ncTh,cot + ncTh);
		rper = __fdividef(cTh - ncot,cTh + ncot);
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
		rat = __fdividef(ph->stokes1*rper2 + ph->stokes2*rpar2,ph->stokes1+ph->stokes2);
		ReflTot = 0;
	}
	else{
		cot = 0.f;
		rpar = 1.f;
		rper = 1.f;
		rat = 0.f;
		rpar2 = rpar*rpar;
		rper2 = rper*rper;
		ReflTot = 1;
	}

	
	if( (ReflTot==1) || (SURd==1) || ( (SURd==3)&&(RAND<rat) ) ){
		//Nouveau parametre pour le photon apres reflexion
		
		// Le photon change de milieu
		if(ph->vz<0){
			if( SIMd==-1 || SIMd==0 ){
				ph->loc = SPACE;
			}
			else{
				ph->loc = ATMOS;
			}
		}
		else{
			if( SIMd==1 ){
				ph->loc = ABSORBED;
			}
			else{
				ph->loc = OCEAN;
			}
		}
		
		
		ph->stokes1 *= rper2;
		ph->stokes2 *= rpar2;
		ph->stokes3 *= -rpar*rper;
		
		ph->vx += 2.F*cTh*nx;
		ph->vy += 2.F*cTh*ny;
		ph->vz += 2.F*cTh*nz;
		ph->ux = __fdividef( nx-cTh*ph->vx,sTh );
		ph->uy = __fdividef( ny-cTh*ph->vy,sTh );
		ph->uz = __fdividef( nz-cTh*ph->vz,sTh );
		
		
		// Suppression des reflexions multiples
		if( (ph->vz<0) && (DIOPTREd==2) && (SIMd!=0 && SIMd!=2 && SIMd!=3) ){
			ph->loc = ABSORBED;
		}
		
		if( SURd==1 ){ /*On pondere le poids du photon par le coefficient de reflexion dans le cas 
			// d'une reflexion speculaire sur le dioptre (mirroir parfait)*/
			ph->weight *= rat;
		}
	}
	else{	// Transmission par le dioptre
		// Le photon change de milieu
		if(ph->vz<0){
			if( SIMd==-1 || SIMd==1 ){
				ph->loc = ABSORBED;
			}
			else{
				ph->loc = OCEAN;
			}
		}
		else{
			if( SIMd==-1 || SIMd==0 ){
				ph->loc = SPACE;
			}
			else{
				ph->loc = ATMOS;
			}
		}
		
		tpar = __fdividef( 2*cTh,ncTh+ cot);
		tper = __fdividef( 2*cTh,cTh+ ncot);
		
		ph->stokes2 *= tpar*tpar;
		ph->stokes1 *= tper*tper;
		ph->stokes3 *= -tpar*tper;
		
		
		alpha  = __fdividef(cTh,nind) - cot;
		ph->vx = __fdividef(ph->vx,nind) + alpha*nx;
		ph->vy = __fdividef(ph->vy,nind) + alpha*ny;
		ph->vz = __fdividef(ph->vz,nind) + alpha*nz;
		ph->ux = __fdividef( nx+cot*ph->vx,sTh )*nind;
		ph->uy = __fdividef( ny+cot*ph->vy,sTh )*nind;
		ph->uz = __fdividef( nz+cot*ph->vz,sTh )*nind;

		
		/* On pondere le poids du photon par le coefficient de transmission dans le cas d'une reflexion
		speculaire sur le dioptre plan (ocean diffusant) */
		if( SURd == 2)
			ph->weight *= (1-rat);
	}
	
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	/** Retour dans le repère d'origine **/
	
	// Re-projection vers le repères de direction de photon. L'angle à prendre pour la projection est -angleImpact
	isp = -isp;
	ist = -ist;
	
	vxn= ict*icp*ph->vx - ict*isp*ph->vy + ist*ph->vz;
	vyn= isp*ph->vx + icp*ph->vy;
	vzn= -icp*ist*ph->vx + ist*isp*ph->vy + ict*ph->vz;
	
	uxn= ict*icp*ph->ux - ict*isp*ph->uy + ist*ph->uz;
	uyn= isp*ph->ux + icp*ph->uy;
	uzn= -icp*ist*ph->ux + ist*isp*ph->uy + ict*ph->uz;
	
	ph->vx = vxn;
	ph->vy = vyn;
	ph->vz = vzn;
	ph->ux = uxn;
	ph->uy = uyn;
	ph->uz = uzn;
	#endif

	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
		int i = 0;
		// On cherche la première action vide du tableau
		while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
		// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
		// "4"représente l'événement "surface" du photon
		evnt[i].action = 4;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
}


/* surfaceLambertienne
* Reflexion sur une surface lambertienne
*/
__device__ void surfaceLambertienne(Photon* ph
						#ifdef RANDMWC
						, unsigned long long* etatThr, unsigned int* configThr
						#endif
						#ifdef RANDCUDA
						, hiprandState_t* etatThr
						#endif
						#ifdef RANDMT
						, EtatMT* etatThr, ConfigMT* configThr
						#endif
						#ifdef TRAJET
						, int idx, Evnt* evnt
						#endif
						){
	
	if( SIMd == -2){ 	// Atmosphère ou océan seuls, la surface absorbe tous les photons
		ph->loc = ABSORBED;
	}
	
	else{
	float thetab;	// angle de diffusion (entre le vecteur avt et après reflexion)
	float uxn,vxn,uyn,vyn,uzn,vzn;	// Vecteur du photon après reflexion
	float cTh2 = RAND;
	float cTh = sqrtf( cTh2 );
	float sTh = sqrtf( 1.0F - cTh2 );
	
	float phi = RAND*DEUXPI;	//angle azimutal
	float cPhi = __cosf(phi);
	float sPhi = __sinf(phi);
	
	float psi;
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	
	ph->locPrec=ph->loc;
	#endif
	
	
	/** calcul u,v new **/
	vxn = cPhi*sTh;
	vyn = sPhi*sTh;
	vzn = cTh;
	
	uxn = cPhi*cTh;
	uyn = sPhi*cTh;
	uzn = -sTh;
	
	/** Calcul angle Psi **/
	float temp;
	// Calcul du produit scalaire V.Vnew
	temp = ph->vx*vxn + ph->vy*vyn + ph->vz*vzn;
	thetab = acosf( fmin( fmax(-1.f,temp),1.f ) );
	if( thetab==0){
		ph->loc=SPACE;
		printf("theta nul\n");
		return;
	}
	
	// (Produit scalaire V.Unew)/sin(theta)
	temp = __fdividef( ph->vx*uxn + ph->vy*uyn + ph->vz*uzn, __sinf(thetab) );
	psi = acosf( fmin( fmax(-1.f,temp),1.f ) );	// angle entre le plan (u,v)old et (u,v)new
	
	if( (ph->vx*(uyn*vzn-uzn*vyn) + ph->vy*(uzn*vxn-uxn*vzn) + ph->vz*(uxn*vyn-uyn*vxn) ) <0 )
	{	// test du signe de v.(unew^vnew) (scalaire et vectoriel)
	psi = -psi;
	}
	
	modifStokes(ph, psi, __cosf(psi) , __sinf(psi), 0 );
	
	ph->vx = vxn;
	ph->vy = vyn;
	ph->vz = vzn;
	ph->ux = uxn;
	ph->uy = uyn;
	ph->uz = uzn;
	
	// Aucun photon n'est absorbés mais on pondère le poids par l'albedo de diffusion de la surface lambertienne.
	ph->weight *= W0LAMd;

	// Si le dioptre est seul, le photon est mis dans l'espace
	bool test_s = ( SIMd == -1);
	ph->loc = SPACE*test_s + ATMOS*(!test_s);
	
	}
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	/** Retour dans le repère d'origine **/
	
	#endif
	
	
	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
		int i = 0;
		// On cherche la première action vide du tableau
		while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
		// Et on remplit la première case vide des tableaux (tableaux de 20 cases)
		// "4"représente l'événement "surface" du photon
		evnt[i].action = 4;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
	
}


/* exit
* Sauve les paramètres des photons sortis dans l'espace dans la boite correspondant à la direction de sortie
*/
__device__ void exit(Photon* ph, Tableaux tab, unsigned long long* nbPhotonsThr
		#ifdef PROGRESSION
		, unsigned int* nbPhotonsSorThr, Variables* var
		#endif
		#ifdef TRAJET
		, int idx, Evnt* evnt
		#endif
		    )
{
	// Remise à zéro de la localisation du photon
	ph->loc = NONE;
	
	
// si son poids est anormalement élevé on le compte comme une erreur. Test effectué uniquement en présence de dioptre
	if( (ph->weight > WEIGHTMAX) && (SIMd!=-2)){
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurpoids), 1);
		#endif
		return;
	}
	
	/* Sinon on traite le photon et on l'ajoute dans le tableau tabPhotons de ce thread
	 * Incrémentation du nombre de photons traités par le thread
	*/
	(*nbPhotonsThr)++;
	
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Il ne faut pas prendre les photons sortant de la demi-sphère de l'atmosphère vers le bas
	if( ph->vz<=0.f ){
		return;
	}
	#endif
	
	// Création d'un float theta qui sert à modifier les nombres de Stokes
	float theta = acosf(fmin(1.F, fmax(-1.F, 0.f * ph->vx + 1.f * ph->vz)) );
	
	// Si theta = 0 on l'ignore (cas où le photon repart dans la direction solaire)
	if(theta == 0.F)
	{
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurtheta), 1);
		#endif
		return;
	}

	// Création d'un angle psi qui sert à modifier les nombres de Stokes
	float psi;
	int ith=0, iphi=0;
	// Initialisation de psi
	calculPsi(ph, &psi, theta);
	
	// Modification des nombres de Stokes
	float cPsi = __cosf(psi);
	float sPsi = __sinf(psi);
	modifStokes(ph, psi, cPsi, sPsi, 0);
	
	// Calcul de la case dans laquelle le photon sort
	calculCase(&ith, &iphi, ph 
			   #ifdef PROGRESSION
			   , var
			   #endif
			   );
	
	/** Modification des paramètres de stokes pour la compatibilité avec le code SOS **/
	if( ph->vy<0.f )
		ph->stokes3 = -ph->stokes3;
	
	if( ph->vx > 0.f )
		ph->stokes3 = -ph->stokes3;
	
	float s1 = ph->stokes1;
	ph->stokes1= ph->stokes2;
	ph->stokes2= s1;
	ph->stokes3 = -ph->stokes3;
	
	
	// On modifie ensuite le poids du photon
	ph->weight = __fdividef(ph->weight, ph->stokes1 + ph->stokes2);
	
	// Rangement du photon dans sa case, et incrémentation de variables
	if(((ith >= 0) && (ith < NBTHETAd)) && ((iphi >= 0) && (iphi < NBPHId)))
	{
		// Rangement dans le tableau des paramètres pondérés du photon

		atomicAdd(tab.tabPhotons+(0 * NBTHETAd * NBPHId + ith * NBPHId + iphi), ph->weight * ph->stokes1);

		atomicAdd(tab.tabPhotons+(1 * NBTHETAd * NBPHId + ith * NBPHId + iphi), ph->weight * ph->stokes2);

		atomicAdd(tab.tabPhotons+(2 * NBTHETAd * NBPHId + ith * NBPHId + iphi), ph->weight * ph->stokes3);
				
// 		atomicAdd(tab.tabPhotons+(3 * NBTHETAd * NBPHId + ith * NBPHId + iphi), ph->weight * ph->stokes4);

		#ifdef PROGRESSION
		// Incrémentation du nombre de photons sortis dans l'espace pour ce thread
		(*nbPhotonsSorThr)++;
		#endif

	}
	else
	{
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurcase), 1);
		#endif
	}

	#ifdef TRAJET
	// Récupération d'informations sur le premier photon traité
	if(idx == 0)
	{
		int i = 0;
		// On cherche la première action vide du tableau
		while(evnt[i].action != 0 && i<NBTRAJET-1) i++;
		// Et on remplit la première case vide des tableaux (tableaux de NBTRAJET cases)
		// "5"représente l'événement "exit" du photon
		evnt[i].action = 5;
		// On récupère le tau et le poids du photon
		evnt[i].poids = ph->weight;
		evnt[i].tau = ph->z;
	}
	#endif
}


/* modifStokes
* Modifie les paramètres de stokes
* Flag permet de tester (si flag=1) ou non la valeur des paramètres avant modification
*/
__device__ void modifStokes(Photon* photon, float psi, float cPsi, float sPsi, int flag)
{
	// On modifie les nombres de Stokes grâce à psi
	if( ((photon->stokes1 != photon->stokes2) || (photon->stokes3 != 0.F) ) || (flag==0))
	{
		float cPsi2 = cPsi * cPsi;
		float sPsi2 = sPsi * sPsi;
		float psi2 = 2.F*psi;
		float stokes1, stokes2, stokes3;
		float a, s2Psi;
		stokes1 = photon->stokes1;
		stokes2 = photon->stokes2;
		stokes3 = photon->stokes3;
		s2Psi = __sinf(psi2);
		a = 0.5f*s2Psi*stokes3;
		photon->stokes1 = cPsi2 * stokes1 + sPsi2 * stokes2 - a;
		photon->stokes2 = sPsi2 * stokes1 + cPsi2 * stokes2 + a;
		photon->stokes3 = s2Psi * (stokes1 - stokes2) + __cosf(psi2) * stokes3;
	}
}


/* calculPsi
* Calcul du psi pour la direction de sortie du photon
*/
__device__ void calculPsi(Photon* photon, float* psi, float theta)
{
	float sign;
// 	if (theta >= 0.05F)
// 	{
		*psi = acosf(fmin(1.F, fmax(-1.F, __fdividef(0.f * photon->ux + 1.f * photon->uz, __sinf(theta)))));
// 	}
// 	else
// 	{
// 		*psi = acosf(fmin(1.F - VALMIN, fmax(-(1.F - VALMIN), - 1.f * photon->ux + 0.f * photon->uz)));
// 	}
	
	sign = 0.f * (photon->uy * photon->vz - photon->uz * photon->vy) + 1.f * (photon->ux * photon->vy - photon->uy * photon->vx);
	if (sign < 0.F) *psi = -(*psi);
}


/* calculCase
* Fonction qui calcule la position (ith, iphi) du photon dans le tableau de sortie
* La position correspond à une boite contenu dans l'espace de sortie
*/
__device__ void calculCase(int* ith, int* iphi, Photon* photon
			#ifdef PROGRESSION
			, Variables* var
			#endif 
			)
{
	// vxy est la projection du vecteur vitesse du photon sur (x,y)
	float vxy = sqrtf(photon->vx * photon->vx + photon->vy * photon->vy);

	// Calcul de la valeur de ithv
	// _rn correspond à round to the nearest integer
	*ith = __float2int_rn(__fdividef(acosf(photon->vz) * NBTHETAd, DEMIPI));

	/* Si le photon ressort très près du zénith on ne peut plus calculer iphi,
	 on est à l'intersection de toutes les cases du haut */
	
	if(vxy >= VALMIN)
	{	//on calcule iphi
	
		// On place d'abord le photon dans un demi-cercle
		float cPhiP = __fdividef(photon->vx, vxy); //cosPhiPhoton
		// Cas limite où phi est très proche de 0, la formule générale ne marche pas
		if(cPhiP >= 1.F) *iphi = 0;
		// Cas limite où phi est très proche de PI, la formule générale ne marche pas
		else if(cPhiP <= -1.F) *iphi = (NBPHId) - 1;
		// Cas général
		else *iphi = __float2int_rd(__fdividef(acosf(cPhiP) * NBPHId, PI));
		
		// Puis on place le photon dans l'autre demi-cercle selon vy, utile uniquement lorsque l'on travail sur tous l'espace
// 		if(photon->vy < 0.F) *iphi = NBPHId - 1 - *iphi;
		#ifdef PROGRESSION
		// Lorsque vy=0 on décide par défaut que le photon reste du côté vy>0
		if(photon->vy == 0.F) atomicAdd(&(var->erreurvy), 1);
		#endif
	}
	
	else{
		// Photon très près du zenith
		#ifdef PROGRESSION
		atomicAdd(&(var->erreurvxy), 1);
		#endif
// 		/*if(photon->vy < 0.F) *iphi = NBPHId - 1;
// 		else*/ *iphi = 0;
		if(photon->vy >= 0.F)  *iphi = 0;
		else *iphi = NBPHId - 1;
	}
	
}


/**********************************************************
*	> Initialisation de données dans le device
***********************************************************/

/* initConstantesDevice
* Fonction qui initialise les constantes du device calculés dans le host
* Elle doit rester dans ce fichier
*/
void initConstantesDevice()
{
	hipMemcpyToSymbol(HIP_SYMBOL(NBPHOTONSd), &NBPHOTONS, sizeof(unsigned long long));
	hipMemcpyToSymbol(HIP_SYMBOL(NBLOOPd), &NBLOOP, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(THSDEGd), &THSDEG, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(LAMBDAd), &LAMBDA, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(TAURAYd), &TAURAY, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(TAUAERd), &TAUAER, sizeof(float));
	
	hipMemcpyToSymbol(HIP_SYMBOL(W0AERd), &W0AER, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(W0LAMd), &W0LAM, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(W0OCEd), &W0OCE, sizeof(float));
	
	hipMemcpyToSymbol(HIP_SYMBOL(HAd), &HA, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(HRd), &HR, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ZMINd), &ZMIN, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(ZMAXd), &ZMAX, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(NATMd), &NATM, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(HATMd), &HATM, sizeof(int));
	
	hipMemcpyToSymbol(HIP_SYMBOL(WINDSPEEDd), &WINDSPEED, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(NH2Od), &NH2O, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(CONPHYd), &CONPHY, sizeof(float));
	hipMemcpyToSymbol(HIP_SYMBOL(XBLOCKd), &XBLOCK, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(YBLOCKd), &YBLOCK, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(XGRIDd), &XGRID, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(YGRIDd), &YGRID, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(NBTHETAd), &NBTHETA, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(NBPHId), &NBPHI, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(PROFILd), &PROFIL, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(SIMd), &SIM, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(SURd), &SUR, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(DIOPTREd), &DIOPTRE, sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(DIFFFd), &DIFFF, sizeof(int));
	
	hipMemcpyToSymbol(HIP_SYMBOL(NFAERd), &NFAER, sizeof(unsigned int));
	hipMemcpyToSymbol(HIP_SYMBOL(NFOCEd), &NFOCE, sizeof(unsigned int));
		
	float THSbis = THSDEG*DEG2RAD; //thetaSolaire en radians
	hipMemcpyToSymbol(HIP_SYMBOL(THSd), &THSbis, sizeof(float));
	
	float CTHSbis = cos(THSbis); //cosThetaSolaire
	hipMemcpyToSymbol(HIP_SYMBOL(CTHSd), &CTHSbis, sizeof(float));
	
	float STHSbis = sin(THSbis); //sinThetaSolaire
	hipMemcpyToSymbol(HIP_SYMBOL(STHSd), &STHSbis, sizeof(float));
	
	float GAMAbis = DEPO / (2.F-DEPO);
	hipMemcpyToSymbol(HIP_SYMBOL(GAMAd), &GAMAbis, sizeof(float));
	
	#ifndef SPHERIQUE
	float TAUATM = TAURAY+TAUAER;
	hipMemcpyToSymbol(HIP_SYMBOL(TAUATMd), &TAUATM, sizeof(float));
	
	float TAUMAX = TAUATM / CTHSbis; //tau initial du photon
	hipMemcpyToSymbol(HIP_SYMBOL(TAUMAXd), &TAUMAX, sizeof(float));
	#endif
}


/**********************************************************
*	> Fonctions liées au générateur aléatoire
***********************************************************/

#ifdef RANDCUDA
/* initRandCUDA
* Fonction qui initialise les generateurs du random cuda
*/
__global__ void initRandCUDA(hiprandState_t* etat, unsigned long long seed)
{
	// Pour chaque thread on initialise son generateur avec le meme seed mais un idx different
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
	hiprand_init(seed, idx, 0, etat+idx);
}
#endif


#ifdef RANDMT
/* initRandMTEtat
* Fonction qui initialise l'etat des generateurs du random Mersenne Twister (generateur = etat + config)
*/
__global__ void initRandMTEtat(EtatMT* etat, ConfigMT* config)
{
	int idx = (blockIdx.x * gridDim.y + blockIdx.y) * blockDim.x * blockDim.y + (threadIdx.x * blockDim.y + threadIdx.y);
	// Initialisation de l'etat du MT de chaque thread avec un seed different et aleatoire
	etat[idx].mt[0] = config[idx].seed;
	for (int i = 1; i < MT_NN; i++)
		etat[idx].mt[i] = (1812433253U * (etat[idx].mt[i - 1] ^ (etat[idx].mt[i - 1] >> 30)) + i) & MT_WMASK;
	etat[idx].iState = 0;
	etat[idx].mti1 = etat[idx].mt[0];
}


/* randomMTfloat
* Fonction random Mersenne Twister qui renvoit un float de ]0.1] à partir d'un generateur (etat+config)
*/
__device__ float randomMTfloat(EtatMT* etat, ConfigMT* config)
{
	//Convert to (0, 1] float
	return __fdividef(__uint2float_rz(randomMTuint(etat, config)) + 1.0f, 4294967296.0f);
}


/* randomMTuint
* Fonction random Mersenne Twister qui renvoit un uint à partir d'un generateur (etat+config)
*/
__device__ unsigned int randomMTuint(EtatMT* etat, ConfigMT* config)
{
	unsigned int mti;
	unsigned int mtiM;
	unsigned int x;
	int iState1;
	int iStateM;
	iState1 = etat->iState + 1;
	iStateM = etat->iState + MT_MM;
	if(iState1 >= MT_NN) iState1 -= MT_NN;
	if(iStateM >= MT_NN) iStateM -= MT_NN;
	mti  = etat->mti1;
	etat->mti1 = etat->mt[iState1];
	mtiM = etat->mt[iStateM];
	
	// MT recurrence
	x = (mti & MT_UMASK) | (etat->mti1 & MT_LMASK);
	x = mtiM ^ (x >> 1) ^ ((x & 1) ? config->matrix_a : 0);
	
	etat->mt[etat->iState] = x;
	etat->iState = iState1;
	
	//Tempering transformation
	x ^= (x >> MT_SHIFT0);
	x ^= (x << MT_SHIFTB) & config->mask_b;
	x ^= (x << MT_SHIFTC) & config->mask_c;
	x ^= (x >> MT_SHIFT1);
	return x;
}
#endif


#ifdef RANDMWC
/* randomMWCfloat
* Fonction random MWC qui renvoit un float de ]0.1] à partir d'un generateur (x+a)
*/
__device__ float randomMWCfloat(unsigned long long* x,unsigned int* a)
{
	//Generate a random number (0,1]
	*x=(*x&0xffffffffull)*(*a)+(*x>>32);
	return __fdividef(__uint2float_rz((unsigned int)(*x)) + 1.0f,(float)0x100000000);
}

#endif
