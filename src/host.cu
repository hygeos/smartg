#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "host.h"
#include "device.h"


/**********************************************************
*
*			host.h
*
*	> Initialisation du générateur de nombres aléatoires MWC
*	> Travail sur les fichiers
*	> Initialisation des différentes structures
*	> Calculs de profils
*	> Fonctions d'affichage
*	> Calcul pour sauvegarde des résultats finaux
*	> Fichier hdf (lecture/écriture témoin, écriture résultats)
*
***********************************************************/


/**********************************************************
*	> Initialisation du générateur de nombres aléatoires MWC
***********************************************************/

/* initRandMWC
* Fonction qui initialise les generateurs du random MWC à partir d'un fichier texte
*/

int initRandMWC(unsigned long long *etat, unsigned int *config, 
	     const unsigned int n_rng, const char *safeprimes_file, unsigned long long xinit)
{
	FILE *fp;
	unsigned int begin=0u;
	unsigned int fora,tmp1,tmp2;
	if (strlen(safeprimes_file) == 0)
	{
        	// Try to find it in the local directory
		safeprimes_file = "MWC.txt";
	}
	fp = fopen(safeprimes_file, "r");
	if(fp == NULL)
	{
		printf("Could not find the file of safeprimes (%s)! Terminating!\n", safeprimes_file);
		return 1;
	}
	fscanf(fp,"%u %u %u",&begin,&tmp1,&tmp2);
	// Here we set up a loop, using the first multiplier in the file to generate x's and c's
	// There are some restictions to these two numbers:
	// 0<=c<a and 0<=x<b, where a is the multiplier and b is the base (2^32)
	// also [x,c]=[0,0] and [b-1,a-1] are not allowed.
	//Make sure xinit is a valid seed (using the above mentioned restrictions)
	if((xinit == 0ull) | (((unsigned int)(xinit>>32))>=(begin-1)) | (((unsigned int)xinit)>=0xfffffffful))
	{
		//xinit (probably) not a valid seed! (we have excluded a few unlikely exceptions)
		printf("%llu not a valid seed! Terminating!\n",xinit);
		return 1;
	}
	for(unsigned int i=0;i < n_rng;i++)
	{
		fscanf(fp,"%u %u %u",&fora,&tmp1,&tmp2);
		config[i]=fora;
		etat[i]=0;
		while( (etat[i]==0) | (((unsigned int)(etat[i]>>32))>=(fora-1)) | (((unsigned int)etat[i])>=0xfffffffful))
		{
			//generate a random number
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);
			//calculate c and store in the upper 32 bits of x[i]
			etat[i]=(unsigned int) floor((((double)((unsigned int)xinit))/(double)0x100000000)*fora);//Make sure 0<=c<a
			etat[i]=etat[i]<<32;
			//generate a random number and store in the lower 32 bits of x[i] (as the initial x of the generator)
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);//x will be 0<=x<b, where b is the base 2^32
			etat[i]+=(unsigned int) xinit;
		}
		//if(i<10)printf("%llu\n",x[i]);
	}
	fclose(fp);
	return 0;
}


/* initRandMTConfig
* Fonction qui initialise en partie les generateurs du random Mersenen Twister
*/
void initRandMTConfig(ConfigMT* config_H, ConfigMT* config_D, int nbThreads)
{
	// Ouverture du fichier
	const char *fname = "MersenneTwister.dat";
	FILE* fd = fopen(fname, "rb");
	if(!fd)
	{
		printf("ERREUR: ouverture fichier MT");
		exit(0);
	}
	// Lecture et initialisation de la config pour chaque generateur (= pour chaque thread)
	for(int i = 0; i < nbThreads; i++)
	{
		/* Le fichier ne contient que 4096 configs, on reutilise donc les memes configs pour les threads en trop mais les nombres
		aléatoires restent independants car les etats des threads sont differents */
		if(i%4096 == 0)
		{
			fseek(fd, 0, 0);
		}
		if(!fread(config_H+i, sizeof(ConfigMT), 1, fd))
		{
			printf("ERREUR: lecture fichier MT");
			exit(0);
		}
	}
	fclose(fd);
	srand((unsigned int)SEED);
	// Creation des seeds aleatoires pour que les threads aient des etats differents
	for(int i = 0; i < nbThreads; i++) config_H[i].seed = (unsigned int)rand();
	
	hipError_t erreur = hipMemcpy(config_D, config_H, nbThreads * sizeof(ConfigMT), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie config_H dans initRandMTConfig\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/**********************************************************
*	> Travail sur les fichiers
***********************************************************/

/* initConstantesHost
* Fonction qui récupère les valeurs des constantes dans le fichier paramètres et initialise les constantes du host
*/
void initConstantesHost(int argc, char** argv)
{
	if(argc < 2)
	{
		printf("ERREUR : lecture argv\n");
		exit(1);
	}
	
	FILE* parametres = fopen( argv[1], "r" );
	
	if( parametres == NULL ){
		printf("ERREUR: Impossible d'ouvrir le fichier %s\n", argv[1] );
		exit(1);
	}
	
	char s[256];

	strcpy(s,"");
	chercheConstante( parametres, "NBPHOTONS", s);
	NBPHOTONS = strtoull(s, NULL, 10);
	
	strcpy(s,"");
	chercheConstante(parametres, "NBLOOP", s);
	NBLOOP = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SEED", s);
	SEED = atoi(s);
	if(SEED == -1) SEED = static_cast<int> (time(NULL));

	strcpy(s,"");
	chercheConstante(parametres, "XBLOCK", s);
	XBLOCK= atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YBLOCK", s);
	YBLOCK = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "XGRID", s);
	XGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YGRID", s);
	YGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBTHETA", s);
	NBTHETA = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBPHI", s);
	NBPHI = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "PROFIL", s);
	PROFIL = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SIM", s);
	SIM = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SUR", s);
	SUR = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "DIOPTRE", s);
	DIOPTRE= atoi(s);
	

	if( SIM!=-2 ) DIFFF = 0;
	else{
		strcpy(s,"");
		chercheConstante(parametres, "DIFFF", s);
		DIFFF = atoi(s);
	}
	
	if( argc>2){ // Il est possible de rentrer theta à la main, utile pour debug et boucle shell
		THSDEG = atof(argv[2]);
	}
	else{
		strcpy(s,"");
		chercheConstante(parametres, "THSDEG", s);
		THSDEG = atof(s);
	}
	
	strcpy(s,"");
	chercheConstante(parametres, "LAMBDA", s);
	LAMBDA = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "TAURAY", s);
	TAURAY = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "TAUAER", s);
	TAUAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "W0AER", s);
	W0AER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "W0LAM", s);
	W0LAM = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "LSAAER", s);
	LSAAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NFAER", s);
	NFAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HA", s);
	HA = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HR", s);
	HR = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "ZMIN", s);
	ZMIN = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "ZMAX", s);
	ZMAX = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NATM", s);
	NATM = atoi(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HATM", s);
	HATM = atoi(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "WINDSPEED", s);
	WINDSPEED = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NH2O", s);
	NH2O = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "CONPHY", s);
	CONPHY = atof(s);
	
	chercheConstante(parametres, "PATHRESULTATSHDF", PATHRESULTATSHDF);
	// Remplir automatiquement le nom complet du fichier
	definirNomFichier(s);
// 	chercheConstante(parametres, "PATHTEMOINHDF", PATHTEMOINHDF);

	chercheConstante( parametres, "PATHDIFFAER", PATHDIFFAER );
	
	chercheConstante( parametres, "PATHPROFILATM", PATHPROFILATM );
	
	fclose( parametres );
}


/* chercheConstante
* Fonction qui cherche nomConstante dans le fichier et met la valeur de la constante dans chaineValeur (en string)
*/
void chercheConstante(FILE* fichier, char* nomConstante, char* chaineValeur)
{
	int longueur = strlen(nomConstante);
	char ligne[100];
	int motTrouve = 0;
	
	// Tant que la constante n'est pas trouvee et qu'on n'est pas à la fin du fichier on lit la ligne
	while(fgets(ligne, 100, fichier) && !motTrouve)
	{
		// Si le debut de la ligne est nomConstante suivi d'un espace ou un egal on va chercher la valeur
		if((strncmp(ligne, nomConstante, longueur) == 0) && (ligne[longueur] == ' ' || ligne[longueur] == '='))
		{
			char* ptr = ligne; //pointeur du debut de la ligne
			// on avance jusqu'au prochain espace ou egal
			while (*ptr != ' ' && *ptr != '=') ptr++;
			// on avance jusqu'à la valeur de la constante
			while(*ptr == ' ' || *ptr == '=') ptr++;
			if (*ptr == '\n')
			{
				printf("ERREUR : lecture Parametre.txt\n");
				exit(1);
			}
			// On met la chaine de la valeur de la constante dans chaineValeur
			strcpy(chaineValeur, ptr);
			chaineValeur[strlen(chaineValeur)-1] = '\0';
			motTrouve = 1;
		}
	}
	rewind(fichier);
	
	if(motTrouve == 0)
	{
		printf("ERREUR : lecture Parametres.txt\n");
		exit(1);
	}
	
}


/* definirNomFichier
* Le nom du fichier de sorti est créé automatiquement en fonction du type de simulation
* Il est également stoké dans un dossier en fonction de la date est du type de simulation
* Le chemin indiqué dans le fichier paramètres est le préfixe du chemin créé ici
*/
void definirNomFichier( char* s ){
	
	// Type de simulation
	definirSimulation(s);
	sprintf(PATHRESULTATSHDF,"%s/%s",PATHRESULTATSHDF,s);
	
	// Date de la simulation
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	sprintf(PATHRESULTATSHDF,"%s/simulation_%02u%02u%04u/",PATHRESULTATSHDF,date->tm_mday, date->tm_mon+1, 1900 + date->tm_year);
	
	// Création du dossier de stockage du résultat
	strcpy(s,"mkdir -p ");
	strcat(s,PATHRESULTATSHDF);
	system(s);
	
	// Création du dossier de stockage du temoin
	strcpy(PATHTEMOINHDF,PATHRESULTATSHDF);
	strcat(PATHTEMOINHDF,"tmp/");
	strcpy(s,"mkdir -p ");
	strcat(s,PATHTEMOINHDF);
	system(s);
	strcat(PATHTEMOINHDF,"tmp_");

	// Nom du fichier de sortie le plus compréhensible et utile possible
	if( SIM==1 ){
		if( DIOPTRE==3 )
		sprintf(s,"out_CUDA_atmos_dioptre_lambertien_ths=%4.2f_tRay=%4.4f_tAer=%4.4f_ws=%3.2f.hdf",THSDEG,TAURAY,TAUAER,WINDSPEED);
		else if( DIOPTRE==2 || DIOPTRE==1 )
			sprintf(s,"out_CUDA_atmos_dioptre_agite_ths=%4.2f_tRay=%4.4f_tAer=%4.4f_ws=%3.2f.hdf",THSDEG,TAURAY,TAUAER,WINDSPEED);
		else
			sprintf(s,"out_CUDA_atmos_dioptre_plan_ths=%4.2f_tRay=%4.4f_tAer=%4.4f.hdf",THSDEG,TAURAY,TAUAER);
	}
	else if( SIM==-1 ){
		if( DIOPTRE==3 )
			sprintf(s,"out_CUDA_dioptre_lambertien_ths=%4.2f_ws=%3.2f.hdf",THSDEG,WINDSPEED);
		else if( DIOPTRE==2 || DIOPTRE==1 )
			sprintf(s,"out_CUDA_dioptre_agite_ths=%4.2f_ws=%3.2f.hdf",THSDEG,WINDSPEED);
		else
			sprintf(s,"out_CUDA_dioptre_plan_ths=%4.2f_ws=%3.2f.hdf",THSDEG,WINDSPEED);
	}
	else if( SIM==-2 )
		sprintf(s,"out_CUDA_atmos_ths=%4.2f_tRay=%4.4f_tAer=%4.4f.hdf",THSDEG,TAURAY,TAUAER);
	else
		sprintf(s,"out_CUDA_ths=%4.2f_tRay=%4.4f_tAer=%4.4f_ws=%3.2f_sim=%d.hdf",THSDEG,TAURAY,TAUAER,WINDSPEED,SIM);
	
	strcat( PATHTEMOINHDF, s);
	strcat(PATHRESULTATSHDF,s);
	
}


/* definirSimulation
* Défini le type de simulation pour la création du chemin et le nom du fichier résultat
*/
void definirSimulation( char* s){

	if( SIM==1 ){
		if( TAUAER==0 && TAURAY!=0){
			if( DIOPTRE==3 )
				sprintf(s,"molecules_dioptre_lambertien");
			else if( DIOPTRE==2 || DIOPTRE==1 )
				sprintf(s,"molecules_dioptre_agite");
			else
				sprintf(s,"molecules_dioptre_plan");
		}
		else if( TAUAER!=0 && TAURAY==0){
			if( DIOPTRE==3 )
				sprintf(s,"aerosols_dioptre_lambertien");
			else if( DIOPTRE==2 || DIOPTRE==1 )
				sprintf(s,"aerosols_dioptre_agite");
			else
				sprintf(s,"aerosols_dioptre_plan");
		}
		else{
			if( DIOPTRE==3 )
				sprintf(s,"atmos_dioptre_lambertien");
			else if( DIOPTRE==2 || DIOPTRE==1 )
				sprintf(s,"atmos_dioptre_agite");
			else
				sprintf(s,"atmos_dioptre_plan");
		}
	}
	
	else if( SIM==-1 ){
		if( DIOPTRE==3 )
			sprintf(s,"dioptre_lambertien");
		else if( DIOPTRE==2 || DIOPTRE==1 )
			sprintf(s,"dioptre_agite");
		else
			sprintf(s,"dioptre_plan");
	}
	
	else if( SIM==-2 ){
		if( TAUAER==0 && TAURAY!=0)
			sprintf(s,"molecules_seules");
		else if( TAURAY==0 && TAUAER!=0)
			sprintf(s,"aerosols_seuls");
		else
			sprintf(s,"atmos_seule");
	}
	
	else
		sprintf(s,"SIM=%d",SIM);
}


/* verifierFichier
* Fonction qui vérifie l'état des fichiers temoin et résultats
* Demande à l'utilisateur s'il veut les supprimer ou non
*/
void verifierFichier(){
	char command[256];
	char res_supp;
	// S'il existe déjà un fichier nommé NOMRESULTATSHDF (Parametres.txt) on arrête le programme
	FILE* fic;
	fic = fopen(PATHTEMOINHDF, "rb");
	if ( fic != NULL)
	{
		printf("ATTENTION: Le fichier temoin %s existe deja.\n",PATHTEMOINHDF);
		printf("Voulez-vous le supprimer? [y/n]\n");
		res_supp=getchar();
		if( res_supp=='y' ){
			sprintf(command,"rm %s",PATHTEMOINHDF);
			system(command);
		}
		getchar();
		fclose(fic);
	}
	
	
	fic = fopen(PATHRESULTATSHDF, "rb");
	if ( fic != NULL)
	{
		printf("ATTENTION: Le fichier resultat %s existe deja.\n",PATHRESULTATSHDF);
		printf("Voulez-vous le supprimer pour continuer? [y/n]\n");
		// 		res_supp=getchar();
		// 		if( res_supp=='y' ){
//    sprintf(command,"rm %s",PATHRESULTATSHDF);
//    system(command);
   // 		}
	   fclose(fic);
	}
	
	
}


/**********************************************************
*	> Initialisation des différentes structures
***********************************************************/

/* initVariables
* Fonction qui initialise les variables à envoyer dans le kernel.
*/
void initVariables(Variables** var_H, Variables** var_D)
{
	// 	Initialisation de la version host des variables
	*var_H = (Variables*)malloc(sizeof(Variables));
	if( var_H == NULL ){
		printf("#--------------------#\n");
		printf("ERREUR: Problème de malloc de var_H dans initVariables\n");
		printf("#--------------------#\n");
		exit(1);
	}
	memset(*var_H, 0, sizeof(Variables));
	
	// Initialisation de la version device des variables
	if( hipMalloc(var_D, sizeof(Variables)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de var_D dans initVariables\n");
		exit(1);
	}
	
	hipError_t err = hipMemset(*(var_D), 0, sizeof(Variables));
	if( err != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de hipMemset var_D dans initVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	
	// var_H est une variable page-locked accessible par le device
// 	if( hipHostAlloc( var_H, sizeof(Variables), hipHostMallocPortable ) != hipSuccess ){
// 		printf("#--------------------#\n");
// 		printf("ERREUR: Problème d'allocation de var_H dans initVariables\n");
// 		printf("#--------------------#\n");
// 		exit(1);
// 	}
	
	// Un pointeur est associé pour travailler sur le device
// 	err = hipHostGetDevicePointer( var_D, *(var_H) ,0); 
// 	if( err != hipSuccess ){
// 		printf("#--------------------#\n");
// 		printf("ERREUR: Problème de mappage de var_D dans initVariables\n");
// 		printf("#--------------------#\n");
// 		exit(1);
// 	}

}


/* reinitVariables
* Fonction qui réinitialise certaines variables avant chaque envoi dans le kernel
*/
void reinitVariables(Variables* var_H, Variables* var_D)
{
	// Le nombre de photons traités pour un appel du Kernel est remis à zéro
	var_H->nbPhotons = 0;
	#ifdef PROGRESSION
	// Le nombre de photons ressortis pour un appel du Kernel est remis à zéro
	var_H->nbPhotonsSor = 0;
	#endif
	// On copie le nouveau var_H dans var_D
	hipError_t erreur = hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie var_H dans reinitVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("# sizeof(*var_D)=%d\tsizeof(*var_H)=%d\tsizeof(*Variables)=%d\n",sizeof(*var_D),sizeof(*var_H),sizeof(Variables));
		printf("# Adresse de var_D : %p\tAdresse de var_H : %p\n", var_H, var_D);
		printf("#--------------------#\n");
		exit(1);
	}
}


#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
/* initInit
* Initialisation de la structure Init contenant les paramètres initiaux du photon rentrant dans l'atmosphère.
* Ces paramètres sont utiles pour une atmosphère sphérique et sont calculés une seule fois dans le host, d'où cette fonction
* et la structure Init
*/
void initInit(Init** init_H, Init** init_D)
{
// 		Initialisation de la version host des variables
	*init_H = (Init*)malloc(sizeof(Init));
	if( init_H == NULL ){
	printf("#--------------------#\n");
	printf("ERREUR: Problème de malloc de init_H dans initInit\n");
	printf("#--------------------#\n");
	exit(1);
   	}
   	memset(*init_H, 0, sizeof(Init));
   
   // Initialisation de la version device des variables
   if( hipMalloc(init_D, sizeof(Init)) == hipErrorOutOfMemory ){
	   printf("ERREUR: Problème de hipMalloc de init_D dans initInit\n");
	   exit(1);
   }
   
   hipError_t err = hipMemset(*(init_D), 0, sizeof(Init));
   if( err != hipSuccess ){
	   printf("#--------------------#\n");
	   printf("# ERREUR: Problème de hipMemset init_D dans initInit\n");
	   printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
	   printf("#--------------------#\n");
	   exit(1);
   }
   
   
   
//    // var_H est une variable page-locked accessible par le device
//    if( hipHostAlloc( var_H, sizeof(Variables), hipHostMallocPortable ) != hipSuccess ){
// 	   printf("#--------------------#\n");
// 	   printf("ERREUR: Problème d'allocation de var_H dans initVariables\n");
// 	   printf("#--------------------#\n");
// 	   exit(1);
//    }
   
   // Un pointeur est associé pour travailler sur le device
   // 	err = hipHostGetDevicePointer( var_D, *(var_H) ,0); 
   // 	if( err != hipSuccess ){
   // 		printf("#--------------------#\n");
   // 		printf("ERREUR: Problème de mappage de var_D dans initVariables\n");
   // 		printf("#--------------------#\n");
   // 		exit(1);
   // 	}
   
}
#endif


/* initTableaux
* Fonction qui initialise les tableaux à envoyer dans le kernel par allocation mémoire et memset
*/
void initTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	hipError_t cudaErreur;	// Permet de tester les erreurs d'allocation mémoire
	
	#ifdef RANDMWC	
	// Création des tableaux de generateurs pour la fonction Random MWC
	tab_H->etat = (unsigned long long*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (unsigned int*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation des tableaux host à l'aide du fichier et du seed
	initRandMWC(tab_H->etat, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID, "MWC.txt", (unsigned long long)SEED);
	
	// Copie dans les tableaux device
	cudaErreur = hipMemcpy(tab_D->etat, tab_H->etat, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->etat dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	
	cudaErreur = hipMemcpy(tab_D->config, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->config dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	#endif
	
	#ifdef RANDCUDA
	// Création du tableau de generateurs (=etat+config) pour la fonction Random Cuda
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(hiprandState_t)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation du tableau dans une fonction du kernel
	initRandCUDA<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, (unsigned long long)SEED);
	#endif
	
	#ifdef RANDMT
	// Création des tableaux de generateurs pour la fonction Random Mersenne Twister
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (ConfigMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	tab_H->etat = (EtatMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
		
	// Initialisation du tableau des configs à l'aide du fichier
	initRandMTConfig(tab_H->config, tab_D->config, XBLOCK * YBLOCK * XGRID * YGRID);
	// Initialisation du tableau des etats dans le kernel
	initRandMTEtat<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, tab_D->config);
	#endif
	
	//if( hipHostAlloc( &(tab_H->tabPhotons), 4*NBTHETA*NBPHI*sizeof(*(tab_H->tabPhotons)), hipHostMallocPortable ) !=hipSuccess){
	// 		printf("#--------------------#\n");
	// 		printf("ERREUR: Problème d'allocation de tab_H->tabPhotons dans initTableaux\n");
	// 		printf("#--------------------#\n");
	// 		exit(1);
	// 	}
	
	// Tableau du poids des photons ressortis
	tab_H->tabPhotons = (float*)malloc(4*NBTHETA * NBPHI * sizeof(*(tab_H->tabPhotons)));
	if( tab_H->tabPhotons == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->tabPhotons dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->tabPhotons,0,4*NBTHETA * NBPHI * sizeof(*(tab_H->tabPhotons)) );
	
	if( hipMalloc(&(tab_D->tabPhotons), 4 * NBTHETA * NBPHI * sizeof(*(tab_D->tabPhotons))) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->tabPhotons dans initTableaux\n");
		exit(1);	
	}
	
	cudaErreur = hipMemset(tab_D->tabPhotons, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D->tabPhotons)));
	if( cudaErreur != hipSuccess ){
	printf("#--------------------#\n");
	printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le initTableaux\n");
	printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
	printf("#--------------------#\n");
	exit(1);
}
	
	
	// Modèle de diffusion des aérosols
	tab_H->faer = (float*)malloc(5 * NFAER * sizeof(float));
	if( tab_H->faer == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->faer dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->faer,0,5 * NFAER*sizeof(float) );
	
	if( hipMalloc(&(tab_D->faer), 5 * NFAER * sizeof(float)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->faer dans initTableaux\n");
		exit(1);	
	}
	
	/** Modèle de l'atmosphère **/
	// Epaisseur optique par couche
	tab_H->h =  (float*)malloc((NATM+1)*sizeof(*(tab_H->h)));
	if( tab_H->h == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->h dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->h,0,(NATM+1)*sizeof(*(tab_H->h)) );
	
	if( hipMalloc( &(tab_D->h), (NATM+1)*sizeof(*(tab_H->h)) ) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->h dans initTableaux\n");
		exit(1);	
	}
	
	// Proportion moléculaire par couche
	tab_H->pMol =  (float*)malloc((NATM+1)*sizeof(float));
	if( tab_H->pMol == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->pMol dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->pMol,0,(NATM+1)*sizeof(float) );
	
	if( hipMalloc( &(tab_D->pMol), (NATM+1)*sizeof(float) ) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->pMol dans initTableaux\n");
		exit(1);	
	}
	
	// Altitude des couches
	tab_H->z =  (float*)malloc((NATM+1)*sizeof(*(tab_H->z)));
	if( tab_H->z == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->z dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->z,0,(NATM+1)*sizeof(*(tab_H->z)) );
	
	if( hipMalloc( &(tab_D->z), (NATM+1)*sizeof(*(tab_H->z)) ) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->z dans initTableaux\n");
		exit(1);	
	}
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	/** Profil initial vu par le photon **/
	tab_H->zph0 =  (float*)malloc((NATM+1)*sizeof(*(tab_H->zph0)));
	if( tab_H->zph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->zph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->zph0,0,(NATM+1)*sizeof(*(tab_H->zph0)) );
	
	if( hipMalloc( &(tab_D->zph0), (NATM+1)*sizeof(*(tab_D->zph0)) ) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->zph0 dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->hph0 =  (float*)malloc((NATM+1)*sizeof(*(tab_H->hph0)));
	if( tab_H->hph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->hph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->hph0,0,(NATM+1)*sizeof(*(tab_H->hph0)) );
	
	if( hipMalloc( &(tab_D->hph0), (NATM+1)*sizeof(*(tab_D->hph0)) ) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->hph0 dans initTableaux\n");
		exit(1);	
	}
	#endif
	
}


/* freeTableaux
* Fonction qui libère l'espace mémoire de tous les tableaux alloués
*/
void freeTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	
	hipError_t erreur;	// Permet de tester le bon déroulement des hipFree
	
	#ifdef RANDMWC
	// Liberation des tableaux de generateurs du random MWC
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->etat);
	
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	#endif
	
	#ifdef RANDCUDA
	// Liberation du tableau de generateurs du random Cuda
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif
	
	#ifdef RANDMT
	// Liberation des tableaux de generateurs du random Mersenen Twister
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	free(tab_H->etat);
	#endif
	
	// Liberation du tableau du poids des photons
	erreur = hipFree(tab_D->tabPhotons);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->tabPhotons dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	// 	hipHostFree(tab_H->tabPhotons);
	free(tab_H->tabPhotons);
	
	// Libération du modèle de diffusion des aérosols
	erreur = hipFree(tab_D->faer);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->faer dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	free(tab_H->faer);
	
	// Libération du modèle atmosphérique
	erreur = hipFree(tab_D->h);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->h dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->h);
	
	//
	erreur = hipFree(tab_D->pMol);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->pMol dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->pMol);
	
	//
	erreur = hipFree(tab_D->z);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->z dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->z);
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Profil initial vu par la photon
	erreur = hipFree(tab_D->zph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->zph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->zph0);
	
	erreur = hipFree(tab_D->hph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->hph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->hph0);
	
	#endif
	
}


/**********************************************************
*	> Calculs de profils
***********************************************************/

/* calculFaer
* Calcul de la fonction de phase des aérosols
*/
void calculFaer( const char* nomFichier, Tableaux* tab_H, Tableaux* tab_D ){
	
	FILE* fichier = fopen(nomFichier, "r");

	double *scum = (double*) malloc(LSAAER*sizeof(*scum));
	if( scum==NULL ){
		printf("ERREUR: Problème de malloc de scum dans calculFaer\n");
		exit(1);
	}
	
	scum[0] = 0;
	int iang = 0, ipf = 0;
	double dtheta, pm1, pm2, sin1, sin2;
	double z, norm;

	/** Allocation de la mémoire des tableaux contenant les données **/
	double *ang;
	double *p1, *p2, *p3, *p4;
	ang = (double*) malloc(LSAAER*sizeof(*ang));
	p1 = (double*) malloc(LSAAER*sizeof(*p1));
	p2 = (double*) malloc(LSAAER*sizeof(*p2));
	p3 = (double*) malloc(LSAAER*sizeof(*p3));
	p4 = (double*) malloc(LSAAER*sizeof(*p4));
	if( ang==NULL || p1==NULL || p2==NULL || p3==NULL || p4==NULL ){
		printf("ERREUR: Problème de malloc de ang ou pi dans calculFaer\n");
		exit(1);
	}
	
	/** Lecture des données sur le modèle de diffusion des aérosols **/
	if(fichier == NULL){
		printf("ERREUR : Ouverture impossible du fichier %s pour la diffusion d'aérosol", nomFichier );
		exit(1);
	}
	
	else{
		for(iang=0; iang<LSAAER; iang++){
			fscanf(fichier, "%lf\t%lf\t%lf\t%lf\t%lf", ang+iang,p1+iang,p2+iang,p3+iang,p4+iang );
			// Conversion en radians
			ang[iang] = ang[iang]*DEG2RAD;
		}
	}
	
	if(fclose(fichier) == EOF){
		printf("ERREUR : Probleme de fermeture du fichier %s", nomFichier);
	}
		
	/** Calcul de scum **/
	for(iang=1; iang<LSAAER; iang++){
		
		dtheta = ang[iang] - ang[iang-1];
		pm1= p1[iang-1] + p2[iang-1];
		pm2= p1[iang] + p2[iang];
		sin1= sin(ang[iang-1]);
		sin2= sin(ang[iang]);
		
		scum[iang] = scum[iang-1] + dtheta*( (sin1*pm1+sin2*pm2)/3 + (sin1*pm2+sin2*pm1)/6 )*DEUXPI; 
	}
	
	// Normalisation
	for(iang=0; iang<LSAAER; iang++){
		scum[iang] = scum[iang]/scum[LSAAER-1];
// 		printf("scum[%d]=%10.10lf\n",iang,scum[iang] );
// 		if( scum[iang] == 1 )
// 			printf("Egal 1, iang=%d\n",iang);
	}
	
	/** Calcul des faer **/
	for(iang=0; iang<NFAER-1; iang++){
		z = double(iang+1)/double(NFAER);
// 		ipf=0;	// NOTE: Surement inutile
		while( (scum[ipf+1]<z) && ipf<(LSAAER-1) )
			ipf++;
		
		tab_H->faer[iang*5+4] = float( ((scum[ipf+1]-z)*ang[ipf] + (z-scum[ipf])*ang[ipf+1])/(scum[ipf+1]-scum[ipf]) );
		norm = p1[ipf]+p2[ipf];			// Angle
		tab_H->faer[iang*5+0] = float( p1[ipf]/norm );	// I paralèlle
		tab_H->faer[iang*5+1] = float( p2[ipf]/norm );	// I perpendiculaire
		tab_H->faer[iang*5+2] = float( p3[ipf]/norm );	// u
		tab_H->faer[iang*5+3] = 0.F;			// v, toujours nul
	}
	
	tab_H->faer[(NFAER-1)*5+4] = PI;
	tab_H->faer[(NFAER-1)*5+0] = 0.5F+00;
	tab_H->faer[(NFAER-1)*5+1] = 0.5F+00;
	tab_H->faer[(NFAER-1)*5+2] = float( p3[LSAAER-1]/(p1[LSAAER-1]+p2[LSAAER-1]) );
	tab_H->faer[(NFAER-1)*5+3] = 0.F+00;
	
	free(scum);
	free(ang);
	free(p1);
	free(p2);
	free(p3);
	free(p4);
	
	/** Allocation des FAER dans la device memory **/		

	hipError_t erreur = hipMemcpy(tab_D->faer, tab_H->faer, 5*NFAER*sizeof(*(tab_H->faer)), hipMemcpyHostToDevice); 
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->faer dans calculFaer\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
}


/* verificationFAER
* Sauvegarde la fonction de phase des aérosols calculée dans un fichier
* Permet de valider le bon calcul de la fonction de phase
*/
void verificationFAER( const char* nomFichier, Tableaux tab){

	FILE* fichier = fopen(nomFichier, "w");
	int i;
	
	fprintf( fichier, "angle\tI//\tIp\n" );
	
	for(i=0; i<NFAER; i++){
		fprintf(fichier, "%f\t%f\t%f\n", tab.faer[i*5+4],tab.faer[i*5+0], tab.faer[i*5+1]);
	}
	
	fclose(fichier);

}


/* profilAtm
* Calcul du profil atmosphérique dans l'atmosphère en fonction de la couche
* Mélange Molécule/Aérosol dans l'atmosphère en fonction de la couche
*/
void profilAtm( Tableaux* tab_H, Tableaux* tab_D ){

	/** Déclaration des variables **/
	/*NOTE: différence avec le code fortran: je n'utilise pas int ncouche */
	
	float tauMol[NATM+1];	// Epaisseur optique des molécules à chaque couche
	float tauAer[NATM+1];	// Epaisseur optique des aérosols à chaque couche
	int i=0;
	float va=0, vr=0;	// Variables tampons
	hipError_t erreur;	// Permet de tester le bon déroulement des opérations mémoires
	
	/** Conditions aux limites au sommet de l'atmosphère **/
	tab_H->z[0] = HATM;
	tauMol[0] = 0.0;
	tauAer[0] = 0.0;
	tab_H->h[0] = 0.0;
	tab_H->pMol[0] = 0.0;	//Je n'utilise pas la proportion d'aérosols car on l'obtient par 1-PMOL

	/** Cas Particuliers **/
	// Épaisseur optique aérosol très faible OU Épaisseur optique moléculaire et aérosol très faible
	// On ne considère une seule sous-couche dans laquelle on trouve toutes les molécules
	if( /*(TAUAER < 0.0001) ||*/ ((TAUAER < 0.0001)&&(TAURAY < 0.0001)) ){
		tauMol[1] = TAURAY;
		tauAer[1] = 0;
		tab_H->z[1]=0;
		tab_H->h[1] = tauMol[1] + tauAer[1];
		tab_H->pMol[1] = 1.0;
		
		/** Envoie des informations dans le device **/
		erreur = hipMemcpy(tab_D->h, tab_H->h, (NATM+1)*sizeof(*(tab_H->h)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->h dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		
		erreur = hipMemcpy(tab_D->pMol, tab_H->pMol, (NATM+1)*sizeof(*(tab_H->pMol)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->pMol dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}		
		
		erreur = hipMemcpy(tab_D->z, tab_H->z, (NATM+1)*sizeof(*(tab_H->z)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->z dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		return;
	}
	
	/** Profil standard avec échelle de hauteur **/
	if( PROFIL == 0 ){
		
		/* Si HA << HR => pas de mélange dans les couches
		On considere alors une atmosphere divisee en deux sous-couches, la  couche superieure contenant toutes les molecules, la
couche inferieure contenant tous les aerosols.
		*/
		if( HA < 0.0001 ){
			tauMol[1] = TAURAY;
			tauAer[1] = 0;
			tab_H->z[1]=0.f;
			tab_H->h[1] = tauMol[1] + tauAer[1];
			tab_H->pMol[1] = 1.0;
			
			tauMol[2] = 0;
			tauAer[2] = TAUAER;
			tab_H->z[2]=0.f;
			tab_H->h[2] = tab_H->h[1] + tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.0;
		}
		
		/* Si HA >> HR => pas de mélange dans les couches
		On considere alors une atmosphere divisee en deux sous-couches, la  couche superieure contenant tous les aérosols, la couche
inferieure contenant toutes les molécules.
		*/
		else if( HA > 499.99 ){
			tauMol[1] = 0.0;
			tauAer[1] = TAUAER;
			tab_H->z[1]=0.f;
			tab_H->h[1] = tauMol[1] + tauAer[1];
			tab_H->pMol[1] = 0.0;
			
			tauMol[2] = TAURAY;
			tauAer[2] = 0.0;
			tab_H->z[2]=0.f;
			tab_H->h[2] = tab_H->h[1] + tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 1.0;
		}
		
		/* Cas Standard avec deux échelles */
		else{
			for( i=0; i<NATM+1; i++){
				if(i!=0){
					tab_H->z[i]=100.F - float(i)*(100.F/NATM);
				}
				vr = TAURAY*exp( -(tab_H->z[i]/HR) );
				va = TAUAER*exp( -(tab_H->z[i]/HA) );
				
				tab_H->h[i] = va+vr;
				
				vr = vr/HR;
				va = va/HA;
				vr = vr/(va+vr);
				tab_H->pMol[i] = vr;
			}
			tab_H->h[0] = 0;
		}
	}
	
	/** Profil à 2 ou 3 couches **/
	else if( PROFIL == 3 ){

		float tauRay1;	// Epaisseur optique moleculaire de la couche 1
		float tauRay2;	// Epaisseur optique moleculaire de la couche 2
		
		tauRay1 = TAURAY*exp(-(ZMAX/HR));	// Epaisseur optique moleculaire de la couche la plus haute
		if( ZMIN < 0.0001 ){
			tauRay2 = TAURAY*(exp(-(ZMIN/HR))-exp(-(ZMAX/HR)));	// Epaisseur optique moleculaire de la couche la plus basse
		}
		
		else{
			tauRay2 = TAURAY*(exp(-(ZMIN/HR))-exp(-(ZMAX/HR)));	// Epaisseur optique moleculaire de la couche intermédiaire
		}
		
		/** Calcul des grandeurs utiles aux OS pour la couche la plus haute **/
		tab_H->z[1]=-( HR*log(tauRay1/TAURAY) );
		tauMol[1] = tauRay1;
		tauAer[1] = 0.F;                                    
		tab_H->h[1] = tauMol[1] + tauAer[1];
		tab_H->pMol[1] = 1.F;

		/** Calcul des grandeurs utiles aux OS pour la deuxieme couche   **/
		if( ZMAX == ZMIN ){ //Uniquement des aerosols dans la couche intermediaire
			tab_H->z[2]=ZMAX;
			tauMol[2] = tauRay1;                                                      
			tauAer[2] = TAUAER;
			tab_H->h[2] = tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.F;                                                      
		}
		
		else{	// Melange homogene d'aerosol et de molecules dans la couche intermediaire
			tab_H->z[2]=ZMIN;
			tauMol[2] = tauRay1+tauRay2;
			tauAer[2] = TAUAER;
			tab_H->h[2] = tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.5F;
		}
		
		/** Calcul des grandeurs utiles aux OS pour la troisieme couche **/
		tab_H->z[3]=0.f;
		tauMol[3] = TAURAY;
		tauAer[3] = TAUAER;
		tab_H->h[3] = tauMol[3] + tauAer[3];
		tab_H->pMol[3] = 1.F;
	}
	
	else if( PROFIL == 2 ){
		// Profil utilisateur
		/* Format du fichier
		=> Ne pas mettre de ligne vide sur la première
		=> n	alt		tauMol		tauAer		h		pAer		pMol
		*/
		FILE* profil = fopen( PATHPROFILATM , "r" );
		float garbage;
		
		int icouche=0;
		char ligne[1024];
	
		if(profil == NULL){
			printf("ERREUR : Ouverture impossible du fichier %s pour le profil atmosphérique\n", PATHPROFILATM );
			exit(1);
		}
		
		else{
			// Passage de la premiere ligne
			fgets(ligne,1024,profil);

			// Extraction des informations
			for( icouche=0; icouche<NATM+1; icouche++ ){
				fscanf(profil, "%d\t%f\t%f\t%f\t%f\t%f\t%f", &i, tab_H->z+icouche, &garbage, &garbage, tab_H->h+icouche,
&garbage,tab_H->pMol+icouche );
			
			}
		}
	
		if(fclose(profil) == EOF){
			printf("ERREUR : Probleme de fermeture du fichier %s", PATHPROFILATM);
		}
		
	}
	
	
		/** Envoie des informations dans le device **/
		erreur = hipMemcpy(tab_D->h, tab_H->h, (NATM+1)*sizeof(*(tab_H->h)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->h dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		
		erreur = hipMemcpy(tab_D->pMol, tab_H->pMol, (NATM+1)*sizeof(*(tab_H->pMol)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->pMol dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}	
		
		erreur = hipMemcpy(tab_D->z, tab_H->z, (NATM+1)*sizeof(*(tab_H->z)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->z dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
	
}


/* verificationAtm
* Sauvegarde du profil atmosphérique dans un fichier
* Permet de valider le bon calcul
*/
void verificationAtm( Tableaux tab_H ){
	
	// Vérification du modèle
	FILE* fichier = fopen("./test/modele_atm_cuda.txt", "w+");
	
	fprintf( fichier, "couche\tz\tpropMol\th\n" );
	
	for( int i=0; i<NATM+1; i++){
		fprintf(fichier, "%d\t%10.8f\t%10.8f\t%10.8f\n",i,tab_H.z[i],tab_H.pMol[i], tab_H.h[i]);
	}
	
	fprintf( fichier, "couche\tz\tpropMol\th\n" );
	
	fclose(fichier);
}

/** Séparation du code pour atmosphère sphérique ou parallèle **/
#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */

/* impactInit
* Calcul du profil que le photon va rencontrer lors de son premier passage dans l'atmosphère
* Sauvegarde de ce profil dans tab et sauvegarde des coordonnées initiales du photon dans init
*/
void impactInit(Init* init_H, Init* init_D, Tableaux* tab_H, Tableaux* tab_D){
	
	double thss, localh;
	double rdelta;
	double xphbis,yphbis,zphbis;	//Coordonnées intermédiaire du photon
	double rsolfi,rsol1,rsol2;
	
	// Correspond aux paramètres initiaux du photon
	double vx = -sin(THSDEG*DEG2RAD);
	double vy = 0.;
	double vz = -cos(THSDEG*DEG2RAD);
	
	/** Calcul du point d'impact **/
	// 	thss = abs(acosf(abs(vz)));
	thss = THSDEG*DEG2RAD;
	
	rdelta = 4.*RTER*RTER + 4.*( tan(thss)*tan(thss)+1. )*( HATM*HATM + 2.*HATM*RTER );
	localh = ( -2.*RTER+sqrt(rdelta) )/( 2.*(tan(thss)*tan(thss)+1.) );
	
	init_H->x0 = localh*tan(thss);
	init_H->y0 = 0.;
	init_H->z0 = RTER + localh;	
	
	tab_H->zph0[0] = 0.;
	tab_H->hph0[0] = 0.;
	
	xphbis = init_H->x0;
	yphbis = init_H->y0;
	zphbis = init_H->z0;
	
	/** Création hphoton et zphoton, chemin optique entre sommet atmosphère et sol pour la direction d'incidence **/
	for(int icouche=1; icouche<NATM+1; icouche++){
		
		rdelta = 4.*(vx*xphbis + vy*yphbis + vz*zphbis)*(vx*xphbis + vy*yphbis + vz*zphbis)
			- 4.*(xphbis*xphbis + yphbis*yphbis + zphbis*zphbis - ((double)tab_H->z[icouche]+RTER)*((double)tab_H->z[icouche]+RTER));
		rsol1 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) + sqrt(rdelta) );
		rsol2 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) - sqrt(rdelta) );
		
		// Il faut choisir la plus petite distance en faisant attention qu'elle soit positive
		if(rsol1>0.){
			if( rsol2>0.)
				rsolfi = min(rsol1,rsol2);
			else
				rsolfi = rsol1;
		}
		else{
			if( rsol2>0. )
				rsolfi=rsol1;
		}
		
		tab_H->zph0[icouche] = tab_H->zph0[icouche-1] + (float)rsolfi;
		tab_H->hph0[icouche] = (float)tab_H->hph0[icouche-1] + 
				(float)( abs( tab_H->h[icouche] - tab_H->h[icouche-1])*rsolfi )/
				(float)( abs( tab_H->z[icouche-1] - tab_H->z[icouche]) );
		
		xphbis+= vx*rsolfi;
		yphbis+= vy*rsolfi;
		zphbis+= vz*rsolfi;
		
	}

	init_H->taumax0 = tab_H->hph0[NATM];
	init_H->zintermax0 = tab_H->zph0[NATM];

	
	/** Envoie des données dans le device **/
	hipError_t erreur = hipMemcpy(init_D, init_H, sizeof(Init), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie init_H dans initInit\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	erreur = hipMemcpy(tab_D->hph0, tab_H->hph0, (NATM+1)*sizeof(*(tab_H->hph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->hph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	erreur = hipMemcpy(tab_D->zph0, tab_H->zph0, (NATM+1)*sizeof(*(tab_H->zph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->zph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
}
#endif


/**********************************************************
*	> Fonctions d'affichage
***********************************************************/

/* afficheParametres
* Affiche les paramètres de la simulation
*/
void afficheParametres()
{
	printf("\n#--------- Paramètres de simulation --------#\n");
	printf(" NBPHOTONS =\t%llu", NBPHOTONS);
	printf("\n");
	printf(" NBTHETA =\t%d", NBTHETA);
	printf("\n");
	printf(" NBPHI\t=\t%d", NBPHI);
	printf("\n");
	printf(" THSDEG\t=\t%f (degrés)", THSDEG);
	printf("\n");
	printf(" LAMBDA\t=\t%f", LAMBDA);
	printf("\n");
	printf(" CONPHY\t=\t%f", CONPHY);
	printf("\n");
	printf(" DIFFF\t=\t%d", DIFFF);
	printf("\n");
	printf(" SIM\t=\t%d", SIM);
	printf("\n");
	printf(" SEED\t=\t%d", SEED);
	printf("\n");
	
	printf("\n#------- Paramètres de performances --------#\n");
	printf(" NBLOOP\t=\t%u", NBLOOP);
	printf("\n");
	printf(" XBLOCK\t=\t%d", XBLOCK);
	printf("\n");
	printf(" YBLOCK\t=\t%d", YBLOCK);
	printf("\n");
	printf(" XGRID\t=\t%d", XGRID);
	printf("\n");
	printf(" YGRID\t=\t%d", YGRID);
	printf("\n");
	
	printf("\n#--------------- Atmosphère ----------------#\n");
	
	#ifdef SPHERIQUE
	printf(" Géométrie de l'atmosphère: \tSphérique");
	printf("\n");
	#endif
	#ifndef SPHERIQUE
	printf(" Géométrie de l'atmosphère: \tParallèle");
	printf("\n");
	#endif
	
	printf(" TAURAY\t=\t%f", TAURAY);
	printf("\n");
	printf(" TAUAER\t=\t%f", TAUAER);
	printf("\n");
	printf(" W0AER\t=\t%f", W0AER);
	printf("\n");
	printf(" LSAAER\t=\t%u", LSAAER);
	printf("\n");
	printf(" NFAER\t=\t%u", NFAER);
	printf("\n");
	printf(" PROFIL\t=\t%d", PROFIL);
	printf("\n");
	printf(" HA\t=\t%f", HA);
	printf("\n");
	printf(" HR\t=\t%f", HR);
	printf("\n");
	printf(" ZMIN\t=\t%f", ZMIN);
	printf("\n");
	printf(" ZMAX\t=\t%f", ZMAX);
	printf("\n");
	printf(" NATM\t=\t%d", NATM);
	printf("\n");
	printf(" HATM\t=\t%d", HATM);
	printf("\n");
	
	printf("\n#--------- Contribution du dioptre ---------#\n");
	printf(" SUR\t=\t%d", SUR);
	printf("\n");
	printf(" DIOPTRE =\t%d", DIOPTRE);
	printf("\n");
	printf(" W0LAM\t=\t%f", W0LAM);
	printf("\n");
	printf(" WINDSPEED =\t%f", WINDSPEED);
	printf("\n");
	printf(" NH2O\t=\t%f", NH2O);
	printf("\n");
	
	printf("\n#----------- Chemin des fichiers -----------#\n");
	printf(" PATHRESULTATSHDF = %s", PATHRESULTATSHDF);
	printf("\n");
	printf(" PATHTEMOINHDF = %s", PATHTEMOINHDF);
	printf("\n");
	printf(" PATHDIFFAER = %s", PATHDIFFAER);
	printf("\n");
	printf(" PATHPROFILATM = %s", PATHPROFILATM);
	printf("\n");
	
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	printf("\n  Date de début  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year,
		   date->tm_hour, date->tm_min, date->tm_sec);

}


/* afficheProgress
* Affiche la progression de la simulation
*/
void afficheProgress(unsigned long long nbPhotonsTot, Variables* var, double tempsPrec
#ifdef PROGRESSION
, unsigned long long nbPhotonsSorTot
#endif
)
{
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	// Calcul du temps ecoule et restant
	double tempsProg = (double)(clock() / CLOCKS_PER_SEC);
	double tempsTot = tempsProg + tempsPrec;
	int tempsEcoule = (int)tempsTot;
	int hEcoulees = tempsEcoule / 3600;
	int minEcoulees = (tempsEcoule%3600) / 60;
	int secEcoulees = tempsEcoule%60;
	int tempsRestant = (int)(tempsTot * ((double)NBPHOTONS / (double)nbPhotonsTot - 1.));
	if(tempsRestant < 0) tempsRestant = 0;
	int hRestantes = tempsRestant / 3600;
	int minRestantes = (tempsRestant%3600) / 60;
	int secRestantes = tempsRestant%60;
	// Calcul du pourcentage de photons traités
	int pourcent = (int)(100 * nbPhotonsTot / NBPHOTONS);
	
	// Affichage
	printf("\n");
	printf(" --------------------------------------\n");
	printf("  Photons lances : %12lu (%3d%%)\n", nbPhotonsTot, pourcent);
	printf("  Photons pb     : %12d\n", var->erreurpoids + var->erreurtheta);
	printf("  Temps ecoule   : %d h %2d min %2d sec\n", hEcoulees, minEcoulees, secEcoulees);
	printf("  Temps restant  : %d h %2d min %2d sec\n", hRestantes, minRestantes, secRestantes);
	printf("  Date actuelle  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year, date->tm_hour,
		   date->tm_min, date->tm_sec);
		   printf(" --------------------------------------\n");
		   
		   #ifdef PROGRESSION
		   printf("%d%% - ", (int)(100*nbPhotonsTot/NBPHOTONS));
		   printf("Temps: %d - ", tempsEcoule);
		   printf("phot sortis: %lu - ", nbPhotonsSorTot);
		   printf("phot traités: %lu - ", nbPhotonsTot);
		   printf("erreur poids/theta/vxy/vy/case: %d/%d/%d/%d/%d", var->erreurpoids, var->erreurtheta, var->erreurvxy,
var->erreurvy,		   var->erreurcase);
		   printf("\n");
		   #endif
}


#ifdef TRAJET
/* initEvnt
* Initialisation des variables à envoyer dans le kernel pour récupérer le trajet d'un photon
*/
void initEvnt(Evnt* evnt_H, Evnt* evnt_D)
{
	for(int i = 0; i < NBTRAJET; i++) evnt_H[i].action = 0;
	hipError_t erreur = hipMemcpy(evnt_D, evnt_H, NBTRAJET * sizeof(Evnt), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie evnt_H dans initEvnt\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/* afficheTrajet
* Fonction qui affiche le début du trajet du premier thread
*/
void afficheTrajet(Evnt* evnt_H)
{
	printf("\nTrajet d'un thread :\n");
	for(int i = 0; i < NBTRAJET; i++)
	{
		if(evnt_H[i].action == 1)
			printf("init : ");
		else if(evnt_H[i].action == 2)
			printf("move : ");
		else if(evnt_H[i].action == 3)
			printf("scat : ");
		else if(evnt_H[i].action == 4)
			printf("surf : ");
		else if(evnt_H[i].action != 5)
		{
			printf("\nERREUR : host afficheTrajet: Aucun trajet a afficher\n");
			return;
		}
		else printf("exit : ");
		printf("tau=%10.9f ", evnt_H[i].tau);
		printf("poids=%10.9f", evnt_H[i].poids);
		printf("\n");
	}
}
#endif


/**********************************************************
*	> Calcul pour sauvegarde des résultats finaux
***********************************************************/

/* calculOmega
* Fonction qui calcule l'aire normalisée de chaque boite, son theta, et son psi, sous forme de 3 tableaux
*/
void calculOmega(float* tabTh, float* tabPhi, float* tabOmega)
{
	// Tableau contenant l'angle theta de chaque morceau de sphère
	memset(tabTh, 0, NBTHETA * sizeof(*tabPhi));
	float dth = DEMIPI / NBTHETA;
	tabTh[0] = dth/4;
	tabTh[1] = dth;
	for(int ith = 2; ith < NBTHETA; ith++){
		tabTh[ith] = tabTh[ith-1] + dth;
	}
	
	// Tableau contenant l'angle psi de chaque morceau de sphère
	memset(tabPhi, 0, NBPHI * sizeof(*tabPhi));
	float dphi = PI / NBPHI;
 	tabPhi[0] = dphi / 2;
	for(int iphi = 1; iphi < NBPHI; iphi++){ 
		tabPhi[iphi] = tabPhi[iphi-1] + dphi;
	}
	// Tableau contenant l'aire de chaque morceau de sphère
	float sumds = 0;
	float tabds[NBTHETA * NBPHI];
	memset(tabds, 0, NBTHETA * NBPHI * sizeof(*tabds));
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		if( ith==0 )
			dth = DEMIPI / (2*NBTHETA);
		else 
			dth = DEMIPI / NBTHETA;
			
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			tabds[ith * NBPHI + iphi] = sin(tabTh[ith]) * dth * dphi;
			sumds += tabds[ith * NBPHI + iphi];
		}
	}
	
	// La derniere demi boite 89.75->90
	for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			sumds += sin( (DEMIPI+tabTh[NBTHETA-1])/2 ) * (dth/2) * dphi;
		}
	
	// Normalisation de l'aire de chaque morceau de sphère
	memset(tabOmega, 0, NBTHETA * NBPHI * sizeof(*tabOmega));
	for(int ith = 0; ith < NBTHETA; ith++)
		for(int iphi = 0; iphi < NBPHI; iphi++){
			tabOmega[ith * NBPHI + iphi] = tabds[ith * NBPHI + iphi] / sumds;
		}
}


/* calculTabFinal
* Fonction qui remplit le tabFinal correspondant à la reflectance (R), Q et U sur tous l'espace de sorti (dans chaque boite)
*/
void calculTabFinal(float* tabFinal, float* tabTh, float* tabPhi, float* tabPhotonsTot, unsigned long long nbPhotonsTot)
{
	
	float tabOmega[NBTHETA * NBPHI]; //tableau contenant l'aire de chaque morceau de sphère
	// Remplissage des tableaux tabTh, tabPhi, et tabOmega
	calculOmega(tabTh, tabPhi, tabOmega);
	
	// Remplissage du tableau final
	for(int iphi = 0; iphi < NBPHI; iphi++)
	{
		for(int ith = 0; ith < NBTHETA; ith++)
		{
			// Reflectance
			tabFinal[0*NBTHETA*NBPHI + iphi*NBTHETA+ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA+ith*NBPHI+iphi] + tabPhotonsTot[1*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi]* cosf(tabTh[ith]));
			
			// Q
			tabFinal[1*NBTHETA*NBPHI + iphi*NBTHETA+ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA+ith*NBPHI+iphi] - tabPhotonsTot[1*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi] * cosf(tabTh[ith]));
			
			// U
			tabFinal[2*NBTHETA*NBPHI + iphi*NBTHETA+ith] = (tabPhotonsTot[2*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi] * cosf(tabTh[ith]));
				
		}
	}
}


/**********************************************************
*	> Fichier hdf (lecture/écriture témoin, écriture résultats)
***********************************************************/

/* creerHDFTemoin
* Fonction qui crée un fichier .hdf contenant les informations nécessaires à la reprise du programme
* //TODO: 	écrire moins régulièrement le témoin (non pas une écriture par appel de kernel)
*			changer le format (écrire un .bin par exemple) pour éventuellement gagner du temps (calculer le gain éventuel)
*/
void creerHDFTemoin(float* tabPhotonsTot, unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	int sdFichier = SDstart(PATHTEMOINHDF, DFACC_CREATE);
	
	char nomTab[20]; //nom du tableau
	sprintf(nomTab,"Temoin (%d%%)", (int)(100 * nbPhotonsTot / NBPHOTONS));
	int nbDimsTab = 1; //nombre de dimensions du tableau
	int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
	valDimsTab[0] = 4 * NBTHETA * NBPHI;
	int typeTab = DFNT_FLOAT32 ; //type des éléments du tableau
	// Création du tableau
	int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	int startTab[nbDimsTab]; //début de la lecture du tableau
	startTab[0]=0;
	// Ecriture du tableau dans le fichier
	int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tabPhotonsTot);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf temoin\n");
		exit(1);
	}
	
	// Ecriture de toutes les informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSdouble = (double)NBPHOTONS; // on convertit en double car le hdf n'accepte pas ull
	double nbPhotonsTotdouble = (double)nbPhotonsTot; // on convertit en double car le hdf n'accepte pas ull
	double tempsEcouledouble = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
	SDsetattr(sdsTab, "NBPHOTONS", DFNT_FLOAT64, 1, &NBPHOTONSdouble);
	SDsetattr(sdsTab, "NBLOOP", DFNT_UINT32, 1, &NBLOOP);
	SDsetattr(sdsTab, "SEED", DFNT_UINT32, 1, &SEED);
	SDsetattr(sdsTab, "XBLOCK", DFNT_INT32, 1, &XBLOCK);
	SDsetattr(sdsTab, "YBLOCK", DFNT_INT32, 1, &YBLOCK);
	SDsetattr(sdsTab, "XGRID", DFNT_INT32, 1, &XGRID);
	SDsetattr(sdsTab, "YGRID", DFNT_INT32, 1, &YGRID);
	SDsetattr(sdsTab, "NBTHETA", DFNT_INT32, 1, &NBTHETA);
	SDsetattr(sdsTab, "NBPHI", DFNT_INT32, 1, &NBPHI);
	SDsetattr(sdsTab, "DIOPTRE", DFNT_INT32, 1, &DIOPTRE);
	SDsetattr(sdsTab, "DIFFF", DFNT_INT32, 1, &DIFFF);
	SDsetattr(sdsTab, "PROFIL", DFNT_INT32, 1, &PROFIL);
	SDsetattr(sdsTab, "SIM", DFNT_INT32, 1, &SIM);
	SDsetattr(sdsTab, "SUR", DFNT_INT32, 1, &SUR);
	SDsetattr(sdsTab, "THSDEG", DFNT_FLOAT32, 1, &THSDEG);
	SDsetattr(sdsTab, "LAMBDA", DFNT_FLOAT32, 1, &LAMBDA);
	SDsetattr(sdsTab, "TAURAY", DFNT_FLOAT32, 1, &TAURAY);
	SDsetattr(sdsTab, "TAUAER", DFNT_FLOAT32, 1, &TAUAER);
	SDsetattr(sdsTab, "W0AER", DFNT_FLOAT32, 1, &W0AER);
	SDsetattr(sdsTab, "W0LAM", DFNT_FLOAT32, 1, &W0LAM);
	
	SDsetattr(sdsTab, "LSAAER", DFNT_UINT32, 1, &LSAAER);
	SDsetattr(sdsTab, "NFAER", DFNT_UINT32, 1, &NFAER);
	
	SDsetattr(sdsTab, "HA", DFNT_FLOAT32, 1, &HA);
	SDsetattr(sdsTab, "HR", DFNT_FLOAT32, 1, &HR);
	SDsetattr(sdsTab, "ZMIN", DFNT_FLOAT32, 1, &ZMIN);
	SDsetattr(sdsTab, "ZMAX", DFNT_FLOAT32, 1, &ZMAX);
	SDsetattr(sdsTab, "NATM", DFNT_INT32, 1, &NATM);
	SDsetattr(sdsTab, "HATM", DFNT_INT32, 1, &HATM);
	SDsetattr(sdsTab, "WINDSPEED", DFNT_FLOAT32, 1, &WINDSPEED);
	SDsetattr(sdsTab, "NH2O", DFNT_FLOAT32, 1, &NH2O);
	SDsetattr(sdsTab, "CONPHY", DFNT_FLOAT32, 1, &CONPHY);
	SDsetattr(sdsTab, "PATHRESULTATSHDF", DFNT_CHAR8, strlen(PATHRESULTATSHDF), PATHRESULTATSHDF);
	SDsetattr(sdsTab, "PATHTEMOINHDF", DFNT_CHAR8, strlen(PATHTEMOINHDF), PATHTEMOINHDF);
	SDsetattr(sdsTab, "PATHDIFFAER", DFNT_CHAR8, strlen(PATHDIFFAER), PATHDIFFAER);
	SDsetattr(sdsTab, "PATHPROFILATM", DFNT_CHAR8, strlen(PATHPROFILATM), PATHPROFILATM);
	
	SDsetattr(sdsTab, "nbPhotonsTot", DFNT_FLOAT64, 1, &nbPhotonsTotdouble);
	SDsetattr(sdsTab, "nbErreursPoids", DFNT_INT32, 1, &(var->erreurpoids));
	SDsetattr(sdsTab, "nbErreursTheta", DFNT_INT32, 1, &(var->erreurtheta));
	SDsetattr(sdsTab, "tempsEcoule", DFNT_FLOAT64, 1, &tempsEcouledouble);
	
	#ifdef PROGRESSION
	SDsetattr(sdsTab, "nbThreads", DFNT_UINT32, 1, &(var->nbThreads));
	SDsetattr(sdsTab, "nbPhotonsSor", DFNT_UINT32, 1, &(var->nbPhotonsSor));
	SDsetattr(sdsTab, "erreurvxy", DFNT_INT32, 1, &(var->erreurvxy));
	SDsetattr(sdsTab, "erreurvy", DFNT_INT32, 1, &(var->erreurvy));
	SDsetattr(sdsTab, "erreurcase", DFNT_INT32, 1, &(var->erreurcase));
	#endif

	// Fermeture du tableau
	SDendaccess(sdsTab);
	// Fermeture du fichier
	SDend(sdFichier);
}


/* lireHDFTemoin
* Si un fichier temoin existe et que les paramètres correspondent à la simulation en cours, cette simulation se poursuit à
* partir de celle sauvée dans le fichier témoin.
*/
void lireHDFTemoin(Variables* var_H, Variables* var_D,
		unsigned long long* nbPhotonsTot, float* tabPhotonsTot, double* tempsEcoule)
{
	// Ouverture du fichier temoin
	int sdFichier = SDstart(PATHTEMOINHDF, DFACC_READ);
	if(sdFichier != -1)
	{
		// Ouverture de l'unique tableau du fichier temoin
		int sdsIndex = 0;
		int sdsTab = SDselect (sdFichier, sdsIndex);
		
		// Recuperation de paramètres du fichier temoin
		int SEEDrecup[1];
		int NBTHETArecup[1];
		int NBPHIrecup[1];
		int DIOPTRErecup[1];
		int DIFFFrecup[1];
		int PROFILrecup[1];
		int SIMrecup[1];
		int SURrecup[1];
		float THSDEGrecup[1];
		float LAMBDArecup[1];
		float TAURAYrecup[1];
		float TAUAERrecup[1];
		float W0AERrecup[1];
		float W0LAMrecup[1];
		float HArecup[1];
		float HRrecup[1];
		float ZMINrecup[1];
		float ZMAXrecup[1];
		int NATMrecup[1];
		int HATMrecup[1];
		float WINDSPEEDrecup[1];
		float NH2Orecup[1];
		float CONPHYrecup[1];
		
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SEED"), (VOIDP)SEEDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBTHETA"), (VOIDP)NBTHETArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBPHI"), (VOIDP)NBPHIrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIOPTRE"), (VOIDP)DIOPTRErecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIFFF"), (VOIDP)DIFFFrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "PROFIL"), (VOIDP)PROFILrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SIM"), (VOIDP)SIMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SUR"), (VOIDP)SURrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "THSDEG"), (VOIDP)THSDEGrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "LAMBDA"), (VOIDP)LAMBDArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAURAY"), (VOIDP)TAURAYrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAUAER"), (VOIDP)TAUAERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "W0AER"), (VOIDP)W0AERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "W0LAM"), (VOIDP)W0LAMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HA"), (VOIDP)HArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HR"), (VOIDP)HRrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMIN"), (VOIDP)ZMINrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMAX"), (VOIDP)ZMAXrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NATM"), (VOIDP)NATMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HATM"), (VOIDP)HATMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "WINDSPEED"), (VOIDP)WINDSPEEDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NH2O"), (VOIDP)NH2Orecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "CONPHY"), (VOIDP)CONPHYrecup);
		
		// Si les parametres sont les memes on recupere des informations pour poursuivre la simulation précédente
		if(NBTHETArecup[0] == NBTHETA
			&& NBPHIrecup[0] == NBPHI
			&& DIOPTRErecup[0] == DIOPTRE
			&& DIFFFrecup[0] == DIFFF
			&& PROFILrecup[0] == PROFIL
			&& SIMrecup[0] == SIM
			&& SURrecup[0] == SUR
			&& THSDEGrecup[0] == THSDEG
			&& LAMBDArecup[0] == LAMBDA
			&& TAURAYrecup[0] == TAURAY
			&& TAUAERrecup[0] == TAUAER
			&& W0AERrecup[0] == W0AER
			&& W0LAMrecup[0] == W0LAM
			&& HArecup[0] == HA
			&& HRrecup[0] == HR
			&& ZMINrecup[0] == ZMIN
			&& ZMAXrecup[0] == ZMAX
			&& NATMrecup[0] == NATM
			&& HATMrecup[0] == HATM
			&& WINDSPEEDrecup[0] == WINDSPEED
			&& NH2Orecup[0] == NH2O
			&& CONPHYrecup[0] == CONPHY)
		{
			
			// Recuperation du nombre de photons traités et du nombre d'erreurs
			double nbPhotonsTotDouble[1]; //on récupère d'abord la variable en double
			unsigned long long nbPhotonsTotRecup[1]; //puis on la passera en unsigned long long
			int nbErreursPoidsRecup[1];
			int nbErreursThetaRecup[1];
			double tempsEcouleRecup[1];
	
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbPhotonsTot"), (VOIDP)nbPhotonsTotDouble);
			nbPhotonsTotRecup[0] = (unsigned long long)nbPhotonsTotDouble[0];
			
			printf("\nPOURSUITE D'UNE SIMULATION ANTERIEURE AU PHOTON %e\n",nbPhotonsTotDouble[0]);
			if(SEEDrecup[0] == SEED) printf("ATTENTION: Nous recommandons SEED=-1 sinon les nombres aleatoires sont\
identiques a chaque lancement.\n");
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursPoids"), (VOIDP)nbErreursPoidsRecup);
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursTheta"), (VOIDP)nbErreursThetaRecup);
			SDreadattr(sdsTab, SDfindattr(sdsTab, "tempsEcoule"), (VOIDP)tempsEcouleRecup);
	
			var_H->erreurpoids = nbErreursPoidsRecup[0];//nombre de photons ayant un poids anormalement élevé
			var_H->erreurtheta = nbErreursThetaRecup[0];//nombre de photons sortant dans la direction solaire
			
			#ifdef PROGRESSION
			unsigned long long nbThreadsRecup[1]; //nombre total de threads lancés
			unsigned long long nbPhotonsSorRecup[1]; //nombre de photons ressortis pour un appel du Kernel
			int erreurvxyRecup[1]; //nombre de photons sortant au zénith et donc difficiles à classer
			int erreurvyRecup[1]; //nombre de photons sortant à phi=0 ou phi=PI et donc difficiles à classer
			int erreurcaseRecup[1]; // nombre de photons rangé dans une case inexistante
			
			SDreadattr(sdsTab, SDfindattr(sdsTab,"nbThreads"), (VOIDP)nbThreadsRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"nbPhotonsSor"), (VOIDP)nbPhotonsSorRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurvxy"), (VOIDP)erreurvxyRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurvy"), (VOIDP)erreurvyRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurcase"), (VOIDP)erreurcaseRecup );
			
			var_H->nbThreads = nbThreadsRecup[0];
			var_H->nbPhotonsSor = nbPhotonsSorRecup[0];
			var_H->erreurvxy = erreurvxyRecup[0];
			var_H->erreurvy = erreurvyRecup[0];
			var_H->erreurcase = erreurcaseRecup[0];
			
			#endif
			
			hipError_t erreur = hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
			if( erreur != hipSuccess ){
				printf( "ERREUR: Problème de copie var_H dans lireHDFTemoin\n");
				printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
				exit(1);
			}
			*(nbPhotonsTot) = nbPhotonsTotRecup[0];
			*(tempsEcoule) = tempsEcouleRecup[0];
	
			// Recuperation du tableau
			int nbDimsTab = 1; //nombre de dimensions du tableau
			int startTab[nbDimsTab], edgesTab[nbDimsTab]; //debut et fin de la lecture du tableau
			startTab[0] = 0;
			edgesTab[0] = 4*NBTHETA * NBPHI;
// 			float tabPhotonsTotRecup[4*NBTHETA * NBPHI]; //tableau de récuperation en float
	
			int status = SDreaddata (sdsTab, startTab, NULL, edgesTab, (VOIDP)tabPhotonsTot/*Recup*/);
			// Vérification du bon fonctionnement de la lecture
			if(status)
			{
				printf("\nERREUR : read hdf temoin\n");
				exit(1);
			}
	
			// Sauvegarde
// 			for(int i = 0; i < 4*NBTHETA * NBPHI; i++)
// 			{
// 				tabPhotonsTot[i] = tabPhotonsTotRecup[i];
// 			}
			
		}
		// Fermeture du tableau
		SDendaccess (sdsTab);
	}
	// Fermeture du fichier
	SDend (sdFichier);
}


/* creerHDFResultats
* Fonction qui crée le fichier .hdf contenant le résultat final pour une demi-sphère
*/
void creerHDFResultats(float* tabFinal, float* tabTh, float* tabPhi,unsigned long long nbPhotonsTot, Variables* var,double tempsPrec)
{
	// Tableau temporaire utile pour la suite
	float tab[NBPHI*NBTHETA];
	// Création du fichier de sortie
	int sdFichier = SDstart(PATHRESULTATSHDF, DFACC_CREATE);
	if( sdFichier == FAIL ){
		printf("ERREUR: Erreur d'ouverture du fichier HDF : %s\nFin du programme\n", PATHRESULTATSHDF);
		exit(1);
	}
	
	// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSdouble = (double)NBPHOTONS;
	double nbPhotonsTotdouble = (double)nbPhotonsTot;
	double tempsEcouledouble = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);

	SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, &NBPHOTONSdouble);
	SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, &NBLOOP);
	SDsetattr(sdFichier, "SEED", DFNT_UINT32, 1, &SEED);
	SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, &XBLOCK);
	SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, &YBLOCK);
	SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, &XGRID);
	SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, &YGRID);
	SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, &NBTHETA);
	SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, &NBPHI);
	SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, &DIOPTRE);
	SDsetattr(sdFichier, "DIFFF", DFNT_INT32, 1, &DIFFF);
	SDsetattr(sdFichier, "PROFIL", DFNT_INT32, 1, &PROFIL);
	SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, &SIM);
	SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, &SUR);
	SDsetattr(sdFichier, "THSDEG", DFNT_FLOAT32, 1, &THSDEG);
	SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, &LAMBDA);
	SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, &TAURAY);
	SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, &TAUAER);
	
	SDsetattr(sdFichier, "LSAAER", DFNT_UINT32, 1, &LSAAER);
	SDsetattr(sdFichier, "NFAER", DFNT_UINT32, 1, &NFAER);
	
	SDsetattr(sdFichier, "W0AER", DFNT_FLOAT32, 1, &W0AER);
	SDsetattr(sdFichier, "W0LAM", DFNT_FLOAT32, 1, &W0LAM);
	SDsetattr(sdFichier, "HA", DFNT_FLOAT32, 1, &HA);
	SDsetattr(sdFichier, "HR", DFNT_FLOAT32, 1, &HR);
	SDsetattr(sdFichier, "ZMIN", DFNT_FLOAT32, 1, &ZMIN);
	SDsetattr(sdFichier, "ZMAX", DFNT_FLOAT32, 1, &ZMAX);
	SDsetattr(sdFichier, "NATM", DFNT_INT32, 1, &NATM);
	SDsetattr(sdFichier, "HATM", DFNT_INT32, 1, &HATM);
	SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, &WINDSPEED);
	SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, &NH2O);
	SDsetattr(sdFichier, "CONPHY", DFNT_FLOAT32, 1, &CONPHY);
	SDsetattr(sdFichier, "PATHRESULTATSHDF", DFNT_CHAR8, strlen(PATHRESULTATSHDF), PATHRESULTATSHDF);
	SDsetattr(sdFichier, "PATHTEMOINHDF", DFNT_CHAR8, strlen(PATHTEMOINHDF), PATHTEMOINHDF);
	SDsetattr(sdFichier, "PATHDIFFAER", DFNT_CHAR8, strlen(PATHDIFFAER), PATHDIFFAER);
	SDsetattr(sdFichier, "PATHPROFILATM", DFNT_CHAR8, strlen(PATHPROFILATM), PATHPROFILATM);
	
	SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, &nbPhotonsTotdouble);
	SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, &(var->erreurpoids));
	SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, &(var->erreurtheta));
	SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, &tempsEcouledouble);
	
	/** 	Création du 1er tableau dans le fichier hdf
		Valeur de la reflectance pour phi et theta donnés		**/
	char* nomTab="Valeurs de la reflectance (I)"; //nom du tableau
	int nbDimsTab = 2; //nombre de dimensions du tableau
	int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
	valDimsTab[1] = NBTHETA;	//colonnes
	valDimsTab[0] = NBPHI;
	int typeTab = DFNT_FLOAT32; //type des éléments du tableau
	
	// Création du tableau
	int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	int startTab[nbDimsTab]; //début de la lecture du tableau
	startTab[0]=0;
	startTab[1]=0;
	// Ecriture du tableau dans le fichier
	int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tabFinal);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats reflectance\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau Q dans le fichier hdf
		Valeur de Q pour phi et theta donnés		**/
	nomTab="Valeurs de Q"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) (tabFinal+NBPHI*NBTHETA) );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats Q\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau U dans le fichier hdf
	Valeur de U pour phi et theta donnés		**/
	nomTab="Valeurs de U"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) (tabFinal+2*NBPHI*NBTHETA) );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats U\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau de lumière polarisée dans le fichier hdf
	Valeur de la lumière polarisée pour phi et theta donnés		**/
	nomTab="Valeurs de la lumiere polarisee (LP)"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	for(int i = 0; i < NBTHETA*NBPHI; i++){
		tab[i] = sqrtf( tabFinal[1*NBTHETA*NBPHI+i]*tabFinal[1*NBTHETA*NBPHI+i] +
						tabFinal[2*NBTHETA*NBPHI+i]*tabFinal[2*NBTHETA*NBPHI+i] );
		// 	printf("[%d]\tQ= %17.17f\tU= %17.17f\tLP=%17.17f\n",i,tabFinal[1*NBTHETA*NBPHI+i],tabFinal[2*NBTHETA*NBPHI+i],tab[i]);
	}
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) tab );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats lumiere polarisee\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	
	/** 	Création du tableau theta
		Valeurs de theta en fonction de l'indice	**/
	//conversion en degrès de theta pour une meilleure visualisation de la sortie
	float tabThBis[NBTHETA];
	for(int i=0; i<NBTHETA; i++)
		tabThBis[i] = tabTh[i]/DEG2RAD;
	
	nomTab = "Valeurs de theta echantillonnees";
	nbDimsTab = 1;
	int valDimsTab2[nbDimsTab];
	valDimsTab2[0] = NBTHETA;
	typeTab = DFNT_FLOAT32;
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab2);
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab2, (VOIDP) tabThBis);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats - tab Theta\n");
		exit(1);
	}
			
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau phi
		Valeurs de phi en fonction de l'indice	**/
	float tabPhiBis[NBPHI];
	for(int i=0; i<NBPHI; i++)
		tabPhiBis[i] = tabPhi[i]/DEG2RAD;
	
	nomTab = "Valeurs de phi echantillonnees";
	nbDimsTab = 1;
	int valDimsTab3[nbDimsTab];
	valDimsTab3[0] = NBPHI;
	typeTab = DFNT_FLOAT32;
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab3);
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab3, (VOIDP)tabPhiBis);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats - tab Phi\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	// Fermeture du fichier
	SDend(sdFichier);
	
}
