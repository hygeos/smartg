#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "host.h"
#include "device.h"

/**********************************************************
*
*			host.h
*
*	> Initialisation du générateur de nombres aléatoires MWC
*	> Travail sur les fichiers
*	> Initialisation des différentes structures
*	> Calculs de profils
*	> Fonctions d'affichage
*	> Calcul pour sauvegarde des résultats finaux
*	> Fichier hdf (lecture/écriture témoin, écriture résultats)
*
***********************************************************/


/**********************************************************
*	> Initialisation du générateur de nombres aléatoires MWC
***********************************************************/

/* initRandMWC
* Fonction qui initialise les generateurs du random MWC à partir d'un fichier texte
*/

int initRandMWC(unsigned long long *etat, unsigned int *config, 
	     const unsigned int n_rng, const char *safeprimes_file, unsigned long long xinit)
{
	FILE *fp;
	unsigned int begin=0u;
	unsigned int fora,tmp1,tmp2;
	if (strlen(safeprimes_file) == 0)
	{
        // Try to find it in the local directory
		safeprimes_file = "MWC.txt";
	}
	fp = fopen(safeprimes_file, "r");
	if(fp == NULL)
	{
		printf("Could not find the file of safeprimes (%s)! Terminating!\n", safeprimes_file);
		return 1;
	}
	fscanf(fp,"%u %u %u",&begin,&tmp1,&tmp2);
	// Here we set up a loop, using the first multiplier in the file to generate x's and c's
	// There are some restictions to these two numbers:
	// 0<=c<a and 0<=x<b, where a is the multiplier and b is the base (2^32)
	// also [x,c]=[0,0] and [b-1,a-1] are not allowed.
	//Make sure xinit is a valid seed (using the above mentioned restrictions)
	if((xinit == 0ull) | (((unsigned int)(xinit>>32))>=(begin-1)) | (((unsigned int)xinit)>=0xfffffffful))
	{
		//xinit (probably) not a valid seed! (we have excluded a few unlikely exceptions)
		printf("%llu not a valid seed! Terminating!\n",xinit);
		return 1;
	}
	for(unsigned int i=0;i < n_rng;i++)
	{
		fscanf(fp,"%u %u %u",&fora,&tmp1,&tmp2);
		config[i]=fora;
		etat[i]=0;
		while( (etat[i]==0) | (((unsigned int)(etat[i]>>32))>=(fora-1)) | (((unsigned int)etat[i])>=0xfffffffful))
		{
			//generate a random number
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);
			//calculate c and store in the upper 32 bits of x[i]
			etat[i]=(unsigned int) floor((((double)((unsigned int)xinit))/(double)0x100000000)*fora);//Make sure 0<=c<a
			etat[i]=etat[i]<<32;
			//generate a random number and store in the lower 32 bits of x[i] (as the initial x of the generator)
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);//x will be 0<=x<b, where b is the base 2^32
			etat[i]+=(unsigned int) xinit;
		}
		//if(i<10)printf("%llu\n",x[i]);
	}
	fclose(fp);
	return 0;
}


/* initRandMTConfig
* Fonction qui initialise en partie les generateurs du random Mersenen Twister
*/
void initRandMTConfig(ConfigMT* config_H, ConfigMT* config_D, int nbThreads)
{
	// Ouverture du fichier
	const char *fname = "auxdata/MT/MersenneTwister.dat";
	FILE* fd = fopen(fname, "rb");
	if(!fd)
	{
		printf("ERREUR: ouverture fichier MT");
		exit(0);
	}
	// Lecture et initialisation de la config pour chaque generateur (= pour chaque thread)
	for(int i = 0; i < nbThreads; i++)
	{
		/* Le fichier ne contient que 4096 configs, on reutilise donc les memes configs pour les threads en trop mais les nombres
		aléatoires restent independants car les etats des threads sont differents */
		if(i%4096 == 0)
		{
			fseek(fd, 0, 0);
		}
		if(!fread(config_H+i, sizeof(ConfigMT), 1, fd))
		{
			printf("ERREUR: lecture fichier MT");
			exit(0);
		}
	}
	fclose(fd);
	srand((unsigned int)SEED);
	// Creation des seeds aleatoires pour que les threads aient des etats differents
	for(int i = 0; i < nbThreads; i++) config_H[i].seed = (unsigned int)rand();
	
	hipError_t erreur = hipMemcpy(config_D, config_H, nbThreads * sizeof(ConfigMT), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie config_H dans initRandMTConfig\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/**********************************************************
*	> Travail sur les fichiers
***********************************************************/

/* initConstantesHost
* Fonction qui récupère les valeurs des constantes dans le fichier paramètres et initialise les constantes du host
*/
void initConstantesHost(int argc, char** argv)
{
	if(argc < 2)
	{
		printf("ERREUR : lecture argv\n");
		exit(1);
	}
	
	FILE* parametres = fopen( argv[1], "r" );
	
	if( parametres == NULL ){
		printf("ERREUR: Impossible d'ouvrir le fichier %s\n", argv[1] );
		exit(1);
	}
	
	char s[256];
    double dbl;

	strcpy(s,"");
	chercheConstante( parametres, "NBPHOTONS", s);
    dbl = atof(s);
    NBPHOTONS = (unsigned long long)dbl;
	
	strcpy(s,"");
	chercheConstante(parametres, "NBLOOP", s);
	NBLOOP = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SEED", s);
	SEED = atoi(s);
	if(SEED == -1) SEED = static_cast<int> (time(NULL));

	strcpy(s,"");
	chercheConstante(parametres, "XBLOCK", s);
	XBLOCK= atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YBLOCK", s);
	YBLOCK = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "XGRID", s);
	XGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YGRID", s);
	YGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBTHETA", s);
	NBTHETA = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBPHI", s);
	NBPHI = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SIM", s);
	SIM = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SUR", s);
	SUR = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "DIOPTRE", s);
	DIOPTRE= atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "ENV", s);
	ENV= atoi(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "THVDEG", s);
	THVDEG = atof(s);

    strcpy(s,"");
	chercheConstante(parametres, "ENV_SIZE", s);
	ENV_SIZE = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "X0", s);
	X0 = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "Y0", s);
	Y0 = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "DEPO", s);
	DEPO = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NFAER", s);
	NFAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NFOCE", s);
	NFOCE = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "WINDSPEED", s);
	WINDSPEED = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NH2O", s);
	NH2O = atof(s);
	
	chercheConstante( parametres, "PATHDIFFOCE", PATHDIFFOCE );
	chercheConstante( parametres, "PATHPROFILOCE", PATHPROFILOCE );

	strcpy(s,"");
	chercheConstante(parametres, "OUTPUT_LAYERS", s);
	OUTPUT_LAYERS = atoi(s);
	
	chercheConstante(parametres, "PATHRESULTATSHDF", PATHRESULTATSHDF);


	chercheConstante( parametres, "PATHDIFFAER", PATHDIFFAER );
	
	chercheConstante( parametres, "PATHPROFILATM", PATHPROFILATM );

	chercheConstante( parametres, "PATHALB", PATHALB );

	chercheConstante( parametres, "DEVICE", s);
    DEVICE = atoi(s);

	fclose( parametres );
}


/* chercheConstante
* Fonction qui cherche nomConstante dans le fichier et met la valeur de la constante dans chaineValeur (en string)
*/

void chercheConstante(FILE* fichier, const char* nomConstante, char* chaineValeur)
{
	int longueur = strlen(nomConstante);
	char ligne[2048];
	int motTrouve = 0;
	
	// Tant que la constante n'est pas trouvee et qu'on n'est pas à la fin du fichier on lit la ligne
	while(fgets(ligne, 2048, fichier) && !motTrouve)
	{
		// Si le debut de la ligne est nomConstante suivi d'un espace ou un egal on va chercher la valeur
		if((strncmp(ligne, nomConstante, longueur) == 0) && (ligne[longueur] == ' ' || ligne[longueur] == '='))
		{
			char* ptr = ligne; //pointeur du debut de la ligne
			// on avance jusqu'au prochain espace ou egal
			while (*ptr != ' ' && *ptr != '=') ptr++;
			// on avance jusqu'à la valeur de la constante
			while(*ptr == ' ' || *ptr == '=') ptr++;
			if (*ptr == '\n')
			{
				printf("ERROR when reading keyword %s in parameter file. Line is:\n", nomConstante);
                printf("%s\n", ligne);
				exit(1);
			}
			// On met la chaine de la valeur de la constante dans chaineValeur
			strcpy(chaineValeur, ptr);
			chaineValeur[strlen(chaineValeur)-1] = '\0';
			motTrouve = 1;
		}
	}
	rewind(fichier);
	
	if(motTrouve == 0)
	{
		printf("ERREUR : lecture fichier paramètres - Constante manquante: %s\n",nomConstante);
		exit(1);
	}
}



void init_profileATM(int *NATM, float *HATM, int *NLAM, char *PATHPROFILATM) {
    //
    // reads the number of layers NATM in the atmosphere profile, and the
    // height of the top layer
    // the profile file contains NATM+1 interfaces from 0 to NATM
    // Eventually reads the number of consecutive profiles NLAM

    printf("Read %s\n", PATHPROFILATM);

    FILE* fp;
    int c, i, read_first=1;
    float H;
    char buffer[2048];
    *HATM = -1;
    *NLAM = 1;

    fp = fopen(PATHPROFILATM, "r");

    if (fp == NULL) {
        printf("ERROR: Cannot open atmospheric profile '%s'\n", PATHPROFILATM);
        exit(1);
    }

    // skip first line
    fgets(buffer, 2048, fp);
    *NATM = -1;

    // read first layer
    while(1) {
        if (fgets(buffer, 2048, fp) == NULL) break;
        if (buffer[0] == '#') {
            *NLAM += 1;
            read_first = 0;
            continue;
        }

        c = sscanf(buffer, "%d\t%f\t", &i, &H);
        if (c != 2) break;
        if (*HATM < 0) *HATM = H;
        if (read_first) *NATM += 1;
    }

    fclose(fp);
}

void init_profileOCE(int *NOCE, int *NLAM, char *PATHPROFILOCE) {
    //
    // reads the number of layers NOCE in the ocean profile
    // the profile file contains NOCE+1 interfaces from 0 to NOCE
    // Eventually reads the number of consecutive profiles NLAM

    printf("Read %s\n", PATHPROFILOCE);

    FILE* fp;
    int c, i, read_first=1;
    float H;
    char buffer[2048];
    *NLAM = 1;

    fp = fopen(PATHPROFILOCE, "r");

    if (fp == NULL) {
        printf("ERROR: Cannot open ocean profile '%s'\n", PATHPROFILOCE);
        exit(1);
    }

    // skip first line
    fgets(buffer, 2048, fp);
    *NOCE = -1;

    // read first layer
    while(1) {
        if (fgets(buffer, 2048, fp) == NULL) break;
        if (buffer[0] == '#') {
            *NLAM += 1;
            read_first = 0;
            continue;
        }


        c = sscanf(buffer, "%d\t%f\t", &i, &H);
        if (c != 2) break;
        if (read_first) *NOCE += 1;
    }

    fclose(fp);
	}



void get_diff(char* chaineValeur, int ilam, char *PATHDIFFAER) {
    //
    // reads the number of angles in the ilamth  phase function (PF)

    FILE* fp;
    int count=0;
    char buffer[2048];

    if (strcmp(PATHDIFFAER, "None") == 0) {
        strcpy(chaineValeur, "None");
        return;
    }

    fp = fopen(PATHDIFFAER, "r");

    if (fp == NULL) {
        printf("ERROR: Cannot open phase function '%s'\n", PATHDIFFAER);
        exit(1);
    }

    while(count < ilam){
        if (fgets(buffer, 2048, fp) == NULL) break;
        count++;
    }

    fgets(buffer, 2048, fp) ;
	char* ptr = buffer; //pointeur du debut de la ligne
	while(*ptr == '\n') ptr++;
	strcpy(chaineValeur, ptr);
	chaineValeur[strlen(chaineValeur)-1] = '\0';

    fclose(fp);
}

int count_lines(char *PATHDIFF) {
    //
    // count the number of uncommented lines in a file
    //

    if (strcmp(PATHDIFF, "None") == 0) {
        return 0;
    }

    FILE *fp;
    int c = 0;
    char buffer[2048];
    fp = fopen(PATHDIFF, "r");
    if (fp == NULL) {
        printf("ERROR: cannot open file '%s'\n", PATHDIFF);
        exit(1);
    }
    while (1) {
        if (fgets(buffer, 2048, fp) == NULL) break;
        if (buffer[0] == '\n') break;
        if (buffer[0] == '#') continue;
        c++;
    }
    fclose(fp);

    return c;
}


/**********************************************************
*	> Initialisation des différentes structures
***********************************************************/

/* initVariables
* Fonction qui initialise les variables à envoyer dans le kernel.
*/
void initVariables(Variables** var_H, Variables** var_D)
{
	// 	Initialisation de la version host des variables
	*var_H = (Variables*)malloc(sizeof(Variables));
	if( var_H == NULL ){
		printf("#--------------------#\n");
		printf("ERREUR: Problème de malloc de var_H dans initVariables\n");
		printf("#--------------------#\n");
		exit(1);
	}
	memset(*var_H, 0, sizeof(Variables));
	
	//	Initialisation de la version device des variables
	if( hipMalloc(var_D, sizeof(Variables)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de var_D dans initVariables\n");
		exit(1);
	}
	
	hipError_t err = hipMemset(*(var_D), 0, sizeof(Variables));
	if( err != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de hipMemset var_D dans initVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
		printf("#--------------------#\n");
		exit(1);
	}

}


/* reinitVariables
* Fonction qui réinitialise certaines variables avant chaque envoi dans le kernel
*/
void reinitVariables(Variables* var_H, Variables* var_D)
{
	// Le nombre de photons traités pour un appel du Kernel est remis à zéro
	var_H->nbPhotons = 0;
	#ifdef PROGRESSION
	// Le nombre de photons ressortis pour un appel du Kernel est remis à zéro
	var_H->nbPhotonsSor = 0;
	#endif
	// On copie le nouveau var_H dans var_D
	hipError_t erreur = hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie var_H dans reinitVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("# sizeof(*var_D)=%d\tsizeof(*var_H)=%d\tsizeof(*Variables)=%d\n",sizeof(*var_D),sizeof(*var_H),sizeof(Variables));
		printf("# Adresse de var_D : %p\tAdresse de var_H : %p\n", var_H, var_D);
		printf("#--------------------#\n");
		exit(1);
	}
}


	/* Code spécifique à une atmosphère sphérique */
/* initInit
* Initialisation de la structure Init contenant les paramètres initiaux du photon rentrant dans l'atmosphère.
* Ces paramètres sont utiles pour une atmosphère sphérique et sont calculés une seule fois dans le host, d'où cette fonction
* et la structure Init
*/
void initInit(Init** init_H, Init** init_D)
{
	// 	Initialisation de la version host des variables
	*init_H = (Init*)malloc(sizeof(Init));
	if( init_H == NULL ){
	printf("#--------------------#\n");
	printf("ERREUR: Problème de malloc de init_H dans initInit\n");
	printf("#--------------------#\n");
	exit(1);
	}
	memset(*init_H, 0, sizeof(Init));
   
	//	Initialisation de la version device des variables
	if( hipMalloc(init_D, sizeof(Init)) == hipErrorOutOfMemory ){
	   printf("ERREUR: Problème de hipMalloc de init_D dans initInit\n");
	   exit(1);
	}

	hipError_t err = hipMemset(*(init_D), 0, sizeof(Init));
	if( err != hipSuccess ){
	   printf("#--------------------#\n");
	   printf("# ERREUR: Problème de hipMemset init_D dans initInit\n");
	   printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
	   printf("#--------------------#\n");
	   exit(1);
	}

}



/* initTableaux
* Fonction qui initialise les tableaux à envoyer dans le kernel par allocation mémoire et memset
*/
void initTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	hipError_t cudaErreur;	// Permet de tester les erreurs d'allocation mémoire
	
	#ifdef RANDMWC	
	// Création des tableaux de generateurs pour la fonction Random MWC
	tab_H->etat = (unsigned long long*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (unsigned int*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation des tableaux host à l'aide du fichier et du seed
	initRandMWC(tab_H->etat, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID, "auxdata/MWC/MWC.txt", (unsigned long long)SEED);
	
	// Copie dans les tableaux device
	cudaErreur = hipMemcpy(tab_D->etat, tab_H->etat, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->etat dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	
	cudaErreur = hipMemcpy(tab_D->config, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->config dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	#endif
	
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
	// Création du tableau de generateurs (=etat+config) pour la fonction Random Cuda
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(curandSTATE)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation du tableau dans une fonction du kernel
        #if defined(RANDCUDA)
	initRandCUDA<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, (unsigned long long)SEED);
        #else
        unsigned long long NbThreads = XGRID * YGRID * XBLOCK * YBLOCK;
        unsigned long long NbDimensions = NbThreads > 20000 ? 20000 : NbThreads;
        hiprandDirectionVectors32_t *d_qrngDirections = 0;
        cudaErreur = hipMalloc((void **)&d_qrngDirections, NbDimensions*sizeof(hiprandDirectionVectors32_t));
        if( cudaErreur != hipSuccess ){
            printf( "ERREUR: Problème d'allocation de d_qrngDirections\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
            exit(1);
        }
        hiprandDirectionVectors32_t *h_rngDirections;
        hiprandGetDirectionVectors32(&h_rngDirections, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
        cudaErreur = hipMemcpy(d_qrngDirections, h_rngDirections, NbDimensions*sizeof(hiprandDirectionVectors32_t), hipMemcpyHostToDevice);
        if( cudaErreur != hipSuccess ){
            printf( "ERREUR: Problème de copie h_rngDirections dans d_qrngDirections\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
            exit(1);
        }
        initRandCUDANDQRNGs<<< dim3(XGRID,YGRID,1), dim3(XBLOCK,YBLOCK,1)>>>(tab_D->etat, d_qrngDirections);
        #endif
	#endif
	
	#ifdef RANDMT
	// Création des tableaux de generateurs pour la fonction Random Mersenne Twister
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (ConfigMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	tab_H->etat = (EtatMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
		
	// Initialisation du tableau des configs à l'aide du fichier
	initRandMTConfig(tab_H->config, tab_D->config, XBLOCK * YBLOCK * XGRID * YGRID);
	// Initialisation du tableau des etats dans le kernel
	initRandMTEtat<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, tab_D->config);
	#endif

        #ifdef RANDPHILOX4x32_7
        //Memset de deux valeurs pour la creation des generateurs philox
        unsigned int compteur_init = 0;
        unsigned int clef_utilisateur = (unsigned int) SEED; /*LDS: eventuellement la conversion ici change la graine mais c'est sans reelle importance il me semble*/
        tab_D->config = clef_utilisateur;
        cudaErreur = hipMalloc((void**)&(tab_D->etat), sizeof(unsigned int)* XBLOCK * YBLOCK * XGRID * YGRID);
        if( cudaErreur != hipSuccess){
            printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
            printf("\t->detail de l'erreur : %s\n", hipGetErrorString(cudaErreur));
            exit(1);
        }
        initPhilox4x32_7Compteur<<<dim3(XGRID,YGRID,1), dim3(XBLOCK,YBLOCK,1)>>>(tab_D->etat, compteur_init);
        #endif
	
	// Tableau du poids des photons ressortis

	
	// Tableau du nombre des photons injectes par interval NLAM
	tab_H->nbPhotonsInter = (unsigned long long*)malloc(NLAM * sizeof(*(tab_H->nbPhotonsInter)));
	if( tab_H->nbPhotonsInter == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->nbPhotonsInter dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->nbPhotonsInter,0,NLAM * sizeof(*(tab_H->nbPhotonsInter)) );

	if( hipMalloc(&(tab_D->nbPhotonsInter), NLAM * sizeof(*(tab_D->nbPhotonsInter))) != hipSuccess){
		printf("ERREUR: Problème de hipMalloc de tab_D->nbPhotonsInter dans initTableaux\n");
		exit(1);
	}

	cudaErreur = hipMemset(tab_D->nbPhotonsInter, 0,  NLAM * sizeof(*(tab_D->nbPhotonsInter)));
	if( cudaErreur != hipSuccess ){
	printf("#--------------------#\n");
	printf("# ERREUR: Problème de hipMemset tab_D.nbPhotonsInter dans le initTableaux\n");
	printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
	printf("#--------------------#\n");
	exit(1);
	}


	// Weight Table of the descending  photons above the surface
	



		// Tableau du poids des photons ressortis
			tab_H->tabPhotons = (float*)malloc(NLVL*4*NBTHETA * NBPHI * NLAM * sizeof(*(tab_H->tabPhotons)));
			if( tab_H->tabPhotons == NULL ){
				printf("ERREUR: Problème de malloc de tab_H->tabPhoton dans initTableaux\n");
				exit(1);
			}
			memset(tab_H->tabPhotons, 0, NLVL*4*NBTHETA * NBPHI * NLAM * sizeof(*(tab_H->tabPhotons)) );

			if( hipMalloc(&(tab_D->tabPhotons), NLVL*4 * NBTHETA * NBPHI * NLAM * sizeof(*(tab_D->tabPhotons))) != hipSuccess){
				printf("ERREUR: Problème de hipMalloc de tab_D->tabPhotons dans initTableaux\n");
				exit(1);
			}

			cudaErreur = hipMemset(tab_D->tabPhotons, 0, NLVL*4*NBTHETA * NBPHI * NLAM *  sizeof(*(tab_D->tabPhotons)));
			if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le initTableaux\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
			}






	/** Modèle de diffusion **/
	// Modèle de diffusion des aérosols
	tab_H->faer = (float*)malloc(5 * NFAER *  NPHAAER * sizeof(float));
	//tab_H->faer = (float*)malloc(5 * NFAER * sizeof(float));
	if( tab_H->faer == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->faer dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->faer,0,5 * NFAER* NPHAAER * sizeof(float) );
	//memset(tab_H->faer,0,5 * NFAER*sizeof(float) );
	
	//if( hipMalloc(&(tab_D->faer), 5 * NFAER * sizeof(float)) != hipSuccess ){
	if( hipMalloc(&(tab_D->faer), 5 * NFAER * NPHAAER * sizeof(float)) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->faer dans initTableaux\n");
		exit(1);	
	}
	
	/** Modèle de l'ocean **/

	// Fonction de phase 

	tab_H->foce = (float*)malloc(5 * NFOCE * NPHAOCE *sizeof(float));

	if( tab_H->foce == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->foce dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->foce,0,5 * NFOCE*NPHAOCE *sizeof(float) );
	
	if( hipMalloc(&(tab_D->foce), 5 * NFOCE * NPHAOCE *sizeof(float)) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->foce dans initTableaux\n");
		exit(1);	
	}

	// Epaisseur optique par couche
	tab_H->ho =  (float*)malloc((NOCE+1)*NLAM*sizeof(*(tab_H->ho)));
	if( tab_H->ho == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->ho dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->ho,0,(NOCE+1)*NLAM*sizeof(*(tab_H->ho)) );

	if( hipMalloc( &(tab_D->ho), (NOCE+1)*NLAM*sizeof(*(tab_H->ho)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->ho dans initTableaux\n");
		exit(1);
	}

    // SSA
	tab_H->sso =  (float*)malloc((NOCE+1)*NLAM*sizeof(float));
	if( tab_H->sso == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->sso dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->sso,0,(NOCE+1)*NLAM*sizeof(float) );

	if( hipMalloc( &(tab_D->sso), (NOCE+1)*NLAM*sizeof(float) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->sso dans initTableaux\n");
		exit(1);
	}

	//garbage
	tab_H->depth=  (float*)malloc((NOCE+1)*sizeof(float));
	if( tab_H->depth == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->sso dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->depth,0,(NOCE+1)*sizeof(float) );


	
	/** Modèle de l'atmosphère **/
	// Epaisseur optique par couche
	tab_H->h =  (float*)malloc((NATM+1)*NLAM*sizeof(*(tab_H->h)));
	if( tab_H->h == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->h dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->h,0,(NATM+1)*NLAM*sizeof(*(tab_H->h)) );
	
	if( hipMalloc( &(tab_D->h), (NATM+1)*NLAM*sizeof(*(tab_H->h)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->h dans initTableaux\n");
		exit(1);	
	}
	
	// Proportion moléculaire par couche
	tab_H->pMol =  (float*)malloc((NATM+1)*NLAM*sizeof(float));
	if( tab_H->pMol == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->pMol dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->pMol,0,(NATM+1)*NLAM*sizeof(float) );
	
	if( hipMalloc( &(tab_D->pMol), (NATM+1)*NLAM*sizeof(float) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->pMol dans initTableaux\n");
		exit(1);	
	}

    //
	tab_H->abs =  (float*)malloc((NATM+1)*NLAM*sizeof(float));
	if( tab_H->abs == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->abs dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->abs,0,(NATM+1)*NLAM*sizeof(float) );
	
	if( hipMalloc( &(tab_D->abs), (NATM+1)*NLAM*sizeof(float) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->abs dans initTableaux\n");
		exit(1);	
	}
	
    //
	tab_H->ssa =  (float*)malloc((NATM+1)*NLAM*sizeof(float));
	if( tab_H->ssa == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->ssa dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->ssa,0,(NATM+1)*NLAM*sizeof(float) );
	
	if( hipMalloc( &(tab_D->ssa), (NATM+1)*NLAM*sizeof(float) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->ssa dans initTableaux\n");
		exit(1);	
	}
	

	tab_H->ip =  (int*)malloc((NATM+1)*NLAM*sizeof(int));
	if( tab_H->ip == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->ip dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->ip,0,(NATM+1)*NLAM*sizeof(int) );
	
	if( hipMalloc( &(tab_D->ip), (NATM+1)*NLAM*sizeof(int) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->ip dans initTableaux\n");
		exit(1);	
	}
	
    //
	tab_H->ipo =  (int*)malloc((NOCE+1)*NLAM*sizeof(int));
	if( tab_H->ipo == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->ipo dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->ipo,0,(NOCE+1)*NLAM*sizeof(int) );
	
	if( hipMalloc( &(tab_D->ipo), (NOCE+1)*NLAM*sizeof(int) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->ipo dans initTableaux\n");
		exit(1);	
	}
	
	

	
	//variable garbage

	//hmol
    tab_H->hmol =  (float*)malloc((NATM+1)*NLAM*sizeof(float));
    	if( tab_H->hmol == NULL ){
    		printf("ERREUR: Problème de malloc de tab_H->hmol dans initTableaux\n");
    		exit(1);
    	}
    memset(tab_H->hmol,0,(NATM+1)*NLAM*sizeof(float) );

    //haer
    tab_H->haer=  (float*)malloc((NATM+1)*NLAM*sizeof(float));
    	if( tab_H->haer == NULL ){
    		printf("ERREUR: Problème de malloc de tab_H->haer dans initTableaux\n");
    		exit(1);
    	}
    memset(tab_H->haer,0,(NATM+1)*NLAM*sizeof(float) );

    //xdel
    tab_H->xdel =  (float*)malloc((NATM+1)*NLAM*sizeof(float));
    	if( tab_H->xdel == NULL ){
    		printf("ERREUR: Problème de malloc de tab_H->xdel dans initTableaux\n");
    		exit(1);
    	}
    memset(tab_H->xdel,0,(NATM+1)*NLAM*sizeof(float) );


    //lambda
    tab_H->lambda =  (float*)malloc(NLAM*sizeof(float));
    if( tab_H->lambda == NULL ){
          printf("ERREUR: Problème de malloc de tab_H->lambda dans initTableaux\n");
          exit(1);
    }

        memset(tab_H->lambda,0,NLAM*sizeof(float));

        if( hipMalloc( &(tab_D->lambda), NLAM*sizeof(float) ) != hipSuccess ){
           printf("ERREUR: Problème de hipMalloc de tab_D->lambda dans initTableaux\n");
           exit(1);
        }

	//test

	// Altitude des couches

		tab_H->z =  (float*)malloc((NATM+1)*sizeof(*(tab_H->z)));
				if( tab_H->z == NULL ){
					printf("ERREUR: Problème de malloc de tab_H->z dans initTableaux\n");
					exit(1);
				}
		memset(tab_H->z,0,(NATM+1)*sizeof(*(tab_H->z)) );

		if( hipMalloc( &(tab_D->z), (NATM+1)*sizeof(*(tab_H->z)) ) != hipSuccess ){
					printf("ERREUR: Problème de hipMalloc de tab_D->z dans initTableaux\n");
					exit(1);
				}


	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */







	/** Profil initial vu par le photon **/
	tab_H->zph0 =  (float*)malloc((NATM+1)*sizeof(*(tab_H->zph0)));
	if( tab_H->zph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->zph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->zph0,0,(NATM+1)*sizeof(*(tab_H->zph0)) );
	
	if( hipMalloc( &(tab_D->zph0), (NATM+1)*sizeof(*(tab_D->zph0)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->zph0 dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->hph0 =  (float*)malloc((NATM+1)*NLAM*sizeof(*(tab_H->hph0)));
	if( tab_H->hph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->hph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->hph0,0,(NATM+1)*NLAM*sizeof(*(tab_H->hph0)) );
	
	if( hipMalloc( &(tab_D->hph0), (NATM+1)*NLAM*sizeof(*(tab_D->hph0)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->hph0 dans initTableaux\n");
		exit(1);	
	}
	#endif

    // Spectral albedo
	tab_H->alb =  (float*)malloc(2*NLAM*sizeof(*(tab_H->alb)));
	if( tab_H->alb == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->alb dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->alb,0,2*NLAM*sizeof(*(tab_H->alb)) );
	
	if( hipMalloc( &(tab_D->alb), 2*NLAM*sizeof(*(tab_D->alb)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->alb dans initTableaux\n");
		exit(1);
	}


}


/* freeTableaux
* Fonction qui libère l'espace mémoire de tous les tableaux alloués
*/
void freeTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	
	hipError_t erreur;	// Permet de tester le bon déroulement des hipFree
	
	#ifdef RANDMWC
	// Liberation des tableaux de generateurs du random MWC
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->etat);
	
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	#endif
	
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
	// Liberation du tableau de generateurs du random Cuda
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif
	
	#ifdef RANDMT
	// Liberation des tableaux de generateurs du random Mersenen Twister
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	free(tab_H->etat);
	#endif
	
        #ifdef RANDPHILOX4x32_7
	// Liberation du tableaux de compteurs des philox
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif

	// Liberation du tableau du poids des photons
	

	erreur = hipFree(tab_D->nbPhotonsInter);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->nbPhotonsInter dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	// 	hipHostFree(tab_H->nbPhotonsInter);
	free(tab_H->nbPhotonsInter);

	/** Modèles de diffusion **/
	// Libération du modèle de diffusion des aérosols
	erreur = hipFree(tab_D->faer);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->faer dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	free(tab_H->faer);
	
	// Libération du modèle ocean
	// Diffusion dans l'océan
	erreur = hipFree(tab_D->foce);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->foce dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	free(tab_H->foce);

	erreur = hipFree(tab_D->ho);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->ho dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}

	free(tab_H->ho);

	erreur = hipFree(tab_D->sso);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->sso dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->sso);
	
	/** Profil amosphèrique **/	
	// Libération du modèle atmosphérique
	erreur = hipFree(tab_D->h);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->h dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->h);
	
	//
	erreur = hipFree(tab_D->pMol);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->pMol dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->pMol);
	
	//
	erreur = hipFree(tab_D->abs);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->abs dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->abs);
	
	//
	erreur = hipFree(tab_D->ssa);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->ssa dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->ssa);
	

	//
	erreur = hipFree(tab_D->ip);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->ip dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->ip);
	
	//
	erreur = hipFree(tab_D->ipo);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->ipo dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->ipo);



	erreur = hipFree(tab_D->lambda);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de hipFree de tab_D->lambda dans freeTableaux\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}

	free(tab_H->lambda);



	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	/* Code spécifique à une atmosphère sphérique */
	


	// Altitude des couches
	erreur = hipFree(tab_D->z);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->z dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}

	free(tab_H->z);


    //garbage
	free(tab_H->hmol);
	free(tab_H->haer);
	free(tab_H->xdel);
	free(tab_H->depth);



	//fusion des tableaux
	erreur = hipFree(tab_D->tabPhotons);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->tabPhotons dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}

	free(tab_H->tabPhotons);
	//fusion des tableaux





	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */

	//test



	// Profil initial vu par la photon
	erreur = hipFree(tab_D->zph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->zph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->zph0);
	
	erreur = hipFree(tab_D->hph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->hph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->hph0);
	
	#endif
	
	//
	erreur = hipFree(tab_D->alb);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->alb dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}

	free(tab_H->alb);




}


/**********************************************************
*	> Calculs de profils
***********************************************************/

/* calculF
* Compute CDF of scattering phase matrices
*/

void calculF( const char* nomFichier, float* phase_H, float* phase_D ,int mlsa,int lsa, int nf,int ip, double *phase){


    // ip is the number of the phase function
    // not necessary when file name is "None"
    if (strcmp(nomFichier, "None") == 0) {
        return;
    }


	double *scum = (double*) malloc(lsa*sizeof(*scum));
	if( scum==NULL ){
		printf("ERREUR: Problème de malloc de scum dans calculF\n");
		exit(1);
	}
	
	scum[0] = 0;
	int iang = 0, ipf = 0;
	double dtheta, pm1, pm2, sin1, sin2;
	double z;
    char buffer[1024];
    char *ptr;

	/** Allocation de la mémoire des tableaux contenant les données **/


    double *ang;
	double *p1, *p2, *p3, *p4;



	ang =&phase[ip*mlsa*5+mlsa*0];
	p2 = &phase[ip*mlsa*5+mlsa*1];
	p1 = &phase[ip*mlsa*5+mlsa*2];
	p3 = &phase[ip*mlsa*5+mlsa*3];
	p4 = &phase[ip*mlsa*5+mlsa*4];



	
	/** Lecture des données sur le modèle de diffusion des aérosols **/
	FILE* fichier = fopen(nomFichier, "r");
	if(fichier == NULL){
		printf("ERREUR : Ouverture impossible du fichier %s pour la diffusion d'aérosol", nomFichier );
		exit(1);
	}
	
	else{
        printf("reading %s\n",nomFichier);
        char c = getc(fichier);
        while(c=='#') {
            while((c=getc(fichier))!='\n');
            c = getc(fichier);
        }
        fseek(fichier, -1, SEEK_CUR);
		for(iang=0; iang<mlsa; iang++){
            fgets(buffer, 1024, fichier);

            // replace all occurences of 'D' by 'E'
            // (compatibility with fortran output)
            ptr = buffer;
            do {
                ptr = strchr(ptr, 'D');
                if (ptr != NULL) {
                    *ptr = 'E';
                }
            } while(ptr != NULL);
            
            if(iang<lsa){
					ptr = buffer;
					while ((*ptr == ' ') || (*ptr == '\t')) ptr++;
					*(ang+iang) = atof(ptr);
					while ((*ptr != ' ') && (*ptr != '\t')) ptr++;
					while ((*ptr == ' ') || (*ptr == '\t')) ptr++;
					*(p2+iang) = atof(ptr);
					while ((*ptr != ' ') && (*ptr != '\t')) ptr++;
					while ((*ptr == ' ') || (*ptr == '\t')) ptr++;
					*(p1+iang) = atof(ptr);
					while ((*ptr != ' ') && (*ptr != '\t')) ptr++;
					while ((*ptr == ' ') || (*ptr == '\t')) ptr++;
					*(p3+iang) = atof(ptr);
					while ((*ptr != ' ') && (*ptr != '\t')) ptr++;
					while ((*ptr == ' ') || (*ptr == '\t')) ptr++;
					*(p4+iang) = atof(ptr);



					// Conversion en radians
					ang[iang] = ang[iang]*DEG2RAD;
            		}
            else{
					*(ang+iang)=  0.0/0.0;
					*(p2+iang) =  0.0/0.0;
					*(p1+iang) =  0.0/0.0;
					*(p3+iang) =  0.0/0.0;
					*(p4+iang) =  0.0/0.0;
            }

		}
	}
	


	if(fclose(fichier) == EOF){
		printf("ERREUR : Probleme de fermeture du fichier %s", nomFichier);
	}
		
	/* Calcul de scum */
	for(iang=1; iang<lsa; iang++){
		
		dtheta = (ang[iang] - ang[iang-1]);
		pm1= p1[iang-1] + p2[iang-1];
		pm2= p1[iang] + p2[iang];
		sin1= sin(ang[iang-1]);
		sin2= sin(ang[iang]);
		
		scum[iang] = scum[iang-1] + dtheta*( (sin1*pm1+sin2*pm2)/3 + (sin1*pm2+sin2*pm1)/6 )*DEUXPI; 
	}
	
	// Normalisation
	for(iang=0; iang<lsa; iang++){
		scum[iang] = scum[iang]/scum[lsa-1];
	}
	
	/* Calcul des faer */
//	for(iang=0; iang<nf-1; iang++){
	for(iang=0; iang<nf; iang++){
		z = double(iang+1)/double(nf);
		while( (scum[ipf+1]<z) )
			ipf++;
		
			phase_H[ip*5*nf+iang*5+4] = float( ((scum[ipf+1]-z)*ang[ipf] + (z-scum[ipf])*ang[ipf+1])/(scum[ipf+1]-scum[ipf]) );
//		norm = p1[ipf]+p2[ipf];			// Angle
//		phase_H[iang*5+0] = float( p1[ipf]/norm );	// I paralèlle
//		phase_H[iang*5+1] = float( p2[ipf]/norm );	// I perpendiculaire
//		phase_H[iang*5+2] = float( p3[ipf]/norm );	// u
//		phase_H[iang*5+3] = 0.F;			// v, toujours nul

			phase_H[ip*5*nf+iang*5+0] = float( p1[ipf] );	// I paralèlle
			phase_H[ip*5*nf+iang*5+1] = float( p2[ipf] );	// I perpendiculaire
			phase_H[ip*5*nf+iang*5+2] = float( p3[ipf] );	// u
			phase_H[ip*5*nf+iang*5+3] = 0.F;	       	// v, toujours nul

	}
	
	free(scum);
	//free(ang);
	//free(p1);
	//free(p2);
	//free(p3);
	//free(p4);

	for(iang=0; iang<lsa; iang++){
		ang[iang] = ang[iang]/DEG2RAD;

	}


	
}

/* Read spectral albedo (for surface ,seafloor and sea reflectance)*/
void profilAlb( Tableaux* tab_H, Tableaux* tab_D ){
    int ilam;
    // Profil utilisateur
    /* Format du fichier
    => alb_surface alb_sea(floor)
    */
    FILE* profil = fopen( PATHALB , "r" );
    char ligne[1024];
    int n, err;
	hipError_t erreur;		// Permet de tester le bon déroulement des opérations mémoires

    if(profil == NULL){
        printf("ERREUR : Ouverture impossible du fichier %s pour le profil albedo\n", PATHALB );
        exit(1);
    }

    else {
        // skip header
        fgets(ligne,1024,profil);

        for( ilam=0; ilam<NLAM; ilam++){
            err = 0;

            // read 1 line
            if (fgets(ligne,1024,profil) == NULL) {
                err += 1;
            }
            n = sscanf(ligne, "%f %f\n",
                    tab_H->alb+0+ilam*2,tab_H->alb+1+ilam*2);

            if (n != 2) { err += 1; }

            if (err) {
                printf("Error, failed reading %d bands in albedo file '%s'\n", NLAM, PATHALB);
                exit(1);
            }
        }
    }

	if(fclose(profil) == EOF){
		printf("ERREUR : Probleme de fermeture du fichier %s", PATHALB);
	}

	erreur = hipMemcpy(tab_D->alb, tab_H->alb, 2*NLAM*sizeof(*(tab_H->alb)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->alb dans profilAlb\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/* Read ocean extinction coefficient and single scattering albedo for ocean*/
void profilOce( Tableaux* tab_H, Tableaux* tab_D){
    int ilam;
    int nscanf;
    int icouche=0;
    float garbage;
    char *ptr;

    // Profil utilisateur
    /* Format du fichier
    => n	alt		ho		sso
    */
    FILE* profil = fopen( PATHPROFILOCE , "r" );
    char buffer[4096];
	hipError_t erreur;		// Permet de tester le bon déroulement des opérations mémoires

	for( ilam=0;ilam<NLAM;ilam++) {
        tab_H->ho[0 + ilam * (NOCE+1)] = 0.;
        tab_H->sso[0 + ilam * (NOCE+1)] = 1.;
    }

    if(profil == NULL){
        printf("ERREUR : Ouverture impossible du fichier %s pour le profil oceanique\n", PATHPROFILOCE );
        exit(1);
    } else {
        for( ilam=0; ilam<NLAM; ilam++){

            // skip comment line
            fgets(buffer,4096,profil);

            // read wavelength
            ptr = strstr(buffer, "LAM=");
            if ((ptr != NULL) && (abs(tab_H->lambda[ilam]) < 1e-5)) {
                ptr += 4;
                while (*ptr == ' ') ptr++;

                tab_H->lambda[ilam]=atof(ptr);
            }

            for( icouche=0; icouche<NOCE+1; icouche++ ){
                fgets(buffer,4096,profil);
                nscanf = sscanf(buffer, "%d\t%f\t%f\t%f\t%d\n", &garbage, tab_H->depth+icouche, tab_H->ho+icouche+ilam*(NOCE+1), tab_H->sso+icouche+ilam*(NOCE+1), tab_H->ipo+icouche+ilam*(NOCE+1));
                if (nscanf != 5) {
                    printf("Error while parsing profile '%s'\n", PATHDIFFOCE);
                    exit(1);
                }
            }

        }
    }


	if(fclose(profil) == EOF){
		printf("ERREUR : Probleme de fermeture du fichier %s", PATHPROFILOCE);
	}

	erreur = hipMemcpy(tab_D->ho, tab_H->ho, (NOCE+1)*NLAM*sizeof(*(tab_H->ho)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->ho dans profilOce\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	erreur = hipMemcpy(tab_D->sso, tab_H->sso, (NOCE+1)*NLAM*sizeof(*(tab_H->sso)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->sso dans profilOce\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
    erreur = hipMemcpy(tab_D->ipo, tab_H->ipo, (NOCE+1)*NLAM*sizeof(*(tab_H->ipo)), hipMemcpyHostToDevice);
    if( erreur != hipSuccess ){
        printf( "ERREUR: Problème de copie tab_D->ipo dans profilOce\n");
        printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
        exit(1);
    }
    erreur = hipMemcpy(tab_D->lambda, tab_H->lambda, NLAM*sizeof(*(tab_H->lambda)), hipMemcpyHostToDevice);
        if( erreur != hipSuccess ){
            printf( "ERREUR: Problème de copie tab_D->lambda dans profilOce\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
            exit(1);
        }

}


/* profilAtm
* Calcul du profil atmosphérique dans l'atmosphère en fonction de la couche
* Mélange Molécule/Aérosol dans l'atmosphère en fonction de la couche
*/
void profilAtm( Tableaux* tab_H, Tableaux* tab_D){

	/** Déclaration des variables **/

	
	int  ilam, nscanf;
	hipError_t erreur;		// Permet de tester le bon déroulement des opérations mémoires
    char buffer[4096];
    char *ptr;
	

	/** Conditions aux limites au sommet de l'atmosphère **/

	#ifdef SPHERIQUE
    tab_H->z[0] = HATM;
	#endif
	for( ilam=0;ilam<NLAM;ilam++) {
        tab_H->h[0 + ilam * (NATM+1)] = 0.;
        tab_H->pMol[0 + ilam * (NATM+1)] = 0.;
      }

    // Profil utilisateur
    /* Format du fichier
    => n	alt		tauMol		tauAer		h		pAer		pMol
    */
    FILE* profil = fopen( PATHPROFILATM , "r" );
    float garbage/*,hmol,haer,xdel*/;


    int icouche=0;


    if(profil == NULL){
        printf("ERREUR : Ouverture impossible du fichier %s pour le profil atmosphérique\n", PATHPROFILATM );
        exit(1);
    }


    else{

        for( ilam=0; ilam<NLAM; ilam++){

            // skip comment line
            fgets(buffer,4096,profil);

            // read wavelength
            ptr = strstr(buffer, "LAM=");
            if ((ptr != NULL) && (abs(tab_H->lambda[ilam]) < 1e-5)) {
                ptr += 4;
                while (*ptr == ' ') ptr++;

                tab_H->lambda[ilam]=atof(ptr);
            }


            for( icouche=0; icouche<NATM+1; icouche++ ){
                fgets(buffer, 4096, profil);
                nscanf = sscanf(buffer, "%d\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%f\t%d\n",
                        &garbage, tab_H->z+icouche, tab_H->hmol+icouche+ilam*(NATM+1), tab_H->haer+icouche+ilam*(NATM+1), tab_H->h+icouche+ilam*(NATM+1),
                        tab_H->xdel+icouche+ilam*(NATM+1),tab_H->pMol+icouche+ilam*(NATM+1), tab_H->ssa+icouche+ilam*(NATM+1), tab_H->abs+icouche+ilam*(NATM+1), tab_H->ip+icouche+ilam*(NATM+1));


                if (nscanf != 10) {
                    printf("Error while parsing profile '%s'\n", PATHPROFILATM);
                    exit(1);
                }
            }
        }





        #if !defined(SPHERIQUE)
         TAUATM = tab_H->h[NATM];
        #endif

        }




		if(fclose(profil) == EOF){
			printf("ERREUR : Probleme de fermeture du fichier %s", PATHPROFILATM);
		}



		/** Envoie des informations dans le device **/
		erreur = hipMemcpy(tab_D->h, tab_H->h, (NATM+1)*NLAM*sizeof(*(tab_H->h)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->h dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}

		erreur = hipMemcpy(tab_D->pMol, tab_H->pMol, (NATM+1)*NLAM*sizeof(*(tab_H->pMol)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->pMol dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}

        erreur = hipMemcpy(tab_D->abs, tab_H->abs, (NATM+1)*NLAM*sizeof(*(tab_H->abs)), hipMemcpyHostToDevice);
        if( erreur != hipSuccess ){
            printf( "ERREUR: Problème de copie tab_D->abs dans profilAtm\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
            exit(1);
        }

        erreur = hipMemcpy(tab_D->ssa, tab_H->ssa, (NATM+1)*NLAM*sizeof(*(tab_H->ssa)), hipMemcpyHostToDevice);
        if( erreur != hipSuccess ){
            printf( "ERREUR: Problème de copie tab_D->ssa dans profilAtm\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
            exit(1);
        }


        erreur = hipMemcpy(tab_D->ip, tab_H->ip, (NATM+1)*NLAM*sizeof(*(tab_H->ip)), hipMemcpyHostToDevice);
        if( erreur != hipSuccess ){
            printf( "ERREUR: Problème de copie tab_D->ip dans profilAtm\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
            exit(1);
        }


		erreur = hipMemcpy(tab_D->z, tab_H->z, (NATM+1)*sizeof(*(tab_H->z)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->z dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}


		erreur = hipMemcpy(tab_D->lambda, tab_H->lambda, NLAM*sizeof(*(tab_H->lambda)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->lambda dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
}


/** Séparation du code pour atmosphère sphérique ou parallèle **/
	/* Code spécifique à une atmosphère sphérique */

/* impactInit
* Calcul du profil que le photon va rencontrer lors de son premier passage dans l'atmosphère
* Sauvegarde de ce profil dans tab et sauvegarde des coordonnées initiales du photon dans init
*/
void impactInit(Init* init_H, Init* init_D, Tableaux* tab_H, Tableaux* tab_D){
	
	double thv, localh;
	double rdelta;
	double xphbis,yphbis,zphbis;	//Coordonnées intermédiaire du photon
	double rsolfi,rsol1,rsol2;
	
	// Correspond aux paramètres initiaux du photon
	double vx = -sin(THVDEG*DEG2RAD);
	double vy = 0.;
	double vz = -cos(THVDEG*DEG2RAD);
	
	#ifdef SPHERIQUE
    int ilam;
	#endif

	/** Calcul du point d'impact **/
	thv = THVDEG*DEG2RAD;
	
	rdelta = 4.*RTER*RTER + 4.*( tan(thv)*tan(thv)+1. )*( HATM*HATM + 2.*HATM*RTER );
	localh = ( -2.*RTER+sqrt(rdelta) )/( 2.*(tan(thv)*tan(thv)+1.) );
	
	init_H->x0 = (float) localh*tan(thv);
	init_H->y0 = 0.f;
	
	init_H->z0 =
			#ifdef SPHERIQUE
			(float) RTER
			#endif
			+ localh;

	#ifdef SPHERIQUE
	tab_H->zph0[0] = 0.;
	for(ilam=0; ilam<NLAM; ilam++){
	   tab_H->hph0[0 + ilam*(NATM+1)] = 0.;
    }
	#endif
	
	xphbis = init_H->x0;
	yphbis = init_H->y0;
	zphbis = init_H->z0;
	
	/** Création hphoton et zphoton, chemin optique entre sommet atmosphère et sol pour la direction d'incidence **/
	for(int icouche=1; icouche<NATM+1; icouche++){
		
		rdelta = 4.*(vx*xphbis + vy*yphbis + vz*zphbis)*(vx*xphbis + vy*yphbis + vz*zphbis)
			- 4.*(xphbis*xphbis + yphbis*yphbis + zphbis*zphbis - ((double)tab_H->z[icouche]+RTER)*((double)tab_H->z[icouche]+RTER));
		rsol1 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) + sqrt(rdelta) );
		rsol2 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) - sqrt(rdelta) );
		
		// solution : la plus petite distance positive
		if(rsol1>0.){
			if( rsol2>0.)
				rsolfi = min(rsol1,rsol2);
			else
				rsolfi = rsol1;
		}
		else{
			if( rsol2>0. )
				rsolfi=rsol2;
		}

		#ifdef SPHERIQUE
		tab_H->zph0[icouche] = tab_H->zph0[icouche-1] + (float)rsolfi;
	    for(ilam=0; ilam<NLAM; ilam++){
		    tab_H->hph0[icouche + ilam*(NATM+1)] = tab_H->hph0[icouche-1+ ilam*(NATM+1)] +
				( abs( tab_H->h[icouche+ ilam*(NATM+1)] - tab_H->h[icouche-1+ ilam*(NATM+1)])*rsolfi )/( abs( tab_H->z[icouche-1] - tab_H->z[icouche]) );
        }
		#endif
		
		xphbis+= vx*rsolfi;
		yphbis+= vy*rsolfi;
		zphbis+= vz*rsolfi;
		
	}

	//for(ilam=0; ilam<NLAM; ilam++){
	//    init_H->taumax0[ilam] = tab_H->hph0[NATM + ilam*(NATM+1)];
    //}
	//init_H->zintermax0 = tab_H->zph0[NATM];

	
	/** Envoie des données dans le device **/
	hipError_t erreur = hipMemcpy(init_D, init_H, sizeof(Init), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie init_H dans initInit\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	#ifdef SPHERIQUE
	erreur = hipMemcpy(tab_D->hph0, tab_H->hph0, (NATM+1)*NLAM*sizeof(*(tab_H->hph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->hph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	

	erreur = hipMemcpy(tab_D->zph0, tab_H->zph0, (NATM+1)*sizeof(*(tab_H->zph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->zph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif
}



/**********************************************************
*	> Fonctions d'affichage
***********************************************************/

/* afficheParametres
* Affiche les paramètres de la simulation
*/
void afficheParametres()
{
	printf("\n#--------- Paramètres de simulation --------#\n");
	printf(" NBPHOTONS =\t%llu", NBPHOTONS);
	printf("\n");
	printf(" NBTHETA =\t%d", NBTHETA);
	printf("\n");
	printf(" NBPHI\t=\t%d", NBPHI);
	printf("\n");
	printf(" THVDEG\t=\t%f (degrés)", THVDEG);
	printf("\n");
	printf(" NLAM\t=\t%d", NLAM);
	printf("\n");
	printf(" SIM\t=\t%d", SIM);
		if( SIM==-2 )
			printf("\t(Atmosphère seule)");
		if( SIM==-1 )
			printf("\t(Dioptre seul)");
		if( SIM==0 )
			printf("\t(Océan et Surface)");
		if( SIM==1 )
			printf("\t(Atmosphère et Surface)");
		if( SIM==2 )
			printf("\t(Atmosphère, Dioptre et Océan)");
		if( SIM==3 )
			printf("\t(Océan seul)");
		
	printf("\n");
	printf(" SEED\t=\t%d", SEED);
	printf("\n");
	
	printf("\n#------- Paramètres de performances --------#\n");
	printf(" NBLOOP\t=\t%u", NBLOOP);
	printf("\n");
	printf(" XBLOCK\t=\t%d", XBLOCK);
	printf("\n");
	printf(" YBLOCK\t=\t%d", YBLOCK);
	printf("\n");
	printf(" XGRID\t=\t%d", XGRID);
	printf("\n");
	printf(" YGRID\t=\t%d", YGRID);
	printf("\n");
	
	
	printf("\n#--------------- Atmosphère ----------------#\n");
	if( SIM==-2 || SIM==1 || SIM==2 ){
		#ifdef SPHERIQUE
		printf(" Géométrie de l'atmosphère: \tSphérique");
		printf("\n");
		#endif
		#ifndef SPHERIQUE
		printf(" Géométrie de l'atmosphère: \tParallèle");
		printf("\n");
		#endif
		
		printf(" LSAAER\t=\t%u", LSAAER);
		printf("\n");
		printf(" NFAER\t=\t%u", NFAER);
		printf("\n");
		printf(" NATM\t=\t%d", NATM);
		printf("\n");
		printf(" HATM\t=\t%f", HATM);
		printf("\n");
	}
	else{
		printf("\tPas de contribution de l'atmosphère\n");
	}
	
	
	printf("\n#--------- Contribution du dioptre ---------#\n");
	if( SIM==-1 || SIM==0 || SIM==1 || SIM==2 ){
		printf(" SUR\t=\t%d", SUR);
		printf("\n");
		printf(" DIOPTRE =\t%d", DIOPTRE);
		printf("\n");
		printf(" WINDSPEED =\t%f", WINDSPEED);
		printf("\n");
	}
	else{
		printf("\tPas de dioptre\n");
	}
   
    printf("\n#--------- Contribution de l'environnement -----#\n");
	if( ENV!=0){
		printf(" ENV_SIZE\t=\t%.1f (km)", ENV_SIZE);
		printf("\n");
		printf(" X0 =\t%.1f (km)", X0);
		printf(" Y0 =\t%.1f (km)", Y0);
		printf("\n");
	}
	else{
		printf("\tPas d'effet d'environnement\n");
	}

	printf("\n#----------------- Océan ------------------#\n");
	printf(" LSAOCE\t=\t%u", LSAOCE);
	printf("\n");
	printf(" NFOCE\t=\t%u", NFOCE);
	printf("\n");
	printf(" NH2O\t=\t%f", NH2O);
	printf("\n");
	printf(" NOCE\t=\t%d", NOCE);
	printf("\n");
	
	printf("\n#----------- Chemin des fichiers -----------#\n");
	printf(" PATHRESULTATSHDF = %s", PATHRESULTATSHDF);
	printf("\n");
	printf(" PATHDIFFAER = %s", PATHDIFFAER);
	printf("\n");
	printf(" PATHPROFILATM = %s", PATHPROFILATM);
	printf("\n");
	printf(" PATHALB = %s", PATHALB);
	printf("\n");
    printf(" PATHDIFFOCE = %s\n", PATHDIFFOCE);
	printf(" PATHPROFILOCE = %s", PATHPROFILOCE);
	printf("\n");
	
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	printf("\n  Date de début  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year,
		   date->tm_hour, date->tm_min, date->tm_sec);

}


/* afficheProgress
* Affiche la progression de la simulation
*/
void afficheProgress(unsigned long long nbPhotonsTot, Variables* var, double tempsPrec
#ifdef PROGRESSION
, unsigned long long nbPhotonsSorTot
#endif
)
{
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	// Calcul du temps ecoule et restant
	double tempsProg = (double)(clock() / CLOCKS_PER_SEC);
	double tempsTot = tempsProg + tempsPrec;
	int tempsEcoule = (int)tempsTot;
	int hEcoulees = tempsEcoule / 3600;
	int minEcoulees = (tempsEcoule%3600) / 60;
	int secEcoulees = tempsEcoule%60;
	int tempsRestant = (int)(tempsTot * ((double)NBPHOTONS / (double)nbPhotonsTot - 1.));
	if(tempsRestant < 0) tempsRestant = 0;
	int hRestantes = tempsRestant / 3600;
	int minRestantes = (tempsRestant%3600) / 60;
	int secRestantes = tempsRestant%60;
	// Calcul du pourcentage de photons traités
	int pourcent = (int)(100 * nbPhotonsTot / NBPHOTONS);
	
	// Affichage
	printf("\n");
	printf(" --------------------------------------\n");
	printf("  Photons lances : %12lu (%3d%%)\n", nbPhotonsTot, pourcent);
	printf("  Photons pb     : %12d\n", var->erreurpoids + var->erreurtheta);
	printf("  Temps ecoule   : %d h %2d min %2d sec\n", hEcoulees, minEcoulees, secEcoulees);
	printf("  Temps restant  : %d h %2d min %2d sec\n", hRestantes, minRestantes, secRestantes);
	printf("  Date actuelle  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year, date->tm_hour,
		   date->tm_min, date->tm_sec);
	printf(" --------------------------------------\n");
   
	#ifdef PROGRESSION
	printf("%d%% - ", (int)(100*nbPhotonsTot/NBPHOTONS));
	printf("Temps: %d - ", tempsEcoule);
	printf("phot sortis: %lu - ", nbPhotonsSorTot);
	printf("phot traités: %lu - ", nbPhotonsTot);
	printf("erreur poids/theta/vxy/vy/case: %d/%d/%d/%d/%d", var->erreurpoids, var->erreurtheta, var->erreurvxy,
		var->erreurvy, var->erreurcase);
	printf("\n");
	#endif
}




/**********************************************************
*	> Calcul pour sauvegarde des résultats finaux
***********************************************************/

/* calculOmega
* Fonction qui calcule l'aire normalisée de chaque boite, son theta, et son psi, sous forme de 3 tableaux
*/
void calculOmega(double* tabTh, double* tabPhi, double* tabOmega)
{
    // Zenith angles of the center of the output angular boxes
	memset(tabTh, 0, NBTHETA * sizeof(*tabPhi));
	double dth = DEMIPI / NBTHETA;
	tabTh[0] = dth / 2.;
	for(int ith = 1; ith < NBTHETA; ith++){
	//tabTh[0] = dth/4;
	//tabTh[1] = dth;
	//for(int ith = 2; ith < NBTHETA; ith++){
		tabTh[ith] = tabTh[ith-1] + dth;
	}
	
    // Azimut angles of the center of the output angular boxes
	memset(tabPhi, 0, NBPHI * sizeof(*tabPhi));
	double dphi = PI / NBPHI;
 	tabPhi[0] = dphi / 2.;
	for(int iphi = 1; iphi < NBPHI; iphi++){ 
		tabPhi[iphi] = tabPhi[iphi-1] + dphi;
	}

	// Solid angles of the output angular boxes 
	double sumds = 0;
	double *tabds;
    tabds = (double*)malloc(NBTHETA * NBPHI * sizeof(double));
	memset(tabds, 0, NBTHETA * NBPHI * sizeof(double));

	for(int ith = 0; ith < NBTHETA; ith++)
	{
		//if( ith==0 )
		//	dth = DEMIPI / (2*NBTHETA);
		//else 
			dth = DEMIPI / NBTHETA;
			
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			tabds[ith * NBPHI + iphi] = sin(tabTh[ith]) * dth * dphi;
			sumds += tabds[ith * NBPHI + iphi];
		}
	}
	
	/*for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			sumds += sin( (DEMIPI+tabTh[NBTHETA-1])/2 ) * (dth/2) * dphi;
		}*/
	
	// Normalisation de l'aire de chaque morceau de sphère
	memset(tabOmega, 0, NBTHETA * NBPHI * sizeof(*tabOmega));
	for(int ith = 0; ith < NBTHETA; ith++)
		for(int iphi = 0; iphi < NBPHI; iphi++){
			tabOmega[ith * NBPHI + iphi] = tabds[ith * NBPHI + iphi] / sumds;
		}
    
    free(tabds);
}


/* calculTabFinal
* Fonction qui remplit le tabFinal correspondant à la reflectance (R), Q et U sur tous l'espace de sorti (dans chaque boite)
*/
void calculTabFinal(double* tabFinal, double* tabTh, double* tabPhi, double* tabPhotonsTot, unsigned long long nbPhotonsTot,
                   unsigned long long* nbPhotonsTotInter)
{
	
    double norm, normInter;
	double *tabOmega;
    tabOmega = (double*)malloc(NBTHETA * NBPHI * sizeof(double));
	// Remplissage des tableaux tabTh, tabPhi, et tabOmega
	calculOmega(tabTh, tabPhi, tabOmega);
	
	// Remplissage du tableau final
	for(int iphi = 0; iphi < NBPHI; iphi++)
	{
		for(int ith = 0; ith < NBTHETA; ith++)
		{
            norm = 2.0 * tabOmega[ith*NBPHI+iphi] * cos(tabTh[ith]);

            for(int i=0;i<NLAM;i++){
               normInter = norm * nbPhotonsTotInter[i];
			  // Reflectance
			          tabFinal[0*NBTHETA*NBPHI*NLAM + i*NBTHETA*NBPHI + iphi*NBTHETA + ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi] +
				 tabPhotonsTot[1*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi]) / normInter;
			
			  // Q
			          tabFinal[1*NBTHETA*NBPHI*NLAM + i*NBTHETA*NBPHI + iphi*NBTHETA + ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi] -
				 tabPhotonsTot[1*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi]) / normInter;
			
			  // U
			          tabFinal[2*NBTHETA*NBPHI*NLAM + i*NBTHETA*NBPHI + iphi*NBTHETA + ith] =
                (tabPhotonsTot[2*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi]) / normInter;
				
			  // N
			          tabFinal[3*NBTHETA*NBPHI*NLAM + i*NBTHETA*NBPHI + iphi*NBTHETA + ith] =
			    (tabPhotonsTot[3*NBPHI*NBTHETA*NLAM + i*NBTHETA*NBPHI + ith*NBPHI    + iphi])  ;
            }
				
		}
	}
    free(tabOmega);
}




//
// write a single sds name in open hdf file sd
//
void write_sds(int sd, const char* name, int ndims, int *dims, int type, char *dim_names, void* values) {

    int start[ndims];
    int sds, i, status;

    // create sds
    sds = SDcreate(sd, name, type, ndims, dims);
    for (i=0 ; i<ndims ; i++) {
        start[i]=0;
    }

    // write in sds
    status = SDwritedata(sds, start, NULL, dims, (VOIDP)values);
    if(status) {
        printf("\nError writing sds %s\n", name);
        exit(1);
    }

    // write "dimensions" attribute
    if (dim_names != NULL) {
        SDsetattr(sds, "dimensions", DFNT_CHAR8, strlen(dim_names), dim_names);
    }

    // close sds
    SDendaccess(sds);
}


/* creerHDFResultats
* Fonction qui crée le fichier .hdf contenant le résultat final pour une demi-sphère
*/



void creerHDFResultats(double* tabFinal,double* tabTh, double* tabPhi, double* tabTransDir, unsigned long long nbPhotonsTot,
                       Variables* var, double tempsPrec,int mlsaoce,int mlsaaer,double *phaseAtm,double *phaseOc,Tableaux tab_H)



{

	// Création du fichier de sortie
	int sdFichier = SDstart(PATHRESULTATSHDF, DFACC_CREATE);
	if (sdFichier == FAIL) {
		printf("ERREUR: création du fichier HDF : %s\n",PATHRESULTATSHDF);
		exit(1);
	}

	// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSdouble = (double)NBPHOTONS;
	double nbPhotonsTotdouble = (double)nbPhotonsTot;
	double tempsEcouledouble = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);

    #ifdef SPHERIQUE
	SDsetattr(sdFichier, "MODE", DFNT_CHAR8, 3, "SSA");
    #else
	SDsetattr(sdFichier, "MODE", DFNT_CHAR8, 3, "PPA");
    #endif
	SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, &NBPHOTONSdouble);
	SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, &NBLOOP);
	SDsetattr(sdFichier, "SEED", DFNT_UINT32, 1, &SEED);
	SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, &XBLOCK);
	SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, &YBLOCK);
	SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, &XGRID);
	SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, &YGRID);
	SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, &NBTHETA);
	SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, &NBPHI);
	SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, &DIOPTRE);
	SDsetattr(sdFichier, "ENV", DFNT_INT32, 1, &ENV);
	SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, &SIM);
	SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, &SUR);
	SDsetattr(sdFichier, "VZA (deg.)", DFNT_FLOAT32, 1, &THVDEG);
	SDsetattr(sdFichier, "NLAM", DFNT_INT32, 1, &NLAM);
	SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, &TAURAY);
	SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, &TAUAER);

	SDsetattr(sdFichier, "LSAAER", DFNT_UINT32, 1, &LSAAER);
	SDsetattr(sdFichier, "NFAER", DFNT_UINT32, 1, &NFAER);
	SDsetattr(sdFichier, "LSAOCE", DFNT_UINT32, 1, &LSAOCE);
	SDsetattr(sdFichier, "NFOCE", DFNT_UINT32, 1, &NFOCE);

	
	SDsetattr(sdFichier, "NPHAAER", DFNT_UINT32, 1, &NPHAAER);
	SDsetattr(sdFichier, "NPHAOCE", DFNT_UINT32, 1, &NPHAOCE);



	SDsetattr(sdFichier, "ENV_SIZE", DFNT_FLOAT32, 1, &ENV_SIZE);
	SDsetattr(sdFichier, "X0", DFNT_FLOAT32, 1, &X0);
	SDsetattr(sdFichier, "Y0", DFNT_FLOAT32, 1, &Y0);
	SDsetattr(sdFichier, "NATM", DFNT_INT32, 1, &NATM);
	SDsetattr(sdFichier, "HATM", DFNT_FLOAT32, 1, &HATM);
	SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, &WINDSPEED);
	SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, &NH2O);
	SDsetattr(sdFichier, "TRANSDIR", DFNT_FLOAT32, 1, &TRANSDIR);
	SDsetattr(sdFichier, "NOCE", DFNT_INT32, 1, &NOCE);
    SDsetattr(sdFichier, "PATHDIFFOCE", DFNT_CHAR8, strlen(PATHDIFFOCE), PATHDIFFOCE);
    SDsetattr(sdFichier, "PATHPROFILOCE", DFNT_CHAR8, strlen(PATHPROFILOCE), PATHPROFILOCE);
    SDsetattr(sdFichier, "PATHRESULTATSHDF", DFNT_CHAR8, strlen(PATHRESULTATSHDF), PATHRESULTATSHDF);
	SDsetattr(sdFichier, "PATHDIFFAER", DFNT_CHAR8, strlen(PATHDIFFAER), PATHDIFFAER);
	SDsetattr(sdFichier, "PATHPROFILATM", DFNT_CHAR8, strlen(PATHPROFILATM), PATHPROFILATM);
	SDsetattr(sdFichier, "PATHALB", DFNT_CHAR8, strlen(PATHALB), PATHALB);

	SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, &nbPhotonsTotdouble);
	SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, &(var->erreurpoids));
	SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, &(var->erreurtheta));
	SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, &tempsEcouledouble);



    //
    // write datasets
    //
    int dims[10];
    int ndims;
    char dim_names[2048];
    if (NLAM == 1) {
        ndims=2;
        dims[1] = NBTHETA;
        dims[0] = NBPHI;
        strncpy(dim_names, "Azimut angles, Zenith angles", 2048);
    } else {
        ndims=3;
        dims[2] = NBTHETA;
        dims[1] = NBPHI;
        dims[0] = NLAM;
        strncpy(dim_names, "LAMBDA, Azimut angles, Zenith angles", 2048);
    }



    write_sds(sdFichier, "I_up (TOA)", ndims, dims, DFNT_FLOAT64, dim_names, (VOIDP)(tabFinal));
    write_sds(sdFichier, "Q_up (TOA)", ndims, dims, DFNT_FLOAT64, dim_names, (VOIDP)(tabFinal+NBPHI*NBTHETA*NLAM));
    write_sds(sdFichier, "U_up (TOA)", ndims, dims, DFNT_FLOAT64, dim_names, (VOIDP)(tabFinal+2*NBPHI*NBTHETA*NLAM));
    write_sds(sdFichier, "N_up (TOA)", ndims, dims, DFNT_FLOAT64, dim_names, (VOIDP)(tabFinal+3*NBPHI*NBTHETA*NLAM));


        if (OUTPUT_LAYERS & OUTPUT_BOA_DOWN_0P_UP_0M) {
            write_sds(sdFichier, "I_down (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+DOWN0P*4*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "Q_down (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0P*4+1)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "U_down (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0P*4+2)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "N_down (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0P*4+3)*NBPHI*NBTHETA*NLAM));

            write_sds(sdFichier, "I_up (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+UP0M*4*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "Q_up (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0M*4+1)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "U_up (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0M*4+2)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "N_up (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0M*4+3)*NBPHI*NBTHETA*NLAM));
        }
        if (OUTPUT_LAYERS & OUTPUT_BOA_DOWN_0M_UP_0P) {
            write_sds(sdFichier, "I_down (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+DOWN0M*4*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "Q_down (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0M*4+1)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "U_down (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0M*4+2)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "N_down (0-)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(DOWN0M*4+3)*NBPHI*NBTHETA*NLAM));

            write_sds(sdFichier, "I_up (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+UP0P*4*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "Q_up (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0P*4+1)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "U_up (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0P*4+2)*NBPHI*NBTHETA*NLAM));
            write_sds(sdFichier, "N_up (0+)", ndims, dims, DFNT_FLOAT64,dim_names, (VOIDP)(tabFinal+(UP0P*4+3)*NBPHI*NBTHETA*NLAM));
        }


    // theta in degrees
    float *tabThBis;
    tabThBis = (float*)malloc(NBTHETA*sizeof(float));
    for(int i=0; i<NBTHETA; i++) {
        tabThBis[i] = tabTh[i]/DEG2RAD;
    }
    dims[0] = NBTHETA;
    write_sds(sdFichier, "Zenith angles", 1, dims, DFNT_FLOAT32, NULL, (VOIDP)tabThBis);
    free(tabThBis);

    // phi in degrees
    float *tabPhiBis;
    tabPhiBis = (float*)malloc(NBPHI*sizeof(float));
    for(int i=0; i<NBPHI; i++) {
        tabPhiBis[i] = tabPhi[i]/DEG2RAD;
    }
    dims[0] = NBPHI;
    write_sds(sdFichier, "Azimut angles", 1, dims, DFNT_FLOAT32, NULL, (VOIDP)tabPhiBis);
    free(tabPhiBis);

    // Direct Transmission
    dims[0] = NLAM;
    write_sds(sdFichier, "Direct Transmission", 1, dims, DFNT_FLOAT64, NULL, (VOIDP)tabTransDir);



    ndims=3;
    if (NPHAAER > 0) {
    	dims[0] = NPHAAER;
    	dims[1]=5;
    	dims[2] = mlsaaer;

    	write_sds(sdFichier, "phaseAtm", ndims, dims, DFNT_FLOAT64,NULL, (VOIDP)(phaseAtm));
    	free(phaseAtm);
    }

	if (strcmp(PATHPROFILATM,"None")!=0){
    	dims[0]=NLAM;
		dims[1]=NATM+1;
		dims[2]=9;

		float *profAtm= (float*)malloc((NLAM*9*(NATM+1))*sizeof(float));

		for(int ilam=0;ilam<NLAM;ilam++) {
			 for(int icouche=0; icouche<NATM+1; icouche++ ){
				 profAtm[ilam*9*(NATM+1)+icouche*9+0]=tab_H.z[icouche];
				 profAtm[ilam*9*(NATM+1)+icouche*9+1]=tab_H.hmol[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+2]=tab_H.haer[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+3]=tab_H.h[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+4]=tab_H.xdel[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+5]=tab_H.pMol[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+6]=tab_H.ssa[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+7]=tab_H.abs[icouche+ilam*(NATM+1)];
				 profAtm[ilam*9*(NATM+1)+icouche*9+8]=tab_H.ip[icouche+ilam*(NATM+1)];

			 }
		}

		write_sds(sdFichier, "profileAtm", ndims, dims, DFNT_FLOAT32,NULL, (VOIDP)(profAtm));

		free(profAtm);



    }

	if(NPHAOCE > 0){

    	dims[0] = NPHAOCE;
    	dims[1]=5;
    	dims[2] = mlsaoce;
		write_sds(sdFichier, "phaseOc", ndims, dims, DFNT_FLOAT64,NULL, (VOIDP)(phaseOc));
		free(phaseOc);
    }


    if (strcmp(PATHPROFILOCE,"None")!=0){
        dims[0]=NLAM;
		dims[1]=NOCE+1;
		dims[2]=4;



        float *profOc= (float*)malloc((4*NLAM*(NOCE+1))*sizeof(float));
		for( int ilam=0;ilam<NLAM;ilam++) {
			for( int icouche=0; icouche<NOCE+1; icouche++ ){
					profOc[ilam*4*(NOCE+1)+icouche*4+0]=tab_H.depth[icouche];
					profOc[ilam*4*(NOCE+1)+1+icouche*4+1]=tab_H.ho[icouche+ilam*(NOCE+1)];
					profOc[ilam*4*(NOCE+1)+icouche*4+2]=tab_H.sso[icouche+ilam*(NOCE+1)];
					profOc[ilam*4*(NOCE+1)+icouche*4+3]=(float)tab_H.ipo[icouche+ilam*(NOCE+1)];
					}
				}
		write_sds(sdFichier, "profileOc", ndims, dims, DFNT_FLOAT32,NULL, (VOIDP)(profOc));

		free(profOc);

    }


    if (NLAM > 1) {
        write_sds(sdFichier, "LAMBDA", 1, dims, DFNT_FLOAT32,NULL, (VOIDP)tab_H.lambda);
    } else {
        SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, tab_H.lambda);
    }


    // closes hdf file
    SDend(sdFichier);

}
//fusion des tableaux

