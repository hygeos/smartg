#include "hip/hip_runtime.h"

	  //////////////
	 // INCLUDES //
	//////////////

#include "communs.h"
#include "host.h"
#include "device.h"

	  ////////////////////
	 // FONCTIONS HOST //
	////////////////////

// Fonction qui initialise les generateurs du random MWC
int initRandMWC(unsigned long long *etat, unsigned int *config, 
	     const unsigned int n_rng, const char *safeprimes_file, unsigned long long xinit)
{
	FILE *fp;
	unsigned int begin=0u;
	unsigned int fora,tmp1,tmp2;
	if (strlen(safeprimes_file) == 0)
	{
        	// Try to find it in the local directory
		safeprimes_file = "MWC.txt";
	}
	fp = fopen(safeprimes_file, "r");
	if(fp == NULL)
	{
		printf("Could not find the file of safeprimes (%s)! Terminating!\n", safeprimes_file);
		return 1;
	}
	fscanf(fp,"%u %u %u",&begin,&tmp1,&tmp2);
	// Here we set up a loop, using the first multiplier in the file to generate x's and c's
	// There are some restictions to these two numbers:
	// 0<=c<a and 0<=x<b, where a is the multiplier and b is the base (2^32)
	// also [x,c]=[0,0] and [b-1,a-1] are not allowed.
	//Make sure xinit is a valid seed (using the above mentioned restrictions)
	if((xinit == 0ull) | (((unsigned int)(xinit>>32))>=(begin-1)) | (((unsigned int)xinit)>=0xfffffffful))
	{
		//xinit (probably) not a valid seed! (we have excluded a few unlikely exceptions)
		printf("%llu not a valid seed! Terminating!\n",xinit);
		return 1;
	}
	for(unsigned int i=0;i < n_rng;i++)
	{
		fscanf(fp,"%u %u %u",&fora,&tmp1,&tmp2);
		config[i]=fora;
		etat[i]=0;
		while( (etat[i]==0) | (((unsigned int)(etat[i]>>32))>=(fora-1)) | (((unsigned int)etat[i])>=0xfffffffful))
		{
			//generate a random number
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);
			//calculate c and store in the upper 32 bits of x[i]
			etat[i]=(unsigned int) floor((((double)((unsigned int)xinit))/(double)0x100000000)*fora);//Make sure 0<=c<a
			etat[i]=etat[i]<<32;
			//generate a random number and store in the lower 32 bits of x[i] (as the initial x of the generator)
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);//x will be 0<=x<b, where b is the base 2^32
			etat[i]+=(unsigned int) xinit;
		}
		//if(i<10)printf("%llu\n",x[i]);
	}
	fclose(fp);
	return 0;
}

// Fonction qui récupère les valeurs des constantes dans Parametres.txt et initialise les constantes du host
void initConstantesHost(int argc, char** argv)
{
	if(argc <2)
	{
		printf("ERREUR : lecture argv");
		exit(1);
	}
	
	char* s = (char*)malloc(100 * sizeof(char));

	strcpy(s,"");
	chercheConstante(argv[1], "NBPHOTONS", s);
	NBPHOTONS = strtoull(s, NULL, 10);

	strcpy(s,"");
	chercheConstante(argv[1], "NBLOOP", s);
	NBLOOP = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "XBLOCK", s);
	XBLOCK= atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "YBLOCK", s);
	YBLOCK = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "XGRID", s);
	XGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "YGRID", s);
	YGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "NBTHETA", s);
	NBTHETA = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "NBPHI", s);
	NBPHI = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "NBSTOKES", s);
	NBSTOKES = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "PROFIL", s);
	PROFIL = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "SIM", s);
	SIM = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "SUR", s);
	SUR = atoi(s);

	strcpy(s,"");
	chercheConstante(argv[1], "DIOPTRE", s);
	DIOPTRE= atoi(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "DIFFF", s);
	DIFFF = atoi(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "THSDEG", s);
	THSDEG = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "LAMBDA", s);
	LAMBDA = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "TAURAY", s);
	TAURAY = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "TAUAER", s);
	TAUAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "W0AER", s);
	W0AER = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "HA", s);
	HA = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "HR", s);
	HR = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "ZMIN", s);
	ZMIN = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "ZMAX", s);
	ZMAX = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "WINDSPEED", s);
	WINDSPEED = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "NH2O", s);
	NH2O = atof(s);
	
	strcpy(s,"");
	chercheConstante(argv[1], "CONPHY", s);
	CONPHY = atof(s);

	free(s);
}

// Fonction qui cherche nomConstante dans le fichier et met la valeur de la constante dans chaineValeur (en string)
void chercheConstante(char* nomFichier, char* nomConstante, char* chaineValeur)
{
	// Ouverture du fichier
	FILE* fichier = fopen(nomFichier, "r");
	if(fichier != NULL)
	{
		int longueur = strlen(nomConstante);
		char ligne[100];
		int motTrouve = 0;
		// Tant que la constante n'est pas trouvee et qu'on n'est pas à la fin du fichier on lit la ligne
		while(fgets(ligne, 100, fichier) && !motTrouve)
		{
			// Si le debut de la ligne est nomConstante suivi d'un espace ou un egal on va chercher la valeur
			if((strncmp(ligne, nomConstante, longueur) == 0) && (ligne[longueur] == ' ' || ligne[longueur] == '='))
			{
				char* ptr = ligne; //pointeur du debut de la ligne
				// on avance jusqu'au prochain espace ou egal
				while (*ptr != ' ' && *ptr != '=') ptr++;
				// on avance jusqu'à la valeur de la constante
				while(*ptr == ' ' || *ptr == '=') ptr++;
				if (*ptr == '\n')
				{
					printf("ERREUR : lecture Parametre.txt");
					exit(1);
				}
				// On met la chaine de la valeur de la constante dans chaineValeur
				strcpy(chaineValeur, ptr);
				chaineValeur[strlen(chaineValeur)-1] = '\0';
				motTrouve = 1;
			}
		}
		fclose(fichier);
		if(motTrouve == 0)
		{
			printf("ERREUR : lecture Parametres.txt");
			exit(1);
		}
	}
	else
	{
		printf("ERREUR : lecture Parametres.txt");
		exit(1);
	}
}

// Fonction qui initialise les variables à envoyer dans le kernel
void initVariables(Variables** var_H, Variables** var_D)
{
	// Initialisation de la version host des variables
	*var_H = (Variables*)malloc(sizeof(Variables));
	memset(*var_H, 0, sizeof(Variables));
	// Initialisation de la version device des variables
	hipMalloc(var_D, sizeof(Variables));
	hipMemset(*(var_D), 0, sizeof(Variables));
}

// Fonction qui initialise les tableaux à envoyer dans le kernel
void initTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
#ifdef RANDMWC
	// Création des tableaux de generateurs pour la fonction Random MWC
	#ifdef NEW
	// La simulation est différente à chaque lancement
	unsigned long long seed = (unsigned long long) time(NULL); //un seul seed pour tous les generateurs
	#else
	// La simulation est identique à chaque lancement
	unsigned long long seed = 777ULL; //un seul seed pour tous les generateurs
	#endif
	tab_H->etat = (unsigned long long*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long));
	hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long));
	tab_H->config = (unsigned int*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int));
	hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int));
	// Initialisation des tableaux host à l'aide du fichier et du seed
	initRandMWC(tab_H->etat, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID, "MWC.txt", seed);
	// Copie dans les tableaux device
	hipMemcpy(tab_D->etat, tab_H->etat, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long), hipMemcpyHostToDevice);
	hipMemcpy(tab_D->config, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int), hipMemcpyHostToDevice);
#endif
#ifdef RANDCUDA
	// Création du tableau de generateurs (=etat+config) pour la fonction Random Cuda
	#ifdef NEW
	// La simulation est différente à chaque lancement
	unsigned long long seed = (unsigned long long) time(NULL); //un seul seed pour tous les generateurs
	#else
	// La simulation est identique à chaque lancement
	unsigned long long seed = 777ULL; //un seul seed pour tous les generateurs
	#endif
	hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(hiprandState_t));
	// Initialisation du tableau dans une fonction du kernel
	initRandCUDA<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, seed);
#endif
#ifdef RANDMT
	// Création des tableaux de generateurs pour la fonction Random Mersenne Twister
	hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT));
	hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT));
	tab_H->config = (ConfigMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT));
	tab_H->etat = (EtatMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT));
	// Initialisation du tableau des configs à l'aide du fichier
	initRandMTConfig(tab_H->config, tab_D->config, XBLOCK * YBLOCK * XGRID * YGRID);
	// Initialisation du tableau des etats dans le kernel
	initRandMTEtat<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, tab_D->config);
#endif
	
	// Tableau du poids des photons ressortis
	tab_H->tabPhotons = (unsigned long long*)malloc(NBTHETA * NBPHI * NBSTOKES * sizeof(unsigned long long));
	hipMalloc(&(tab_D->tabPhotons), NBTHETA * NBPHI * NBSTOKES * sizeof(unsigned long long));
}

// Fonction qui initialise en partie les generateurs du random Mersenen Twister
void initRandMTConfig(ConfigMT* config_H, ConfigMT* config_D, int nbThreads)
{
	// Ouverture du fichier
	const char *fname = "MersenneTwister.dat";
	FILE* fd = fopen(fname, "rb");
	if(!fd)
	{
		printf("ERREUR: ouverture fichier MT");
		exit(0);
	}
	// Lecture et initialisation de la config pour chaque generateur (= pour chaque thread)
	for(int i = 0; i < nbThreads; i++)
	{
		// Le fichier ne contient que 4096 configs, on reutilise donc les memes configs pour les threads en trop mais les nombres aléatoires restent independants car les etats des threads sont differents
		if(i%4096 == 0)
		{
			fseek(fd, 0, 0);
		}
		if(!fread(config_H+i, sizeof(ConfigMT), 1, fd))
		{
			printf("ERREUR: lecture fichier MT");
			exit(0);
		}
	}
	fclose(fd);
	#ifdef NEW
	// La simulation est différente à chaque lancement
	srand(time(NULL));
	#else
	// La simulation est identique à chaque lancement
	srand(777);
	#endif
	// Creation des seeds aleatoires pour que les threads aient des etats differents
	for(int i = 0; i < nbThreads; i++) config_H[i].seed = (unsigned int)rand();
	hipMemcpy(config_D, config_H, nbThreads * sizeof(ConfigMT), hipMemcpyHostToDevice);
}

// DEBUG : Initialisation des variables à envoyer dans le kernel pour récupérer le trajet d'un photon
void initEvnt(Evnt* evnt_H, Evnt* evnt_D)
{
	for(int i = 0; i < 20; i++) evnt_H[i].action = 0;
	hipMemcpy(evnt_D, evnt_H, 20 * sizeof(Evnt), hipMemcpyHostToDevice);
}

// Fonction qui réinitialise certaines variables avant chaque envoi dans le kernel
void reinitVariables(Variables* var_H, Variables* var_D)
{
	// Le nombre de photons traités pour un appel du Kernel est remis à zéro
	var_H->nbPhotons = 0;
	#ifdef PROGRESSION
	// Le nombre de photons ressortis pour un appel du Kernel est remis à zéro
	var_H->nbPhotonsSor = 0;
	#endif
	// On copie le nouveau var_H dans var_D
	hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
}

// Fonction qui calcule pour chaque l'aire normalisée de chaque boite, son theta, et son psi, sous forme de 3 tableaux
void calculOmega(float* tabTh, float* tabPhi, float* tabOmega)
{
	// Tableau contenant l'angle theta de chaque morceau de sphère
	memset(tabTh, 0, NBTHETA * sizeof(float));
	float dth = DEMIPI / NBTHETA;
	tabTh[0] = dth / 2;
	for(int ith = 1; ith < NBTHETA; ith++) tabTh[ith] = tabTh[ith-1] + dth;
	// Tableau contenant l'angle psi de chaque morceau de sphère
	memset(tabPhi, 0, NBPHI * sizeof(float));
	float dphi = DEUXPI / NBPHI;
	tabPhi[0] = dphi / 2;
	for(int iphi = 1; iphi < NBPHI; iphi++) tabPhi[iphi] = tabPhi[iphi-1] + dphi;
	// Tableau contenant l'aire de chaque morceau de sphère
	float sumds = 0;
	float tabds[NBTHETA * NBPHI];
	memset(tabds, 0, NBTHETA * NBPHI * sizeof(float));
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			tabds[ith * NBPHI + iphi] = sin(tabTh[ith]) * dth * dphi;
			sumds += tabds[ith * NBPHI + iphi];
		}
	}
	// Normalisation de l'aire de chaque morceau de sphère
	memset(tabOmega, 0, NBTHETA * NBPHI * sizeof(float));
	for(int ith = 0; ith < NBTHETA; ith++)
		for(int iphi = 0; iphi < NBPHI; iphi++)
			tabOmega[ith * NBPHI + iphi] = tabds[ith * NBPHI + iphi] / sumds;
}

// Fonction qui remplit le tabFinal, tabTh et tabPhi
void calculTabFinal(float* tabFinal, float* tabTh, float* tabPhi, unsigned long long* tabPhotonsTot, unsigned long long nbPhotonsTot)
{
	float tabOmega[NBTHETA * NBPHI]; //tableau contenant l'aire de chaque morceau de sphère
	// Remplissage des tableaux tabTh, tabPhi, et tabOmega
	calculOmega(tabTh, tabPhi, tabOmega);
	// Remplissage du tableau final
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			tabFinal[ith*NBPHI+iphi] = (tabPhotonsTot[0*NBPHI*NBTHETA+ith*NBPHI+iphi] + tabPhotonsTot[1*NBPHI*NBTHETA+ith*NBPHI+iphi]) / (2 * nbPhotonsTot * tabOmega[ith*NBPHI+iphi] * SCALEFACTOR * cosf(tabTh[ith]));
		}
	}
}

// Fonction qui affiche les paramètres de la simulation
void afficheParametres()
{
	printf("\n");
	printf("NBPHOTONS = %lu", NBPHOTONS);
	printf("\n");
	printf("NBLOOP = %u", NBLOOP);
	printf("\n");
	printf("XBLOCK = %d", XBLOCK);
	printf("\n");
	printf("YBLOCK = %d", YBLOCK);
	printf("\n");
	printf("XGRID = %d", XGRID);
	printf("\n");
	printf("YGRID = %d", YGRID);
	printf("\n");
	printf("NBTHETA = %d", NBTHETA);
	printf("\n");
	printf("NBPHI = %d", NBPHI);
	printf("\n");
	printf("NBSTOKES = %d", NBSTOKES);
	printf("\n");
	printf("THSDEG = %f (degrés)", THSDEG);
	printf("\n");
	printf("LAMBDA = %f", LAMBDA);
	printf("\n");
	printf("TAURAY = %f", TAURAY);
	printf("\n");
	printf("TAUAER = %f", TAUAER);
	printf("\n");
	printf("W0AER = %f", W0AER);
	printf("\n");
	printf("PROFIL = %d", PROFIL);
	printf("\n");
	printf("HA = %f", HA);
	printf("\n");
	printf("HR = %f", HR);
	printf("\n");
	printf("ZMIN = %f", ZMIN);
	printf("\n");
	printf("ZMAX = %f", ZMAX);
	printf("\n");
	printf("WINDSPEED = %f", WINDSPEED);
	printf("\n");
	printf("NH2O = %f", NH2O);
	printf("\n");
	printf("SIM = %d", SIM);
	printf("\n");
	printf("SUR = %d", SUR);
	printf("\n");
	printf("DIOPTRE = %d", DIOPTRE);
	printf("\n");
	printf("CONPHY = %f", CONPHY);
	printf("\n");
	printf("DIFFF = %d", DIFFF);
	printf("\n");
}

// Fonction qui affiche la progression de la simulation
void afficheProgress(unsigned long long nbPhotonsTot, Variables* var, double tempsPrec
		#ifdef PROGRESSION
		, unsigned long long nbPhotonsSorTot
		#endif
		    )
{
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	// Calcul du temps ecoule et restant
	double tempsProg = (double)(clock() / CLOCKS_PER_SEC);
	double tempsTot = tempsProg + tempsPrec;
	int tempsEcoule = (int)tempsTot;
	int hEcoulees = tempsEcoule / 3600;
	int minEcoulees = (tempsEcoule%3600) / 60;
	int secEcoulees = tempsEcoule%60;
	int tempsRestant = (int)(tempsTot * ((double)NBPHOTONS / (double)nbPhotonsTot - 1.));
	if(tempsRestant < 0) tempsRestant = 0;
	int hRestantes = tempsRestant / 3600;
	int minRestantes = (tempsRestant%3600) / 60;
	int secRestantes = tempsRestant%60;
	// Calcul du pourcentage de photons traités
	int pourcent = (int)(100 * nbPhotonsTot / NBPHOTONS);
	
	// Affichage
	printf("\n");
	printf("-------------------------------------------\n");
	printf("Photons lances : %12lu (%3d%%)\n", nbPhotonsTot, pourcent);
	printf("Photons pb     : %12d\n", var->erreurpoids + var->erreurtheta);
	printf("Temps ecoule   : %d h %2d min %2d sec\n", hEcoulees, minEcoulees, secEcoulees);
	printf("Temps restant  : %d h %2d min %2d sec\n", hRestantes, minRestantes, secRestantes);
	printf("Date actuelle  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year, date->tm_hour, date->tm_min, date->tm_sec);
	printf("-------------------------------------------\n");
	
	#ifdef PROGRESSION
	printf("%d%% - ", (int)(100*nbPhotonsTot/NBPHOTONS));
	printf("Temps: %d - ", tempsEcoule);
	printf("phot sortis: %lu - ", nbPhotonsSorTot);
	printf("phot traités: %lu - ", nbPhotonsTot);
	printf("erreur poids/theta/vxy/vy/case: %d/%d/%d/%d/%d", var->erreurpoids, var->erreurtheta, var->erreurvxy, var->erreurvy, var->erreurcase);
	printf("\n");
	#endif
}

// Fonction qui affiche le début du trajet du premier thread
void afficheTrajet(Evnt* evnt_H)
{
	printf("\nTrajet d'un thread :\n");
	for(int i = 0; i < 20; i++)
	{
		if(evnt_H[i].action == 1)
			printf("init : ");
		else if(evnt_H[i].action == 2)
			printf("move : ");
		else if(evnt_H[i].action == 3)
			printf("scat : ");
		else if(evnt_H[i].action == 4)
			printf("surf : ");
		else if(evnt_H[i].action != 5)
		{
			printf("\nERREUR : host afficheTrajet\n");
			exit(1);
		}
		else printf("exit : ");
		printf("tau=%f ", evnt_H[i].tau);
		printf("poids=%f", evnt_H[i].poids);
		printf("\n");
	}
}

// Fonction qui affiche les tableaux "finaux" pour chaque nombre de Stokes
void afficheTabStokes(unsigned long long* tabPhotonsTot)
{
	printf("\nTableau Stokes1 :\n");
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			printf("%llu ", tabPhotonsTot[0 * NBPHI * NBTHETA + ith * NBPHI + iphi]);
		}
		printf("\n");
	}
	printf("\nTableau Stokes2 :\n");
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			printf("%llu ", tabPhotonsTot[1 * NBPHI * NBTHETA + ith * NBPHI + iphi]);
		}
		printf("\n");
	}
}

// Fonction qui affiche le tableau final
void afficheTabFinal(float* tabFinal)
{
	// Affichage du tableau final
	printf("\nTableau Final :\n");
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			printf("%f ", tabFinal[ith * NBPHI + iphi]);
		}
		printf("\n");
	}
}

// Fonction qui crée un fichier .hdf contenant les informations nécessaires à la reprise du programme
void creerHDFTemoin(unsigned long long* tabPhotonsTot, unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	char nomFichier[20] = "tmp/Temoin.hdf";
	int sdFichier = SDstart(nomFichier, DFACC_CREATE);
	// Création et remplissage du tableau du fichier (en double car le format hdf n'accepte pas int64)
	double* tab;
	tab = (double*)malloc(NBTHETA * NBPHI * NBSTOKES * sizeof(double));
	memset(tab, 0, NBTHETA * NBPHI * NBSTOKES * sizeof(double));
	// On remplit le tableau en convertissant de unsigned long long a double car le hdf n'accepte pas ull
	for(int i = 0; i < NBTHETA * NBPHI * NBSTOKES; i++)
		tab[i] = (double)tabPhotonsTot[i];
	char nomTab[20]; //nom du tableau
	sprintf(nomTab,"Temoin (%d%%)", (int)(100 * nbPhotonsTot / NBPHOTONS));
	int nbDimsTab = 1; //nombre de dimensions du tableau
	int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
	valDimsTab[0] = NBTHETA * NBPHI * NBSTOKES;
	int typeTab = DFNT_FLOAT64 ; //type des éléments du tableau
	// Création du tableau
	int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	int startTab[nbDimsTab]; //début de la lecture du tableau
	startTab[0]=0;
	// Ecriture du tableau dans le fichier
	int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tab);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf temoin\n");
		exit(1);
	}
	// Liberation du tableau
	free(tab);
	
	// Ecriture de toutes les informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSsave[1];
	unsigned int NBLOOPsave[1];
	float THSDEGsave[1];
	float LAMBDAsave[1];
	float TAURAYsave[1];
	float TAUAERsave[1];
	float W0AERsave[1];
	float HAsave[1];
	float HRsave[1];
	float ZMINsave[1];
	float ZMAXsave[1];
	float WINDSPEEDsave[1];
	float NH2Osave[1];
	float CONPHYsave[1];
	int XBLOCKsave[1];
	int YBLOCKsave[1];
	int XGRIDsave[1];
	int YGRIDsave[1];
	int NBTHETAsave[1];
	int NBPHIsave[1];
	int NBSTOKESsave[1];
	int DIOPTREsave[1];
	int DIFFFsave[1];
	int PROFILsave[1];
	int SIMsave[1];
	int SURsave[1];
	double nbPhotonsTotSave[1];
	int nbErreursPoidsSave[1];
	int nbErreursThetaSave[1];
	double tempsEcouleSave[1];
	
	NBPHOTONSsave[0] = (double)NBPHOTONS; // on convertit en double car le hdf n'accepte pas ull
	NBLOOPsave[0] = NBLOOP;
	THSDEGsave[0] = THSDEG;
	LAMBDAsave[0] = LAMBDA;
	TAURAYsave[0] = TAURAY;
	TAUAERsave[0] = TAUAER;
	W0AERsave[0] = W0AER;
	HAsave[0] = HA;
	HRsave[0] = HR;
	ZMINsave[0] = ZMIN;
	ZMAXsave[0] = ZMAX;
	WINDSPEEDsave[0] = WINDSPEED;
	NH2Osave[0] = NH2O;
	CONPHYsave[0] = CONPHY;
	XBLOCKsave[0] = XBLOCK;
	YBLOCKsave[0] = YBLOCK;
	XGRIDsave[0] = XGRID;
	YGRIDsave[0] = YGRID;
	NBTHETAsave[0] = NBTHETA;
	NBPHIsave[0] = NBPHI;
	NBSTOKESsave[0] = NBSTOKES;
	DIOPTREsave[0] = DIOPTRE;
	DIFFFsave[0] = DIFFF;
	PROFILsave[0] = PROFIL;
	SIMsave[0] = SIM;
	SURsave[0] = SUR;
	nbPhotonsTotSave[0] = (double)nbPhotonsTot; // on convertit en double car le hdf n'accepte pas ull
	nbErreursPoidsSave[0] = var->erreurpoids;
	nbErreursThetaSave[0] = var->erreurtheta;
	tempsEcouleSave[0] = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
	SDsetattr(sdsTab, "NBPHOTONS", DFNT_FLOAT64, 1, NBPHOTONSsave);
	SDsetattr(sdsTab, "NBLOOP", DFNT_UINT32, 1, NBLOOPsave);
	SDsetattr(sdsTab, "THSDEG", DFNT_FLOAT32, 1, THSDEGsave);
	SDsetattr(sdsTab, "LAMBDA", DFNT_FLOAT32, 1, LAMBDAsave);
	SDsetattr(sdsTab, "TAURAY", DFNT_FLOAT32, 1, TAURAYsave);
	SDsetattr(sdsTab, "TAUAER", DFNT_FLOAT32, 1, TAUAERsave);
	SDsetattr(sdsTab, "W0AER", DFNT_FLOAT32, 1, W0AERsave);
	SDsetattr(sdsTab, "HA", DFNT_FLOAT32, 1, HAsave);
	SDsetattr(sdsTab, "HR", DFNT_FLOAT32, 1, HRsave);
	SDsetattr(sdsTab, "ZMIN", DFNT_FLOAT32, 1, ZMINsave);
	SDsetattr(sdsTab, "ZMAX", DFNT_FLOAT32, 1, ZMAXsave);
	SDsetattr(sdsTab, "WINDSPEED", DFNT_FLOAT32, 1, WINDSPEEDsave);
	SDsetattr(sdsTab, "NH2O", DFNT_FLOAT32, 1, NH2Osave);
	SDsetattr(sdsTab, "CONPHY", DFNT_FLOAT32, 1, CONPHYsave);
	SDsetattr(sdsTab, "XBLOCK", DFNT_INT32, 1, XBLOCKsave);
	SDsetattr(sdsTab, "YBLOCK", DFNT_INT32, 1, YBLOCKsave);
	SDsetattr(sdsTab, "XGRID", DFNT_INT32, 1, XGRIDsave);
	SDsetattr(sdsTab, "YGRID", DFNT_INT32, 1, YGRIDsave);
	SDsetattr(sdsTab, "NBTHETA", DFNT_INT32, 1, NBTHETAsave);
	SDsetattr(sdsTab, "NBPHI", DFNT_INT32, 1, NBPHIsave);
	SDsetattr(sdsTab, "NBSTOKES", DFNT_INT32, 1, NBSTOKESsave);
	SDsetattr(sdsTab, "DIOPTRE", DFNT_INT32, 1, DIOPTREsave);
	SDsetattr(sdsTab, "DIFFF", DFNT_INT32, 1, DIFFFsave);
	SDsetattr(sdsTab, "PROFIL", DFNT_INT32, 1, PROFILsave);
	SDsetattr(sdsTab, "SIM", DFNT_INT32, 1, SIMsave);
	SDsetattr(sdsTab, "SUR", DFNT_INT32, 1, SURsave);
	SDsetattr(sdsTab, "nbPhotonsTot", DFNT_FLOAT64, 1, nbPhotonsTotSave);
	SDsetattr(sdsTab, "nbErreursPoids", DFNT_INT32, 1, nbErreursPoidsSave);
	SDsetattr(sdsTab, "nbErreursTheta", DFNT_INT32, 1, nbErreursThetaSave);
	SDsetattr(sdsTab, "tempsEcoule", DFNT_FLOAT64, 1, tempsEcouleSave);

	// Fermeture du tableau
	SDendaccess(sdsTab);
	// Fermeture du fichier
	SDend(sdFichier);
}

void lireHDFTemoin(Variables* var_H, Variables* var_D,
		unsigned long long* nbPhotonsTot, unsigned long long* tabPhotonsTot, double* tempsEcoule)
{
	// Ouverture du fichier temoin
	char nomFichier[20] = "tmp/Temoin.hdf";
	int sdFichier = SDstart(nomFichier, DFACC_READ);
	if(sdFichier != -1)
	{
		// Ouverture de l'unique tableau du fichier temoin
		int sdsIndex = 0;
		int sdsTab = SDselect (sdFichier, sdsIndex);
		
		// Recuperation des paramètres du fichier temoin
		double NBPHOTONSdouble[1]; //on récupère d'abord la variable en double
		unsigned long long NBPHOTONSrecup[1]; //puis on la passera en unsigned long long
		unsigned int NBLOOPrecup[1];
		float THSDEGrecup[1];
		float LAMBDArecup[1];
		float TAURAYrecup[1];
		float TAUAERrecup[1];
		float W0AERrecup[1];
		float HArecup[1];
		float HRrecup[1];
		float ZMINrecup[1];
		float ZMAXrecup[1];
		float WINDSPEEDrecup[1];
		float NH2Orecup[1];
		float CONPHYrecup[1];
		int XBLOCKrecup[1];
		int YBLOCKrecup[1];
		int XGRIDrecup[1];
		int YGRIDrecup[1];
		int NBTHETArecup[1];
		int NBPHIrecup[1];
		int NBSTOKESrecup[1];
		int DIOPTRErecup[1];
		int DIFFFrecup[1];
		int PROFILrecup[1];
		int SIMrecup[1];
		int SURrecup[1];
		
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBPHOTONS"), (VOIDP)NBPHOTONSdouble);
		NBPHOTONSrecup[0] = (unsigned long long)NBPHOTONSdouble[0];
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBLOOP"), (VOIDP)NBLOOPrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "THSDEG"), (VOIDP)THSDEGrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "LAMBDA"), (VOIDP)LAMBDArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAURAY"), (VOIDP)TAURAYrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAUAER"), (VOIDP)TAUAERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "W0AER"), (VOIDP)W0AERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HA"), (VOIDP)HArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HR"), (VOIDP)HRrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMIN"), (VOIDP)ZMINrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMAX"), (VOIDP)ZMAXrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "WINDSPEED"), (VOIDP)WINDSPEEDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NH2O"), (VOIDP)NH2Orecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "CONPHY"), (VOIDP)CONPHYrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "XBLOCK"), (VOIDP)XBLOCKrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "YBLOCK"), (VOIDP)YBLOCKrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "XGRID"), (VOIDP)XGRIDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "YGRID"), (VOIDP)YGRIDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBTHETA"), (VOIDP)NBTHETArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBPHI"), (VOIDP)NBPHIrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBSTOKES"), (VOIDP)NBSTOKESrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIOPTRE"), (VOIDP)DIOPTRErecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIFFF"), (VOIDP)DIFFFrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "PROFIL"), (VOIDP)PROFILrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SIM"), (VOIDP)SIMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SUR"), (VOIDP)SURrecup);
		
		// Si les parametres sont les memes on recupere des informations pour poursuivre la simulation précédente
		if(NBPHOTONSrecup[0] == NBPHOTONS
			&& NBLOOPrecup[0] == NBLOOP
			&& THSDEGrecup[0] == THSDEG
			&& LAMBDArecup[0] == LAMBDA
			&& TAURAYrecup[0] == TAURAY
			&& TAUAERrecup[0] == TAUAER
			&& W0AERrecup[0] == W0AER
			&& HArecup[0] == HA
			&& HRrecup[0] == HR
			&& ZMINrecup[0] == ZMIN
			&& ZMAXrecup[0] == ZMAX
			&& WINDSPEEDrecup[0] == WINDSPEED
			&& NH2Orecup[0] == NH2O
			&& CONPHYrecup[0] == CONPHY
			&& XBLOCKrecup[0] == XBLOCK
			&& YBLOCKrecup[0] == YBLOCK
			&& XGRIDrecup[0] == XGRID
			&& YGRIDrecup[0] == YGRID
			&& NBTHETArecup[0] == NBTHETA
			&& NBPHIrecup[0] == NBPHI
			&& NBSTOKESrecup[0] == NBSTOKES
			&& DIOPTRErecup[0] == DIOPTRE
			&& DIFFFrecup[0] == DIFFF
			&& PROFILrecup[0] == PROFIL
			&& SIMrecup[0] == SIM
			&& SURrecup[0] == SUR)
		{
		/*
			char reponse;
			int cont = 1;
			while(cont)
			{
				cout << "Continuer avec les simulations sauvegardees? [Y/n]\n";
				cin >> reponse;
				if (reponse == 'Y' || reponse == 'y' || reponse == '')
				{
					cont = 0;
					*/
					// Recuperation du nombre de photons traités et du nombre d'erreurs
					double nbPhotonsTotDouble[1]; //on récupère d'abord la variable en double
					unsigned long long nbPhotonsTotRecup[1]; //puis on la passera en unsigned long long
					int nbErreursPoidsRecup[1];
					int nbErreursThetaRecup[1];
					double tempsEcouleRecup[1];
			
					SDreadattr(sdsTab, SDfindattr(sdsTab, "nbPhotonsTot"), (VOIDP)nbPhotonsTotDouble);
					nbPhotonsTotRecup[0] = (unsigned long long)nbPhotonsTotDouble[0];
					SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursPoids"), (VOIDP)nbErreursPoidsRecup);
					SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursTheta"), (VOIDP)nbErreursThetaRecup);
					SDreadattr(sdsTab, SDfindattr(sdsTab, "tempsEcoule"), (VOIDP)tempsEcouleRecup);
			
					var_H->erreurpoids = nbErreursPoidsRecup[0];//nombre de photons ayant un poids anormalement élevé
					var_H->erreurtheta = nbErreursThetaRecup[0];//nombre de photons sortant dans la direction solaire
					hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
					*(nbPhotonsTot) = nbPhotonsTotRecup[0];
					*(tempsEcoule) = tempsEcouleRecup[0];
			
					// Recuperation du tableau
					int nbDimsTab = 1; //nombre de dimensions du tableau
					int startTab[nbDimsTab], edgesTab[nbDimsTab]; //debut et fin de la lecture du tableau
					startTab[0] = 0;
					edgesTab[0] = NBTHETA * NBPHI * NBSTOKES;
					double tabPhotonsTotRecup[NBTHETA * NBPHI * NBSTOKES]; //tableau de récuperation en double
			
					int status = SDreaddata (sdsTab, startTab, NULL, edgesTab, (VOIDP)tabPhotonsTotRecup);
					// Vérification du bon fonctionnement de la lecture
					if(status)
					{
						printf("\nERREUR : read hdf resultats\n");
						exit(1);
					}
			
					// Conversion en unsigned long long
					for(int i = 0; i < NBTHETA * NBPHI * NBSTOKES; i++)
					{
						tabPhotonsTot[i] = (unsigned long long)tabPhotonsTotRecup[i]; //conversion en ull
					}
					/*
				}
				else if (reponse == 'N' || reponse == 'n')
				{
					cont = 0;
					remove("tmp/Temoin.hdf");
				}
			}*/
			
		}
		// Fermeture du tableau
		SDendaccess (sdsTab);
	}
	// Fermeture du fichier
	SDend (sdFichier);
}
	
// Fonction qui crée le fichier .hdf contenant le résultat final pour une demi-sphère
void creerHDFResultats(float* tabFinal, float* tabTh, float* tabPhi,
		unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	char nomFichier[50] = "out_prog/Resultats.hdf";
	int sdFichier = SDstart(nomFichier, DFACC_CREATE);
	// Pour chaque phi on ajoute au fichier le tableau représentant le résultat final en fonction de theta
	for(int iphi = 0; iphi < NBPHI; iphi++)
	{
		// Création et remplissage d'un tableau du fichier
		// tab[ith*2+0] est le résultat final en ith et tab[ith*2+1] est la valeur de theta
		float* tab;
		tab = (float*)malloc(2 * NBTHETA * sizeof(float));
		memset(tab, 0, 2 * NBTHETA * sizeof(float));
		for(int ith = 0; ith < NBTHETA; ith++)
		{
			tab[ith*2+0] = tabFinal[ith*NBPHI+iphi];
			tab[ith*2+1] = tabTh[ith];
		}
		char nomTab[20]; //nom du tableau
		sprintf(nomTab,"Resultats (iphi = %d)", iphi);
		int nbDimsTab = 2; //nombre de dimensions du tableau
		int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
		valDimsTab[0] = NBTHETA;
		valDimsTab[1] = 2;
		int typeTab = DFNT_FLOAT32; //type des éléments du tableau
		// Création du tableau
		int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
		int startTab[nbDimsTab]; //début de la lecture du tableau
		startTab[0]=0;
		startTab[1]=0;
		// Ecriture du tableau dans le fichier
		int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tab);
		// Vérification du bon fonctionnement de l'écriture
		if(status)
		{
			printf("\nERREUR : write hdf resultats\n");
			exit(1);
		}
		
		// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
		double NBPHOTONSsave[1];
		unsigned int NBLOOPsave[1];
		float THSDEGsave[1];
		float LAMBDAsave[1];
		float TAURAYsave[1];
		float TAUAERsave[1];
		float W0AERsave[1];
		float HAsave[1];
		float HRsave[1];
		float ZMINsave[1];
		float ZMAXsave[1];
		float WINDSPEEDsave[1];
		float NH2Osave[1];
		float CONPHYsave[1];
		int XBLOCKsave[1];
		int YBLOCKsave[1];
		int XGRIDsave[1];
		int YGRIDsave[1];
		int NBTHETAsave[1];
		int NBPHIsave[1];
		int NBSTOKESsave[1];
		int DIOPTREsave[1];
		int DIFFFsave[1];
		int PROFILsave[1];
		int SIMsave[1];
		int SURsave[1];
		double nbPhotonsTotSave[1];
		int nbErreursPoidsSave[1];
		int nbErreursThetaSave[1];
		double tempsEcouleSave[1];
	
		NBPHOTONSsave[0] = (double)NBPHOTONS;
		NBLOOPsave[0] = NBLOOP;
		THSDEGsave[0] = THSDEG;
		LAMBDAsave[0] = LAMBDA;
		TAURAYsave[0] = TAURAY;
		TAUAERsave[0] = TAUAER;
		W0AERsave[0] = W0AER;
		HAsave[0] = HA;
		HRsave[0] = HR;
		ZMINsave[0] = ZMIN;
		ZMAXsave[0] = ZMAX;
		WINDSPEEDsave[0] = WINDSPEED;
		NH2Osave[0] = NH2O;
		CONPHYsave[0] = CONPHY;
		XBLOCKsave[0] = XBLOCK;
		YBLOCKsave[0] = YBLOCK;
		XGRIDsave[0] = XGRID;
		YGRIDsave[0] = YGRID;
		NBTHETAsave[0] = NBTHETA;
		NBPHIsave[0] = NBPHI;
		NBSTOKESsave[0] = NBSTOKES;
		DIOPTREsave[0] = DIOPTRE;
		DIFFFsave[0] = DIFFF;
		PROFILsave[0] = PROFIL;
		SIMsave[0] = SIM;
		SURsave[0] = SUR;
		nbPhotonsTotSave[0] = (double)nbPhotonsTot;
		nbErreursPoidsSave[0] = var->erreurpoids;
		nbErreursThetaSave[0] = var->erreurtheta;
		tempsEcouleSave[0] = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
		SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, NBPHOTONSsave);
		SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, NBLOOPsave);
		SDsetattr(sdFichier, "THSDEG", DFNT_FLOAT32, 1, THSDEGsave);
		SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, LAMBDAsave);
		SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, TAURAYsave);
		SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, TAUAERsave);
		SDsetattr(sdFichier, "W0AER", DFNT_FLOAT32, 1, W0AERsave);
		SDsetattr(sdFichier, "HA", DFNT_FLOAT32, 1, HAsave);
		SDsetattr(sdFichier, "HR", DFNT_FLOAT32, 1, HRsave);
		SDsetattr(sdFichier, "ZMIN", DFNT_FLOAT32, 1, ZMINsave);
		SDsetattr(sdFichier, "ZMAX", DFNT_FLOAT32, 1, ZMAXsave);
		SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, WINDSPEEDsave);
		SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, NH2Osave);
		SDsetattr(sdFichier, "CONPHY", DFNT_FLOAT32, 1, CONPHYsave);
		SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, XBLOCKsave);
		SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, YBLOCKsave);
		SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, XGRIDsave);
		SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, YGRIDsave);
		SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, NBTHETAsave);
		SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, NBPHIsave);
		SDsetattr(sdFichier, "NBSTOKES", DFNT_INT32, 1, NBSTOKESsave);
		SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, DIOPTREsave);
		SDsetattr(sdFichier, "DIFFF", DFNT_INT32, 1, DIFFFsave);
		SDsetattr(sdFichier, "PROFIL", DFNT_INT32, 1, PROFILsave);
		SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, SIMsave);
		SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, SURsave);
		SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, nbPhotonsTotSave);
		SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, nbErreursPoidsSave);
		SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, nbErreursThetaSave);
		SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, tempsEcouleSave);
		
		// Ecriture d'informations sur le tableau
		char description[20];
		sprintf(description, "%f", tabPhi[iphi]);
		if(strcmp(description, "") != 0)
			SDsetattr(sdsTab, "phi", DFNT_CHAR8, strlen(description), description);
		
		// Fermeture du tableau
		SDendaccess(sdsTab);
		// Liberation du tableau
		free(tab);
	}
	// Fermeture du fichier
	SDend(sdFichier);
}

// Fonction qui crée le fichier .hdf contenant le résultat final répertorié sur un quart de sphère
void creerHDFResultatsQuartsphere(float* tabFinal, float* tabTh, float* tabPhi,
		unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	char nomFichier[20] = "out_prog/Quart.hdf";
	int sdFichier = SDstart(nomFichier, DFACC_CREATE);
	// Pour chaque phi de la demi-sphère on ajoute au fichier le tableau représentant la moyenne du résultat final des 2 demi-sphères en fonction de theta
	for(int iphi = 0; iphi < NBPHI/2; iphi++)
	{
		// Création et remplissage d'un tableau du fichier
		// tab[ith*2+0] est le résultat final en ith et tab[ith*2+1] est la valeur de theta
		float* tab;
		tab = (float*)malloc(2 * NBTHETA * sizeof(float));
		memset(tab, 0, 2 * NBTHETA * sizeof(float));
		for(int ith = 0; ith < NBTHETA; ith++)
		{
			int jphi = NBPHI - 1 - iphi;
			tab[ith*2+0] = (tabFinal[ith*NBPHI+iphi]+tabFinal[ith*NBPHI+jphi])/2;
			tab[ith*2+1] = tabTh[ith];
		}
		char nomTab[20]; //nom du tableau
		sprintf(nomTab,"Quart (iphi = %d)", iphi);
		int nbDimsTab = 2; //nombre de dimensions du tableau
		int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
		valDimsTab[0] = NBTHETA;
		valDimsTab[1] = 2;
		int typeTab = DFNT_FLOAT32; //type des éléments du tableau
		// Création du tableau
		int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
		int startTab[nbDimsTab]; //début de la lecture du tableau
		startTab[0]=0;
		startTab[1]=0;
		// Ecriture du tableau dans le fichier
		int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tab);
		// Vérification du bon fonctionnement de l'écriture
		if(status)
		{
			printf("\nERREUR : write hdf quart\n");
			exit(1);
		}
		
		// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
		double NBPHOTONSsave[1];
		unsigned int NBLOOPsave[1];
		float THSDEGsave[1];
		float LAMBDAsave[1];
		float TAURAYsave[1];
		float TAUAERsave[1];
		float W0AERsave[1];
		float HAsave[1];
		float HRsave[1];
		float ZMINsave[1];
		float ZMAXsave[1];
		float WINDSPEEDsave[1];
		float NH2Osave[1];
		float CONPHYsave[1];
		int XBLOCKsave[1];
		int YBLOCKsave[1];
		int XGRIDsave[1];
		int YGRIDsave[1];
		int NBTHETAsave[1];
		int NBPHIsave[1];
		int NBSTOKESsave[1];
		int DIOPTREsave[1];
		int DIFFFsave[1];
		int PROFILsave[1];
		int SIMsave[1];
		int SURsave[1];
		double nbPhotonsTotSave[1];
		int nbErreursPoidsSave[1];
		int nbErreursThetaSave[1];
		double tempsEcouleSave[1];
	
		NBPHOTONSsave[0] = (double)NBPHOTONS;
		NBLOOPsave[0] = NBLOOP;
		THSDEGsave[0] = THSDEG;
		LAMBDAsave[0] = LAMBDA;
		TAURAYsave[0] = TAURAY;
		TAUAERsave[0] = TAUAER;
		W0AERsave[0] = W0AER;
		HAsave[0] = HA;
		HRsave[0] = HR;
		ZMINsave[0] = ZMIN;
		ZMAXsave[0] = ZMAX;
		WINDSPEEDsave[0] = WINDSPEED;
		NH2Osave[0] = NH2O;
		CONPHYsave[0] = CONPHY;
		XBLOCKsave[0] = XBLOCK;
		YBLOCKsave[0] = YBLOCK;
		XGRIDsave[0] = XGRID;
		YGRIDsave[0] = YGRID;
		NBTHETAsave[0] = NBTHETA;
		NBPHIsave[0] = NBPHI;
		NBSTOKESsave[0] = NBSTOKES;
		DIOPTREsave[0] = DIOPTRE;
		DIFFFsave[0] = DIFFF;
		PROFILsave[0] = PROFIL;
		SIMsave[0] = SIM;
		SURsave[0] = SUR;
		nbPhotonsTotSave[0] = (double)nbPhotonsTot;
		nbErreursPoidsSave[0] = var->erreurpoids;
		nbErreursThetaSave[0] = var->erreurtheta;
		tempsEcouleSave[0] = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
		SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, NBPHOTONSsave);
		SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, NBLOOPsave);
		SDsetattr(sdFichier, "THSDEG", DFNT_FLOAT32, 1, THSDEGsave);
		SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, LAMBDAsave);
		SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, TAURAYsave);
		SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, TAUAERsave);
		SDsetattr(sdFichier, "W0AER", DFNT_FLOAT32, 1, W0AERsave);
		SDsetattr(sdFichier, "HA", DFNT_FLOAT32, 1, HAsave);
		SDsetattr(sdFichier, "HR", DFNT_FLOAT32, 1, HRsave);
		SDsetattr(sdFichier, "ZMIN", DFNT_FLOAT32, 1, ZMINsave);
		SDsetattr(sdFichier, "ZMAX", DFNT_FLOAT32, 1, ZMAXsave);
		SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, WINDSPEEDsave);
		SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, NH2Osave);
		SDsetattr(sdFichier, "CONPHY", DFNT_FLOAT32, 1, CONPHYsave);
		SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, XBLOCKsave);
		SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, YBLOCKsave);
		SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, XGRIDsave);
		SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, YGRIDsave);
		SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, NBTHETAsave);
		SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, NBPHIsave);
		SDsetattr(sdFichier, "NBSTOKES", DFNT_INT32, 1, NBSTOKESsave);
		SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, DIOPTREsave);
		SDsetattr(sdFichier, "DIFFF", DFNT_INT32, 1, DIFFFsave);
		SDsetattr(sdFichier, "PROFIL", DFNT_INT32, 1, PROFILsave);
		SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, SIMsave);
		SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, SURsave);
		SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, nbPhotonsTotSave);
		SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, nbErreursPoidsSave);
		SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, nbErreursThetaSave);
		SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, tempsEcouleSave);
		
		// Ecriture d'informations sur le tableau
		char description[20];
		sprintf(description, "%f", tabPhi[iphi]);
		if(strcmp(description, "") != 0)
			SDsetattr(sdsTab, "phi", DFNT_CHAR8, strlen(description), description);
		// Fermeture du tableau
		SDendaccess(sdsTab);
		// Liberation du tableau
		free(tab);
	}
	// Fermeture du fichier
	SDend(sdFichier);
}

// Fonction qui crée le fichier .hdf comparant les résultats finaux pour chaque quart de sphère
void creerHDFComparaison(float* tabFinal, float* tabTh, float* tabPhi,
		unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	char nomFichier[50] = "out_prog/Comparaison.hdf";
	int sdFichier = SDstart(nomFichier, DFACC_CREATE);
	// Création du tableau à insérer dans le fichier
	// tab[ith*2+0] est la valeur de theta, tab[ith*2+1] est le résultat final en (ith,iphi), et tab[ith*2+2] est le resultat final en (ith,NBPHI-1-iphi)
	float* tab;
	tab = (float*)malloc(3 * NBTHETA * sizeof(float));
	memset(tab, 0, 3 * NBTHETA * sizeof(float));
	// Pour chaque phi de la demi-sphère on ajoute au fichier le tableau représentant le résultat final du quart de sphère gauche en fonction de theta puis de celui de droite en fonction de theta
	for(int iphi = 0; iphi < NBPHI/2; iphi++)
	{
		// Remplissage du tableau
		for(int ith = 0; ith < NBTHETA; ith++)
		{
			int jphi = NBPHI - 1 - iphi;
			tab[ith*3+0] = tabTh[ith];
			tab[ith*3+1] = tabFinal[ith*NBPHI+iphi];
			tab[ith*3+2] = tabFinal[ith*NBPHI+jphi];
		}
		
		char nomTab[20]; //nom du tableau
		sprintf(nomTab,"Comparaison (iphi = %d)", iphi);
		int nbDimsTab = 2; //nombre de dimensions du tableau
		int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
		valDimsTab[0] = NBTHETA;
		valDimsTab[1] = 3;
		int typeTab = DFNT_FLOAT32; //type des éléments du tableau
		// Création du tableau
		int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
		// Départ de la lecture du tableau
		int startTab[nbDimsTab];
		startTab[0]=0;
		startTab[1]=0;
		// Ecriture du tableau dans le fichier
		int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tab);
		// Vérification du bon fonctionnement de l'écriture
		if(status)
		{
			printf("\nERREUR : write hdf comparaison\n");
			exit(1);
		}
		
		// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
		double NBPHOTONSsave[1];
		unsigned int NBLOOPsave[1];
		float THSDEGsave[1];
		float LAMBDAsave[1];
		float TAURAYsave[1];
		float TAUAERsave[1];
		float W0AERsave[1];
		float HAsave[1];
		float HRsave[1];
		float ZMINsave[1];
		float ZMAXsave[1];
		float WINDSPEEDsave[1];
		float NH2Osave[1];
		float CONPHYsave[1];
		int XBLOCKsave[1];
		int YBLOCKsave[1];
		int XGRIDsave[1];
		int YGRIDsave[1];
		int NBTHETAsave[1];
		int NBPHIsave[1];
		int NBSTOKESsave[1];
		int DIOPTREsave[1];
		int DIFFFsave[1];
		int PROFILsave[1];
		int SIMsave[1];
		int SURsave[1];
		double nbPhotonsTotSave[1];
		int nbErreursPoidsSave[1];
		int nbErreursThetaSave[1];
		double tempsEcouleSave[1];
	
		NBPHOTONSsave[0] = (double)NBPHOTONS;
		NBLOOPsave[0] = NBLOOP;
		THSDEGsave[0] = THSDEG;
		LAMBDAsave[0] = LAMBDA;
		TAURAYsave[0] = TAURAY;
		TAUAERsave[0] = TAUAER;
		W0AERsave[0] = W0AER;
		HAsave[0] = HA;
		HRsave[0] = HR;
		ZMINsave[0] = ZMIN;
		ZMAXsave[0] = ZMAX;
		WINDSPEEDsave[0] = WINDSPEED;
		NH2Osave[0] = NH2O;
		CONPHYsave[0] = CONPHY;
		XBLOCKsave[0] = XBLOCK;
		YBLOCKsave[0] = YBLOCK;
		XGRIDsave[0] = XGRID;
		YGRIDsave[0] = YGRID;
		NBTHETAsave[0] = NBTHETA;
		NBPHIsave[0] = NBPHI;
		NBSTOKESsave[0] = NBSTOKES;
		DIOPTREsave[0] = DIOPTRE;
		DIFFFsave[0] = DIFFF;
		PROFILsave[0] = PROFIL;
		SIMsave[0] = SIM;
		SURsave[0] = SUR;
		nbPhotonsTotSave[0] = (double)nbPhotonsTot;
		nbErreursPoidsSave[0] = var->erreurpoids;
		nbErreursThetaSave[0] = var->erreurtheta;
		tempsEcouleSave[0] = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
		SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, NBPHOTONSsave);
		SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, NBLOOPsave);
		SDsetattr(sdFichier, "THSDEG", DFNT_FLOAT32, 1, THSDEGsave);
		SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, LAMBDAsave);
		SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, TAURAYsave);
		SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, TAUAERsave);
		SDsetattr(sdFichier, "W0AER", DFNT_FLOAT32, 1, W0AERsave);
		SDsetattr(sdFichier, "HA", DFNT_FLOAT32, 1, HAsave);
		SDsetattr(sdFichier, "HR", DFNT_FLOAT32, 1, HRsave);
		SDsetattr(sdFichier, "ZMIN", DFNT_FLOAT32, 1, ZMINsave);
		SDsetattr(sdFichier, "ZMAX", DFNT_FLOAT32, 1, ZMAXsave);
		SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, WINDSPEEDsave);
		SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, NH2Osave);
		SDsetattr(sdFichier, "CONPHY", DFNT_FLOAT32, 1, CONPHYsave);
		SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, XBLOCKsave);
		SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, YBLOCKsave);
		SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, XGRIDsave);
		SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, YGRIDsave);
		SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, NBTHETAsave);
		SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, NBPHIsave);
		SDsetattr(sdFichier, "NBSTOKES", DFNT_INT32, 1, NBSTOKESsave);
		SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, DIOPTREsave);
		SDsetattr(sdFichier, "DIFFF", DFNT_INT32, 1, DIFFFsave);
		SDsetattr(sdFichier, "PROFIL", DFNT_INT32, 1, PROFILsave);
		SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, SIMsave);
		SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, SURsave);
		SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, nbPhotonsTotSave);
		SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, nbErreursPoidsSave);
		SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, nbErreursThetaSave);
		SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, tempsEcouleSave);
		
		// Ecriture d'informations sur le tableau
		char description[20];
		sprintf(description, "%f", tabPhi[iphi]);
		if (strcmp(description, "") != 0) {
			SDsetattr(sdsTab, "phi", DFNT_CHAR8, strlen(description), description);
		}
		// Fermeture du tableau
		SDendaccess(sdsTab);
	}
	// Fermeture du fichier
	SDend(sdFichier);
	// Liberation du tableau
	free(tab);
}

// Fonction qui libère les tableaux envoyés dans le kernel
void freeTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	#ifdef RANDMWC
	// Liberation des tableaux de generateurs du random MWC
	hipFree(tab_D->etat);
	free(tab_H->etat);
	hipFree(tab_D->config);
	free(tab_H->config);
	#endif
	#ifdef RANDCUDA
	// Liberation du tableau de generateurs du random Cuda
	hipFree(tab_D->etat);
	#endif
	#ifdef RANDMT
	// Liberation des tableaux de generateurs du random Mersenen Twister
	hipFree(tab_D->config);
	hipFree(tab_D->etat);
	free(tab_H->config);
	#endif
	// Liberation du tableau du poids des photons
	hipFree(tab_D->tabPhotons);
	free(tab_H->tabPhotons);
}
