#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "communs.h"
#include "host.h"
#include "device.h"

// __constant__ float foce_c[5*10000000];
/**********************************************************
*
*			host.h
*
*	> Initialisation du générateur de nombres aléatoires MWC
*	> Travail sur les fichiers
*	> Initialisation des différentes structures
*	> Calculs de profils
*	> Fonctions d'affichage
*	> Calcul pour sauvegarde des résultats finaux
*	> Fichier hdf (lecture/écriture témoin, écriture résultats)
*
***********************************************************/


/**********************************************************
*	> Initialisation du générateur de nombres aléatoires MWC
***********************************************************/

/* initRandMWC
* Fonction qui initialise les generateurs du random MWC à partir d'un fichier texte
*/

int initRandMWC(unsigned long long *etat, unsigned int *config, 
	     const unsigned int n_rng, const char *safeprimes_file, unsigned long long xinit)
{
	FILE *fp;
	unsigned int begin=0u;
	unsigned int fora,tmp1,tmp2;
	if (strlen(safeprimes_file) == 0)
	{
        // Try to find it in the local directory
		safeprimes_file = "MWC.txt";
	}
	fp = fopen(safeprimes_file, "r");
	if(fp == NULL)
	{
		printf("Could not find the file of safeprimes (%s)! Terminating!\n", safeprimes_file);
		return 1;
	}
	fscanf(fp,"%u %u %u",&begin,&tmp1,&tmp2);
	// Here we set up a loop, using the first multiplier in the file to generate x's and c's
	// There are some restictions to these two numbers:
	// 0<=c<a and 0<=x<b, where a is the multiplier and b is the base (2^32)
	// also [x,c]=[0,0] and [b-1,a-1] are not allowed.
	//Make sure xinit is a valid seed (using the above mentioned restrictions)
	if((xinit == 0ull) | (((unsigned int)(xinit>>32))>=(begin-1)) | (((unsigned int)xinit)>=0xfffffffful))
	{
		//xinit (probably) not a valid seed! (we have excluded a few unlikely exceptions)
		printf("%llu not a valid seed! Terminating!\n",xinit);
		return 1;
	}
	for(unsigned int i=0;i < n_rng;i++)
	{
		fscanf(fp,"%u %u %u",&fora,&tmp1,&tmp2);
		config[i]=fora;
		etat[i]=0;
		while( (etat[i]==0) | (((unsigned int)(etat[i]>>32))>=(fora-1)) | (((unsigned int)etat[i])>=0xfffffffful))
		{
			//generate a random number
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);
			//calculate c and store in the upper 32 bits of x[i]
			etat[i]=(unsigned int) floor((((double)((unsigned int)xinit))/(double)0x100000000)*fora);//Make sure 0<=c<a
			etat[i]=etat[i]<<32;
			//generate a random number and store in the lower 32 bits of x[i] (as the initial x of the generator)
			xinit=(xinit&0xffffffffull)*(begin)+(xinit>>32);//x will be 0<=x<b, where b is the base 2^32
			etat[i]+=(unsigned int) xinit;
		}
		//if(i<10)printf("%llu\n",x[i]);
	}
	fclose(fp);
	return 0;
}


/* initRandMTConfig
* Fonction qui initialise en partie les generateurs du random Mersenen Twister
*/
void initRandMTConfig(ConfigMT* config_H, ConfigMT* config_D, int nbThreads)
{
	// Ouverture du fichier
	const char *fname = "auxdata/MT/MersenneTwister.dat";
	FILE* fd = fopen(fname, "rb");
	if(!fd)
	{
		printf("ERREUR: ouverture fichier MT");
		exit(0);
	}
	// Lecture et initialisation de la config pour chaque generateur (= pour chaque thread)
	for(int i = 0; i < nbThreads; i++)
	{
		/* Le fichier ne contient que 4096 configs, on reutilise donc les memes configs pour les threads en trop mais les nombres
		aléatoires restent independants car les etats des threads sont differents */
		if(i%4096 == 0)
		{
			fseek(fd, 0, 0);
		}
		if(!fread(config_H+i, sizeof(ConfigMT), 1, fd))
		{
			printf("ERREUR: lecture fichier MT");
			exit(0);
		}
	}
	fclose(fd);
	srand((unsigned int)SEED);
	// Creation des seeds aleatoires pour que les threads aient des etats differents
	for(int i = 0; i < nbThreads; i++) config_H[i].seed = (unsigned int)rand();
	
	hipError_t erreur = hipMemcpy(config_D, config_H, nbThreads * sizeof(ConfigMT), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie config_H dans initRandMTConfig\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/**********************************************************
*	> Travail sur les fichiers
***********************************************************/

/* initConstantesHost
* Fonction qui récupère les valeurs des constantes dans le fichier paramètres et initialise les constantes du host
*/
void initConstantesHost(int argc, char** argv)
{
	if(argc < 2)
	{
		printf("ERREUR : lecture argv\n");
		exit(1);
	}
	
	FILE* parametres = fopen( argv[1], "r" );
	
	if( parametres == NULL ){
		printf("ERREUR: Impossible d'ouvrir le fichier %s\n", argv[1] );
		exit(1);
	}
	
	char s[256];

	strcpy(s,"");
	chercheConstante( parametres, "NBPHOTONS", s);
	NBPHOTONS = strtoull(s, NULL, 10);
	
	strcpy(s,"");
	chercheConstante(parametres, "NBLOOP", s);
	NBLOOP = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SEED", s);
	SEED = atoi(s);
	if(SEED == -1) SEED = static_cast<int> (time(NULL));

	strcpy(s,"");
	chercheConstante(parametres, "XBLOCK", s);
	XBLOCK= atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YBLOCK", s);
	YBLOCK = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "XGRID", s);
	XGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "YGRID", s);
	YGRID = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBTHETA", s);
	NBTHETA = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "NBPHI", s);
	NBPHI = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "PROFIL", s);
	PROFIL = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SIM", s);
	SIM = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "SUR", s);
	SUR = atoi(s);

	strcpy(s,"");
	chercheConstante(parametres, "DIOPTRE", s);
	DIOPTRE= atoi(s);
	

	if( SIM!=-2 ) DIFFF = 0;
	else{
		strcpy(s,"");
		chercheConstante(parametres, "DIFFF", s);
		DIFFF = atoi(s);
	}
	
	strcpy(s,"");
	chercheConstante(parametres, "THSDEG", s);
	THSDEG = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "LAMBDA", s);
	LAMBDA = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "TAURAY", s);
	TAURAY = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "TAUAER", s);
	TAUAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "W0AER", s);
	W0AER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "W0LAM", s);
	W0LAM = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "LSAAER", s);
	LSAAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NFAER", s);
	NFAER = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "LSAOCE", s);
	LSAOCE = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NFOCE", s);
	NFOCE = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HA", s);
	HA = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HR", s);
	HR = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "ZMIN", s);
	ZMIN = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "ZMAX", s);
	ZMAX = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NATM", s);
	NATM = atoi(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "HATM", s);
	HATM = atoi(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "WINDSPEED", s);
	WINDSPEED = atof(s);
	
	strcpy(s,"");
	chercheConstante(parametres, "NH2O", s);
	NH2O = atof(s);
	
    #ifdef FLAGOCEAN
	strcpy(s,"");
	chercheConstante(parametres, "CONPHY", s);
	CONPHY = atof(s);
    #endif
	
	chercheConstante(parametres, "PATHRESULTATSHDF", PATHRESULTATSHDF);
    
    // nom du fichier témoin
    strcpy(PATHTEMOINHDF, PATHRESULTATSHDF);
    strcat(PATHTEMOINHDF, ".temoin");
	
	chercheConstante( parametres, "PATHDIFFAER", PATHDIFFAER );
	
	chercheConstante( parametres, "PATHPROFILATM", PATHPROFILATM );
	
	fclose( parametres );
}


/* chercheConstante
* Fonction qui cherche nomConstante dans le fichier et met la valeur de la constante dans chaineValeur (en string)
*/
void chercheConstante(FILE* fichier, char* nomConstante, char* chaineValeur)
{
	int longueur = strlen(nomConstante);
	char ligne[100];
	int motTrouve = 0;
	
	// Tant que la constante n'est pas trouvee et qu'on n'est pas à la fin du fichier on lit la ligne
	while(fgets(ligne, 100, fichier) && !motTrouve)
	{
		// Si le debut de la ligne est nomConstante suivi d'un espace ou un egal on va chercher la valeur
		if((strncmp(ligne, nomConstante, longueur) == 0) && (ligne[longueur] == ' ' || ligne[longueur] == '='))
		{
			char* ptr = ligne; //pointeur du debut de la ligne
			// on avance jusqu'au prochain espace ou egal
			while (*ptr != ' ' && *ptr != '=') ptr++;
			// on avance jusqu'à la valeur de la constante
			while(*ptr == ' ' || *ptr == '=') ptr++;
			if (*ptr == '\n')
			{
				printf("ERREUR : lecture Parametre.txt\n");
				exit(1);
			}
			// On met la chaine de la valeur de la constante dans chaineValeur
			strcpy(chaineValeur, ptr);
			chaineValeur[strlen(chaineValeur)-1] = '\0';
			motTrouve = 1;
		}
	}
	rewind(fichier);
	
	if(motTrouve == 0)
	{
		printf("ERREUR : lecture fichier paramètres - Constante manquante: %s\n",nomConstante);
		exit(1);
	}
}


/* verifierFichier
* Fonction qui vérifie l'état des fichiers temoin et résultats
* Demande à l'utilisateur s'il veut les supprimer ou non
*/
void verifierFichier(){
	char command[256];
	char res_supp='n';
	// S'il existe déjà un fichier nommé NOMRESULTATSHDF (Parametres.txt) on arrête le programme
	FILE* fic;
	fic = fopen(PATHTEMOINHDF, "rb");
	if ( fic != NULL)
	{
		printf("ATTENTION: Le fichier temoin %s existe deja.\n",PATHTEMOINHDF);
		printf("Voulez-vous le supprimer? [y/n]\n");
		while(1){
 			res_supp=getchar();
 			getchar();
			if( res_supp=='y' ){
				sprintf(command,"rm %s",PATHTEMOINHDF);
				system(command);
				break;
			}
			else if( res_supp=='n' ){
				break;
			}
			else{
				printf("Retapez votre choix SVP.\n");
			}
			
		}
		fclose(fic);
	}
	
	
// 	getchar();

// 	fic = fopen(PATHRESULTATSHDF, "rb");
// 	if ( fic != NULL)
// 	{
// 		printf("ATTENTION: Le fichier resultat %s existe deja.\n",PATHRESULTATSHDF);
// 		printf("Voulez-vous le supprimer pour continuer? [y/n]\n");
		// 		res_supp=getchar();
		// 		if( res_supp=='y' ){
//    sprintf(command,"rm %s",PATHRESULTATSHDF);
//    system(command);
   // 		}
// 	   fclose(fic);
// 	}
	
	
}


/**********************************************************
*	> Initialisation des différentes structures
***********************************************************/

/* initVariables
* Fonction qui initialise les variables à envoyer dans le kernel.
*/
void initVariables(Variables** var_H, Variables** var_D)
{
	// 	Initialisation de la version host des variables
	*var_H = (Variables*)malloc(sizeof(Variables));
	if( var_H == NULL ){
		printf("#--------------------#\n");
		printf("ERREUR: Problème de malloc de var_H dans initVariables\n");
		printf("#--------------------#\n");
		exit(1);
	}
	memset(*var_H, 0, sizeof(Variables));
	
	//	Initialisation de la version device des variables
	if( hipMalloc(var_D, sizeof(Variables)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de var_D dans initVariables\n");
		exit(1);
	}
	
	hipError_t err = hipMemset(*(var_D), 0, sizeof(Variables));
	if( err != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de hipMemset var_D dans initVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
		printf("#--------------------#\n");
		exit(1);
	}

}


/* reinitVariables
* Fonction qui réinitialise certaines variables avant chaque envoi dans le kernel
*/
void reinitVariables(Variables* var_H, Variables* var_D)
{
	// Le nombre de photons traités pour un appel du Kernel est remis à zéro
	var_H->nbPhotons = 0;
	#ifdef PROGRESSION
	// Le nombre de photons ressortis pour un appel du Kernel est remis à zéro
	var_H->nbPhotonsSor = 0;
	#endif
	// On copie le nouveau var_H dans var_D
	hipError_t erreur = hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie var_H dans reinitVariables\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("# sizeof(*var_D)=%d\tsizeof(*var_H)=%d\tsizeof(*Variables)=%d\n",sizeof(*var_D),sizeof(*var_H),sizeof(Variables));
		printf("# Adresse de var_D : %p\tAdresse de var_H : %p\n", var_H, var_D);
		printf("#--------------------#\n");
		exit(1);
	}
}


#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
/* initInit
* Initialisation de la structure Init contenant les paramètres initiaux du photon rentrant dans l'atmosphère.
* Ces paramètres sont utiles pour une atmosphère sphérique et sont calculés une seule fois dans le host, d'où cette fonction
* et la structure Init
*/
void initInit(Init** init_H, Init** init_D)
{
	// 	Initialisation de la version host des variables
	*init_H = (Init*)malloc(sizeof(Init));
	if( init_H == NULL ){
	printf("#--------------------#\n");
	printf("ERREUR: Problème de malloc de init_H dans initInit\n");
	printf("#--------------------#\n");
	exit(1);
	}
	memset(*init_H, 0, sizeof(Init));
   
	//	Initialisation de la version device des variables
	if( hipMalloc(init_D, sizeof(Init)) == hipErrorOutOfMemory ){
	   printf("ERREUR: Problème de hipMalloc de init_D dans initInit\n");
	   exit(1);
	}

	hipError_t err = hipMemset(*(init_D), 0, sizeof(Init));
	if( err != hipSuccess ){
	   printf("#--------------------#\n");
	   printf("# ERREUR: Problème de hipMemset init_D dans initInit\n");
	   printf("# Nature de l'erreur: %s\n",hipGetErrorString(err) );
	   printf("#--------------------#\n");
	   exit(1);
	}

}
#endif


/* initTableaux
* Fonction qui initialise les tableaux à envoyer dans le kernel par allocation mémoire et memset
*/
void initTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	hipError_t cudaErreur;	// Permet de tester les erreurs d'allocation mémoire
	
	#ifdef RANDMWC	
	// Création des tableaux de generateurs pour la fonction Random MWC
	tab_H->etat = (unsigned long long*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (unsigned int*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int)) == hipErrorOutOfMemory){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation des tableaux host à l'aide du fichier et du seed
	initRandMWC(tab_H->etat, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID, "auxdata/MWC/MWC.txt", (unsigned long long)SEED);
	
	// Copie dans les tableaux device
	cudaErreur = hipMemcpy(tab_D->etat, tab_H->etat, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned long long), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->etat dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	
	cudaErreur = hipMemcpy(tab_D->config, tab_H->config, XBLOCK * YBLOCK * XGRID * YGRID * sizeof(unsigned int), hipMemcpyHostToDevice);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->config dans initTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	#endif
	
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
	// Création du tableau de generateurs (=etat+config) pour la fonction Random Cuda
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(curandSTATE)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	// Initialisation du tableau dans une fonction du kernel
        #if defined(RANDCUDA)
	initRandCUDA<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, (unsigned long long)SEED);
        #else
        unsigned long long NbThreads = XGRID * YGRID * XBLOCK * YBLOCK;
        unsigned long long NbDimensions = NbThreads > 20000 ? 20000 : NbThreads;
        hiprandDirectionVectors32_t *d_qrngDirections = 0;
        cudaErreur = hipMalloc((void **)&d_qrngDirections, NbDimensions*sizeof(hiprandDirectionVectors32_t));
        if( cudaErreur != hipSuccess ){
            printf( "ERREUR: Problème d'allocation de d_qrngDirections\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
            exit(1);
        }
        hiprandDirectionVectors32_t *h_rngDirections;
        hiprandGetDirectionVectors32(&h_rngDirections, HIPRAND_SCRAMBLED_DIRECTION_VECTORS_32_JOEKUO6);
        cudaErreur = hipMemcpy(d_qrngDirections, h_rngDirections, NbDimensions*sizeof(hiprandDirectionVectors32_t), hipMemcpyHostToDevice);
        if( cudaErreur != hipSuccess ){
            printf( "ERREUR: Problème de copie h_rngDirections dans d_qrngDirections\n");
            printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
            exit(1);
        }
        initRandCUDANDQRNGs<<< dim3(XGRID,YGRID,1), dim3(XBLOCK,YBLOCK,1)>>>(tab_D->etat, d_qrngDirections);
        #endif
	#endif
	
	#ifdef RANDMT
	// Création des tableaux de generateurs pour la fonction Random Mersenne Twister
	if( hipMalloc(&(tab_D->config), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->config dans initTableaux\n");
		exit(1);	
	}
	
	if( hipMalloc(&(tab_D->etat), XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT)) == hipErrorOutOfMemory ){
		printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->config = (ConfigMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(ConfigMT));
	if( tab_H->config == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->config dans initTableaux\n");
		exit(1);
	}
	
	tab_H->etat = (EtatMT*)malloc(XBLOCK * YBLOCK * XGRID * YGRID * sizeof(EtatMT));
	if( tab_H->etat == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->etat dans initTableaux\n");
		exit(1);
	}
		
	// Initialisation du tableau des configs à l'aide du fichier
	initRandMTConfig(tab_H->config, tab_D->config, XBLOCK * YBLOCK * XGRID * YGRID);
	// Initialisation du tableau des etats dans le kernel
	initRandMTEtat<<<XGRID * YGRID, XBLOCK * YBLOCK>>>(tab_D->etat, tab_D->config);
	#endif

        #ifdef RANDPHILOX4x32_7
        //Memset de deux valeurs pour la creation des generateurs philox
        unsigned int compteur_init = 0;
        unsigned int clef_utilisateur = (unsigned int) SEED; /*LDS: eventuellement la conversion ici change la graine mais c'est sans reelle importance il me semble*/
        tab_D->config = clef_utilisateur;
        cudaErreur = hipMalloc((void**)&(tab_D->etat), sizeof(unsigned int)* XBLOCK * YBLOCK * XGRID * YGRID);
        if( cudaErreur != hipSuccess){
            printf("ERREUR: Problème de hipMalloc de tab_D->etat dans initTableaux\n");
            printf("\t->detail de l'erreur : %s\n", hipGetErrorString(cudaErreur));
            exit(1);
        }
        initPhilox4x32_7Compteur<<<dim3(XGRID,YGRID,1), dim3(XBLOCK,YBLOCK,1)>>>(tab_D->etat, compteur_init);
        #endif
	
	// Tableau du poids des photons ressortis
	tab_H->tabPhotons = (float*)malloc(4*NBTHETA * NBPHI * sizeof(*(tab_H->tabPhotons)));
	if( tab_H->tabPhotons == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->tabPhotons dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->tabPhotons,0,4*NBTHETA * NBPHI * sizeof(*(tab_H->tabPhotons)) );
	
	if( hipMalloc(&(tab_D->tabPhotons), 4 * NBTHETA * NBPHI * sizeof(*(tab_D->tabPhotons))) != hipSuccess){
		printf("ERREUR: Problème de hipMalloc de tab_D->tabPhotons dans initTableaux\n");
		exit(1);	
	}
	
	cudaErreur = hipMemset(tab_D->tabPhotons, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D->tabPhotons)));
	if( cudaErreur != hipSuccess ){
	printf("#--------------------#\n");
	printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le initTableaux\n");
	printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
	printf("#--------------------#\n");
	exit(1);
	}
	
	/** Modèle de diffusion **/
	// Modèle de diffusion des aérosols
	tab_H->faer = (float*)malloc(5 * NFAER * sizeof(float));
	if( tab_H->faer == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->faer dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->faer,0,5 * NFAER*sizeof(float) );
	
	if( hipMalloc(&(tab_D->faer), 5 * NFAER * sizeof(float)) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->faer dans initTableaux\n");
		exit(1);	
	}
	
	#ifdef FLAGOCEAN
	// Modèle de diffusion dans l'océan
	tab_H->foce = (float*)malloc(5 * NFOCE * sizeof(float));
	if( tab_H->foce == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->foce dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->foce,0,5 * NFOCE*sizeof(float) );
	
	if( hipMalloc(&(tab_D->foce), 5 * NFOCE * sizeof(float)) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->foce dans initTableaux\n");
		exit(1);	
	}
	#endif
	
	
	/** Modèle de l'atmosphère **/
	// Epaisseur optique par couche
	tab_H->h =  (float*)malloc((NATM+1)*sizeof(*(tab_H->h)));
	if( tab_H->h == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->h dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->h,0,(NATM+1)*sizeof(*(tab_H->h)) );
	
	if( hipMalloc( &(tab_D->h), (NATM+1)*sizeof(*(tab_H->h)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->h dans initTableaux\n");
		exit(1);	
	}
	
	// Proportion moléculaire par couche
	tab_H->pMol =  (float*)malloc((NATM+1)*sizeof(float));
	if( tab_H->pMol == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->pMol dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->pMol,0,(NATM+1)*sizeof(float) );
	
	if( hipMalloc( &(tab_D->pMol), (NATM+1)*sizeof(float) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->pMol dans initTableaux\n");
		exit(1);	
	}
	
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	
	// Altitude des couches
	tab_H->z =  (float*)malloc((NATM+1)*sizeof(*(tab_H->z)));
	if( tab_H->z == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->z dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->z,0,(NATM+1)*sizeof(*(tab_H->z)) );
	
	if( hipMalloc( &(tab_D->z), (NATM+1)*sizeof(*(tab_H->z)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->z dans initTableaux\n");
		exit(1);	
	}
	
	/** Profil initial vu par le photon **/
	tab_H->zph0 =  (float*)malloc((NATM+1)*sizeof(*(tab_H->zph0)));
	if( tab_H->zph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->zph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->zph0,0,(NATM+1)*sizeof(*(tab_H->zph0)) );
	
	if( hipMalloc( &(tab_D->zph0), (NATM+1)*sizeof(*(tab_D->zph0)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->zph0 dans initTableaux\n");
		exit(1);	
	}
	
	tab_H->hph0 =  (float*)malloc((NATM+1)*sizeof(*(tab_H->hph0)));
	if( tab_H->hph0 == NULL ){
		printf("ERREUR: Problème de malloc de tab_H->hph0 dans initTableaux\n");
		exit(1);
	}
	memset(tab_H->hph0,0,(NATM+1)*sizeof(*(tab_H->hph0)) );
	
	if( hipMalloc( &(tab_D->hph0), (NATM+1)*sizeof(*(tab_D->hph0)) ) != hipSuccess ){
		printf("ERREUR: Problème de hipMalloc de tab_D->hph0 dans initTableaux\n");
		exit(1);	
	}
	#endif
	
}


/* freeTableaux
* Fonction qui libère l'espace mémoire de tous les tableaux alloués
*/
void freeTableaux(Tableaux* tab_H, Tableaux* tab_D)
{
	
	hipError_t erreur;	// Permet de tester le bon déroulement des hipFree
	
	#ifdef RANDMWC
	// Liberation des tableaux de generateurs du random MWC
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->etat);
	
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	#endif
	
        #if defined(RANDCUDA) || defined (RANDCURANDSOBOL32) || defined (RANDCURANDSCRAMBLEDSOBOL32)
	// Liberation du tableau de generateurs du random Cuda
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif
	
	#ifdef RANDMT
	// Liberation des tableaux de generateurs du random Mersenen Twister
	erreur = hipFree(tab_D->config);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->etat dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->config);
	free(tab_H->etat);
	#endif
	
        #ifdef RANDPHILOX4x32_7
	// Liberation du tableaux de compteurs des philox
	erreur = hipFree(tab_D->etat);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->config dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	#endif

	// Liberation du tableau du poids des photons
	erreur = hipFree(tab_D->tabPhotons);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->tabPhotons dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	// 	hipHostFree(tab_H->tabPhotons);
	free(tab_H->tabPhotons);
	
	/** Modèles de diffusion **/
	// Libération du modèle de diffusion des aérosols
	erreur = hipFree(tab_D->faer);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->faer dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	free(tab_H->faer);
	
	#ifdef FLAGOCEAN
	// Diffusion dans l'océan
	erreur = hipFree(tab_D->foce);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->foce dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	free(tab_H->foce);
	#endif
	
	
	/** Profil amosphèrique **/	
	// Libération du modèle atmosphérique
	erreur = hipFree(tab_D->h);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->h dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->h);
	
	//
	erreur = hipFree(tab_D->pMol);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->pMol dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->pMol);
	
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	
	// Altitude des couches
	erreur = hipFree(tab_D->z);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->z dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->z);
	
	// Profil initial vu par la photon
	erreur = hipFree(tab_D->zph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->zph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->zph0);
	
	erreur = hipFree(tab_D->hph0);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de hipFree de tab_D->hph0 dans freeTableaux\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	free(tab_H->hph0);
	
	#endif
	
}


/**********************************************************
*	> Calculs de profils
***********************************************************/

/* calculFaer
* Calcul de la fonction de phase des aérosols
*/
void calculFaer( const char* nomFichier, Tableaux* tab_H, Tableaux* tab_D ){
	
	FILE* fichier = fopen(nomFichier, "r");

	double *scum = (double*) malloc(LSAAER*sizeof(*scum));
	if( scum==NULL ){
		printf("ERREUR: Problème de malloc de scum dans calculFaer\n");
		exit(1);
	}
	
	scum[0] = 0;
	int iang = 0, ipf = 0;
	double dtheta, pm1, pm2, sin1, sin2;
	double z, norm;

	/** Allocation de la mémoire des tableaux contenant les données **/
	double *ang;
	double *p1, *p2, *p3, *p4;
	ang = (double*) malloc(LSAAER*sizeof(*ang));
	p1 = (double*) malloc(LSAAER*sizeof(*p1));
	p2 = (double*) malloc(LSAAER*sizeof(*p2));
	p3 = (double*) malloc(LSAAER*sizeof(*p3));
	p4 = (double*) malloc(LSAAER*sizeof(*p4));
	if( ang==NULL || p1==NULL || p2==NULL || p3==NULL || p4==NULL ){
		printf("ERREUR: Problème de malloc de ang ou pi dans calculFaer\n");
		exit(1);
	}
	
	/** Lecture des données sur le modèle de diffusion des aérosols **/
	if(fichier == NULL){
		printf("ERREUR : Ouverture impossible du fichier %s pour la diffusion d'aérosol", nomFichier );
		exit(1);
	}
	
	else{
		for(iang=0; iang<LSAAER; iang++){
			fscanf(fichier, "%lf\t%lf\t%lf\t%lf\t%lf", ang+iang,p2+iang,p1+iang,p3+iang,p4+iang );
			// Conversion en radians
			ang[iang] = ang[iang]*DEG2RAD;
		}
	}
	
	if(fclose(fichier) == EOF){
		printf("ERREUR : Probleme de fermeture du fichier %s", nomFichier);
	}
		
	/** Calcul de scum **/
	for(iang=1; iang<LSAAER; iang++){
		
		dtheta = ang[iang] - ang[iang-1];
		pm1= p1[iang-1] + p2[iang-1];
		pm2= p1[iang] + p2[iang];
		sin1= sin(ang[iang-1]);
		sin2= sin(ang[iang]);
		
		scum[iang] = scum[iang-1] + dtheta*( (sin1*pm1+sin2*pm2)/3 + (sin1*pm2+sin2*pm1)/6 )*DEUXPI; 
	}
	
	// Normalisation
	for(iang=0; iang<LSAAER; iang++){
		scum[iang] = scum[iang]/scum[LSAAER-1];
	}
	
	/** Calcul des faer **/
	for(iang=0; iang<NFAER-1; iang++){
		z = double(iang+1)/double(NFAER);
		while( (scum[ipf+1]<z) && ipf<(LSAAER-1) )
			ipf++;
		
		tab_H->faer[iang*5+4] = float( ((scum[ipf+1]-z)*ang[ipf] + (z-scum[ipf])*ang[ipf+1])/(scum[ipf+1]-scum[ipf]) );
		norm = p1[ipf]+p2[ipf];			// Angle
		tab_H->faer[iang*5+0] = float( p1[ipf]/norm );	// I paralèlle
		tab_H->faer[iang*5+1] = float( p2[ipf]/norm );	// I perpendiculaire
		tab_H->faer[iang*5+2] = float( p3[ipf]/norm );	// u
		tab_H->faer[iang*5+3] = 0.F;			// v, toujours nul
	}
	
	tab_H->faer[(NFAER-1)*5+4] = PI;
	tab_H->faer[(NFAER-1)*5+0] = 0.5F+00;
	tab_H->faer[(NFAER-1)*5+1] = 0.5F+00;
	tab_H->faer[(NFAER-1)*5+2] = float( p3[LSAAER-1]/(p1[LSAAER-1]+p2[LSAAER-1]) );
	tab_H->faer[(NFAER-1)*5+3] = 0.F+00;
	
	free(scum);
	free(ang);
	free(p1);
	free(p2);
	free(p3);
	free(p4);
	
	/** Allocation des FAER dans la device memory **/		

	hipError_t erreur = hipMemcpy(tab_D->faer, tab_H->faer, 5*NFAER*sizeof(*(tab_H->faer)), hipMemcpyHostToDevice); 
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->faer dans calculFaer\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
}


/* verificationFAER
* Sauvegarde la fonction de phase des aérosols calculée dans un fichier
* Permet de valider le bon calcul de la fonction de phase
*/
void verificationFAER( const char* nomFichier, Tableaux tab){

	FILE* fichier = fopen(nomFichier, "w");
	int i;
	
	fprintf( fichier, "angle\tI//\tIp\n" );
	
	for(i=0; i<NFAER; i++){
		fprintf(fichier, "%f\t%f\t%f\n", tab.faer[i*5+4],tab.faer[i*5+0], tab.faer[i*5+1]);
	}
	
	fclose(fichier);

}


#ifdef FLAGOCEAN
/* calculFoce
* Calcul de la fonction de phase dans l'océan
*/
void calculFoce( Tableaux* tab_H, Tableaux* tab_D ){

	/** Déclaration **/
	// Données utiles pour le calcul
	double lamb0[NWAV]={350., 355., 360., 365, 370., 375, 380., 385, 390., 395, 400., 405, 410., 415, 420., 425, 430., 435, 440.,
						445, 450., 455, 460., 465, 470., 475, 480., 485, 490., 495, 500., 505, 510., 515, 520., 525, 
						530., 535, 540., 545, 550., 555, 560., 565, 570., 575, 580., 585, 590., 595, 600., 605, 610., 615, 620.,
						625, 630., 635, 640., 645, 650., 655, 660., 665, 670., 675, 680., 685, 690., 695, 700.};

	double ah2o[NWAV]={	0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.01137, 0.00941, 0.00851, 0.00813, 0.00663, 0.0053, 0.00473, 
						0.00444, 0.00454, 0.00478, 0.00495, 0.0053,	0.00635, 0.00751, 0.00922, 0.00962, 0.00979, 
						0.01011, 0.0106, 0.0114, 0.0127, 0.0136, 0.015, 0.0173, 0.0204, 0.0256, 0.0325, 0.0396, 
						0.0409, 0.0417, 0.0434, 0.0452, 0.0474, 0.0511, 0.0565, 0.0596, 0.0619, 0.0642, 0.0695, 
						0.0772, 0.0896, 0.11, 0.1351, 0.1672, 0.2224, 0.2577, 0.2644, 0.2678, 0.2755, 0.2834, 
						0.2916, 0.3012, 0.3108, 0.325, 0.34, 0.371, 0.41, 0.429 , 0.439, 0.448, 0.465, 0.486,
						0.516, 0.559, 0.624 };

     double A_bricaud95[NWAV]={0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 
							  0.0263, 0.0285, 0.0313, 0.03375, 0.0356, 0.03655, 0.0386, 0.0397, 0.0403, 0.03865, 
							  0.0371, 0.0356, 0.035, 0.0341, 0.0332, 0.0315, 0.0301, 0.02875, 0.0274, 0.02535, 
							  0.023, 0.0204, 0.018, 0.01595, 0.0143, 0.01285, 0.0117, 0.0106, 0.0097, 0.0088 , 
							  0.008, 0.007, 0.0062, 0.0056, 0.0053, 0.0052, 0.0053, 0.0055, 0.0056, 0.0056 , 
							  0.0054, 0.0055, 0.0057, 0.0061, 0.0065, 0.00675, 0.0071, 0.00745, 0.0077, 0.00795, 
							  0.0083, 0.0092, 0.0115, 0.01525, 0.0189, 0.0201, 0.0182, 0.01345, 0.0083, 0.0049, 0.003 };

	double B_bricaud95[NWAV]={0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 0.0, 
							  0.282, 0.2815, 0.283 , 0.292 , 0.299 , 0.3145, 0.314 , 0.326 , 0.332 , 0.3515, 
							  0.359, 0.3665, 0.365 , 0.3685, 0.368 , 0.3765, 0.377 , 0.3745, 0.361 , 0.345 , 
							  0.321, 0.294, 0.26  , 0.2305, 0.196 , 0.1675, 0.139 , 0.114 , 0.09  , 0.0695, 
							  0.052, 0.0315, 0.016 , 0.0085, 0.005 , 0.02  , 0.035 , 0.053 , 0.073 , 0.0905, 
							  0.092, 0.084, 0.071 , 0.0645, 0.064 , 0.0725, 0.078 , 0.086 , 0.098 , 0.116 , 
							  0.124, 0.123, 0.121 , 0.134 , 0.149 , 0.158 , 0.155 , 0.131 , 0.086 , 0.0285, -0.034 };

	double ee[NWAV]={0.77800, 0.76700, 0.75600, 0.73700, 0.72000, 0.70000, 0.68500, 0.67300, 0.67000, 0.66000,
					  0.64358, 0.64776, 0.65175, 0.65555, 0.65917, 0.66259, 0.66583, 0.66889, 0.67175, 0.67443,
					  0.67692, 0.67923, 0.68134, 0.68327, 0.68501, 0.68657, 0.68794, 0.68903, 0.68955, 0.68947,
					  0.68880, 0.68753, 0.68567, 0.68320, 0.68015, 0.67649, 0.67224, 0.66739, 0.66195, 0.65591,
					  0.64927, 0.64204, 0.64000, 0.63000, 0.62300, 0.61500, 0.61000, 0.61400, 0.61800, 0.62200,
					  0.62600, 0.63000, 0.63400, 0.63800, 0.64200, 0.64700, 0.65300, 0.65800, 0.66300, 0.66700,
					  0.67200, 0.67700, 0.68200, 0.68700, 0.69500, 0.69700, 0.69300, 0.66500, 0.64000,0.62000,0.60000 };

	double Chi[NWAV]={0.15300, 0.14900, 0.14400, 0.14000, 0.13600, 0.13100, 0.12700, 0.12300, 0.11900, 0.11800,
					  0.11748, 0.12066, 0.12259, 0.12326, 0.12269, 0.12086, 0.11779, 0.11372, 0.10963, 0.10560,
					  0.10165, 0.09776, 0.09393, 0.09018, 0.08649, 0.08287, 0.07932, 0.07584, 0.07242, 0.06907,
					  0.06579, 0.06257, 0.05943, 0.05635, 0.05341, 0.05072, 0.04829, 0.04611, 0.04419, 0.04253,
					  0.04111, 0.03996, 0.03900, 0.03750, 0.03600, 0.03400, 0.03300, 0.03280, 0.03250, 0.03300,
					  0.03400, 0.03500, 0.03600, 0.03750, 0.03850, 0.04000, 0.04200, 0.04300, 0.04400, 0.04450,
					  0.04500, 0.04600, 0.04750, 0.04900, 0.05150, 0.05200, 0.05050, 0.04400, 0.03900,0.03400,0.03000 };

	double Kw[NWAV]={0.02710, 0.02380, 0.02160, 0.01880, 0.01770, 0.01595, 0.01510, 0.01376, 0.01271, 0.01208,
					  0.01042, 0.00890, 0.00812, 0.00765, 0.00758, 0.00768, 0.00770, 0.00792, 0.00885, 0.00990,
					  0.01148, 0.01182, 0.01188, 0.01211, 0.01251, 0.01320, 0.01444, 0.01526, 0.01660, 0.01885,
					  0.02188, 0.02701, 0.03385, 0.04090, 0.04214, 0.04287, 0.04454, 0.04630, 0.04846, 0.05212,
					  0.05746, 0.06053, 0.06280, 0.06507, 0.07034, 0.07801, 0.09038, 0.11076, 0.13584, 0.16792,
					  0.22310, 0.25838, 0.26506, 0.26843, 0.27612, 0.28400, 0.29218, 0.30176, 0.31134, 0.32553,
					  0.34052, 0.37150, 0.41048, 0.42947, 0.43946, 0.44844, 0.46543, 0.48642, 0.51640,0.55939,0.62438 };


	int ilambda, iang, ipf;
	double a0, b0, a1, b1, a2, b2;	// Coefficients d'absorption et de diffusion
	double r1;
	double anap440, anap, aphi;
	double bb1, g1, g2;				// Coefficients liés à la fonction Henyey greenstein
	
	double integ_ff;
	double rat1;				// Utilisé pour la troncature de la fonction de phase
	double extoce;			//
	double atot, btot;
	double Kd;
	
	double delta;
	double dtheta;
	double pm1, pm2;				// Variable intermédiaire de calcul
	double sin1, sin2;				// Variable intermédiaire de calcul
	
	double z, norm, v;
	
	double* scum;
	scum = (double*) malloc(NFOCE*sizeof(*scum));
	if( scum==NULL){
		printf("Probleme d'allocation de scum dans calculFoce\n");
		exit(1);
	}
	
	double* ang;
	ang = (double*) malloc(NFOCE*sizeof(*ang));
	if( ang==NULL){
		printf("Probleme d'allocation de ang dans calculFoce\n");
		exit(1);
	}
	
	double* pf = (double*) malloc(4*LSAOCE*sizeof(*pf));
 	/* pf[iang + i] donne accès aux nombre de stokes i+1 pour l'indice d'angle iang */
 	if( pf==NULL){
		printf("Probleme d'allocation de pf dans calculFoce\n");
		exit(1);
 	}
	
	double* pf0 = (double*) malloc(4*LSAOCE*sizeof(*pf0));
	/* pf0[iang + i] donne accès aux nombre de stokes i+1 pour l'indice d'angle iang */
	if( pf0==NULL){
		printf("Probleme d'allocation de pf0 dans calculFoce\n");
		exit(1);
	}
	
	double* pf1 = (double*) malloc(4*LSAOCE*sizeof(*pf1));
	/* pf1[iang + i] donne accès aux nombre de stokes i+1 pour l'indice d'angle iang */
	if( pf1==NULL){
		printf("Probleme d'allocation de pf1 dans calculFoce\n");
		exit(1);
	}
	
	double* pf2 = (double*) malloc(4*LSAOCE*sizeof(*pf2));
	/* pf2[iang + i] donne accès aux nombre de stokes i+1 pour l'indice d'angle iang */
	if( pf2==NULL){
		printf("Probleme d'allocation de pf2 dans calculFoce\n");
		exit(1);
	}
	
	/** Calculs **/
	ilambda = int( (LAMBDA - lamb0[0])/(lamb0[1]-lamb0[0]) );
	if( ilambda < 0 ){
		printf("Lambda est out of range");
		exit(1);
	}
	ilambda = min( ilambda, NWAV-1 );
	
	// Coefficients pour l'eau
	a0 = ah2o[ilambda];
	b0 = 19.3e-4*pow(LAMBDA/550.,-4.3);

	// Coefficients d'absorption et de diffusion pour le phytoplancton
	anap440 = 0.0124*pow(CONPHY,0.724);
	anap = anap440*exp( -0.011*(LAMBDA-440) );
	aphi = A_bricaud95[ilambda]*pow( CONPHY,1.-B_bricaud95[ilambda] );
	a1 = anap + aphi;
	b1 = 0.416*pow( CONPHY,0.766 )*550/LAMBDA;
	
	// Backscatterring part
	if( CONPHY<2 ){
		v = 0.5*( log10(CONPHY) - 0.3 );
	}
	else{
		v = 0;
	}

	bb1 = 0.002 + 0.01*( 0.5-0.25*log10(CONPHY))*pow(LAMBDA/550,v);
	r1 = (bb1 - 0.002)/0.028;
	g1 = 1 - (2*bb1)/(bb1 + 0.414);
	
	// Coefficients pour les 2ème particules
	a2 = 0;
	b2 = 0;
	g2 = 0.9;
	
	
	/* 	Hereafter, we define h2o, part1 and part2 phase function.  They can be
		d here by other means (Mie scattering) provided that:
		1) A value is given for every degree angle
		2) The P.F. spherical integral is 4 PI (i.e., the mean value for the
			two first terms sum is 1)
		3) The first term is for perpend. polarisation, the second for parallel */
	for( iang=0; iang<LSAOCE; iang++ ){
		ang[iang] = 180*double(iang)/double(LSAOCE-1)*DEG2RAD;
	}
	
	for( iang=0; iang<LSAOCE; iang++ ){

		/** Fonction de phase **/
		/* Pour l'eau
		* Ici on suppose que le facteur de dépolarisation est nul
		*/
		pf0[iang*4 + 0] = 0.75;
		pf0[iang*4 + 1] = 0.75*cos(ang[iang])*cos(ang[iang]);
		pf0[iang*4 + 2] = 0.75*cos(ang[iang]);
		pf0[iang*4 + 3] = 0.;

		
		/* Pour les deux autres particules */
		if( iang>=ANGTRONC ){
			pf1[iang*4 + 0]=0.5*(r1*fournierForand(ang[iang],1.117,3.695) +(1-r1)*fournierForand(ang[iang],1.05,3.259));
		}
		else{
			pf1[iang*4 + 0] = 0.5*(r1*fournierForand(ang[ANGTRONC],1.117,3.695)
				+ (1-r1)*fournierForand(ang[ANGTRONC],1.05,3.259));
		}
		
		pf1[iang*4 + 1] = pf1[iang*4 + 0];
		pf1[iang*4 + 2] = 0.;
		pf1[iang*4 + 3] = 0.;
		
		pf2[iang*4 + 0] = henyeyGreenstein( g2, ang[iang] )/2;
		pf2[iang*4 + 1] = pf2[iang*4 + 0];
		pf2[iang*4 + 2] = 0.;
		pf2[iang*4 + 3] = 0.;
	}
	
	/** Renormalisation après troncature de la fonction de phase **/
	integ_ff = 0;
	for( iang=1; iang<LSAOCE; iang++ ){
		dtheta = ang[iang] - ang[iang-1];
		pm1 = pf1[(iang-1)*4 + 0] + pf1[(iang-1)*4 + 1];
		pm2 = pf1[iang*4 + 0] + pf1[iang*4 + 1];
		sin1 = sin(ang[iang-1]);
		sin2 = sin(ang[iang]);
		integ_ff = integ_ff + dtheta*( (sin1*pm1+sin2*pm2)/3. + (sin1*pm2+sin2*pm1)/6. );
	}
	
	rat1 = integ_ff/2;
	
	for( iang=0; iang<LSAOCE; iang++ ){
		pf1[iang*4 + 0] *= 1/rat1;
		pf1[iang*4 + 1] *= 1/rat1;
	}
	
	b1 *= rat1;
	
	
	/** Coefficients d'extinction et scattering albedo globaux **/
	btot = b0 + b1 + b2;
	atot = a0 + a1 + a2;
	
	/** Absorption totale déduite du coefficient d'atténuation de Morel **/
	Kd = Kw[ilambda] + Chi[ilambda]*pow(CONPHY,ee[ilambda]);
	delta = (0.256*(b0+b1/rat1+b2))*(0.256*(b0+b1/rat1+b2)) + 4*Kd*Kd;
	atot = 0.5*(-0.256*(b0+b1/rat1+b2) + sqrt(delta));
	
	extoce = atot + btot;
	W0OCE = btot/extoce;
	
	/** Calcul de la fonction de phase globale de diffusion **/
	for( iang=0; iang<LSAOCE ; iang++ ){
		pf[iang*4 + 0] = (b0*pf0[iang*4 + 0] + b1*pf1[iang*4 + 0] + b2*pf2[iang*4 + 0])/btot;
		pf[iang*4 + 1] = (b0*pf0[iang*4 + 1] + b1*pf1[iang*4 + 1] + b2*pf2[iang*4 + 1])/btot;
		pf[iang*4 + 2] = (b0*pf0[iang*4 + 2] + b1*pf1[iang*4 + 2] + b2*pf2[iang*4 + 2])/btot;
		pf[iang*4 + 3] = (b0*pf0[iang*4 + 3] + b1*pf1[iang*4 + 3] + b2*pf2[iang*4 + 3])/btot;
	}
	
	/* scum est une fonction s'accroissant entre 0 et 1 telle que d(scum)/dthe
	* est proportiennelle a la luminance diffusee entre THE et THE+dthe
	*/
	scum[0] = 0;
	for( iang = 1; iang<LSAOCE; iang++ ){
		dtheta = ang[iang] - ang[iang-1];
		pm1 = pf[(iang-1)*4 + 0] + pf[(iang-1)*4 + 1];
		pm2 = pf[iang*4 + 0] + pf[iang*4 + 1];
		sin1 = sin(ang[iang-1]);
		sin2 = sin(ang[iang]);
		scum[iang] = scum[iang-1] + dtheta*(( sin1*pm1+sin2*pm2 )/3 + (sin1*pm2+sin2*pm1)/6.)*DEUXPI;
	}
	
	if( abs(scum[LSAOCE-1]-4*PI)>0.1 ){
		printf("ERREUR lors de la dérivation de la foncion de phase océanique, scum = %lf\n", scum[LSAOCE-1]);
		exit(1);		
	}
	
	for( iang = 0; iang<LSAOCE; iang++ ){
		scum[iang] = scum[iang]/scum[LSAOCE-1];
	}
	
	/* foce gives NFOCE angles increasing from 0 to 180, and distributed according to the statistic scum
	*/
	ipf = 0;
	for( iang = 0; iang<NFOCE-1; iang++ ){
		z = double(iang)/double(NFOCE);
		while( scum[ipf+1]<z )
			ipf++;
		tab_H->foce[iang*5 + 4] = (float) ( (scum[ipf+1]-z)*ang[ipf] + (z-scum[ipf])*ang[ipf+1] )/(scum[ipf+1]-scum[ipf]);
		norm = pf[ipf*4 + 0] + pf[ipf*4 + 1];
		tab_H->foce[iang*5 + 0] = (float) pf[ipf*4 + 0]/norm;
		tab_H->foce[iang*5 + 1] = (float) pf[ipf*4 + 1]/norm;
		tab_H->foce[iang*5 + 2] = (float) pf[ipf*4 + 2]/norm;
		tab_H->foce[iang*5 + 3] = (float) pf[ipf*4 + 3]/norm;
	}
	
	tab_H->foce[(NFOCE-1)*5 + 4] = PI;
	tab_H->foce[(NFOCE-1)*5 + 0] = 0.5f;
	tab_H->foce[(NFOCE-1)*5 + 1] = 0.5f;
	tab_H->foce[(NFOCE-1)*5 + 2] = (float) pf[(LSAOCE-1)*4 + 2]/(pf[(LSAOCE-1)*4 + 0]+pf[(LSAOCE-1)*4 + 1]);
	tab_H->foce[(NFOCE-1)*5 + 3] = 0.f;
	
	/** Transfert de foce dans le device **/
	hipError_t erreur = hipMemcpy(tab_D->foce, tab_H->foce, 5*NFOCE*sizeof(*(tab_H->foce)), hipMemcpyHostToDevice); 
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->foce dans calculFoce\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	
	/** Libération de la mémoire allouée **/
	free( scum );
	free( ang );
	free( pf );
	free( pf0 );
	free( pf1 );
	free( pf2 );
}


/* henyeyGreenstein
* 
*/
double henyeyGreenstein( double asym, double angle ){

	return  (1 - asym*asym)/pow(1 + asym*asym - 2*asym*cos(angle),1.5);
}


/* fournierForand
 * 
 */
 double fournierForand(double ang, double n, double mu){

	double v;
	double delta, delta180;
	double res;
	
	v = (3-mu)/2;
	delta = 4/( 3*(n-1)*(n-1) )*sin(ang/2)*sin(ang/2);
	delta180 = 4/( 3*(n-1)*(n-1) )*sin(PI/2)*sin(PI/2);

	res = 1/( 4*PI*(1-delta)*(1-delta)*pow(delta,v) )*( v*(1-delta) - (1-pow(delta,v)) +
		( delta*(1-pow(delta,v)) - v*(1-delta) )*1/(sin(ang/2)*sin(ang/2)) )
		+ (1-pow(delta180,v))/(16*PI*(delta180-1)*pow(delta180,v)) * (3*cos(ang)*cos(ang) - 1);
	res *= 4*PI;
	
	return res;
}


/* verificationFoce
* Sauvegarde la fonction de phase dans l'océan calculée dans un fichier
* Permet de valider le bon calcul de la fonction de phase
*/
void verificationFoce( const char* nomFichier, Tableaux tab){
	
	FILE* fichier = fopen(nomFichier, "w");
	int i;
	
	fprintf( fichier, "angle\tI//\tIp\n" );
	
	for(i=0; i<NFOCE; i++){
		fprintf(fichier, "%f\t%20.16f\t%20.16f\n", tab.foce[i*5+4],tab.foce[i*5+0], tab.foce[i*5+1]);
	}
	
	fclose(fichier);
	
}
#endif

/* profilAtm
* Calcul du profil atmosphérique dans l'atmosphère en fonction de la couche
* Mélange Molécule/Aérosol dans l'atmosphère en fonction de la couche
*/
void profilAtm( Tableaux* tab_H, Tableaux* tab_D ){

	/** Déclaration des variables **/
	
	float tauMol[NATM+1];	// Epaisseur optique des molécules à chaque couche
	float tauAer[NATM+1];	// Epaisseur optique des aérosols à chaque couche
	int i=0;
	float va=0, vr=0;		// Variables tampons
	hipError_t erreur;		// Permet de tester le bon déroulement des opérations mémoires
	#ifndef SPHERIQUE
	float z;				// Variable représentant l'altitude
	z = HATM;
	#endif
	
	/** Conditions aux limites au sommet de l'atmosphère **/
	#ifdef SPHERIQUE
	tab_H->z[0] = HATM;
	#endif
	tauMol[0] = 0.0;
	tauAer[0] = 0.0;
	tab_H->h[0] = 0.0;
	tab_H->pMol[0] = 0.0;	//Je n'utilise pas la proportion d'aérosols car on l'obtient par 1-PMOL

	/** Cas Particuliers **/
	// Épaisseur optique aérosol très faible OU Épaisseur optique moléculaire et aérosol très faible
	// On ne considère une seule sous-couche dans laquelle on trouve toutes les molécules
	if( /*(TAUAER < 0.0001) ||*/ ((TAUAER < 0.0001)&&(TAURAY < 0.0001)) ){
		tauMol[1] = TAURAY;
		tauAer[1] = 0;
		#ifdef SPHERIQUE
		tab_H->z[1]=0;
		#endif
		tab_H->h[1] = tauMol[1] + tauAer[1];
		tab_H->pMol[1] = 1.0;
		
		/** Envoie des informations dans le device **/
		erreur = hipMemcpy(tab_D->h, tab_H->h, (NATM+1)*sizeof(*(tab_H->h)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->h dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		
		erreur = hipMemcpy(tab_D->pMol, tab_H->pMol, (NATM+1)*sizeof(*(tab_H->pMol)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->pMol dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}		
		
		#ifdef SPHERIQUE
		erreur = hipMemcpy(tab_D->z, tab_H->z, (NATM+1)*sizeof(*(tab_H->z)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->z dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		#endif
		return;
	}
	
	/** Profil standard avec échelle de hauteur **/
	if( PROFIL == 0 ){
		
		/* Si HA << HR => pas de mélange dans les couches
		On considere alors une atmosphere divisee en deux sous-couches, la  couche superieure contenant toutes les molecules, la
couche inferieure contenant tous les aerosols.
		*/
		if( HA < 0.0001 ){
			tauMol[1] = TAURAY;
			tauAer[1] = 0;
			#ifdef SPHERIQUE
			tab_H->z[1]=0.f;
			#endif
			tab_H->h[1] = tauMol[1] + tauAer[1];
			tab_H->pMol[1] = 1.0;
			
			tauMol[2] = 0;
			tauAer[2] = TAUAER;
			#ifdef SPHERIQUE
			tab_H->z[2]=0.f;
			#endif
			tab_H->h[2] = tab_H->h[1] + tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.0;
		}
		
		/* Si HA >> HR => pas de mélange dans les couches
		On considere alors une atmosphere divisee en deux sous-couches, la  couche superieure contenant tous les aérosols, la couche
inferieure contenant toutes les molécules.
		*/
		else if( HA > 499.99 ){
			tauMol[1] = 0.0;
			tauAer[1] = TAUAER;
			#ifdef SPHERIQUE
			tab_H->z[1]=0.f;
			#endif
			tab_H->h[1] = tauMol[1] + tauAer[1];
			tab_H->pMol[1] = 0.0;
			
			tauMol[2] = TAURAY;
			tauAer[2] = 0.0;
			#ifdef SPHERIQUE
			tab_H->z[2]=0.f;
			#endif
			tab_H->h[2] = tab_H->h[1] + tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 1.0;
		}
		
		/* Cas Standard avec deux échelles */
		else{
			for( i=0; i<NATM+1; i++){
				
				#ifdef SPHERIQUE
				if(i!=0){
					tab_H->z[i]=100.F - float(i)*(100.F/NATM);
				}
				vr = TAURAY*exp( -(tab_H->z[i]/HR) );
				va = TAUAER*exp( -(tab_H->z[i]/HA) );
				#endif
				#ifndef SPHERIQUE
				if(i!=0){
					z = 100.F - float(i)*(100.F/NATM);
				}
				vr = TAURAY*exp( -(z/HR) );
				va = TAUAER*exp( -(z/HA) );
				#endif
				
				tab_H->h[i] = va+vr;
				
				vr = vr/HR;
				va = va/HA;
				vr = vr/(va+vr);
				tab_H->pMol[i] = vr;
			}
			tab_H->h[0] = 0;
		}
	}
	
	/** Profil à 2 ou 3 couches **/
	else if( PROFIL == 1 ){

		float tauRay1;	// Epaisseur optique moleculaire de la couche 1
		float tauRay2;	// Epaisseur optique moleculaire de la couche 2
		
		tauRay1 = TAURAY*exp(-(ZMAX/HR));	// Epaisseur optique moleculaire de la couche la plus haute
		if( ZMIN < 0.0001 ){
			tauRay2 = TAURAY*(exp(-(ZMIN/HR))-exp(-(ZMAX/HR)));	// Epaisseur optique moleculaire de la couche la plus basse
		}
		
		else{
			tauRay2 = TAURAY*(exp(-(ZMIN/HR))-exp(-(ZMAX/HR)));	// Epaisseur optique moleculaire de la couche intermédiaire
		}
		
		/** Calcul des grandeurs utiles aux OS pour la couche la plus haute **/
		#ifdef SPHERIQUE
		tab_H->z[1]=-( HR*log(tauRay1/TAURAY) );
		#endif
		tauMol[1] = tauRay1;
		tauAer[1] = 0.F;                                    
		tab_H->h[1] = tauMol[1] + tauAer[1];
		tab_H->pMol[1] = 1.F;

		/** Calcul des grandeurs utiles aux OS pour la deuxieme couche   **/
		if( ZMAX == ZMIN ){ //Uniquement des aerosols dans la couche intermediaire
			#ifdef SPHERIQUE
			tab_H->z[2]=ZMAX;
			#endif
			tauMol[2] = tauRay1;                                                      
			tauAer[2] = TAUAER;
			tab_H->h[2] = tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.F;                                                      
		}
		
		else{	// Melange homogene d'aerosol et de molecules dans la couche intermediaire
			#ifdef SPHERIQUE
			tab_H->z[2]=ZMIN;
			#endif
			tauMol[2] = tauRay1+tauRay2;
			tauAer[2] = TAUAER;
			tab_H->h[2] = tauMol[2] + tauAer[2];
			tab_H->pMol[2] = 0.5F;
		}
		
		/** Calcul des grandeurs utiles aux OS pour la troisieme couche **/
		#ifdef SPHERIQUE
		tab_H->z[3]=0.f;
		#endif
		tauMol[3] = TAURAY;
		tauAer[3] = TAUAER;
		tab_H->h[3] = tauMol[3] + tauAer[3];
		tab_H->pMol[3] = 1.F;
	}
	
	else if( PROFIL == 2 ){
		// Profil utilisateur
		/* Format du fichier
		=> n	alt		tauMol		tauAer		h		pAer		pMol
		*/
		FILE* profil = fopen( PATHPROFILATM , "r" );
		float garbage;
		
		int icouche=0;
		char ligne[1024];
	
		if(profil == NULL){
			printf("ERREUR : Ouverture impossible du fichier %s pour le profil atmosphérique\n", PATHPROFILATM );
			exit(1);
		}
		
		else{
			// Passage de la premiere ligne
			fgets(ligne,1024,profil);

			// Extraction des informations
			#ifdef SPHERIQUE
			for( icouche=0; icouche<NATM+1; icouche++ ){
				fscanf(profil, "%d\t%f\t%f\t%f\t%f\t%f\t%f", &i, tab_H->z+icouche, &garbage, &garbage, tab_H->h+icouche,
&garbage,tab_H->pMol+icouche );
			}
			#endif
			#ifndef SPHERIQUE
			for( icouche=0; icouche<NATM+1; icouche++ ){
				fscanf(profil, "%d\t%f\t%f\t%f\t%f\t%f\t%f", &i, &garbage, &garbage, &garbage, tab_H->h+icouche,
					   &garbage,tab_H->pMol+icouche );
			}
			#endif
		}
	
		if(fclose(profil) == EOF){
			printf("ERREUR : Probleme de fermeture du fichier %s", PATHPROFILATM);
		}
		
	}
	
	
		/** Envoie des informations dans le device **/
		erreur = hipMemcpy(tab_D->h, tab_H->h, (NATM+1)*sizeof(*(tab_H->h)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->h dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		
		erreur = hipMemcpy(tab_D->pMol, tab_H->pMol, (NATM+1)*sizeof(*(tab_H->pMol)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->pMol dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}	
		
		#ifdef SPHERIQUE
		erreur = hipMemcpy(tab_D->z, tab_H->z, (NATM+1)*sizeof(*(tab_H->z)), hipMemcpyHostToDevice);
		if( erreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_D->z dans profilAtm\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
			exit(1);
		}
		#endif
	
}


/* verificationAtm
* Sauvegarde du profil atmosphérique dans un fichier
* Permet de valider le bon calcul
*/
void verificationAtm( Tableaux tab_H ){
	
	// Vérification du modèle
	FILE* fichier = fopen("./test/modele_atm_cuda.txt", "w+");
	
	#ifdef SPHERIQUE
	fprintf( fichier, "couche\tz\tpropMol\th\n" );
	
	for( int i=0; i<NATM+1; i++){
		fprintf(fichier, "%d\t%10.8f\t%10.8f\t%10.8f\n",i,tab_H.z[i],tab_H.pMol[i], tab_H.h[i]);
	}
	
	fprintf( fichier, "couche\tz\tpropMol\th\n" );
	#endif
	
	#ifndef SPHERIQUE
	fprintf( fichier, "couche\tpropMol\th\n" );
	
	for( int i=0; i<NATM+1; i++){
		fprintf(fichier, "%d\t%10.8f\t%10.8f\n",i,tab_H.pMol[i], tab_H.h[i]);
	}
	
	fprintf( fichier, "couche\tpropMol\th\n" );
	#endif
	
	fclose(fichier);
}

/** Séparation du code pour atmosphère sphérique ou parallèle **/
#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */

/* impactInit
* Calcul du profil que le photon va rencontrer lors de son premier passage dans l'atmosphère
* Sauvegarde de ce profil dans tab et sauvegarde des coordonnées initiales du photon dans init
*/
void impactInit(Init* init_H, Init* init_D, Tableaux* tab_H, Tableaux* tab_D){
	
	double thss, localh;
	double rdelta;
	double xphbis,yphbis,zphbis;	//Coordonnées intermédiaire du photon
	double rsolfi,rsol1,rsol2;
	
	// Correspond aux paramètres initiaux du photon
	double vx = -sin(THSDEG*DEG2RAD);
	double vy = 0.;
	double vz = -cos(THSDEG*DEG2RAD);
	
	/** Calcul du point d'impact **/
	// 	thss = abs(acosf(abs(vz)));
	thss = THSDEG*DEG2RAD;
	
	rdelta = 4.*RTER*RTER + 4.*( tan(thss)*tan(thss)+1. )*( HATM*HATM + 2.*HATM*RTER );
	localh = ( -2.*RTER+sqrt(rdelta) )/( 2.*(tan(thss)*tan(thss)+1.) );
	
	init_H->x0 = (float) localh*tan(thss);
	init_H->y0 = 0.f;
	init_H->z0 = (float) RTER + localh;	
	
	tab_H->zph0[0] = 0.;
	tab_H->hph0[0] = 0.;
	
	xphbis = init_H->x0;
	yphbis = init_H->y0;
	zphbis = init_H->z0;
	
	/** Création hphoton et zphoton, chemin optique entre sommet atmosphère et sol pour la direction d'incidence **/
	for(int icouche=1; icouche<NATM+1; icouche++){
		
		rdelta = 4.*(vx*xphbis + vy*yphbis + vz*zphbis)*(vx*xphbis + vy*yphbis + vz*zphbis)
			- 4.*(xphbis*xphbis + yphbis*yphbis + zphbis*zphbis - ((double)tab_H->z[icouche]+RTER)*((double)tab_H->z[icouche]+RTER));
		rsol1 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) + sqrt(rdelta) );
		rsol2 = 0.5*( -2.*(vx*xphbis + vy*yphbis + vz*zphbis) - sqrt(rdelta) );
		
		// Il faut choisir la plus petite distance en faisant attention qu'elle soit positive
		if(rsol1>0.){
			if( rsol2>0.)
				rsolfi = min(rsol1,rsol2);
			else
				rsolfi = rsol1;
		}
		else{
			if( rsol2>0. )
				rsolfi=rsol1;
		}
		
		tab_H->zph0[icouche] = tab_H->zph0[icouche-1] + (float)rsolfi;
		tab_H->hph0[icouche] = tab_H->hph0[icouche-1] + 
				( abs( tab_H->h[icouche] - tab_H->h[icouche-1])*rsolfi )/( abs( tab_H->z[icouche-1] - tab_H->z[icouche]) );
		
		xphbis+= vx*rsolfi;
		yphbis+= vy*rsolfi;
		zphbis+= vz*rsolfi;
		
	}

	init_H->taumax0 = tab_H->hph0[NATM];
	init_H->zintermax0 = tab_H->zph0[NATM];

	
	/** Envoie des données dans le device **/
	hipError_t erreur = hipMemcpy(init_D, init_H, sizeof(Init), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème de copie init_H dans initInit\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	erreur = hipMemcpy(tab_D->hph0, tab_H->hph0, (NATM+1)*sizeof(*(tab_H->hph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->hph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
	erreur = hipMemcpy(tab_D->zph0, tab_H->zph0, (NATM+1)*sizeof(*(tab_H->zph0)), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tab_D->zph0 dans initInit\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
	
}
#endif


/**********************************************************
*	> Fonctions d'affichage
***********************************************************/

/* afficheParametres
* Affiche les paramètres de la simulation
*/
void afficheParametres()
{
	printf("\n#--------- Paramètres de simulation --------#\n");
	printf(" NBPHOTONS =\t%llu", NBPHOTONS);
	printf("\n");
	printf(" NBTHETA =\t%d", NBTHETA);
	printf("\n");
	printf(" NBPHI\t=\t%d", NBPHI);
	printf("\n");
	printf(" THSDEG\t=\t%f (degrés)", THSDEG);
	printf("\n");
	printf(" LAMBDA\t=\t%f", LAMBDA);
	printf("\n");
	printf(" SIM\t=\t%d", SIM);
		if( SIM==-2 )
			printf("\t(Atmosphère seule)");
		if( SIM==-1 )
			printf("\t(Dioptre seul)");
		if( SIM==0 )
			printf("\t(Océan et Surface)");
		if( SIM==1 )
			printf("\t(Atmosphère et Surface)");
		if( SIM==2 )
			printf("\t(Atmosphère, Dioptre et Océan)");
		if( SIM==3 )
			printf("\t(Océan seul)");
		
	printf("\n");
	printf(" SEED\t=\t%d", SEED);
	printf("\n");
	
	printf("\n#------- Paramètres de performances --------#\n");
	printf(" NBLOOP\t=\t%u", NBLOOP);
	printf("\n");
	printf(" XBLOCK\t=\t%d", XBLOCK);
	printf("\n");
	printf(" YBLOCK\t=\t%d", YBLOCK);
	printf("\n");
	printf(" XGRID\t=\t%d", XGRID);
	printf("\n");
	printf(" YGRID\t=\t%d", YGRID);
	printf("\n");
	
	
	printf("\n#--------------- Atmosphère ----------------#\n");
	if( SIM==-2 || SIM==1 || SIM==2 ){
		#ifdef SPHERIQUE
		printf(" Géométrie de l'atmosphère: \tSphérique");
		printf("\n");
		#endif
		#ifndef SPHERIQUE
		printf(" Géométrie de l'atmosphère: \tParallèle");
		printf("\n");
		#endif
		
		printf(" TAURAY\t=\t%f", TAURAY);
		printf("\n");
		printf(" TAUAER\t=\t%f", TAUAER);
		printf("\n");
		printf(" W0AER\t=\t%f", W0AER);
		printf("\n");
		printf(" LSAAER\t=\t%u", LSAAER);
		printf("\n");
		printf(" NFAER\t=\t%u", NFAER);
		printf("\n");
		printf(" PROFIL\t=\t%d", PROFIL);
		printf("\n");
		printf(" HA\t=\t%f", HA);
		printf("\n");
		printf(" HR\t=\t%f", HR);
		printf("\n");
		printf(" ZMIN\t=\t%f", ZMIN);
		printf("\n");
		printf(" ZMAX\t=\t%f", ZMAX);
		printf("\n");
		printf(" NATM\t=\t%d", NATM);
		printf("\n");
		printf(" HATM\t=\t%d", HATM);
		printf("\n");
		printf(" DIFFF\t=\t%d", DIFFF);
		printf("\n");
	}
	else{
		printf("\tPas de contribution de l'atmosphère\n");
	}
	
	
	printf("\n#--------- Contribution du dioptre ---------#\n");
	if( SIM==-1 || SIM==0 || SIM==1 || SIM==2 ){
		printf(" SUR\t=\t%d", SUR);
		printf("\n");
		printf(" DIOPTRE =\t%d", DIOPTRE);
		printf("\n");
		printf(" W0LAM\t=\t%f", W0LAM);
		printf("\n");
		printf(" WINDSPEED =\t%f", WINDSPEED);
		printf("\n");
	}
	else{
		printf("\tPas de dioptre\n");
	}
	
	#ifdef FLAGOCEAN
	printf("\n#----------------- Océan ------------------#\n");
	printf(" LSAOCE\t=\t%u", LSAOCE);
	printf("\n");
	printf(" NFOCE\t=\t%u", NFOCE);
	printf("\n");
	printf(" CONPHY\t=\t%f", CONPHY);
	printf("\n");
	printf(" NH2O\t=\t%f", NH2O);
	printf("\n");
	#endif
	
	printf("\n#----------- Chemin des fichiers -----------#\n");
	printf(" PATHRESULTATSHDF = %s", PATHRESULTATSHDF);
	printf("\n");
	printf(" PATHTEMOINHDF = %s", PATHTEMOINHDF);
	printf("\n");
	printf(" PATHDIFFAER = %s", PATHDIFFAER);
	printf("\n");
	printf(" PATHPROFILATM = %s", PATHPROFILATM);
	printf("\n");
	
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	printf("\n  Date de début  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year,
		   date->tm_hour, date->tm_min, date->tm_sec);

}


/* afficheProgress
* Affiche la progression de la simulation
*/
void afficheProgress(unsigned long long nbPhotonsTot, Variables* var, double tempsPrec
#ifdef PROGRESSION
, unsigned long long nbPhotonsSorTot
#endif
)
{
	// Calcul la date et l'heure courante
	time_t dateTime = time(NULL);
	struct tm* date = localtime(&dateTime);
	// Calcul du temps ecoule et restant
	double tempsProg = (double)(clock() / CLOCKS_PER_SEC);
	double tempsTot = tempsProg + tempsPrec;
	int tempsEcoule = (int)tempsTot;
	int hEcoulees = tempsEcoule / 3600;
	int minEcoulees = (tempsEcoule%3600) / 60;
	int secEcoulees = tempsEcoule%60;
	int tempsRestant = (int)(tempsTot * ((double)NBPHOTONS / (double)nbPhotonsTot - 1.));
	if(tempsRestant < 0) tempsRestant = 0;
	int hRestantes = tempsRestant / 3600;
	int minRestantes = (tempsRestant%3600) / 60;
	int secRestantes = tempsRestant%60;
	// Calcul du pourcentage de photons traités
	int pourcent = (int)(100 * nbPhotonsTot / NBPHOTONS);
	
	// Affichage
	printf("\n");
	printf(" --------------------------------------\n");
	printf("  Photons lances : %12lu (%3d%%)\n", nbPhotonsTot, pourcent);
	printf("  Photons pb     : %12d\n", var->erreurpoids + var->erreurtheta);
	printf("  Temps ecoule   : %d h %2d min %2d sec\n", hEcoulees, minEcoulees, secEcoulees);
	printf("  Temps restant  : %d h %2d min %2d sec\n", hRestantes, minRestantes, secRestantes);
	printf("  Date actuelle  : %02u/%02u/%04u %02u:%02u:%02u\n", date->tm_mday, date->tm_mon+1, 1900 + date->tm_year, date->tm_hour,
		   date->tm_min, date->tm_sec);
	printf(" --------------------------------------\n");
   
	#ifdef PROGRESSION
	printf("%d%% - ", (int)(100*nbPhotonsTot/NBPHOTONS));
	printf("Temps: %d - ", tempsEcoule);
	printf("phot sortis: %lu - ", nbPhotonsSorTot);
	printf("phot traités: %lu - ", nbPhotonsTot);
	printf("erreur poids/theta/vxy/vy/case: %d/%d/%d/%d/%d", var->erreurpoids, var->erreurtheta, var->erreurvxy,
		var->erreurvy, var->erreurcase);
	printf("\n");
	#endif
}


#ifdef TRAJET
/* initEvnt
* Initialisation des variables à envoyer dans le kernel pour récupérer le trajet d'un photon
*/
void initEvnt(Evnt* evnt_H, Evnt* evnt_D)
{
	for(int i = 0; i < NBTRAJET; i++) evnt_H[i].action = 0;
	hipError_t erreur = hipMemcpy(evnt_D, evnt_H, NBTRAJET * sizeof(Evnt), hipMemcpyHostToDevice);
	if( erreur != hipSuccess ){
		printf( "ERREUR: Problème de copie evnt_H dans initEvnt\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
		exit(1);
	}
}


/* afficheTrajet
* Fonction qui affiche le début du trajet du premier thread
*/
void afficheTrajet(Evnt* evnt_H)
{
	printf("\nTrajet d'un thread :\n");
	for(int i = 0; i < NBTRAJET; i++)
	{
		if(evnt_H[i].action == 1)
			printf("init : ");
		else if(evnt_H[i].action == 2)
			printf("move : ");
		else if(evnt_H[i].action == 3)
			printf("scat : ");
		else if(evnt_H[i].action == 4)
			printf("surf : ");
		else if(evnt_H[i].action != 5)
		{
			printf("\nERREUR : host afficheTrajet: Aucun trajet a afficher\n");
			return;
		}
		else printf("exit : ");
		printf("tau=%10.9f ", evnt_H[i].tau);
		printf("poids=%10.9f", evnt_H[i].poids);
		printf("\n");
	}
}
#endif


/**********************************************************
*	> Calcul pour sauvegarde des résultats finaux
***********************************************************/

/* calculOmega
* Fonction qui calcule l'aire normalisée de chaque boite, son theta, et son psi, sous forme de 3 tableaux
*/
void calculOmega(double* tabTh, double* tabPhi, double* tabOmega)
{
	// Tableau contenant l'angle theta de chaque morceau de sphère
	memset(tabTh, 0, NBTHETA * sizeof(*tabPhi));
	double dth = DEMIPI / NBTHETA;
	tabTh[0] = dth/4;
	tabTh[1] = dth;
	for(int ith = 2; ith < NBTHETA; ith++){
		tabTh[ith] = tabTh[ith-1] + dth;
	}
	
	// Tableau contenant l'angle psi de chaque morceau de sphère
	memset(tabPhi, 0, NBPHI * sizeof(*tabPhi));
	double dphi = PI / NBPHI;
 	tabPhi[0] = dphi / 2;
	for(int iphi = 1; iphi < NBPHI; iphi++){ 
		tabPhi[iphi] = tabPhi[iphi-1] + dphi;
	}
	// Tableau contenant l'aire de chaque morceau de sphère
	double sumds = 0;
	double tabds[NBTHETA * NBPHI];
	memset(tabds, 0, NBTHETA * NBPHI * sizeof(*tabds));
	for(int ith = 0; ith < NBTHETA; ith++)
	{
		if( ith==0 )
			dth = DEMIPI / (2*NBTHETA);
		else 
			dth = DEMIPI / NBTHETA;
			
		for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			tabds[ith * NBPHI + iphi] = sin(tabTh[ith]) * dth * dphi;
			sumds += tabds[ith * NBPHI + iphi];
		}
	}
	
	// La derniere demi boite 89.75->90
	for(int iphi = 0; iphi < NBPHI; iphi++)
		{
			sumds += sin( (DEMIPI+tabTh[NBTHETA-1])/2 ) * (dth/2) * dphi;
		}
	
	// Normalisation de l'aire de chaque morceau de sphère
	memset(tabOmega, 0, NBTHETA * NBPHI * sizeof(*tabOmega));
	for(int ith = 0; ith < NBTHETA; ith++)
		for(int iphi = 0; iphi < NBPHI; iphi++){
			tabOmega[ith * NBPHI + iphi] = tabds[ith * NBPHI + iphi] / sumds;
		}
}


/* calculTabFinal
* Fonction qui remplit le tabFinal correspondant à la reflectance (R), Q et U sur tous l'espace de sorti (dans chaque boite)
*/
void calculTabFinal(double* tabFinal, double* tabTh, double* tabPhi, double* tabPhotonsTot, unsigned long long nbPhotonsTot)
{
	
	double tabOmega[NBTHETA * NBPHI]; //tableau contenant l'aire de chaque morceau de sphère
	// Remplissage des tableaux tabTh, tabPhi, et tabOmega
	calculOmega(tabTh, tabPhi, tabOmega);
	
	// Remplissage du tableau final
	for(int iphi = 0; iphi < NBPHI; iphi++)
	{
		for(int ith = 0; ith < NBTHETA; ith++)
		{
			// Reflectance
			tabFinal[0*NBTHETA*NBPHI + iphi*NBTHETA+ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA+ith*NBPHI+iphi] + tabPhotonsTot[1*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi]* cos(tabTh[ith]));
			
			// Q
			tabFinal[1*NBTHETA*NBPHI + iphi*NBTHETA+ith] =
				(tabPhotonsTot[0*NBPHI*NBTHETA+ith*NBPHI+iphi] - tabPhotonsTot[1*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi] * cos(tabTh[ith]));
			
			// U
			tabFinal[2*NBTHETA*NBPHI + iphi*NBTHETA+ith] = (tabPhotonsTot[2*NBPHI*NBTHETA+ith*NBPHI+iphi]) / 
				(2* nbPhotonsTot * tabOmega[ith*NBPHI+iphi] * cos(tabTh[ith]));
				
		}
	}
}


/**********************************************************
*	> Fichier hdf (lecture/écriture témoin, écriture résultats)
***********************************************************/

/* creerHDFTemoin
* Fonction qui crée un fichier .hdf contenant les informations nécessaires à la reprise du programme
* //TODO: 	écrire moins régulièrement le témoin (non pas une écriture par appel de kernel)
*			changer le format (écrire un .bin par exemple) pour éventuellement gagner du temps (calculer le gain éventuel)
*/
void creerHDFTemoin(double* tabPhotonsTot, unsigned long long nbPhotonsTot, Variables* var, double tempsPrec)
{
	// Création du fichier de sortie
	int sdFichier = SDstart(PATHTEMOINHDF, DFACC_CREATE);
    if (sdFichier == FAIL) {
        printf("ERREUR : création fichier témoin (%s)\n", PATHTEMOINHDF);
        exit(1);
    }
	
	char nomTab[20]; //nom du tableau
	sprintf(nomTab,"Temoin (%d%%)", (int)(100 * nbPhotonsTot / NBPHOTONS));
	int nbDimsTab = 1; //nombre de dimensions du tableau
	int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
	valDimsTab[0] = 4 * NBTHETA * NBPHI;
	int typeTab = DFNT_FLOAT64 ; //type des éléments du tableau
	// Création du tableau
	int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	int startTab[nbDimsTab]; //début de la lecture du tableau
	startTab[0]=0;
	// Ecriture du tableau dans le fichier
	int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tabPhotonsTot);
	// Vérification du bon fonctionnement de l'écriture
	if (status)
	{
		printf("\nERREUR : write hdf temoin (%s)\n", sdFichier);
		exit(1);
	}
	
	// Ecriture de toutes les informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSdouble = (double)NBPHOTONS; // on convertit en double car le hdf n'accepte pas ull
	double nbPhotonsTotdouble = (double)nbPhotonsTot; // on convertit en double car le hdf n'accepte pas ull
	double tempsEcouledouble = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);
	
	SDsetattr(sdsTab, "NBPHOTONS", DFNT_FLOAT64, 1, &NBPHOTONSdouble);
	SDsetattr(sdsTab, "NBLOOP", DFNT_UINT32, 1, &NBLOOP);
	SDsetattr(sdsTab, "SEED", DFNT_UINT32, 1, &SEED);
	SDsetattr(sdsTab, "XBLOCK", DFNT_INT32, 1, &XBLOCK);
	SDsetattr(sdsTab, "YBLOCK", DFNT_INT32, 1, &YBLOCK);
	SDsetattr(sdsTab, "XGRID", DFNT_INT32, 1, &XGRID);
	SDsetattr(sdsTab, "YGRID", DFNT_INT32, 1, &YGRID);
	SDsetattr(sdsTab, "NBTHETA", DFNT_INT32, 1, &NBTHETA);
	SDsetattr(sdsTab, "NBPHI", DFNT_INT32, 1, &NBPHI);
	SDsetattr(sdsTab, "DIOPTRE", DFNT_INT32, 1, &DIOPTRE);
	SDsetattr(sdsTab, "DIFFF", DFNT_INT32, 1, &DIFFF);
	SDsetattr(sdsTab, "PROFIL", DFNT_INT32, 1, &PROFIL);
	SDsetattr(sdsTab, "SIM", DFNT_INT32, 1, &SIM);
	SDsetattr(sdsTab, "SUR", DFNT_INT32, 1, &SUR);
	SDsetattr(sdsTab, "THSDEG", DFNT_FLOAT32, 1, &THSDEG);
	SDsetattr(sdsTab, "LAMBDA", DFNT_FLOAT32, 1, &LAMBDA);
	SDsetattr(sdsTab, "TAURAY", DFNT_FLOAT32, 1, &TAURAY);
	SDsetattr(sdsTab, "TAUAER", DFNT_FLOAT32, 1, &TAUAER);
	SDsetattr(sdsTab, "W0AER", DFNT_FLOAT32, 1, &W0AER);
	SDsetattr(sdsTab, "W0LAM", DFNT_FLOAT32, 1, &W0LAM);
	
	SDsetattr(sdsTab, "LSAAER", DFNT_UINT32, 1, &LSAAER);
	SDsetattr(sdsTab, "NFAER", DFNT_UINT32, 1, &NFAER);
	SDsetattr(sdsTab, "LSAOCE", DFNT_UINT32, 1, &LSAOCE);
	SDsetattr(sdsTab, "NFOCE", DFNT_UINT32, 1, &NFOCE);
	
	SDsetattr(sdsTab, "HA", DFNT_FLOAT32, 1, &HA);
	SDsetattr(sdsTab, "HR", DFNT_FLOAT32, 1, &HR);
	SDsetattr(sdsTab, "ZMIN", DFNT_FLOAT32, 1, &ZMIN);
	SDsetattr(sdsTab, "ZMAX", DFNT_FLOAT32, 1, &ZMAX);
	SDsetattr(sdsTab, "NATM", DFNT_INT32, 1, &NATM);
	SDsetattr(sdsTab, "HATM", DFNT_INT32, 1, &HATM);
	SDsetattr(sdsTab, "WINDSPEED", DFNT_FLOAT32, 1, &WINDSPEED);
	SDsetattr(sdsTab, "NH2O", DFNT_FLOAT32, 1, &NH2O);
    #ifdef FLAGOCEAN
	SDsetattr(sdsTab, "CONPHY", DFNT_FLOAT32, 1, &CONPHY);
    #endif
	SDsetattr(sdsTab, "PATHRESULTATSHDF", DFNT_CHAR8, strlen(PATHRESULTATSHDF), PATHRESULTATSHDF);
	SDsetattr(sdsTab, "PATHTEMOINHDF", DFNT_CHAR8, strlen(PATHTEMOINHDF), PATHTEMOINHDF);
	SDsetattr(sdsTab, "PATHDIFFAER", DFNT_CHAR8, strlen(PATHDIFFAER), PATHDIFFAER);
	SDsetattr(sdsTab, "PATHPROFILATM", DFNT_CHAR8, strlen(PATHPROFILATM), PATHPROFILATM);
	
	SDsetattr(sdsTab, "nbPhotonsTot", DFNT_FLOAT64, 1, &nbPhotonsTotdouble);
	SDsetattr(sdsTab, "nbErreursPoids", DFNT_INT32, 1, &(var->erreurpoids));
	SDsetattr(sdsTab, "nbErreursTheta", DFNT_INT32, 1, &(var->erreurtheta));
	SDsetattr(sdsTab, "tempsEcoule", DFNT_FLOAT64, 1, &tempsEcouledouble);
	
	#ifdef PROGRESSION
	SDsetattr(sdsTab, "nbThreads", DFNT_UINT32, 1, &(var->nbThreads));
	SDsetattr(sdsTab, "nbPhotonsSor", DFNT_UINT32, 1, &(var->nbPhotonsSor));
	SDsetattr(sdsTab, "erreurvxy", DFNT_INT32, 1, &(var->erreurvxy));
	SDsetattr(sdsTab, "erreurvy", DFNT_INT32, 1, &(var->erreurvy));
	SDsetattr(sdsTab, "erreurcase", DFNT_INT32, 1, &(var->erreurcase));
	#endif

	// Fermeture du tableau
	SDendaccess(sdsTab);
	// Fermeture du fichier
	SDend(sdFichier);
}


/* lireHDFTemoin
* Si un fichier temoin existe et que les paramètres correspondent à la simulation en cours, cette simulation se poursuit à
* partir de celle sauvée dans le fichier témoin.
*/
void lireHDFTemoin(Variables* var_H, Variables* var_D,
		unsigned long long* nbPhotonsTot, double* tabPhotonsTot, double* tempsEcoule)
{
	// Ouverture du fichier temoin
	int sdFichier = SDstart(PATHTEMOINHDF, DFACC_READ);
	if(sdFichier != -1)
	{
		// Ouverture de l'unique tableau du fichier temoin
		int sdsIndex = 0;
		int sdsTab = SDselect (sdFichier, sdsIndex);
		
		// Recuperation de paramètres du fichier temoin
		int SEEDrecup[1];
		int NBTHETArecup[1];
		int NBPHIrecup[1];
		int DIOPTRErecup[1];
		int DIFFFrecup[1];
		int PROFILrecup[1];
		int SIMrecup[1];
		int SURrecup[1];
		float THSDEGrecup[1];
		float LAMBDArecup[1];
		float TAURAYrecup[1];
		float TAUAERrecup[1];
		float W0AERrecup[1];
		float W0LAMrecup[1];
		float HArecup[1];
		float HRrecup[1];
		float ZMINrecup[1];
		float ZMAXrecup[1];
		int NATMrecup[1];
		int HATMrecup[1];
		float WINDSPEEDrecup[1];
		float NH2Orecup[1];
        #ifdef FLAGOCEAN
		float CONPHYrecup[1];
        #endif
		
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SEED"), (VOIDP)SEEDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBTHETA"), (VOIDP)NBTHETArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NBPHI"), (VOIDP)NBPHIrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIOPTRE"), (VOIDP)DIOPTRErecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "DIFFF"), (VOIDP)DIFFFrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "PROFIL"), (VOIDP)PROFILrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SIM"), (VOIDP)SIMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "SUR"), (VOIDP)SURrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "THSDEG"), (VOIDP)THSDEGrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "LAMBDA"), (VOIDP)LAMBDArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAURAY"), (VOIDP)TAURAYrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "TAUAER"), (VOIDP)TAUAERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "W0AER"), (VOIDP)W0AERrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "W0LAM"), (VOIDP)W0LAMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HA"), (VOIDP)HArecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HR"), (VOIDP)HRrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMIN"), (VOIDP)ZMINrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "ZMAX"), (VOIDP)ZMAXrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NATM"), (VOIDP)NATMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "HATM"), (VOIDP)HATMrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "WINDSPEED"), (VOIDP)WINDSPEEDrecup);
		SDreadattr(sdsTab, SDfindattr(sdsTab, "NH2O"), (VOIDP)NH2Orecup);
        #ifdef FLAGOCEAN
		SDreadattr(sdsTab, SDfindattr(sdsTab, "CONPHY"), (VOIDP)CONPHYrecup);
        #endif
		
		// Si les parametres sont les memes on recupere des informations pour poursuivre la simulation précédente
		if(NBTHETArecup[0] == NBTHETA
			&& NBPHIrecup[0] == NBPHI
			&& DIOPTRErecup[0] == DIOPTRE
			&& DIFFFrecup[0] == DIFFF
			&& PROFILrecup[0] == PROFIL
			&& SIMrecup[0] == SIM
			&& SURrecup[0] == SUR
			&& THSDEGrecup[0] == THSDEG
			&& LAMBDArecup[0] == LAMBDA
			&& TAURAYrecup[0] == TAURAY
			&& TAUAERrecup[0] == TAUAER
			&& W0AERrecup[0] == W0AER
			&& W0LAMrecup[0] == W0LAM
			&& HArecup[0] == HA
			&& HRrecup[0] == HR
			&& ZMINrecup[0] == ZMIN
			&& ZMAXrecup[0] == ZMAX
			&& NATMrecup[0] == NATM
			&& HATMrecup[0] == HATM
			&& WINDSPEEDrecup[0] == WINDSPEED
			&& NH2Orecup[0] == NH2O
            #ifdef FLAGOCEAN
			&& CONPHYrecup[0] == CONPHY
            #endif
            )
		{
			
			// Recuperation du nombre de photons traités et du nombre d'erreurs
			double nbPhotonsTotDouble[1]; //on récupère d'abord la variable en double
			unsigned long long nbPhotonsTotRecup[1]; //puis on la passera en unsigned long long
			int nbErreursPoidsRecup[1];
			int nbErreursThetaRecup[1];
			double tempsEcouleRecup[1];
	
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbPhotonsTot"), (VOIDP)nbPhotonsTotDouble);
			nbPhotonsTotRecup[0] = (unsigned long long)nbPhotonsTotDouble[0];
			
			printf("\nPOURSUITE D'UNE SIMULATION ANTERIEURE AU PHOTON %e\n",nbPhotonsTotDouble[0]);
			if(SEEDrecup[0] == SEED) printf("ATTENTION: Nous recommandons SEED=-1 sinon les nombres aleatoires sont\
identiques a chaque lancement.\n");
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursPoids"), (VOIDP)nbErreursPoidsRecup);
			SDreadattr(sdsTab, SDfindattr(sdsTab, "nbErreursTheta"), (VOIDP)nbErreursThetaRecup);
			SDreadattr(sdsTab, SDfindattr(sdsTab, "tempsEcoule"), (VOIDP)tempsEcouleRecup);
	
			var_H->erreurpoids = nbErreursPoidsRecup[0];//nombre de photons ayant un poids anormalement élevé
			var_H->erreurtheta = nbErreursThetaRecup[0];//nombre de photons sortant dans la direction solaire
			
			#ifdef PROGRESSION
			unsigned long long nbThreadsRecup[1]; //nombre total de threads lancés
			unsigned long long nbPhotonsSorRecup[1]; //nombre de photons ressortis pour un appel du Kernel
			int erreurvxyRecup[1]; //nombre de photons sortant au zénith et donc difficiles à classer
			int erreurvyRecup[1]; //nombre de photons sortant à phi=0 ou phi=PI et donc difficiles à classer
			int erreurcaseRecup[1]; // nombre de photons rangé dans une case inexistante
			
			SDreadattr(sdsTab, SDfindattr(sdsTab,"nbThreads"), (VOIDP)nbThreadsRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"nbPhotonsSor"), (VOIDP)nbPhotonsSorRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurvxy"), (VOIDP)erreurvxyRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurvy"), (VOIDP)erreurvyRecup );
			SDreadattr(sdsTab, SDfindattr(sdsTab,"erreurcase"), (VOIDP)erreurcaseRecup );
			
			var_H->nbThreads = nbThreadsRecup[0];
			var_H->nbPhotonsSor = nbPhotonsSorRecup[0];
			var_H->erreurvxy = erreurvxyRecup[0];
			var_H->erreurvy = erreurvyRecup[0];
			var_H->erreurcase = erreurcaseRecup[0];
			
			#endif
			
			hipError_t erreur = hipMemcpy(var_D, var_H, sizeof(Variables), hipMemcpyHostToDevice);
			if( erreur != hipSuccess ){
				printf( "ERREUR: Problème de copie var_H dans lireHDFTemoin\n");
				printf( "Nature de l'erreur: %s\n",hipGetErrorString(erreur) );
				exit(1);
			}
			*(nbPhotonsTot) = nbPhotonsTotRecup[0];
			*(tempsEcoule) = tempsEcouleRecup[0];
	
			// Recuperation du tableau
			int nbDimsTab = 1; //nombre de dimensions du tableau
			int startTab[nbDimsTab], edgesTab[nbDimsTab]; //debut et fin de la lecture du tableau
			startTab[0] = 0;
			edgesTab[0] = 4*NBTHETA * NBPHI;
	
			int status = SDreaddata (sdsTab, startTab, NULL, edgesTab, (VOIDP)tabPhotonsTot);
			// Vérification du bon fonctionnement de la lecture
			if(status)
			{
				printf("\nERREUR : read hdf temoin\n");
				exit(1);
			}
			
		}
		// Fermeture du tableau
		SDendaccess (sdsTab);
	}
	// Fermeture du fichier
	SDend (sdFichier);
}


/* creerHDFResultats
* Fonction qui crée le fichier .hdf contenant le résultat final pour une demi-sphère
*/
void creerHDFResultats(double* tabFinal, double* tabTh, double* tabPhi,unsigned long long nbPhotonsTot, Variables* var,double
tempsPrec)
{
	// Tableau temporaire utile pour la suite
	double tab[NBPHI*NBTHETA];

	// Création du fichier de sortie
	int sdFichier = SDstart(PATHRESULTATSHDF, DFACC_CREATE);
	if (sdFichier == FAIL) {
		printf("ERREUR: création du fichier HDF : %s\n", PATHRESULTATSHDF);
		exit(1);
	}
	
	// Ecriture des informations sur la simulation : paramètres, nbphotons, nbErreurs, tempsEcoule
	double NBPHOTONSdouble = (double)NBPHOTONS;
	double nbPhotonsTotdouble = (double)nbPhotonsTot;
	double tempsEcouledouble = tempsPrec + (double)(clock() / CLOCKS_PER_SEC);

	SDsetattr(sdFichier, "NBPHOTONS", DFNT_FLOAT64, 1, &NBPHOTONSdouble);
	SDsetattr(sdFichier, "NBLOOP", DFNT_UINT32, 1, &NBLOOP);
	SDsetattr(sdFichier, "SEED", DFNT_UINT32, 1, &SEED);
	SDsetattr(sdFichier, "XBLOCK", DFNT_INT32, 1, &XBLOCK);
	SDsetattr(sdFichier, "YBLOCK", DFNT_INT32, 1, &YBLOCK);
	SDsetattr(sdFichier, "XGRID", DFNT_INT32, 1, &XGRID);
	SDsetattr(sdFichier, "YGRID", DFNT_INT32, 1, &YGRID);
	SDsetattr(sdFichier, "NBTHETA", DFNT_INT32, 1, &NBTHETA);
	SDsetattr(sdFichier, "NBPHI", DFNT_INT32, 1, &NBPHI);
	SDsetattr(sdFichier, "DIOPTRE", DFNT_INT32, 1, &DIOPTRE);
	SDsetattr(sdFichier, "DIFFF", DFNT_INT32, 1, &DIFFF);
	SDsetattr(sdFichier, "PROFIL", DFNT_INT32, 1, &PROFIL);
	SDsetattr(sdFichier, "SIM", DFNT_INT32, 1, &SIM);
	SDsetattr(sdFichier, "SUR", DFNT_INT32, 1, &SUR);
	SDsetattr(sdFichier, "THSDEG", DFNT_FLOAT32, 1, &THSDEG);
	SDsetattr(sdFichier, "LAMBDA", DFNT_FLOAT32, 1, &LAMBDA);
	SDsetattr(sdFichier, "TAURAY", DFNT_FLOAT32, 1, &TAURAY);
	SDsetattr(sdFichier, "TAUAER", DFNT_FLOAT32, 1, &TAUAER);
	
	SDsetattr(sdFichier, "LSAAER", DFNT_UINT32, 1, &LSAAER);
	SDsetattr(sdFichier, "NFAER", DFNT_UINT32, 1, &NFAER);
	SDsetattr(sdFichier, "LSAOCE", DFNT_UINT32, 1, &LSAOCE);
	SDsetattr(sdFichier, "NFOCE", DFNT_UINT32, 1, &NFOCE);
	
	SDsetattr(sdFichier, "W0AER", DFNT_FLOAT32, 1, &W0AER);
	SDsetattr(sdFichier, "W0LAM", DFNT_FLOAT32, 1, &W0LAM);
	SDsetattr(sdFichier, "HA", DFNT_FLOAT32, 1, &HA);
	SDsetattr(sdFichier, "HR", DFNT_FLOAT32, 1, &HR);
	SDsetattr(sdFichier, "ZMIN", DFNT_FLOAT32, 1, &ZMIN);
	SDsetattr(sdFichier, "ZMAX", DFNT_FLOAT32, 1, &ZMAX);
	SDsetattr(sdFichier, "NATM", DFNT_INT32, 1, &NATM);
	SDsetattr(sdFichier, "HATM", DFNT_INT32, 1, &HATM);
	SDsetattr(sdFichier, "WINDSPEED", DFNT_FLOAT32, 1, &WINDSPEED);
	SDsetattr(sdFichier, "NH2O", DFNT_FLOAT32, 1, &NH2O);
    #ifdef FLAGOCEAN
	SDsetattr(sdFichier, "CONPHY", DFNT_FLOAT32, 1, &CONPHY);
    #endif
	SDsetattr(sdFichier, "PATHRESULTATSHDF", DFNT_CHAR8, strlen(PATHRESULTATSHDF), PATHRESULTATSHDF);
	SDsetattr(sdFichier, "PATHTEMOINHDF", DFNT_CHAR8, strlen(PATHTEMOINHDF), PATHTEMOINHDF);
	SDsetattr(sdFichier, "PATHDIFFAER", DFNT_CHAR8, strlen(PATHDIFFAER), PATHDIFFAER);
	SDsetattr(sdFichier, "PATHPROFILATM", DFNT_CHAR8, strlen(PATHPROFILATM), PATHPROFILATM);
	
	SDsetattr(sdFichier, "nbPhotonsTot", DFNT_FLOAT64, 1, &nbPhotonsTotdouble);
	SDsetattr(sdFichier, "nbErreursPoids", DFNT_INT32, 1, &(var->erreurpoids));
	SDsetattr(sdFichier, "nbErreursTheta", DFNT_INT32, 1, &(var->erreurtheta));
	SDsetattr(sdFichier, "tempsEcoule", DFNT_FLOAT64, 1, &tempsEcouledouble);
	
	/** 	Création du 1er tableau dans le fichier hdf
		Valeur de la reflectance pour phi et theta donnés		**/
	char* nomTab="Valeurs de la reflectance (I)"; //nom du tableau
	int nbDimsTab = 2; //nombre de dimensions du tableau
	int valDimsTab[nbDimsTab]; //valeurs des dimensions du tableau
	valDimsTab[1] = NBTHETA;	//colonnes
	valDimsTab[0] = NBPHI;
	int typeTab = DFNT_FLOAT64; //type des éléments du tableau
	
	// Création du tableau
	int sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	int startTab[nbDimsTab]; //début de la lecture du tableau
	startTab[0]=0;
	startTab[1]=0;
	// Ecriture du tableau dans le fichier
	int status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP)tabFinal);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats reflectance\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau Q dans le fichier hdf
		Valeur de Q pour phi et theta donnés		**/
	nomTab="Valeurs de Q"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) (tabFinal+NBPHI*NBTHETA) );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats Q\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau U dans le fichier hdf
	Valeur de U pour phi et theta donnés		**/
	nomTab="Valeurs de U"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) (tabFinal+2*NBPHI*NBTHETA) );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats U\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau de lumière polarisée dans le fichier hdf
	Valeur de la lumière polarisée pour phi et theta donnés		**/
	nomTab="Valeurs de la lumiere polarisee (LP)"; //nom du tableau
	// La plupart des paramètres restent les mêmes, pas besoin de les réinitialiser
	
	for(int i = 0; i < NBTHETA*NBPHI; i++){
		tab[i] = sqrt( tabFinal[1*NBTHETA*NBPHI+i]*tabFinal[1*NBTHETA*NBPHI+i] +
						tabFinal[2*NBTHETA*NBPHI+i]*tabFinal[2*NBTHETA*NBPHI+i] );
	}
	
	// Création du tableau
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab);
	// Ecriture du tableau dans le fichier
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab, (VOIDP) tab );
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats lumiere polarisee\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	
	/** 	Création du tableau theta
		Valeurs de theta en fonction de l'indice	**/
	//conversion en degrès de theta pour une meilleure visualisation de la sortie
	float tabThBis[NBTHETA];
	for(int i=0; i<NBTHETA; i++)
		tabThBis[i] = tabTh[i]/DEG2RAD;
	
	nomTab = "Valeurs de theta echantillonnees";
	nbDimsTab = 1;
	int valDimsTab2[nbDimsTab];
	valDimsTab2[0] = NBTHETA;
	typeTab = DFNT_FLOAT32;
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab2);
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab2, (VOIDP) tabThBis);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats - tab Theta\n");
		exit(1);
	}
			
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	/** 	Création du tableau phi
		Valeurs de phi en fonction de l'indice	**/
	float tabPhiBis[NBPHI];
	for(int i=0; i<NBPHI; i++)
		tabPhiBis[i] = tabPhi[i]/DEG2RAD;
	
	nomTab = "Valeurs de phi echantillonnees";
	nbDimsTab = 1;
	int valDimsTab3[nbDimsTab];
	valDimsTab3[0] = NBPHI;
	typeTab = DFNT_FLOAT32;
	sdsTab = SDcreate(sdFichier, nomTab, typeTab, nbDimsTab, valDimsTab3);
	status = SDwritedata(sdsTab, startTab, NULL, valDimsTab3, (VOIDP)tabPhiBis);
	// Vérification du bon fonctionnement de l'écriture
	if(status)
	{
		printf("\nERREUR : write hdf resultats - tab Phi\n");
		exit(1);
	}
	
	// Fermeture du tableau
	SDendaccess(sdsTab);
	
	// Fermeture du fichier
	SDend(sdFichier);
	
}
