#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "main.h"
#include "communs.h"
#include "device.h"
#include "host.h"
#include "checkGPUcontext.h"

#ifdef RANDPHILOX4x32_7
//Cette partie est necessitee par les generateurs "1,2,3" pour leur version 64bits.
//Le Philox ne nous interesse pas en 64bits mais a moins de revoir tous les includes fournis avec le Philox (!),
//on est plutot contraint de rajouter ces trois lignes...
#ifndef __STDC_CONSTANT_MACROS
#define __STDC_CONSTANT_MACROS
#endif
#endif

#ifdef _PERF
#include "perfo.h"
static SPerf* perfPrint;
static SPerf* perfInitG;
static SPerf* perfKernel;
static SPerf* perfMemcpyH2DVar;
static SPerf* perfMemcpyH2DTab;
static SPerf* perfMemcpyD2HVar;
static SPerf* perfMemcpyD2HTab;
static SPerf* perfCreateWitness;
static SPerf* perfFree;
static SPerf* perfCreateFinalTab;
#endif


/**********************************************************
*	> Fonction main
***********************************************************/

// Fonction principale
int main (int argc, char *argv[])
{
        /** Initialisation des timers **/
        long int time_current;
        long int time_lastwrite = 0;

#ifdef _PERF
        perfPrint = NULL;
        perfInitG = NULL;
        perfKernel = NULL;
        perfMemcpyH2DVar = NULL;
        perfMemcpyH2DTab = NULL;
        perfMemcpyD2HVar = NULL;
        perfMemcpyD2HTab = NULL;
        perfCreateWitness = NULL;
        perfCreateFinalTab = NULL;
        perfFree = NULL;
//         perfPrint = CreateSPerf("Affichage");
        perfInitG = CreateSPerf("Init Totale");
        perfKernel = CreateSPerf("Kernel");
        perfMemcpyH2DVar = CreateSPerf("Copie Host vers Device (variables)");
        perfMemcpyH2DTab = CreateSPerf("Copie Host vers Device (tableaux)");
        perfMemcpyD2HVar = CreateSPerf("Copie Device vers Host (variables)");
        perfMemcpyD2HTab = CreateSPerf("Copie Device vers Host (tableaux)");
        perfCreateWitness = CreateSPerf("Creation du fichier temoin (calcul + ecriture)");
        perfCreateFinalTab = CreateSPerf("Creation du fichier final (calcul + ecriture)");
        perfFree = CreateSPerf("Liberation de la memoire");
        StartProcessing(perfInitG);
#endif

	/** Initialisation des constantes du host (en partie recuperees dans le fichier Parametres.txt) **/
	initConstantesHost(argc, argv);


    // read NATM and HATM in profile
    init_profile(&NATM, &HATM, PATHPROFILATM);

    // read LSAAER and LSAOCE
    LSAAER = count_lines(PATHDIFFAER);
    LSAOCE = count_lines(PATHDIFFOCE);
	

	/** Initialisation de la carte graphique **/
	hipError_t cudaErreur;	// Permet de vérifier les allocations mémoire
	
        // Verification de l'environnement GPU
        DEVICE = CheckGPUContext(DEVICE);
        if (DEVICE < 0){
            printf("\n!!MCCUDA Erreur!! main : erreur au sein de CheckGPUContext()\n");
            exit(1);
        }

	//
	hipDeviceReset();
	
	// Préférer utiliser plus de mémoire cache que de shared memory
	cudaErreur = hipFuncSetCacheConfig(reinterpret_cast<const void*>(lancementKernel),  hipFuncCachePreferL1);
	if( cudaErreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème cuFuncSetCacheConfig dans le main\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	/** Vérification que le code compilé est compatible avec la simulation demandée **/
	#ifndef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		printf("Veuillez compiler avec le flag FLAGOCEAN afin d'utiliser ce milieu \n");
		exit(1);
	}
	#endif
	
	
	/** Variables du main **/
	
	double tempsPrec = 0.; 	//temps ecoule de la simulation precedente
	
	// Regroupement et initialisation des variables a envoyer dans le kernel (structure de variables)
	Variables* var_H; //variables version host
	Variables* var_D; //variables version device
	initVariables(&var_H, &var_D);
	
	// Regroupement et initialisation des tableaux a envoyer dans le kernel (structure de pointeurs)
	Tableaux tab_H; //tableaux version host
	Tableaux tab_D; //tableaux version device
	initTableaux(&tab_H, &tab_D);
	
	// Variables et tableaux qui restent dans le host et se remplissent petit à petit
	unsigned long long nbPhotonsTot = 0; //nombre total de photons traités
	
	#ifdef PROGRESSION
	unsigned long long nbPhotonsSorTot = 0; //nombre total de photons ressortis
	#endif
	
	double* tabPhotonsTot; //tableau du poids total des photons sortis
	tabPhotonsTot = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTot)));
	if( tabPhotonsTot == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTot dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTot,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTot)));
	
	double* tabPhotonsTotDown; //tableau du poids total des photons sortis
	tabPhotonsTotDown = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotDown)));
	if( tabPhotonsTotDown == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotDown dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotDown,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotDown)));
	
	// Variables permettant le calcul du résultat final
    double *tabFinal;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinal = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

    double *tabFinalDown;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinalDown = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

	double *tabTh;
    tabTh = (double*)malloc(NBTHETA*sizeof(double));
	double *tabPhi;
    tabPhi = (double*)malloc(NBPHI*sizeof(double));
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Définition et initialisation des constantes initiales du photon
	Init* init_H;
	Init* init_D;
	initInit(&init_H, &init_D);
	#endif
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	
	/** Vérification de l'existence ou non d'un fichier témoin **/
    verifierFichier();
	
	#ifdef PARAMETRES
	/** Affichage des paramètres de la simulation **/
	afficheParametres();
	#endif
	
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	/** Calcul des modèles utiles à l'algorithme **/
	// Calcul de faer, modèle de diffusion des aérosols
    calculF( PATHDIFFAER, tab_H.faer, tab_D.faer, LSAAER, NFAER);
#ifdef _PERF
                StopProcessing(perfInitG);
                GetElapsedTime(perfInitG);
#endif
	
	// Calcul de foce, modèle de diffusion dans l'océan
	#ifdef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		calculF( PATHDIFFOCE, tab_H.foce, tab_D.foce, LSAOCE, NFOCE);
        float extoce = ATOT + BTOT;
        W0OCE = BTOT/extoce;
	}
	#endif

#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	// Calcul du mélange Molécule/Aérosol dans l'atmosphère en fonction de la couche
	profilAtm( &tab_H, &tab_D );
#ifdef _PERF
        StopProcessing(perfInitG);
        GetElapsedTime(perfInitG);
#endif
	
	
	/** Initialisation des constantes du device à partir des constantes du host **/
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	initConstantesDevice();

	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Calcul du point d'impact du photon
	impactInit(init_H, init_D, &tab_H, &tab_D);

	#ifdef DEBUG
	printf("Paramètres initiaux du photon: taumax0=%lf - zintermax=%lf - (%lf,%lf,%lf)\n",\
		   init_H->taumax0, init_H->zintermax0, init_H->x0, init_H->y0, init_H->z0 );
// 	for(int i=0; i<NATM+1; i++)
// 		printf("zph0[%d]=%10.7f - hph0[%d]=%10.7f\n",i, tab_H.zph0[i], i ,tab_H.hph0[i] );
	#endif
	
	#endif /* Fin de la spécification atmosphère sphérique */
	
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	/** Fonction qui permet de poursuivre la simulation précédente si elle n'est pas terminee **/
	lireHDFTemoin(var_H, var_D, &nbPhotonsTot, tabPhotonsTot, tabPhotonsTotDown, &tempsPrec);
	
	
	/** Organisation des threads en blocks de threads et en grids de blocks **/
	dim3 blockSize(XBLOCK,YBLOCK);
	dim3 gridSize(XGRID,YGRID);


	// Variable permettant de savoir si on est passé dans la boucle ou non
	bool passageBoucle = false;
	if(nbPhotonsTot < NBPHOTONS) 
		passageBoucle = true;
	
	// Tant qu'il n'y a pas assez de photons traités on relance le kernel
	while(nbPhotonsTot < NBPHOTONS)
	{
		/** Remise à zéro de certaines variables et certains tableaux **/
#ifdef _PERF
        StartProcessing(perfMemcpyH2DVar);
#endif
// 		reinitVariables(var_H, var_D);
		cudaErreur = hipMemset(&(var_D->nbPhotons), 0, sizeof(var_D->nbPhotons));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
#ifdef _PERF
        StopProcessing(perfMemcpyH2DVar);
        GetElapsedTime(perfMemcpyH2DVar);
#endif
		#ifdef PROGRESSION
		cudaErreur = hipMemset(&(var_D->nbPhotonsSor), 0, sizeof(var_D->nbPhotonsSor));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
		#endif
		
		
		/** Réinitialisation des données de la simulation **/
#ifdef _PERF
                StartProcessing(perfMemcpyH2DTab);
#endif
		cudaErreur = hipMemset(tab_D.tabPhotons, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotons)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsDown, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsDown)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsDown dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                StopProcessing(perfMemcpyH2DTab);
                GetElapsedTime(perfMemcpyH2DTab);
#endif
		
#ifdef _PERF
                StartProcessing(perfKernel);
#endif
		/** Lancement du kernel **/
		lancementKernel<<<gridSize, blockSize>>>(var_D, tab_D
				#ifdef SPHERIQUE
				, init_D
				#endif
							);
		// Attend que tous les threads aient fini avant de faire autre chose
// 		hipDeviceSynchronize();
#ifdef _PERF
                hipDeviceSynchronize();
                StopProcessing(perfKernel);
                GetElapsedTime(perfKernel);
#endif
		
#ifdef _PERF
                StartProcessing(perfMemcpyD2HVar);
#endif
		/** Récupération des variables et d'un tableau envoyés dans le kernel **/
		cudaErreur = hipMemcpy(var_H, var_D, sizeof(Variables), hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de copie var_D dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HVar);
                        GetElapsedTime(perfMemcpyD2HVar);
#endif

		// On remplit les variables et tableau qui restent dans le host
		nbPhotonsTot += var_H->nbPhotons;

#ifdef _PERF
                StartProcessing(perfMemcpyD2HTab);
#endif
		/** Copie des informations du device pour la création du fichier témoin **/
		/* Il a été remarqué que sans cette copie et remise à zéro du tableau tab_D.tabPhotons, des erreurs apparaissent si les
valeurs stockées sont élevées. Ceci doit venir du fait que l'on somme une grosse valeur à une plus faible */
		cudaErreur = hipMemcpy(tab_H.tabPhotons, tab_D.tabPhotons, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotons)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotons dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsDown, tab_D.tabPhotonsDown, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsDown)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsDown dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
        }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HTab);
                        GetElapsedTime(perfMemcpyD2HTab);
#endif

#ifdef _PERF
                StartProcessing(perfCreateWitness);
#endif
		/** Creation d'un fichier témoin pour pouvoir reprendre la simulation en cas d'arrêt **/
		for(int i = 0; i < 4*NBTHETA*NBPHI; i++) {
			tabPhotonsTot[i] += (double) tab_H.tabPhotons[i];
			tabPhotonsTotDown[i] += (double) tab_H.tabPhotonsDown[i];
        }
		
        time_current = clock() / CLOCKS_PER_SEC;
		if ((WRITE_PERIOD > 0) && ((time_current - time_lastwrite > 60*WRITE_PERIOD) || (time_lastwrite == 0))) {
            printf("== WRITING WITNESS FILE ==\n"); // FIXME
            creerHDFTemoin(tabPhotonsTot, tabPhotonsTotDown, nbPhotonsTot,var_H, tempsPrec);
            time_lastwrite = time_current;
        }
#ifdef _PERF
                        StopProcessing(perfCreateWitness);
                        GetElapsedTime(perfCreateWitness);
#endif
		
		#ifdef PROGRESSION
		nbPhotonsSorTot += var_H->nbPhotonsSor;
		#endif
		
		
		/** Affichage de l'avancement de la simulation **/
		afficheProgress(nbPhotonsTot, var_H, tempsPrec
			#ifdef PROGRESSION
			, nbPhotonsSorTot
			#endif
			);
	}
	
	// Si on n'est pas passé dans la boucle on affiche quand-même l'avancement de la simulation
	if(!passageBoucle) afficheProgress(nbPhotonsTot, var_H, tempsPrec
					#ifdef PROGRESSION
					, nbPhotonsSorTot
					#endif
					  );
	
	
#ifdef _PERF
        StartProcessing(perfCreateFinalTab);
#endif
	/** Création et calcul du tableau final (regroupant le poids de tous les photons ressortis sur une demi-sphère,
	 * par unité de surface) 
	**/	
	// Remplissage des 3 tableaux
	calculTabFinal(tabFinal, tabTh, tabPhi, tabPhotonsTot, nbPhotonsTot);
	calculTabFinal(tabFinalDown, tabTh, tabPhi, tabPhotonsTotDown, nbPhotonsTot);

	
	/** Fonction qui crée le fichier .hdf contenant le résultat final sur la demi-sphère **/
//	creerHDFResultats(tabFinal, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
	creerHDFResultats(tabFinal, tabFinalDown, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
#ifdef _PERF
        StopProcessing(perfCreateFinalTab);
        GetElapsedTime(perfCreateFinalTab);
#endif
	printf(" Fin de l'execution du programme. Resultats stockes dans %s\n",PATHRESULTATSHDF);
	

#ifdef _PERF
        StartProcessing(perfFree);
#endif
	/** Libération de la mémoire allouée **/
	// Libération du groupe de variables envoyé dans le kernel
	cudaErreur = hipFree(var_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de var_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(var_H);
    free(tabFinal);
    free(tabFinalDown);
    free(tabPhi);
    free(tabTh);

	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	cudaErreur = hipFree(init_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de init_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(init_H);
	#endif
	
	// Libération des tableaux envoyés dans le kernel
	freeTableaux(&tab_H, &tab_D);
	// Libération du tableau du host
	free( tabPhotonsTot );
	free( tabPhotonsTotDown );

	
#ifdef _PERF
        StopProcessing(perfFree);
        GetElapsedTime(perfFree);
#endif

#ifdef _PERF
        if ( perfPrint != NULL )
            printf("\n %s...%u us", GetName(perfPrint), GetTotalElapsedTime(perfPrint));
        if ( perfInitG != NULL )
            printf("\n %s...%u us", GetName(perfInitG), GetTotalElapsedTime(perfInitG));
        if ( perfKernel != NULL )
            printf("\n %s...%u us", GetName(perfKernel), GetTotalElapsedTime(perfKernel));
        if ( perfMemcpyH2DVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DVar), GetTotalElapsedTime(perfMemcpyH2DVar));
        if ( perfMemcpyH2DTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DTab), GetTotalElapsedTime(perfMemcpyH2DTab));
        if ( perfMemcpyD2HVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HVar), GetTotalElapsedTime(perfMemcpyD2HVar));
        if ( perfMemcpyD2HTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HTab), GetTotalElapsedTime(perfMemcpyD2HTab));
        if ( perfCreateWitness != NULL )
            printf("\n %s...%u us", GetName(perfCreateWitness), GetTotalElapsedTime(perfCreateWitness));
        if ( perfCreateFinalTab != NULL )
            printf("\n %s...%u us", GetName(perfCreateFinalTab), GetTotalElapsedTime(perfCreateFinalTab));
        if ( perfFree != NULL )
            printf("\n %s...%u us", GetName(perfFree), GetTotalElapsedTime(perfFree));
        DeleteSPerf(perfPrint);
        DeleteSPerf(perfInitG);
        DeleteSPerf(perfKernel);
        DeleteSPerf(perfMemcpyH2DVar);
        DeleteSPerf(perfMemcpyH2DTab);
        DeleteSPerf(perfMemcpyD2HVar);
        DeleteSPerf(perfMemcpyD2HTab);
        DeleteSPerf(perfCreateWitness);
        DeleteSPerf(perfCreateFinalTab);
        DeleteSPerf(perfFree);
        printf("\n");
#endif

    message_end(DEVICE);

        //
        hipDeviceReset();
	return 0;

}
