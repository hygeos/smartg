#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "main.h"
#include "communs.h"
#include "device.h"
#include "host.h"
#include "checkGPUcontext.h"

#ifdef RANDPHILOX4x32_7
//Cette partie est necessitee par les generateurs "1,2,3" pour leur version 64bits.
//Le Philox ne nous interesse pas en 64bits mais a moins de revoir tous les includes fournis avec le Philox (!),
//on est plutot contraint de rajouter ces trois lignes...
#ifndef __STDC_CONSTANT_MACROS
#define __STDC_CONSTANT_MACROS
#endif
#endif

#ifdef _PERF
#include "perfo.h"
static SPerf* perfPrint;
static SPerf* perfInitG;
static SPerf* perfKernel;
static SPerf* perfMemcpyH2DVar;
static SPerf* perfMemcpyH2DTab;
static SPerf* perfMemcpyD2HVar;
static SPerf* perfMemcpyD2HTab;
static SPerf* perfCreateWitness;
static SPerf* perfFree;
static SPerf* perfCreateFinalTab;
#endif


/**********************************************************
*	> Fonction main
***********************************************************/

// Fonction principale
int main (int argc, char *argv[])
{
        /** Initialisation des timers **/
        long int time_current;
        long int time_lastwrite = 0;

#ifdef _PERF
        perfPrint = NULL;
        perfInitG = NULL;
        perfKernel = NULL;
        perfMemcpyH2DVar = NULL;
        perfMemcpyH2DTab = NULL;
        perfMemcpyD2HVar = NULL;
        perfMemcpyD2HTab = NULL;
        perfCreateWitness = NULL;
        perfCreateFinalTab = NULL;
        perfFree = NULL;
//         perfPrint = CreateSPerf("Affichage");
        perfInitG = CreateSPerf("Init Totale");
        perfKernel = CreateSPerf("Kernel");
        perfMemcpyH2DVar = CreateSPerf("Copie Host vers Device (variables)");
        perfMemcpyH2DTab = CreateSPerf("Copie Host vers Device (tableaux)");
        perfMemcpyD2HVar = CreateSPerf("Copie Device vers Host (variables)");
        perfMemcpyD2HTab = CreateSPerf("Copie Device vers Host (tableaux)");
        perfCreateWitness = CreateSPerf("Creation du fichier temoin (calcul + ecriture)");
        perfCreateFinalTab = CreateSPerf("Creation du fichier final (calcul + ecriture)");
        perfFree = CreateSPerf("Liberation de la memoire");
        StartProcessing(perfInitG);
#endif

	/** Initialisation des constantes du host (en partie recuperees dans le fichier Parametres.txt) **/
	initConstantesHost(argc, argv);


    // read NATM and HATM in profile
    // (if simulation includes atmosphere)
    if ((SIM == -2) || (SIM == 1) || (SIM == 2)) {
        init_profile(&NATM, &HATM, PATHPROFILATM);
    } else {
        HATM = 0;
        NATM = 0;
        TRANSDIR=1.;
    }


    // read LSAAER and LSAOCE
    LSAAER = count_lines(PATHDIFFAER);
    LSAOCE = count_lines(PATHDIFFOCE);
	

	/** Initialisation de la carte graphique **/
	hipError_t cudaErreur;	// Permet de vérifier les allocations mémoire
	
        // Verification de l'environnement GPU
        DEVICE = CheckGPUContext(DEVICE);
        if (DEVICE < 0){
            printf("\n!!MCCUDA Erreur!! main : erreur au sein de CheckGPUContext()\n");
            exit(1);
        }

	//
	hipDeviceReset();
	
	// Préférer utiliser plus de mémoire cache que de shared memory
	cudaErreur = hipFuncSetCacheConfig(reinterpret_cast<const void*>(lancementKernel),  hipFuncCachePreferL1);
	if( cudaErreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème cuFuncSetCacheConfig dans le main\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	/** Vérification que le code compilé est compatible avec la simulation demandée **/
	#ifndef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		printf("Veuillez compiler avec le flag FLAGOCEAN afin d'utiliser ce milieu \n");
		exit(1);
	}
	#endif
	
	
	/** Variables du main **/
	
	double tempsPrec = 0.; 	//temps ecoule de la simulation precedente
	
	// Regroupement et initialisation des variables a envoyer dans le kernel (structure de variables)
	Variables* var_H; //variables version host
	Variables* var_D; //variables version device
	initVariables(&var_H, &var_D);
	
	// Regroupement et initialisation des tableaux a envoyer dans le kernel (structure de pointeurs)
	Tableaux tab_H; //tableaux version host
	Tableaux tab_D; //tableaux version device
	initTableaux(&tab_H, &tab_D);
	
	// Variables et tableaux qui restent dans le host et se remplissent petit à petit
	unsigned long long nbPhotonsTot = 0; //nombre total de photons traités
	
	#ifdef PROGRESSION
	unsigned long long nbPhotonsSorTot = 0; //nombre total de photons ressortis
	#endif
	
	double* tabPhotonsTot; //tableau du poids total des photons sortis
	tabPhotonsTot = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTot)));
	if( tabPhotonsTot == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTot dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTot,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTot)));
	
	double* tabPhotonsTotDown0P; //tableau du poids total des photons sortis
	tabPhotonsTotDown0P = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotDown0P)));
	if( tabPhotonsTotDown0P == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotDown0P dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotDown0P,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotDown0P)));
	
	double* tabPhotonsTotDown0M; //tableau du poids total des photons sortis
	tabPhotonsTotDown0M = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotDown0M)));
	if( tabPhotonsTotDown0M == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotDown0M dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotDown0M,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotDown0M)));
	
	double* tabPhotonsTotUp0M; //tableau du poids total des photons sortis
	tabPhotonsTotUp0M = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotUp0M)));
	if( tabPhotonsTotUp0M == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotUp0M dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotUp0M,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotUp0M)));
	
	double* tabPhotonsTotUp0P; //tableau du poids total des photons sortis
	tabPhotonsTotUp0P = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotUp0P)));
	if( tabPhotonsTotUp0P == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotUp0P dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotUp0P,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotUp0P)));
	
	// Variables permettant le calcul du résultat final
    double *tabFinal;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinal = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

    double *tabFinalDown0P;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinalDown0P = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

    double *tabFinalDown0M; 
    tabFinalDown0M = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));
    double *tabFinalUp0P; 
    tabFinalUp0P = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));
    double *tabFinalUp0M; 
    tabFinalUp0M = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

	double *tabTh;
    tabTh = (double*)malloc(NBTHETA*sizeof(double));
	double *tabPhi;
    tabPhi = (double*)malloc(NBPHI*sizeof(double));
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Définition et initialisation des constantes initiales du photon
	Init* init_H;
	Init* init_D;
	initInit(&init_H, &init_D);
	#endif
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	
	/** Vérification de l'existence ou non d'un fichier témoin **/
    verifierFichier();
	
	#ifdef PARAMETRES
	/** Affichage des paramètres de la simulation **/
	afficheParametres();
	#endif
	
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	/** Calcul des modèles utiles à l'algorithme **/
	// Calcul de faer, modèle de diffusion des aérosols
    calculF( PATHDIFFAER, tab_H.faer, tab_D.faer, LSAAER, NFAER);
#ifdef _PERF
                StopProcessing(perfInitG);
                GetElapsedTime(perfInitG);
#endif
	
	// Calcul de foce, modèle de diffusion dans l'océan
	#ifdef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		calculF( PATHDIFFOCE, tab_H.foce, tab_D.foce, LSAOCE, NFOCE);
        float extoce = ATOT + BTOT;
        W0OCE = BTOT/extoce;
	}
	#endif

#ifdef _PERF
        StartProcessing(perfInitG);
#endif
    if ((SIM == -2) || (SIM == 1) || (SIM == 2)) {
        // Read atmospheric profile
        profilAtm(&tab_H, &tab_D);
        TRANSDIR = exp(-TAUATM/cos(THVDEG*PI/180.));
    }
#ifdef _PERF
        StopProcessing(perfInitG);
        GetElapsedTime(perfInitG);
#endif
	
	
	/** Initialisation des constantes du device à partir des constantes du host **/
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	initConstantesDevice();

	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Calcul du point d'impact du photon
	impactInit(init_H, init_D, &tab_H, &tab_D);
    TRANSDIR = exp(-init_H->taumax0);

	#ifdef DEBUG
	printf("Paramètres initiaux du photon: taumax0=%lf - zintermax=%lf - (%lf,%lf,%lf)\n",\
		   init_H->taumax0, init_H->zintermax0, init_H->x0, init_H->y0, init_H->z0 );
// 	for(int i=0; i<NATM+1; i++)
// 		printf("zph0[%d]=%10.7f - hph0[%d]=%10.7f\n",i, tab_H.zph0[i], i ,tab_H.hph0[i] );
	#endif
	
	#endif /* Fin de la spécification atmosphère sphérique */
	
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	/** Fonction qui permet de poursuivre la simulation précédente si elle n'est pas terminee **/
	//lireHDFTemoin(var_H, var_D, &nbPhotonsTot, tabPhotonsTot, tabPhotonsTotDown0P, &tempsPrec);
	
	
	/** Organisation des threads en blocks de threads et en grids de blocks **/
	dim3 blockSize(XBLOCK,YBLOCK);
	dim3 gridSize(XGRID,YGRID);


	// Variable permettant de savoir si on est passé dans la boucle ou non
	bool passageBoucle = false;
	if(nbPhotonsTot < NBPHOTONS) 
		passageBoucle = true;
	
	// Tant qu'il n'y a pas assez de photons traités on relance le kernel
	while(nbPhotonsTot < NBPHOTONS)
	{
		/** Remise à zéro de certaines variables et certains tableaux **/
#ifdef _PERF
        StartProcessing(perfMemcpyH2DVar);
#endif
// 		reinitVariables(var_H, var_D);
		cudaErreur = hipMemset(&(var_D->nbPhotons), 0, sizeof(var_D->nbPhotons));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
#ifdef _PERF
        StopProcessing(perfMemcpyH2DVar);
        GetElapsedTime(perfMemcpyH2DVar);
#endif
		#ifdef PROGRESSION
		cudaErreur = hipMemset(&(var_D->nbPhotonsSor), 0, sizeof(var_D->nbPhotonsSor));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
		#endif
		
		
		/** Réinitialisation des données de la simulation **/
#ifdef _PERF
                StartProcessing(perfMemcpyH2DTab);
#endif
		cudaErreur = hipMemset(tab_D.tabPhotons, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotons)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsDown0P, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsDown0P)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsDown0P dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsDown0M, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsDown0M)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsDown0M dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsUp0P, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsUp0P)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsUp0P dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsUp0M, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsUp0M)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsUp0M dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                StopProcessing(perfMemcpyH2DTab);
                GetElapsedTime(perfMemcpyH2DTab);
#endif
		
#ifdef _PERF
                StartProcessing(perfKernel);
#endif
		/** Lancement du kernel **/
		lancementKernel<<<gridSize, blockSize>>>(var_D, tab_D
				#ifdef SPHERIQUE
				, init_D
				#endif
							);
		// Attend que tous les threads aient fini avant de faire autre chose
// 		hipDeviceSynchronize();
#ifdef _PERF
                hipDeviceSynchronize();
                StopProcessing(perfKernel);
                GetElapsedTime(perfKernel);
#endif
		
#ifdef _PERF
                StartProcessing(perfMemcpyD2HVar);
#endif
		/** Récupération des variables et d'un tableau envoyés dans le kernel **/
		cudaErreur = hipMemcpy(var_H, var_D, sizeof(Variables), hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de copie var_D dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HVar);
                        GetElapsedTime(perfMemcpyD2HVar);
#endif

		// On remplit les variables et tableau qui restent dans le host
		nbPhotonsTot += var_H->nbPhotons;

#ifdef _PERF
                StartProcessing(perfMemcpyD2HTab);
#endif
		/** Copie des informations du device pour la création du fichier témoin **/
		/* Il a été remarqué que sans cette copie et remise à zéro du tableau tab_D.tabPhotons, des erreurs apparaissent si les
valeurs stockées sont élevées. Ceci doit venir du fait que l'on somme une grosse valeur à une plus faible */
		cudaErreur = hipMemcpy(tab_H.tabPhotons, tab_D.tabPhotons, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotons)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotons dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsDown0P, tab_D.tabPhotonsDown0P, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsDown0P)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsDown0P dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsDown0M, tab_D.tabPhotonsDown0M, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsDown0M)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsDown0M dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsUp0P, tab_D.tabPhotonsUp0P, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsUp0P)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsUp0P dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsUp0M, tab_D.tabPhotonsUp0M, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsUp0M)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsUp0M dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
        }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HTab);
                        GetElapsedTime(perfMemcpyD2HTab);
#endif

#ifdef _PERF
                StartProcessing(perfCreateWitness);
#endif
		/** Creation d'un fichier témoin pour pouvoir reprendre la simulation en cas d'arrêt **/
		for(int i = 0; i < 4*NBTHETA*NBPHI; i++) {
			tabPhotonsTot[i] += (double) tab_H.tabPhotons[i];
			tabPhotonsTotDown0P[i] += (double) tab_H.tabPhotonsDown0P[i];
			tabPhotonsTotDown0M[i] += (double) tab_H.tabPhotonsDown0M[i];
			tabPhotonsTotUp0P[i] += (double) tab_H.tabPhotonsUp0P[i];
			tabPhotonsTotUp0M[i] += (double) tab_H.tabPhotonsUp0M[i];
        }
		
        time_current = clock() / CLOCKS_PER_SEC;
		if ((WRITE_PERIOD > 0) && ((time_current - time_lastwrite > 60*WRITE_PERIOD) || (time_lastwrite == 0))) {
            printf("== WRITING WITNESS FILE ==\n"); // FIXME
            //creerHDFTemoin(tabPhotonsTot, tabPhotonsTotDown0P, nbPhotonsTot,var_H, tempsPrec);
            time_lastwrite = time_current;
        }
#ifdef _PERF
                        StopProcessing(perfCreateWitness);
                        GetElapsedTime(perfCreateWitness);
#endif
		
		#ifdef PROGRESSION
		nbPhotonsSorTot += var_H->nbPhotonsSor;
		#endif
		
		
		/** Affichage de l'avancement de la simulation **/
		afficheProgress(nbPhotonsTot, var_H, tempsPrec
			#ifdef PROGRESSION
			, nbPhotonsSorTot
			#endif
			);
	}
	
	// Si on n'est pas passé dans la boucle on affiche quand-même l'avancement de la simulation
	if(!passageBoucle) afficheProgress(nbPhotonsTot, var_H, tempsPrec
					#ifdef PROGRESSION
					, nbPhotonsSorTot
					#endif
					  );
	
	
#ifdef _PERF
        StartProcessing(perfCreateFinalTab);
#endif
	/** Création et calcul du tableau final (regroupant le poids de tous les photons ressortis sur une demi-sphère,
	 * par unité de surface) 
	**/	
	// Remplissage des 3 tableaux
	calculTabFinal(tabFinal, tabTh, tabPhi, tabPhotonsTot, nbPhotonsTot);
	calculTabFinal(tabFinalDown0P, tabTh, tabPhi, tabPhotonsTotDown0P, nbPhotonsTot);
	calculTabFinal(tabFinalDown0M, tabTh, tabPhi, tabPhotonsTotDown0M, nbPhotonsTot);
	calculTabFinal(tabFinalUp0P, tabTh, tabPhi, tabPhotonsTotUp0P, nbPhotonsTot);
	calculTabFinal(tabFinalUp0M, tabTh, tabPhi, tabPhotonsTotUp0M, nbPhotonsTot);

	
	/** Fonction qui crée le fichier .hdf contenant le résultat final sur la demi-sphère **/
//	creerHDFResultats(tabFinal, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
//	creerHDFResultats(tabFinal, tabFinalDown0P, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
	creerHDFResultats(tabFinal, tabFinalDown0P, tabFinalDown0M, tabFinalUp0P, tabFinalUp0M, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
#ifdef _PERF
        StopProcessing(perfCreateFinalTab);
        GetElapsedTime(perfCreateFinalTab);
#endif
	printf(" Fin de l'execution du programme. Resultats stockes dans %s\n",PATHRESULTATSHDF);
	

#ifdef _PERF
        StartProcessing(perfFree);
#endif
	/** Libération de la mémoire allouée **/
	// Libération du groupe de variables envoyé dans le kernel
	cudaErreur = hipFree(var_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de var_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(var_H);
    free(tabFinal);
    free(tabFinalDown0P);
    free(tabFinalDown0M);
    free(tabFinalUp0P);
    free(tabFinalUp0M);
    free(tabPhi);
    free(tabTh);

	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	cudaErreur = hipFree(init_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de init_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(init_H);
	#endif
	
	// Libération des tableaux envoyés dans le kernel
	freeTableaux(&tab_H, &tab_D);
	// Libération du tableau du host
	free( tabPhotonsTot );
	free( tabPhotonsTotDown0P );
	free( tabPhotonsTotDown0M );
	free( tabPhotonsTotUp0P );
	free( tabPhotonsTotUp0M );

	
#ifdef _PERF
        StopProcessing(perfFree);
        GetElapsedTime(perfFree);
#endif

#ifdef _PERF
        if ( perfPrint != NULL )
            printf("\n %s...%u us", GetName(perfPrint), GetTotalElapsedTime(perfPrint));
        if ( perfInitG != NULL )
            printf("\n %s...%u us", GetName(perfInitG), GetTotalElapsedTime(perfInitG));
        if ( perfKernel != NULL )
            printf("\n %s...%u us", GetName(perfKernel), GetTotalElapsedTime(perfKernel));
        if ( perfMemcpyH2DVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DVar), GetTotalElapsedTime(perfMemcpyH2DVar));
        if ( perfMemcpyH2DTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DTab), GetTotalElapsedTime(perfMemcpyH2DTab));
        if ( perfMemcpyD2HVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HVar), GetTotalElapsedTime(perfMemcpyD2HVar));
        if ( perfMemcpyD2HTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HTab), GetTotalElapsedTime(perfMemcpyD2HTab));
        if ( perfCreateWitness != NULL )
            printf("\n %s...%u us", GetName(perfCreateWitness), GetTotalElapsedTime(perfCreateWitness));
        if ( perfCreateFinalTab != NULL )
            printf("\n %s...%u us", GetName(perfCreateFinalTab), GetTotalElapsedTime(perfCreateFinalTab));
        if ( perfFree != NULL )
            printf("\n %s...%u us", GetName(perfFree), GetTotalElapsedTime(perfFree));
        DeleteSPerf(perfPrint);
        DeleteSPerf(perfInitG);
        DeleteSPerf(perfKernel);
        DeleteSPerf(perfMemcpyH2DVar);
        DeleteSPerf(perfMemcpyH2DTab);
        DeleteSPerf(perfMemcpyD2HVar);
        DeleteSPerf(perfMemcpyD2HTab);
        DeleteSPerf(perfCreateWitness);
        DeleteSPerf(perfCreateFinalTab);
        DeleteSPerf(perfFree);
        printf("\n");
#endif

    message_end(DEVICE);

        //
        hipDeviceReset();
	return 0;

}
