#include "hip/hip_runtime.h"

/**********************************************************
*	> Includes
***********************************************************/

#include "main.h"
#include "communs.h"
#include "device.h"
#include "host.h"
#include "checkGPUcontext.h"

#ifdef RANDPHILOX4x32_7
//Cette partie est necessitee par les generateurs "1,2,3" pour leur version 64bits.
//Le Philox ne nous interesse pas en 64bits mais a moins de revoir tous les includes fournis avec le Philox (!),
//on est plutot contraint de rajouter ces trois lignes...
#ifndef __STDC_CONSTANT_MACROS
#define __STDC_CONSTANT_MACROS
#endif
#endif

#ifdef _PERF
#include "perfo.h"
static SPerf* perfPrint;
static SPerf* perfInitG;
static SPerf* perfKernel;
static SPerf* perfMemcpyH2DVar;
static SPerf* perfMemcpyH2DTab;
static SPerf* perfMemcpyD2HVar;
static SPerf* perfMemcpyD2HTab;
static SPerf* perfCreateWitness;
static SPerf* perfFree;
static SPerf* perfCreateFinalTab;
#endif


/**********************************************************
*	> Fonction main
***********************************************************/

// Fonction principale
int main (int argc, char *argv[])
{
        /** Initialisation des timers **/
        long int time_current;
        long int time_lastwrite = 0;

#ifdef _PERF
        perfPrint = NULL;
        perfInitG = NULL;
        perfKernel = NULL;
        perfMemcpyH2DVar = NULL;
        perfMemcpyH2DTab = NULL;
        perfMemcpyD2HVar = NULL;
        perfMemcpyD2HTab = NULL;
        perfCreateWitness = NULL;
        perfCreateFinalTab = NULL;
        perfFree = NULL;
//         perfPrint = CreateSPerf("Affichage");
        perfInitG = CreateSPerf("Init Totale");
        perfKernel = CreateSPerf("Kernel");
        perfMemcpyH2DVar = CreateSPerf("Copie Host vers Device (variables)");
        perfMemcpyH2DTab = CreateSPerf("Copie Host vers Device (tableaux)");
        perfMemcpyD2HVar = CreateSPerf("Copie Device vers Host (variables)");
        perfMemcpyD2HTab = CreateSPerf("Copie Device vers Host (tableaux)");
        perfCreateWitness = CreateSPerf("Creation du fichier temoin (calcul + ecriture)");
        perfCreateFinalTab = CreateSPerf("Creation du fichier final (calcul + ecriture)");
        perfFree = CreateSPerf("Liberation de la memoire");
        StartProcessing(perfInitG);
#endif

	/** Initialisation de la carte graphique **/
	hipError_t cudaErreur;	// Permet de vérifier les allocations mémoire
	
        // Verification de l'environnement GPU
        if ( CheckGPUContext() != MCCUDA_OK ){
            printf("\n!!MCCUDA Erreur!! main : erreur au sein de CheckGPUContext()\n");
            exit(1);
        }

	//
	hipDeviceReset();
	
	// Préférer utiliser plus de mémoire cache que de shared memory
	cudaErreur = hipFuncSetCacheConfig(reinterpret_cast<const void*>(lancementKernel),  hipFuncCachePreferL1);
	if( cudaErreur != hipSuccess ){
		printf("#--------------------#\n");
		printf("# ERREUR: Problème cuFuncSetCacheConfig dans le main\n");
		printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		printf("#--------------------#\n");
		exit(1);
	}
	
	/** Initialisation des constantes du host (en partie recuperees dans le fichier Parametres.txt) **/
	initConstantesHost(argc, argv);
	
	
	/** Vérification que le code compilé est compatible avec la simulation demandée **/
	#ifndef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		printf("Veuillez compiler avec le flag FLAGOCEAN afin d'utiliser ce milieu \n");
		exit(1);
	}
	#endif
	
	
	/** Variables du main **/
	
	double tempsPrec = 0.; 	//temps ecoule de la simulation precedente
	
	// Regroupement et initialisation des variables a envoyer dans le kernel (structure de variables)
	Variables* var_H; //variables version host
	Variables* var_D; //variables version device
	initVariables(&var_H, &var_D);
	
	// Regroupement et initialisation des tableaux a envoyer dans le kernel (structure de pointeurs)
	Tableaux tab_H; //tableaux version host
	Tableaux tab_D; //tableaux version device
	initTableaux(&tab_H, &tab_D);
	
	// Variables et tableaux qui restent dans le host et se remplissent petit à petit
	unsigned long long nbPhotonsTot = 0; //nombre total de photons traités
	
	#ifdef PROGRESSION
	unsigned long long nbPhotonsSorTot = 0; //nombre total de photons ressortis
	#endif
	
	double* tabPhotonsTot; //tableau du poids total des photons sortis
	tabPhotonsTot = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTot)));
	if( tabPhotonsTot == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTot dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTot,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTot)));
	
	double* tabPhotonsTotDown; //tableau du poids total des photons sortis
	tabPhotonsTotDown = (double*)malloc(4*NBTHETA * NBPHI * sizeof(*(tabPhotonsTotDown)));
	if( tabPhotonsTotDown == NULL ){
		printf("ERREUR: Problème de malloc de tabPhotonsTotDown dans le main\n");
		exit(1);
	}
	
	memset(tabPhotonsTotDown,0,4*NBTHETA*NBPHI*sizeof(*(tabPhotonsTotDown)));
	
	// Variables permettant le calcul du résultat final
    double *tabFinal;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinal = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

    double *tabFinalDown;   // tableau final: 4 dimensions pour
                        // R=stokes1+stokes2(dim0) , Q=stokes1-stokes2(dim1),
                        // U=stokes3(dim2)  et Nbphoton(dim4)
    tabFinalDown = (double*)malloc(4*NBTHETA*NBPHI*sizeof(double));

	double *tabTh;
    tabTh = (double*)malloc(NBTHETA*sizeof(double));
	double *tabPhi;
    tabPhi = (double*)malloc(NBPHI*sizeof(double));
	
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Définition et initialisation des constantes initiales du photon
	Init* init_H;
	Init* init_D;
	initInit(&init_H, &init_D);
	#endif
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	#ifdef TABRAND
	// DEBUG Recuperations des nombres aleatoires du random en place
	float tableauRand_H[100] = {0};
	float* tableauRand_D;
	if( hipMalloc(&tableauRand_D, 100 * sizeof(float)) != hipSuccess){
		printf("ERREUR: Problème de hipMalloc de tableauRand_D dans le main\n");
		exit(1);
	}
	hipMemset(tableauRand_D, 0, 100 * sizeof(float));
	#endif
	
	
	/** Vérification de l'existence ou non d'un fichier témoin **/
    verifierFichier();
	
	#ifdef PARAMETRES
	/** Affichage des paramètres de la simulation **/
	afficheParametres();
	#endif
	
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	/** Calcul des modèles utiles à l'algorithme **/
	// Calcul de faer, modèle de diffusion des aérosols
	if( TAUAER > 0 ){
		calculFaer( PATHDIFFAER, &tab_H, &tab_D );
// 		verificationFAER( "./test/FAER_test.txt", tab_H );
        }
#ifdef _PERF
                StopProcessing(perfInitG);
                GetElapsedTime(perfInitG);
#endif
	
	// Calcul de foce, modèle de diffusion dans l'océan
	#ifdef FLAGOCEAN
	if( SIM==0 || SIM==2 || SIM==3 ){
		calculFoce( &tab_H, &tab_D );
// 		verificationFoce( "./test/Foce_test.txt", tab_H );
	}
	#endif

#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	// Calcul du mélange Molécule/Aérosol dans l'atmosphère en fonction de la couche
	profilAtm( &tab_H, &tab_D );
// 	verificationAtm( tab_H );
#ifdef _PERF
        StopProcessing(perfInitG);
        GetElapsedTime(perfInitG);
#endif
	
	
	/** Initialisation des constantes du device à partir des constantes du host **/
#ifdef _PERF
        StartProcessing(perfInitG);
#endif
	initConstantesDevice();
	
	/** Séparation du code pour atmosphère sphérique ou parallèle **/
	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	// Calcul du point d'impact du photon
	impactInit(init_H, init_D, &tab_H, &tab_D);

	#ifdef DEBUG
	printf("Paramètres initiaux du photon: taumax0=%lf - zintermax=%lf - (%lf,%lf,%lf)\n",\
		   init_H->taumax0, init_H->zintermax0, init_H->x0, init_H->y0, init_H->z0 );
// 	for(int i=0; i<NATM+1; i++)
// 		printf("zph0[%d]=%10.7f - hph0[%d]=%10.7f\n",i, tab_H.zph0[i], i ,tab_H.hph0[i] );
	#endif
	
	#endif /* Fin de la spécification atmosphère sphérique */
	
	
#ifdef _PERF
    StopProcessing(perfInitG);
    GetElapsedTime(perfInitG);
#endif

	/** Fonction qui permet de poursuivre la simulation précédente si elle n'est pas terminee **/
	lireHDFTemoin(var_H, var_D, &nbPhotonsTot, tabPhotonsTot, tabPhotonsTotDown, &tempsPrec);
	
	
	#ifdef TRAJET
	// DEBUG : Variables permettant de récupérer le début du trajet d'un photon
	Evnt evnt_H[NBTRAJET];
	Evnt* evnt_D;
	if( hipMalloc(&evnt_D, NBTRAJET* sizeof(Evnt)) != hipSuccess){
		printf("ERREUR: Problème de hipMalloc de evnt_D dans le main\n");
		exit(1);
	}

	initEvnt(evnt_H, evnt_D);
	#endif
	
	/** Organisation des threads en blocks de threads et en grids de blocks **/
	dim3 blockSize(XBLOCK,YBLOCK);
	dim3 gridSize(XGRID,YGRID);


	// Variable permettant de savoir si on est passé dans la boucle ou non
	bool passageBoucle = false;
	if(nbPhotonsTot < NBPHOTONS) 
		passageBoucle = true;
	
	// Tant qu'il n'y a pas assez de photons traités on relance le kernel
	while(nbPhotonsTot < NBPHOTONS)
	{
		/** Remise à zéro de certaines variables et certains tableaux **/
#ifdef _PERF
        StartProcessing(perfMemcpyH2DVar);
#endif
// 		reinitVariables(var_H, var_D);
		cudaErreur = hipMemset(&(var_D->nbPhotons), 0, sizeof(var_D->nbPhotons));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
#ifdef _PERF
        StopProcessing(perfMemcpyH2DVar);
        GetElapsedTime(perfMemcpyH2DVar);
#endif
		#ifdef PROGRESSION
		cudaErreur = hipMemset(&(var_D->nbPhotonsSor), 0, sizeof(var_D->nbPhotonsSor));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset var_D.nbPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
		}
		#endif
		
		
		/** Réinitialisation des données de la simulation **/
#ifdef _PERF
                StartProcessing(perfMemcpyH2DTab);
#endif
		cudaErreur = hipMemset(tab_D.tabPhotons, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotons)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotons dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }

		cudaErreur = hipMemset(tab_D.tabPhotonsDown, 0, 4*NBTHETA * NBPHI * sizeof(*(tab_D.tabPhotonsDown)));
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de hipMemset tab_D.tabPhotonsDown dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                StopProcessing(perfMemcpyH2DTab);
                GetElapsedTime(perfMemcpyH2DTab);
#endif
		
#ifdef _PERF
                StartProcessing(perfKernel);
#endif
		/** Lancement du kernel **/
		lancementKernel<<<gridSize, blockSize>>>(var_D, tab_D
				#ifdef SPHERIQUE
				, init_D
				#endif
				#ifdef TABRAND
				, tableauRand_D
				#endif
				#ifdef TRAJET
				, evnt_D //récupération d'un trajet de photons
				#endif
							);
		// Attend que tous les threads aient fini avant de faire autre chose
// 		hipDeviceSynchronize();
#ifdef _PERF
                hipDeviceSynchronize();
                StopProcessing(perfKernel);
                GetElapsedTime(perfKernel);
#endif
		
#ifdef _PERF
                StartProcessing(perfMemcpyD2HVar);
#endif
		/** Récupération des variables et d'un tableau envoyés dans le kernel **/
		cudaErreur = hipMemcpy(var_H, var_D, sizeof(Variables), hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf("#--------------------#\n");
			printf("# ERREUR: Problème de copie var_D dans le main\n");
			printf("# Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			printf("#--------------------#\n");
			exit(1);
                }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HVar);
                        GetElapsedTime(perfMemcpyD2HVar);
#endif

		// On remplit les variables et tableau qui restent dans le host
		nbPhotonsTot += var_H->nbPhotons;

#ifdef _PERF
                StartProcessing(perfMemcpyD2HTab);
#endif
		/** Copie des informations du device pour la création du fichier témoin **/
		/* Il a été remarqué que sans cette copie et remise à zéro du tableau tab_D.tabPhotons, des erreurs apparaissent si les
valeurs stockées sont élevées. Ceci doit venir du fait que l'on somme une grosse valeur à une plus faible */
		cudaErreur = hipMemcpy(tab_H.tabPhotons, tab_D.tabPhotons, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotons)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotons dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
		}

		cudaErreur = hipMemcpy(tab_H.tabPhotonsDown, tab_D.tabPhotonsDown, 4*NBTHETA * NBPHI * sizeof(*(tab_H.tabPhotonsDown)),
hipMemcpyDeviceToHost);
		if( cudaErreur != hipSuccess ){
			printf( "ERREUR: Problème de copie tab_H.tabPhotonsDown dans le main\n");
			printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
			exit(1);
        }
#ifdef _PERF
                        StopProcessing(perfMemcpyD2HTab);
                        GetElapsedTime(perfMemcpyD2HTab);
#endif

#ifdef _PERF
                StartProcessing(perfCreateWitness);
#endif
		/** Creation d'un fichier témoin pour pouvoir reprendre la simulation en cas d'arrêt **/
		for(int i = 0; i < 4*NBTHETA*NBPHI; i++) {
			tabPhotonsTot[i] += (double) tab_H.tabPhotons[i];
			tabPhotonsTotDown[i] += (double) tab_H.tabPhotonsDown[i];
        }
		
        time_current = clock() / CLOCKS_PER_SEC;
		if ((WRITE_PERIOD > 0) && ((time_current - time_lastwrite > 60*WRITE_PERIOD) || (time_lastwrite == 0))) {
            printf("== WRITING WITNESS FILE ==\n"); // FIXME
            creerHDFTemoin(tabPhotonsTot, tabPhotonsTotDown, nbPhotonsTot,var_H, tempsPrec);
            time_lastwrite = time_current;
        }
#ifdef _PERF
                        StopProcessing(perfCreateWitness);
                        GetElapsedTime(perfCreateWitness);
#endif
		
		#ifdef PROGRESSION
		nbPhotonsSorTot += var_H->nbPhotonsSor;
		#endif
		
		
		/** Affichage de l'avancement de la simulation **/
		afficheProgress(nbPhotonsTot, var_H, tempsPrec
			#ifdef PROGRESSION
			, nbPhotonsSorTot
			#endif
			);
	}
	
	// Si on n'est pas passé dans la boucle on affiche quand-même l'avancement de la simulation
	if(!passageBoucle) afficheProgress(nbPhotonsTot, var_H, tempsPrec
					#ifdef PROGRESSION
					, nbPhotonsSorTot
					#endif
					  );
	
	
	#ifdef TABRAND
	// DEBUG Recuperations et affichage des nombres aleatoires du random
	cudaErreur = hipMemcpy(tableauRand_H, tableauRand_D, 100 * sizeof(float), hipMemcpyDeviceToHost);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie tableauRand_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}


	printf("\n=====RAND========================\n");
	for(int i = 0; i < 10; i++)
	{
		printf("thread%d : ", i%5);
		for(int j = 0; j < 10; j++)
		{
			printf("%f - ", tableauRand_H[i*10+j]);
		}
		printf("\n");
	}
	printf("==================================\n");
	#endif
	
	
#ifdef _PERF
        StartProcessing(perfCreateFinalTab);
#endif
	/** Création et calcul du tableau final (regroupant le poids de tous les photons ressortis sur une demi-sphère,
	 * par unité de surface) 
	**/	
	// Remplissage des 3 tableaux
	calculTabFinal(tabFinal, tabTh, tabPhi, tabPhotonsTot, nbPhotonsTot);
	calculTabFinal(tabFinalDown, tabTh, tabPhi, tabPhotonsTotDown, nbPhotonsTot);

	
	/** Fonction qui crée le fichier .hdf contenant le résultat final sur la demi-sphère **/
//	creerHDFResultats(tabFinal, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
	creerHDFResultats(tabFinal, tabFinalDown, tabTh, tabPhi, nbPhotonsTot, var_H, tempsPrec);
#ifdef _PERF
        StopProcessing(perfCreateFinalTab);
        GetElapsedTime(perfCreateFinalTab);
#endif
	printf(" Fin de l'execution du programme. Resultats stockes dans %s\n",PATHRESULTATSHDF);
	
	#ifdef TRAJET
	/** Affichage du trajet du photon **/
	// Récupération des variables envoyées dans le kernel
	cudaErreur = hipMemcpy(evnt_H, evnt_D, NBTRAJET * sizeof(Evnt), hipMemcpyDeviceToHost);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de copie evnt_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	
	// Affichage du trajet du premier thread
	afficheTrajet(evnt_H);
	#endif

#ifdef _PERF
        StartProcessing(perfFree);
#endif
	/** Libération de la mémoire allouée **/
	// Libération du groupe de variables envoyé dans le kernel
	cudaErreur = hipFree(var_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de var_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(var_H);
    free(tabFinal);
    free(tabFinalDown);
    free(tabPhi);
    free(tabTh);

	#ifdef SPHERIQUE	/* Code spécifique à une atmosphère sphérique */
	cudaErreur = hipFree(init_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de init_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}

	free(init_H);
	#endif
	
	// Libération des tableaux envoyés dans le kernel
	freeTableaux(&tab_H, &tab_D);
	// Libération du tableau du host
	free( tabPhotonsTot );
	free( tabPhotonsTotDown );

	// Libération des variables qui récupèrent le trajet d'un photon
	#ifdef TRAJET
	cudaErreur = hipFree(evnt_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free de evnt_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	#endif
	
	#ifdef TABRAND
	//DEBUG random
	cudaErreur = hipFree(tableauRand_D);
	if( cudaErreur != hipSuccess ){
		printf( "ERREUR: Problème de free tableauRand_D dans le main\n");
		printf( "Nature de l'erreur: %s\n",hipGetErrorString(cudaErreur) );
		exit(1);
	}
	#endif
#ifdef _PERF
        StopProcessing(perfFree);
        GetElapsedTime(perfFree);
#endif

#ifdef _PERF
        if ( perfPrint != NULL )
            printf("\n %s...%u us", GetName(perfPrint), GetTotalElapsedTime(perfPrint));
        if ( perfInitG != NULL )
            printf("\n %s...%u us", GetName(perfInitG), GetTotalElapsedTime(perfInitG));
        if ( perfKernel != NULL )
            printf("\n %s...%u us", GetName(perfKernel), GetTotalElapsedTime(perfKernel));
        if ( perfMemcpyH2DVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DVar), GetTotalElapsedTime(perfMemcpyH2DVar));
        if ( perfMemcpyH2DTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyH2DTab), GetTotalElapsedTime(perfMemcpyH2DTab));
        if ( perfMemcpyD2HVar != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HVar), GetTotalElapsedTime(perfMemcpyD2HVar));
        if ( perfMemcpyD2HTab != NULL )
            printf("\n %s...%u us", GetName(perfMemcpyD2HTab), GetTotalElapsedTime(perfMemcpyD2HTab));
        if ( perfCreateWitness != NULL )
            printf("\n %s...%u us", GetName(perfCreateWitness), GetTotalElapsedTime(perfCreateWitness));
        if ( perfCreateFinalTab != NULL )
            printf("\n %s...%u us", GetName(perfCreateFinalTab), GetTotalElapsedTime(perfCreateFinalTab));
        if ( perfFree != NULL )
            printf("\n %s...%u us", GetName(perfFree), GetTotalElapsedTime(perfFree));
        DeleteSPerf(perfPrint);
        DeleteSPerf(perfInitG);
        DeleteSPerf(perfKernel);
        DeleteSPerf(perfMemcpyH2DVar);
        DeleteSPerf(perfMemcpyH2DTab);
        DeleteSPerf(perfMemcpyD2HVar);
        DeleteSPerf(perfMemcpyD2HTab);
        DeleteSPerf(perfCreateWitness);
        DeleteSPerf(perfCreateFinalTab);
        DeleteSPerf(perfFree);
        printf("\n");
#endif

	return 0;

}
